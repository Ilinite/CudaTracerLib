#include "hip/hip_runtime.h"
#include "Vec2fPixelDebugVisualizer.h"
#include <Engine/Image.h>
#include "PixelDebugVisualizerHelpers.h"
#include <Kernel/TraceHelper.h>
#include <Math/Sampling.h>

namespace CudaTracerLib {

struct Vec2f_op
{
	CUDA_DEVICE void operator()(unsigned int x, unsigned int y, Image& img, PixelDebugVisualizer<Vec2f>& buffer, bool arg)
	{
		auto val = buffer.getScaledValue(x, y);
		Spectrum col;
		if (arg)
			col = Spectrum((val.x + 1) / 2, (val.y + 1) / 2, 0.0f);
		else col = Spectrum(val.x, val.y, 0.0f);

		img.getProcessedData(x, y) = col.toRGBCOL();
	}
};

void PixelDebugVisualizer<Vec2f>::Visualize(Image& img)
{
	m_buffer.Synchronize();

	Launch(img, *this, m_normalize, Vec2f_op());
}


void PixelDebugVisualizer<Vec2f>::VisualizePixel(unsigned int x, unsigned int y, const IDebugDrawer& drawer)
{
	auto prim_ray = g_SceneData.GenerateSensorRay(x, y);
	auto res = traceRay(prim_ray);
	if (!res.hasHit())
		return;

	DifferentialGeometry dg;
	res.fillDG(dg);

	auto v = getScaledValue(x, y);

	if (m_pixelType == VisualizePixelType::OnSurface)
	{
		drawer.DrawLine(dg.P, dg.P + dg.sys.toWorld(Vec3f(v.x, v.y, 0.0f)));
	}
	else if (m_pixelType == VisualizePixelType::Ellipse)
	{
		drawer.DrawEllipse(dg.P, dg.sys.t, dg.sys.s, v.x, v.y);
	}
	else if (m_pixelType == VisualizePixelType::PolarCoordinates)
	{
		Vec2f t(v.y * math::cos(v.x), v.y * math::sin(v.x));
		drawer.DrawLine(dg.P, dg.P + dg.sys.toLocal(Vec3f(t.x, t.y, 0.0f)));
	}
	else if (m_pixelType == VisualizePixelType::SphericalCoordinates)
	{
		auto dir = MonteCarlo::SphericalDirection(v.x, v.y);
		drawer.DrawLine(dg.P, dg.P + dg.sys.toWorld(dir));
	}
}

}