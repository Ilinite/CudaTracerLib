#include "hip/hip_runtime.h"
#include "k_PathTracer.h"
#include "k_TraceHelper.h"
#include <time.h>
#include "k_TraceAlgorithms.h"

CUDA_ALIGN(16) CUDA_DEVICE unsigned int g_NextRayCounter;

__global__ void pathKernel(unsigned int width, unsigned int height, unsigned int a_PassIndex, e_Image g_Image)
{
	CudaRNG rng = g_RNGData();
	int rayidx;
	int N = width * height;
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        const bool          terminated     = 1;//nodeAddr == EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));	

        if(terminated)
        {			
            if (idxTerminated == 0)
				rayBase = atomicAdd(&g_NextRayCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
			if (rayidx >= N)
                break;
		}

		unsigned int x = rayidx % width, y = rayidx / width;
		Ray r;
		Spectrum imp = g_CameraData.sampleRay(r, make_float2(x, y), rng.randomFloat2());

		Spectrum col = imp * PathTrace(r.direction, r.origin, rng);
		
		g_Image.AddSample(x, y, col);
	}
	while(true);
	g_RNGData(rng);
}

__global__ void debugPixel(unsigned int width, unsigned int height, int2 p)
{
	CudaRNG rng = g_RNGData();
	Ray r = g_CameraData.GenRay(p.x, p.y);	
	PathTrace(r.direction, r.origin, rng);
}

void k_PathTracer::DoRender(e_Image* I)
{
	k_ProgressiveTracer::DoRender(I);
	unsigned int zero = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(g_NextRayCounter), &zero, sizeof(unsigned int));
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, g_sRngs);
	if(m_Direct)
		pathKernel<<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, m_uPassesDone, *I);
	else pathKernel<<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, m_uPassesDone, *I);
	m_uPassesDone++;
	k_TracerBase_update_TracedRays
	I->DoUpdateDisplay();
}

void k_PathTracer::Debug(int2 pixel)
{
	m_pScene->UpdateInvalidated();
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, g_sRngs);
	debugPixel<<<1,1>>>(w,h,pixel);
}