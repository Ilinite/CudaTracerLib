#include "hip/hip_runtime.h"
#include "k_IrradianceCache.h"
#include "k_TraceAlgorithms.h"
#include "k_TraceHelper.h"
#include "k_IntegrateHelper.h"

CUDA_DEVICE k_HashGrid_Irreg g_sHash;
CUDA_DEVICE unsigned int g_sEntryCount;

template<bool DIRECT> CUDA_DEVICE float3 E1(Ray& r, TraceResult& r2, CudaRNG& rng, e_KernelBSDF* bsdf, float* dist, float3* awi = 0)
{
	float3 wo = -r.direction;
	float pdf;
	float3 wi;
	bsdf->Sample_f(wo, &wi, BSDFSample(rng), &pdf, BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_DIFFUSE));
	if(awi)
		*awi = wi;
	return PathTrace<DIRECT>(wi, r(r2.m_fDist), rng, dist);
}

template<bool DIRECT, int N> CUDA_DEVICE void E2(Ray& r, TraceResult& r2, CudaRNG& rng, e_KernelBSDF* bsdf, float3* ae, float* ar, float3* awi = 0)
{
	float3 e = make_float3(0);
	float air = 0;
	float3 w = make_float3(0);
	for(int i = 0; i < N; i++)
	{
		float q;
		e += E1<DIRECT>(r, r2, rng, bsdf, &q, &w);
		air += 1.0f / q;
	}
	*ar = float(N) / air;
	if(awi)
		*awi = w / float(N);
	*ae = e / float(N) * PI;
}

template<bool DIRECT, int N> CUDA_DEVICE float3 E(Ray& r, TraceResult& r2, CudaRNG& rng, e_KernelBSDF* bsdf, k_IrrEntry* entries, unsigned int entryNum, unsigned int* grid, float rScale, float3* awi = 0)
{/*
		float3 ae;
		float ar;
		float3 wi;
		E2<DIRECT, N>(r, r2, rng, bsdf, &ae, &ar, &wi);
		if(awi)
			*awi = wi;
		return ae;*/

	uint3 i0 = g_sHash.Transform(r(r2.m_fDist));
	unsigned int i = g_sHash.Hash(i0);
	unsigned int j = atomicInc(&g_sEntryCount, -1);
	if(j < entryNum)
	{
		unsigned int k = atomicExch(grid + i, j);
		float3 ae;
		float ar;
		float3 wi;
		E2<DIRECT, N>(r, r2, rng, bsdf, &ae, &ar, &wi);
		entries[j] = k_IrrEntry(r(r2.m_fDist), ae, bsdf->sys.m_normal, ar * rScale, k, wi);
		if(awi)
			*awi = wi;
		return ae;
	}
	return make_float3(0);
}

template<bool DIRECT, int N> __global__ void kFirstPass(int w, int h, k_IrrEntry* entries, unsigned int entryNum, unsigned int* grid, float rScale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	CudaRNG rng = g_RNGData();
	if(x < w && y < h)
	{
		Ray r = g_CameraData.GenRay(x, y, w, h, rng.randomFloat(), rng.randomFloat());
		TraceResult r2 = k_TraceRay(r);
		if(r2)
		{
			e_KernelBSDF bsdf = r2.m_pTri->GetBSDF(r2.m_fUV, r2.m_pNode->getWorldMatrix(), g_SceneData.m_sMatData.Data, r2.m_pNode->m_uMaterialOffset);
			if(bsdf.NumComponents(BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_DIFFUSE)) && g_sHash.IsValidHash(r(r2.m_fDist)))
				E<DIRECT, N>(r, r2, rng, &bsdf, entries, entryNum, grid, rScale);
		}
	}
}

template<bool DIRECT, int N, int M, int O> __global__ void kScndPass(int w, int h, RGBCOL* a_Target, k_IrrEntry* entries, unsigned int entryNum, unsigned int* grid, unsigned int gridLength, float rScale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	CudaRNG rng = g_RNGData();
	if(x < w && y < h)
	{
		Ray ro = g_CameraData.GenRay(x, y, w, h, rng.randomFloat(), rng.randomFloat());

		struct stackEntry
		{
			Ray r;
			float3 fs;
			unsigned int d;
			CUDA_FUNC_IN stackEntry(){}
			CUDA_FUNC_IN stackEntry(Ray _r, float3 _fs, unsigned int _d)
			{
				r = _r;
				fs = _fs;
				d = _d;
			}
		};
		float3 L = make_float3(0);
		const unsigned int stackN = 16;
		stackEntry stack[stackN];
		stack[0] = stackEntry(ro, make_float3(1), 0);
		unsigned int stackPos = 1;
		while(stackPos)
		{
			stackEntry s = stack[--stackPos];
			TraceResult r2 = k_TraceRay(s.r);
			if(r2)
			{
				e_KernelBSDF bsdf = r2.m_pTri->GetBSDF(r2.m_fUV, r2.m_pNode->getWorldMatrix(), g_SceneData.m_sMatData.Data, r2.m_pNode->m_uMaterialOffset);

				float3 p = s.r(r2.m_fDist);
				L += s.fs * Le(p, bsdf.ng, -s.r.direction, r2, g_SceneData);
				if(bsdf.NumComponents(BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_DIFFUSE)) && g_sHash.IsValidHash(p))
				{
					uint3 i0 = g_sHash.Transform(p);
					int num = 0, si = -1;
					float3 EAcc = make_float3(0), wiAcc = make_float3(0);
					float wAcc = 0;
					while(num < M && si++ < O)
					{
						uint3 low = i0 - make_uint3(si), high = i0 + make_uint3(si);
#define ITERATE(fixedCoord, a, b, c) \
	for(int i = low.a; i < high.a; i++) \
		for(int j = low.b; j < high.b; j++) \
		{ \
			uint3 co; \
			co.a = i; \
			co.b = j; \
			co.c = fixedCoord.c; \
			unsigned int hi = g_sHash.Hash(co), e = grid[hi]; \
			while(e != -1 && e < entryNum) \
			{ \
				k_IrrEntry e2 = entries[e]; \
				e = e2.next; \
				float d = dot(e2.nor, bsdf.ng), d2 = DistanceSquared(e2.pos, p); \
				float perr = Distance(p, e2.pos) / e2.rad; \
				float nerr = sqrtf((1.f - dot(e2.nor, bsdf.ng)) / (1.f - cosf(Radians(30)))); \
				if(perr < 1 && dot(e2.nor, bsdf.ng) > 0.99f) \
				{ \
					float wi = 1 ; \
					EAcc += wi * e2.E; \
					wiAcc += wi * e2.wi; \
					wAcc += wi; \
					num++; \
				} \
			} \
		}
						ITERATE(low, x, z, y)
						if(si)
						{
							ITERATE(high, x, z, y)

							ITERATE(low, x, y, z)
							ITERATE(high, x, y, z)						

							ITERATE(low, z, y, x)
							ITERATE(high, z, y, x)
						}
#undef ITERATE
					}
					float3 e, wi;
					if(num >= M)
					{
						e = EAcc / wAcc;
						wi = normalize(wiAcc / wAcc);
						L = make_float3(0,0,0);
						break;
					}
					else
					{
						e = E<DIRECT, N>(s.r, r2, rng, &bsdf, entries, entryNum, grid, rScale, &wi);
						L = make_float3(1,0,0);
						break;
					}
					L += s.fs * bsdf.f(-s.r.direction, wi, BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_DIFFUSE)) * e;
				}
				if(s.d < 5 && stackPos < stackN - 1)
				{
					float3 r_wi;
					float r_pdf;
					float3 r_f = bsdf.Sample_f(-s.r.direction, &r_wi, BSDFSample(rng), &r_pdf, BxDFType(BSDF_REFLECTION | BSDF_SPECULAR | BSDF_GLOSSY));
					if(r_pdf && fsumf(r_f) != 0)
						stack[stackPos++] = stackEntry(Ray(p, r_wi), bsdf.IntegratePdf(r_f, r_pdf, r_wi) * s.fs, s.d + 1);
					float3 t_wi;
					float t_pdf;
					float3 t_f = bsdf.Sample_f(-s.r.direction, &t_wi, BSDFSample(rng), &t_pdf, BxDFType(BSDF_TRANSMISSION | BSDF_SPECULAR | BSDF_GLOSSY));
					if(t_pdf && fsumf(t_f) != 0)
						stack[stackPos++] = stackEntry(Ray(p, t_wi), bsdf.IntegratePdf(t_f, t_pdf, t_wi) * s.fs, s.d + 1);
				}
			}
		}

		unsigned int i2 = y * w + x;
		a_Target[i2] = Float3ToCOLORREF(L);
	}
	g_RNGData(rng);
}

void k_IrradianceCache::DoRender(RGBCOL* a_Buf)
{	
	m_uPassesDone++;
}

void k_IrradianceCache::StartNewTrace(RGBCOL* a_Buf)
{
	unsigned int abc = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(g_sEntryCount), &abc, 4);
	hipMemset(m_pGrid, -1, sizeof(unsigned int) * m_uGridLength);
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, m_sRngs);
	AABB m_sEyeBox = GetEyeHitPointBox();
	float r = fsumf(m_sEyeBox.maxV - m_sEyeBox.minV) / w;
	m_sGrid = k_HashGrid_Irreg(m_sEyeBox, r, m_uGridLength);
	hipMemcpyToSymbol(HIP_SYMBOL(g_sHash), &m_sGrid, sizeof(m_sGrid));
	int p = 16, p2 = 64;
	//kFirstPass<false, 4><<<dim3( p2 / p, p2 / p, 1), dim3(p, p, 1)>>>(p2, p2, m_pEntries, m_uEntryNum, m_pGrid, rScale);
	//hipDeviceSynchronize();
	kScndPass<false, 16, 4, 10><<<dim3( w / p + 1, h / p + 1, 1), dim3(p, p, 1)>>>(w, h, a_Buf, m_pEntries, m_uEntryNum, m_pGrid, m_uGridLength, rScale);
	hipDeviceSynchronize();
}

void k_IrradianceCache::Resize(unsigned int _w, unsigned int _h)
{
	k_TracerBase::Resize(_w, _h);

}

void k_IrradianceCache::Debug(int2 pixel)
{

}