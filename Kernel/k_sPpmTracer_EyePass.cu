#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include "k_TraceHelper.h"
#include "k_IntegrateHelper.h"

//texture<uint4, 1> t_PhotonTex;
//texture<unsigned int, 1> t_HashTex;

void k_PhotonMapCollection::StartNewRendering(const AABB& sbox, const AABB& vbox, float a_R)
{
	//hipChannelFormatDesc cd0 = hipCreateChannelDesc<uint4>();
	//hipBindTexture(0, &t_PhotonTex, m_pPhotons, &cd0, m_uPhotonBufferLength * sizeof(k_pPpmPhoton));
	m_sVolumeMap.StartNewRendering(vbox, a_R);
	m_sSurfaceMap.StartNewRendering(sbox, a_R);
}

template<typename HASH> void k_PhotonMap<HASH>::StartNewRendering(const AABB& box, float a_InitRadius)
{
	m_sHash = HASH(box, a_InitRadius, m_uGridLength);
	hipMemset(m_pDeviceHashGrid, -1, sizeof(unsigned int) * m_uGridLength);

	//hipChannelFormatDesc cd1 = hipCreateChannelDesc<unsigned int>();		
	//hipBindTexture(0, &t_HashTex, m_pDeviceHashGrid, &cd1, m_uGridLength * sizeof(unsigned int));
}

template<typename HASH> CUDA_ONLY_FUNC float3 k_PhotonMap<HASH>::L_Surface(float a_r, float a_NumPhotonEmitted, CudaRNG& rng, const e_KernelBSDF* bsdf, const float3& n, const float3& p, const float3& wo) const
{
	Onb sys(n);
	sys.m_tangent *= a_r;
	sys.m_binormal *= a_r;
	float3 low = fminf(p - sys.m_tangent + sys.m_binormal, p + sys.m_tangent - sys.m_binormal), high = fmaxf(p - sys.m_tangent + sys.m_binormal, p + sys.m_tangent - sys.m_binormal);
	const float r2 = a_r * a_r, r3 = 1.0f / (r2 * a_NumPhotonEmitted), r4 = 1.0f / r2;
	float3 L = make_float3(0), Lr = make_float3(0), Lt = make_float3(0);
	//uint3 lo = m_sHash.Transform(p - make_float3(a_r)), hi = m_sHash.Transform(p + make_float3(a_r));
	uint3 lo = m_sHash.Transform(low), hi = m_sHash.Transform(high);
	const bool glossy = bsdf->NumComponents(BxDFType(BSDF_ALL_TRANSMISSION | BSDF_ALL_REFLECTION | BSDF_GLOSSY));
	for(int a = lo.x; a <= hi.x; a++)
		for(int b = lo.y; b <= hi.y; b++)
			for(int c = lo.z; c <= hi.z; c++)
			{
				unsigned int i0 = m_sHash.Hash(make_uint3(a,b,c)), i = m_pDeviceHashGrid[i0], q = 0;//tex1Dfetch(t_HashTex, i0)
				while(i != -1 && q++ < 1000)
				{
					k_pPpmPhoton e = m_pDevicePhotons[i];
					//k_pPpmPhoton e(tex1Dfetch(t_PhotonTex, i));
					float3 nor = e.getNormal(), wi = e.getWi(), l = e.getL(), P = e.Pos;//m_sHash.DecodePos(e.Pos, make_uint3(a,b,c))
					float dist2 = dot(P - p, P - p);
					if(dist2 < r2 && AbsDot(nor, n) > 0.95f)//
					{
						float s = 1.0f - dist2 * r4, k = 3.0f * INV_PI * s * s * r3;
						if(glossy)
							L += bsdf->f(wo, wi) * k * l;
						else if(dot(n, wi) > 0.0f)
							Lr += k * l;
						else Lt += k * l;
					}
					i = e.next;
				}
			}
	float buf[6 * 6 * 2];
	L += Lr * bsdf->rho(wo, rng, (unsigned char*)&buf, BSDF_ALL_REFLECTION)   * INV_PI +
		 Lt * bsdf->rho(wo, rng, (unsigned char*)&buf, BSDF_ALL_TRANSMISSION) * INV_PI;
	return L;
}

CUDA_DEVICE k_PhotonMapCollection g_Map;

template<typename HASH> template<bool VOL> CUDA_ONLY_FUNC float3 k_PhotonMap<HASH>::L_Volume(float a_r, float a_NumPhotonEmitted, CudaRNG& rng, const Ray& r, float tmin, float tmax, const float3& sigt) const
{
	float Vs = 1.0f / ((4.0f / 3.0f) * PI * a_r * a_r * a_r * a_NumPhotonEmitted), r2 = a_r * a_r;
	float3 L_n = make_float3(0);
	float a,b;
	if(!m_sHash.getAABB().Intersect(r, &a, &b))
		return L_n;//that would be dumb
	a = clamp(a, tmin , tmax );
	b = clamp(b, tmin , tmax );
	float d = 2.0f * a_r;
	while(b > a)
	{
		float3 L = make_float3(0);
		float3 x = r(b);
		uint3 lo = m_sHash.Transform(x - make_float3(a_r)), hi = m_sHash.Transform(x + make_float3(a_r));
		for(unsigned int ac = lo.x; ac <= hi.x; ac++)
			for(unsigned int bc = lo.y; bc <= hi.y; bc++)
				for(unsigned int cc = lo.z; cc <= hi.z; cc++)
				{
					unsigned int i0 = m_sHash.Hash(make_uint3(ac,bc,cc)), i = m_pDeviceHashGrid[i0];
					while(i != -1 && i < m_uMaxPhotonCount)
					{
						k_pPpmPhoton e = m_pDevicePhotons[i];
						float3 wi = e.getWi(), l = e.getL(), P = e.Pos;
						if(dot(P - x, P - x) < r2)
						{
							float p;
							if(VOL)
								p = g_SceneData.m_sVolume.p(x, -wi, r.direction);
							else p = 1.f / (4.f * PI);
							L += p * l * Vs;
						}
						i = e.next;
					}
				}
		if(VOL)
			L_n = L * d + L_n * exp(-g_SceneData.m_sVolume.tau(r, b - d, b)) + g_SceneData.m_sVolume.Lve(x, -1.0f * r.direction) * d;
		else L_n = L * d + L_n * exp(sigt * -d);
		b -= d;
	}
	return L_n;
}

template<bool DIRECT> __global__ void k_EyePass(int2 off, int w, int h, RGBCOL* a_Target, k_sPpmPixel* a_Pixels, float a_PassIndex, float a_rSurface, float a_rVolume)
{
	if(off.x)
		a_PassIndex++;
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	CudaRNG rng = g_RNGData();
	x += off.x; y += off.y;
	if(x < w && y < h)
	{
		Ray ro = g_CameraData.GenRay(x, y, w, h, rng.randomFloat(), rng.randomFloat());

		struct stackEntry
		{
			Ray r;
			float3 fs;
			unsigned int d;
			CUDA_FUNC_IN stackEntry(){}
			CUDA_FUNC_IN stackEntry(Ray _r, float3 _fs, unsigned int _d)
			{
				r = _r;
				fs = _fs;
				d = _d;
			}
		};
		float3 L = make_float3(0);
		const unsigned int stackN = 16;
		stackEntry stack[stackN];
		stack[0] = stackEntry(ro, make_float3(1), 0);
		unsigned int stackPos = 1;
		while(stackPos)
		{
			stackEntry s = stack[--stackPos];
			TraceResult r2;
			r2.Init();
			if(k_TraceRay<true>(s.r.direction, s.r.origin, &r2))
			{
				e_KernelBSDF bsdf = r2.m_pTri->GetBSDF(r2.m_fUV, r2.m_pNode->getWorldMatrix(), g_SceneData.m_sMatData.Data, r2.m_pNode->m_uMaterialOffset);

				if(g_SceneData.m_sVolume.HasVolumes())
				{
					float tmin, tmax;
					g_SceneData.m_sVolume.IntersectP(s.r, 0, r2.m_fDist, &tmin, &tmax);
					L += s.fs * g_Map.L<true>(a_rVolume, rng, s.r, tmin, tmax, make_float3(0));
					s.fs = s.fs * exp(-g_SceneData.m_sVolume.tau(s.r, tmin, tmax));
				}

				float3 p = s.r(r2.m_fDist);
				if(DIRECT)
					L += s.fs * UniformSampleAllLights(p, bsdf.ng, -s.r.direction, &bsdf, rng, 4);
				L += s.fs * Le(s.r(r2.m_fDist), bsdf.ng, -s.r.direction, r2, g_SceneData);
				e_KernelBSSRDF bssrdf;
				if(r2.m_pTri->GetBSSRDF(r2.m_fUV, r2.m_pNode->getWorldMatrix(), g_SceneData.m_sMatData.Data, r2.m_pNode->m_uMaterialOffset, &bssrdf))
				{
					float3 dir = refract(s.r.direction, bsdf.sys.m_normal, 1.0f / bssrdf.e);
					TraceResult r3;
					r3.Init();
					k_TraceRay<true>(dir, p, &r3);
					L += s.fs * g_Map.L<false>(a_rVolume, rng, Ray(p, dir), 0, r3.m_fDist, bssrdf.sigp_s + bssrdf.sig_a);
				}
				if(bsdf.NumComponents(BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_DIFFUSE)))
					L += s.fs * g_Map.L(a_rSurface, rng, &bsdf, bsdf.sys.m_normal, p, -s.r.direction);
				if(s.d < 5 && stackPos < stackN - 1)
				{/*
					if(bsdf.NumComponents(BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_GLOSSY)))
					{
						float3 wi;
						float pdf;
						const int N = clamp(2u, 0u, stackN - stackPos);
						for(int i = 0; i < N; i++)
						{
							float3 f = bsdf.Sample_f(-s.r.direction, &wi, BSDFSample(rng), &pdf, BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_GLOSSY));
							if(pdf && fsumf(f) != 0)
								stack[stackPos++] = stackEntry(Ray(p, wi), bsdf.IntegratePdf(f, pdf, wi) * s.fs / float(N), s.d + 1);
						}
					}
					else*/
					{
						float3 r_wi;
						float r_pdf;
						float3 r_f = bsdf.Sample_f(-s.r.direction, &r_wi, BSDFSample(rng), &r_pdf, BxDFType(BSDF_REFLECTION | BSDF_SPECULAR | BSDF_GLOSSY));
						if(r_pdf && fsumf(r_f) != 0)
							stack[stackPos++] = stackEntry(Ray(p, r_wi), bsdf.IntegratePdf(r_f, r_pdf, r_wi) * s.fs, s.d + 1);
						float3 t_wi;
						float t_pdf;
						float3 t_f = bsdf.Sample_f(-s.r.direction, &t_wi, BSDFSample(rng), &t_pdf, BxDFType(BSDF_TRANSMISSION | BSDF_SPECULAR | BSDF_GLOSSY));
						if(t_pdf && fsumf(t_f) != 0)
							stack[stackPos++] = stackEntry(Ray(p, t_wi), bsdf.IntegratePdf(t_f, t_pdf, t_wi) * s.fs, s.d + 1);
					}
				}
			}
			else if(g_SceneData.m_sVolume.HasVolumes())
			{
				float tmin, tmax;
				g_SceneData.m_sVolume.IntersectP(s.r, 0, r2.m_fDist, &tmin, &tmax);
				L += s.fs * g_Map.L<true>(a_rVolume, rng, s.r, tmin, tmax, make_float3(0));
			}
		}
		a_Pixels[y * w + x].m_vPixelColor += L;
		RGBCOL c = Float3ToCOLORREF(a_Pixels[y * w + x].m_vPixelColor / a_PassIndex);
		unsigned int i2 = y * w + x;
		a_Target[i2] = c;
	}
	g_RNGData(rng);
}

/*
__global__ void k_EyePass(int2 off, int w, int h, RGBCOL* a_Target, k_sPpmPixel* a_Pixels, float a_PassIndex, float a_rSurface, float a_rVolume)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	CudaRNG rng = g_RNGData();
	hiprand_init(1234, y * w + x, a_PassIndex, (hiprandState*)&rng);
	x += off.x; y += off.y;
	if(x < w && y < h)
	{
		Ray r = g_CameraData.GenRay(x, y, w, h, rng.randomFloat(), rng.randomFloat());

		float3 L = make_float3(0), throughput = make_float3(1);
		TraceResult r2;
		r2.Init(); int d = 0;
		while(k_TraceRay<true>(r.direction, r.origin, &r2) && d++ < 10)
		{
			float3 p = r(r2.m_fDist);
			e_KernelBSDF bsdf = r2.m_pTri->GetBSDF(r2.m_fUV, r2.m_pNode->getWorldMatrix(), g_SceneData.m_sMatData.Data, r2.m_pNode->m_uMaterialOffset);
			L += throughput * r2.m_pTri->Le(r2.m_fUV, bsdf.ng, -r.direction, g_SceneData.m_sMatData.Data, r2.m_pNode->m_uMaterialOffset);
			if(bsdf.NumComponents(BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_DIFFUSE)))
			{
				L += throughput * g_Map.L(a_rSurface, rng, &bsdf, bsdf.sys.m_normal, p, -r.direction);
				break;
			}
			else
			{
				float3 wi;
				float pdf;
				BxDFType sampledType;
				float3 f = bsdf.Sample_f(-r.direction, &wi, BSDFSample(rng), &pdf, BxDFType(BSDF_REFLECTION | BSDF_TRANSMISSION | BSDF_SPECULAR | BSDF_GLOSSY), &sampledType);
				if(pdf > 0 && fsumf(f) != 0)
				{
					throughput = throughput * f * AbsDot(wi, bsdf.sys.m_normal) / pdf;
					r = Ray(p, wi);
				}
				else break;
			}
			r2.Init();
		}

		a_Pixels[y * w + x].m_vPixelColor += L;
		RGBCOL c = Float3ToCOLORREF(a_Pixels[y * w + x].m_vPixelColor / a_PassIndex);
		unsigned int i2 = y * w + x;
		a_Target[i2] = c;
	}
	g_RNGData(rng);
}
*/
void k_sPpmTracer::doEyePass(RGBCOL* a_Buf)
{
	hipMemcpyToSymbol(HIP_SYMBOL(g_Map), &m_sMaps, sizeof(k_PhotonMapCollection));
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, m_sRngs);
	const unsigned int p = 16;
	if(m_bDirect)
		k_EyePass<true><<<dim3( w / p + 1, h / p + 1, 1), dim3(p, p, 1)>>>(make_int2(0,0), w, h, a_Buf, m_pDevicePixels, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3));
	else k_EyePass<false><<<dim3( w / p + 1, h / p + 1, 1), dim3(p, p, 1)>>>(make_int2(0,0), w, h, a_Buf, m_pDevicePixels, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3));
}

void k_sPpmTracer::Debug(int2 pixel)
{
	hipMemcpyToSymbol(HIP_SYMBOL(g_Map), &m_sMaps, sizeof(k_PhotonMapCollection));
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, m_sRngs);
	const unsigned int p = 16;
	k_EyePass<true><<<1, 1>>>(pixel, w, h, (RGBCOL*)m_pDevicePixels, m_pDevicePixels, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3));
}