#include "hip/hip_runtime.h"
#include "PathSpaceFilteringBuffer.h"
#include "Tracer.h"
#include "TraceAlgorithms.h"

namespace CudaTracerLib {

void PathSpaceFilteringBuffer::PrepareForRendering(Image& I, DynamicScene* scene)
{
	auto eye_hit_points = TracerBase::GetEyeHitPointBox(scene, false);
	m_hitPointBuffer.SetGridDimensions(eye_hit_points);
}

template<bool USE_DEPTH_IMAGE> CUDA_FUNC_IN void computePixel(Image& I, DeviceDepthImage& depthImg, SpatialGridList_Linked<PathSpaceFilteringBuffer::path_entry>& path_buffer, float rad, unsigned int x, unsigned int y)
{
	auto rng = g_SamplerData(y * I.getWidth() + x);
	NormalizedT<Ray> ray, rayX, rayY;
	auto W = g_SceneData.sampleSensorRay(ray, rayX, rayY, Vec2f((float)x, (float)y) + rng.randomFloat2(), rng.randomFloat2());
	auto res = traceRay(ray);
	int depth = 0;
	while (res.hasHit() && depth++ < 4)
	{
		BSDFSamplingRecord bRec;
		res.getBsdfSample(ray, bRec, ERadiance);

		if (depth == 1)
		{
			bRec.dg.computePartials(ray, rayX, rayY);
			Vec3f dp_dx, dp_dy;
			bRec.dg.compute_dp_ds(dp_dx, dp_dy);
			float avg_dist_next_pixel = fmaxf(dp_dx.length(), dp_dy.length());
			//rad = fmaxf(rad, avg_dist_next_pixel);

			if (USE_DEPTH_IMAGE)
				depthImg.Store(x, y, res.m_fDist);
		}

		if (res.getMat().bsdf.hasComponent(EDelta) || res.getMat().bsdf.hasComponent(EGlossy))
		{
			W *= res.getMat().bsdf.sample(bRec, rng.randomFloat2());
			ray = NormalizedT<Ray>(bRec.dg.P, bRec.getOutgoing());
			res = traceRay(ray);
		}
		else
		{
			auto query_box = bRec.dg.ComputeOnSurfaceDiskBounds(rad);
			Spectrum L_o = 0.0f;
			int n_found = 0;
			path_buffer.ForAll(query_box.minV, query_box.maxV, [&](const Vec3u& cell_idx, unsigned int idx, const PathSpaceFilteringBuffer::path_entry& ent)
			{
				if (distanceSquared(ent.p, bRec.dg.P) < rad * rad && dot(Uchar2ToNormalizedFloat3(ent.nor), bRec.dg.sys.n) > 0.75f)
				{
					bRec.wo = Uchar2ToNormalizedFloat3(ent.wi);
					auto f_r = res.getMat().bsdf.f(bRec);
					Spectrum L_i;
					L_i.fromRGBE(ent.Li);
					L_o += f_r * L_i;
					n_found++;
				}
			});
			if(n_found != 0)
				L_o /= (float)n_found;

			L_o += UniformSampleOneLight(bRec, res.getMat(), rng);

			I.AddSample(x, y, W * L_o);
			break;
		}
	}
}

template<bool USE_DEPTH_IMAGE> CUDA_GLOBAL void computePixelsKernel(Image I, DeviceDepthImage depthImg, SpatialGridList_Linked<PathSpaceFilteringBuffer::path_entry> path_buffer, float rad)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x, y = threadIdx.y + blockDim.y * blockIdx.y;
	if (x < I.getWidth() && y < I.getHeight())
		computePixel<USE_DEPTH_IMAGE>(I, depthImg, path_buffer, rad, x, y);
}

void PathSpaceFilteringBuffer::ComputePixelValues(Image& I, DynamicScene* scene, DeviceDepthImage* depthImage)
{
	auto rad_i = m_pixelRad0 * math::pow((float)m_numIteration, ((2.0f / 3.0f) - 1) / 2.0f);
	UpdateKernel(scene);
	int p0 = 16;
	if(depthImage)
		computePixelsKernel<true> << <dim3(I.getWidth() / p0 + 1, I.getHeight() / p0 + 1, 1), dim3(p0, p0, 1) >> >(I, *depthImage, m_hitPointBuffer, rad_i);
	else computePixelsKernel<false> << <dim3(I.getWidth() / p0 + 1, I.getHeight() / p0 + 1, 1), dim3(p0, p0, 1) >> >(I, DeviceDepthImage(), m_hitPointBuffer, rad_i);
	ThrowCudaErrors(hipDeviceSynchronize());
}

}