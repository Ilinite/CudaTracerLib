#include "hip/hip_runtime.h"
#include "VarianceBlockSampler.h"
#include <Kernel/Tracer.h>

namespace CudaTracerLib
{

CUDA_GLOBAL void updateInfo(VarianceBlockSampler::TmpBlockInfo* a_pTmpBlockInfoDevice, IBlockSampler::BlockInfo* a_pPersBlockInfoDevice, const PixelVarianceBuffer varBuffer, Image img, float splatScale, unsigned int numTotalBlocksX, float numPasses)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x, y = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int b_x = x / BLOCK_SAMPLER_BlockSize, b_y = y / BLOCK_SAMPLER_BlockSize, bIdx = b_y * numTotalBlocksX + b_x;

	if (x < img.getWidth() && y < img.getHeight())
	{
		auto num_passes_block = a_pPersBlockInfoDevice[bIdx].passesDone;

		auto I_N = img.getPixelData(x, y).toSpectrum(splatScale);

		auto pInfo = varBuffer(x, y);
		auto var = pInfo.computeVariance();
		auto e = pInfo.computeAverage();

		auto& bInfo = a_pTmpBlockInfoDevice[bIdx];
		if (var >= 0 && !math::IsNaN(var))
		{
			atomicAdd(&bInfo.BLOCK_VAR_I, var);
			atomicInc(&bInfo.NUM_PIXELS_VAR, 0xffffffff);
		}
		atomicAdd(&bInfo.BLOCK_E_I, e);
		atomicAdd(&bInfo.BLOCK_E_I2, e * e);
		atomicInc(&bInfo.NUM_PIXELS_E, 0xffffffff);
	}
}

CUDA_GLOBAL void visualizeWeights(VarianceBlockSampler::TmpBlockInfo* a_pTmpBlockInfoDevice, IBlockSampler::BlockInfo* a_pPersBlockInfoDevice, const PixelVarianceBuffer varBuffer, Image img, float splatScale, unsigned int numTotalBlocksX,
								  float min_block, float max_block, float min_est, float max_est)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x, y = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int b_x = x / BLOCK_SAMPLER_BlockSize, b_y = y / BLOCK_SAMPLER_BlockSize, bIdx = b_y * numTotalBlocksX + b_x;

	if (x < img.getWidth() && y < img.getHeight())
	{
		auto& bInfo = a_pTmpBlockInfoDevice[bIdx];
		float w = bInfo.getWeight(min_block, max_block, min_est, max_est);
		img.getProcessedData(x, y) = Spectrum(w).toRGBCOL();
	}
}

void VarianceBlockSampler::StartNewRendering(DynamicScene* a_Scene, Image* img)
{
	IUserPreferenceSampler::StartNewRendering(a_Scene, img);
	m_uPassesDone = 0;
}

void VarianceBlockSampler::AddPass(Image* img, TracerBase* tracer, const PixelVarianceBuffer& varBuffer)
{
	m_uPassesDone++;

	const int cBlock = 16;
	int nx = (img->getWidth() + cBlock - 1) / cBlock, ny = (img->getHeight() + cBlock - 1) / cBlock;

	m_blockInfo.Memset(0);
	updateInfo << <dim3(nx, ny), dim3(cBlock, cBlock) >> > (m_blockInfo.getDevicePtr(), m_sBlockInfo.getDevicePtr(), varBuffer, *img, tracer->getSplatScale(), getTotalBlocksXDim(), (float)m_uPassesDone);
	m_blockInfo.setOnGPU();
	m_blockInfo.Synchronize();

	float min_block = FLT_MAX, max_block = -FLT_MAX;
	float min_est = FLT_MAX, max_est = -FLT_MAX;
	for (unsigned int i = 0; i < m_blockInfo.getLength(); i++)
	{
		auto& b = m_blockInfo[i];
		float est_var = b.get_w1();
		float block_var = b.get_w2();
		min_block = min(min_block, block_var); max_block = max(max_block, block_var);
		min_est = min(min_est, est_var); max_est = max(max_est, est_var);
	}

	//visualizeWeights << <dim3(nx, ny), dim3(cBlock, cBlock) >> > (m_blockInfo.getDevicePtr(), m_sBlockInfo.getDevicePtr(), varBuffer, *img, tracer->getSplatScale(), getTotalBlocksXDim(), min_block, max_block, min_est, max_est);

	std::sort(std::begin(m_indices), std::end(m_indices), [&](int i1, int i2)
	{
		return m_blockInfo[i1].getWeight(min_block, max_block, min_est, max_est) * math::sqr(m_userWeights[i1]) > m_blockInfo[i2].getWeight(min_block, max_block, min_est, max_est) * math::sqr(m_userWeights[i2]);
	});

	IUserPreferenceSampler::AddPass(img, tracer, varBuffer);
}

void VarianceBlockSampler::IterateBlocks(iterate_blocks_clb_t clb) const
{
	if(m_uPassesDone < 10)
		IterateAllBlocksUniform(clb);
	else MixedBlockIterate(m_indices, clb, m_uPassesDone);
}

}