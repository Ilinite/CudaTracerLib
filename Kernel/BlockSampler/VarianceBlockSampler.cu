#include "hip/hip_runtime.h"
#include "VarianceBlockSampler.h"
#include <Kernel/Tracer.h>

namespace CudaTracerLib
{

CUDA_GLOBAL void updateInfo(VarianceBlockSampler::TmpBlockInfo* a_pTmpBlockInfoDevice, IBlockSampler::BlockInfo* a_pPersBlockInfoDevice, const PixelVarianceBuffer varBuffer, Image img, float splatScale, unsigned int numTotalBlocksX)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x, y = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int b_x = x / BLOCK_SAMPLER_BlockSize, b_y = y / BLOCK_SAMPLER_BlockSize, bIdx = b_y * numTotalBlocksX + b_x;

	if (x < img.getWidth() && y < img.getHeight())
	{
		auto num_passes_block = a_pPersBlockInfoDevice[bIdx].passesDone;

		auto I_N = img.getPixelData(x, y).toSpectrum(splatScale);

		auto pInfo = varBuffer(x, y);
		auto var = pInfo.I.Var(pInfo.numSamples);
		auto e = pInfo.I.E(pInfo.numSamples);

		auto& bInfo = a_pTmpBlockInfoDevice[bIdx];
		if (var >= 0 && !math::IsNaN(var))
		{
			atomicAdd(&bInfo.BLOCK_VAR_I, var);
			atomicInc(&bInfo.NUM_PIXELS_VAR, 0xffffffff);
		}
		atomicAdd(&bInfo.BLOCK_E_I, e);
		atomicAdd(&bInfo.BLOCK_E_I2, e * e);
		atomicInc(&bInfo.NUM_PIXELS_E, 0xffffffff);
	}
}

void VarianceBlockSampler::StartNewRendering(DynamicScene* a_Scene, Image* img)
{
	IUserPreferenceSampler::StartNewRendering(a_Scene, img);
	m_uPassesDone = 0;
}

void VarianceBlockSampler::AddPass(Image* img, TracerBase* tracer, const PixelVarianceBuffer& varBuffer)
{
	m_uPassesDone++;

	const int cBlock = 32;
	int nx = (img->getWidth() + cBlock - 1) / cBlock, ny = (img->getHeight() + cBlock - 1) / cBlock;

	m_blockInfo.Memset(0);
	updateInfo << <dim3(nx, ny), dim3(cBlock, cBlock) >> > (m_blockInfo.getDevicePtr(), m_sBlockInfo.getDevicePtr(), varBuffer, *img, tracer->getSplatScale(), getTotalBlocksXDim());
	m_blockInfo.setOnGPU();
	m_blockInfo.Synchronize();

	float min_block = FLT_MAX, max_block = -FLT_MAX;
	float min_est = FLT_MAX, max_est = -FLT_MAX;
	for (unsigned int i = 0; i < m_blockInfo.getLength(); i++)
	{
		auto& b = m_blockInfo[i];
		float est_var = b.get_w1();
		float block_var = b.get_w2();
		min_block = min(min_block, block_var); max_block = max(max_block, block_var);
		min_est = min(min_est, est_var); max_est = max(max_est, est_var);
	}

	std::sort(std::begin(m_indices), std::end(m_indices), [&](int i1, int i2)
	{
		return m_blockInfo[i1].getWeight(min_block, max_block, min_est, max_est) * math::sqr(m_userWeights[i1]) > m_blockInfo[i2].getWeight(min_block, max_block, min_est, max_est) * math::sqr(m_userWeights[i2]);
	});

	IUserPreferenceSampler::AddPass(img, tracer, varBuffer);
}

void VarianceBlockSampler::IterateBlocks(iterate_blocks_clb_t clb)
{
	if(m_uPassesDone < 10)
		IterateAllBlocksUniform(clb);
	else
	{
		for (int i = 0; i < getNumTotalBlocks() / 4; i++)
		{
			auto flattened_idx = m_indices[i];
			int block_x, block_y, x, y, bw, bh;
			getIdxComponents(flattened_idx, block_x, block_y);

			getBlockRect(block_x, block_y, x, y, bw, bh);

			clb(flattened_idx, x, y, bw, bh);
		}

		int num_deterministic = 2;
		int start_deterministic = m_uPassesDone % num_deterministic;//deterministically sample the same number of blocks every n passes
		for (int i = start_deterministic; i < getNumTotalBlocks(); i += num_deterministic)
		{
			int block_x, block_y, x, y, bw, bh;
			getIdxComponents(i, block_x, block_y);

			getBlockRect(block_x, block_y, x, y, bw, bh);

			clb(i, x, y, bw, bh);
		}
	}
}

}