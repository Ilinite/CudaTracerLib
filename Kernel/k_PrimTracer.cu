#include "hip/hip_runtime.h"
#include "k_PrimTracer.h"
#include "k_TraceHelper.h"
#include "k_IntegrateHelper.h"

CUDA_ONLY_FUNC float3 trace(Ray& r, CudaRNG& rng)
{
	MapParameters pm(float3(), make_float2(0.5f + atan2(r.direction.z, r.direction.x) * 0.5f * INV_PI, 0.5f - asin(r.direction.y) * INV_PI), Onb());
	TraceResult r2;
	r2.Init();
	float3 c = make_float3(1);
	unsigned int depth = 0;
	e_KernelBSDF bsdf;
	while(k_TraceRay<true>(r.direction, r.origin, &r2) && depth++ < 5)
	{
		if(g_SceneData.m_sVolume.HasVolumes())
			c = c * exp(-g_SceneData.m_sVolume.tau(r, 0, r2.m_fDist));
		float3 wi;
		float pdf;/*
		r2.GetBSDF(r(r2.m_fDist), g_SceneData.m_sMatData.Data, &bsdf);
		//((float3*)&bsdf.sys.m_tangent) = cross(bsdf.sys.m_binormal, bsdf.sys.m_normal);
		//return make_float3(dot(-r.direction, bsdf.sys.m_normal));
		BxDFType sampledType;
		float3 f = bsdf.Sample_f(-r.direction, &wi, BSDFSample(rng), &pdf, BSDF_ALL, &sampledType);
		f = f * AbsDot(wi, bsdf.sys.m_normal) / pdf;
		c = c * f;
		if((sampledType & BSDF_SPECULAR) != BSDF_SPECULAR)
			break;
		r.origin = r(r2.m_fDist);
		r.direction = wi;
		r2.Init();*/
	}
	if(r2.hasHit())
	{
		return c;// * UniformSampleAllLights(r(r2.m_fDist), bsdf.sys.m_normal, -r.direction, &bsdf, rng, 1);
	}
	else if(g_SceneData.m_sEnvMap.CanSample())
		c = c * g_SceneData.m_sEnvMap.Sample(r);
		//c = g_SceneData.m_sEnvMap.Map.Evaluate(pm);
	else c = make_float3(0);
	return c;
}

__global__ void primaryKernel(long long width, long long height, e_Image I)
{
	CudaRNG rng = g_RNGData();
	int rayidx;
	int N = width * height;
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        const bool          terminated     = 1;//nodeAddr == EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));	

        if(terminated)
        {			
            if (idxTerminated == 0)
				rayBase = atomicAdd(&g_NextRayCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
			if (rayidx >= N)
                break;
		}
		unsigned int x = rayidx % width, y = rayidx / width;
		
		float3 c = make_float3(0);
		float N2 = 1;
		for(float f = 0; f < N2; f++)
		{
			CameraSample s = nextSample(x, y, rng);
			Ray r = g_CameraData.GenRay(s, width, height);
			c += trace(r, rng);
		}
		I.SetSampleDirect(nextSample(x, y, rng), c / N2);
		
		//Ray r = g_CameraData.GenRay(x, y, width, height, rng.randomFloat(), rng.randomFloat());
		//TraceResult r2 = k_TraceRay(r);
		//float3 c = make_float3(r2.m_fDist/length(g_SceneData.m_sBox.Size())*2.0f);
	}
	while(true);
	g_RNGData(rng);
}

__global__ void debugPixe2l(unsigned int width, unsigned int height, int2 p)
{
	Ray r = g_CameraData.GenRay(p, make_int2(width, height));
	//dir = make_float3(-0.98181188f, 0.18984018f, -0.0024534566f);
	//ori = make_float3(68790.375f, -12297.199f, 57510.383f);
	//ori += make_float3(g_SceneData.m_sTerrain.m_sMin.x, 0, g_SceneData.m_sTerrain.m_sMin.z);
	trace(r, g_RNGData());
}

static bool init = false;
void k_PrimTracer::DoRender(e_Image* I)
{
	if(!init)
	{
		init = true;
		cudaThreadSetLimit(hipLimitStackSize, 2048);
	}
	m_sRngs.m_uOffset++;
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, m_sRngs);
	primaryKernel<<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, *I);
	hipError_t r = hipDeviceSynchronize();
	m_uRaysTraced = w * h;
	m_uPassesDone = 1;
	I->UpdateDisplay();
}

void k_PrimTracer::Debug(int2 pixel)
{
	m_pScene->UpdateInvalidated();
	e_KernelDynamicScene d2 = m_pScene->getKernelSceneData();
	k_INITIALIZE(d2);
	k_STARTPASS(m_pScene, m_pCamera, m_sRngs);
	debugPixe2l<<<1,1>>>(w,h,pixel);
}

void k_PrimTracer::CreateSliders(SliderCreateCallback a_Callback)
{
	
}