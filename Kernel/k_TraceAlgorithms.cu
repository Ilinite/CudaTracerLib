#include "hip/hip_runtime.h"
#include "k_TraceAlgorithms.h"

Spectrum EstimateDirect(BSDFSamplingRecord& bRec, const e_KernelMaterial& mat, const e_KernelLight* light, unsigned int li, EBSDFType flags)
{
	DirectSamplingRecord dRec(bRec.map.P, bRec.map.sys.n, bRec.map.uv);
	Spectrum value = light->sampleDirect(dRec, bRec.rng->randomFloat2());
	if(!value.isZero())
	{
		bRec.wo = bRec.map.sys.toLocal(dRec.d);
		bRec.typeMask = flags;
		Spectrum bsdfVal = mat.bsdf.f(bRec);
		if (!bsdfVal.isZero() && !g_SceneData.Occluded(Ray(dRec.ref, dRec.d), 0, dRec.dist))
		{
			const float bsdfPdf = mat.bsdf.pdf(bRec);
			const float weight = MonteCarlo::PowerHeuristic(1, dRec.pdf, 1, bsdfPdf);
			return value * bsdfVal * weight;
		}
	}
	bRec.typeMask = EAll;
	return 0.0f;
	/*
	Spectrum Ld = make_float3(0.0f);
	float lightPdf, bsdfPdf;
	DirectSamplingRecord dRec(bRec.map.P, bRec.map.sys.n, bRec.map.uv);
	Spectrum Li = light->sampleDirect(dRec, bRec.rng->randomFloat2());
	lightPdf = dRec.pdf;
	if(lightPdf > 0.0f && !Li.isZero())
	{
		bRec.wo = bRec.map.sys.toLocal(dRec.d);
		Spectrum f = mat.bsdf.f(bRec);
		Ray r(dRec.ref, dRec.d);
		if(!f.isZero() && !Occluded(r, 0, dRec.dist))
		{
			Li = Li * Transmittance(r, 0, dRec.dist);
			if(light->IsDeltaLight())
				Ld += f * Li * AbsDot(r.direction, bRec.map.sys.n);
			else
			{
				bRec.typeMask = flags;
				bsdfPdf = mat.bsdf.pdf(bRec);
				float weight = MonteCarlo::PowerHeuristic(1, lightPdf, 1, bsdfPdf);
				Ld += f * Li * AbsDot(r.direction, bRec.map.sys.n) * weight;
				bRec.typeMask = EAll;
			}
		}
	}
	
	if(!light->IsDeltaLight())
	{
		bRec.typeMask = flags;
		Spectrum f = mat.bsdf.sample(bRec, bRec.rng->randomFloat2());
		float3 wi = bRec.map.sys.toWorld(bRec.wo);
		if(!f.isZero() && bsdfPdf > 0.0f)
		{
			float weight = 1.0f;
			if (!(bRec.sampledType & EDelta))
			{
                if (lightPdf == 0.0f)
                    return Ld;
                weight = MonteCarlo::PowerHeuristic(1, bsdfPdf, 1, lightPdf);
            }
			Spectrum Li = make_float3(0.0f);
			TraceResult r2;
			r2.Init();
			if(k_TraceRay(wi, bRec.map.P, &r2) && r2.LightIndex() == li)
				Li = r2.Le(bRec.map.P, bRec.map.sys.n, -wi);
			else Li = light->eval(bRec.map.P, bRec.map.sys, wi);
			if(!Li.isZero())
			{
				Li = Li * Transmittance(Ray(bRec.map.P, wi), 0, r2.m_fDist);
				Ld += Li * f * AbsDot(wi, bRec.map.sys.n) * weight;
			}
		}
	}

	return Ld;*/
}

Spectrum UniformSampleAllLights(BSDFSamplingRecord& bRec, const e_KernelMaterial& mat, int nSamples)
{
	//only sample the relevant lights and assume the others emit the same
	Spectrum L = Spectrum(0.0f);
	for(unsigned int i = 0; i < g_SceneData.m_uEmitterCount; i++)
	{
		unsigned int l = g_SceneData.m_uEmitterIndices[i];
		e_KernelLight* light = g_SceneData.m_sLightData.Data + l;
		Spectrum Ld = Spectrum(0.0f);
		for(int j = 0; j < nSamples; j++)
		{
			Ld += EstimateDirect(bRec, mat, light, l, EBSDFType(EAll & ~EDelta));
		}
		L += Ld / float(nSamples);
	}
	return L * float(g_SceneData.m_sLightData.UsedCount) / float(g_SceneData.m_uEmitterCount);
}

Spectrum UniformSampleOneLight(BSDFSamplingRecord& bRec, const e_KernelMaterial& mat)
{
	if(!g_SceneData.m_uEmitterCount)
		return 0.0f;
	float emitpdf;
	unsigned int index = g_SceneData.m_uEmitterIndices[g_SceneData.m_emitterPDF.SampleDiscrete(bRec.rng->randomFloat(), &emitpdf)];
	return float(g_SceneData.m_sLightData.UsedCount) * EstimateDirect(bRec, mat, g_SceneData.m_sLightData.Data + index, index, EBSDFType(EAll & ~EDelta));
}

Spectrum PathTrace(float3& a_Dir, float3& a_Ori, CudaRNG& rnd, float* distTravalled)
{
	const bool DIRECT = 1;
	Ray r0 = Ray(a_Ori, a_Dir);
	TraceResult r;
	r.Init(true);
	Spectrum cl = Spectrum(0.0f);   // accumulated color
	Spectrum cf = Spectrum(1.0f);  // accumulated reflectance
	int depth = 0;
	bool specularBounce = false;
	BSDFSamplingRecord bRec;
	while (k_TraceRay(r0.direction, r0.origin, &r) && depth++ < 7)
	{
		if(distTravalled && depth == 1)
			*distTravalled = r.m_fDist;
		r.getBsdfSample(r0, rnd, &bRec);// return (Spectrum(bRec.map.sys.n) + Spectrum(1)) / 2.0f;return bRec.map.sys.n;
		if(!DIRECT || (depth == 1 || specularBounce))
			cl += cf * r.Le(r0(r.m_fDist), bRec.map.sys, -r0.direction);
		if(DIRECT)
			cl += cf * UniformSampleAllLights(bRec, r.getMat(), 1);
		Spectrum f = r.getMat().bsdf.sample(bRec, rnd.randomFloat2());
		specularBounce = (bRec.sampledType & EDelta) != 0;
		float p = f.max(); 
		if (depth > 5)
			if (rnd.randomFloat() < p)
				f = f / p;
			else break;
		if(f.isZero())
			break;
		cf = cf * f;
		r0 = Ray(r0(r.m_fDist), bRec.getOutgoing());
		r.Init();
	}
	if(!r.hasHit())
		cl += cf * g_SceneData.EvalEnvironment(r0);
	return cl;
}