#include "hip/hip_runtime.h"
#include "k_TraceAlgorithms.h"

//#define EXT_EST

CUDA_FUNC_IN float G(const float3& N_x, const float3& N_y, const float3& x, const float3& y)
{
	float3 theta = normalize(y - x);
	return AbsDot(N_x, theta) * AbsDot(N_y, -theta) / DistanceSquared(x, y);
}

CUDA_FUNC_IN Spectrum EstimateDirect(BSDFSamplingRecord bRec, const e_KernelMaterial& mat, const e_KernelLight* light, unsigned int li, EBSDFType flags)
{
#ifndef EXT_EST
	DirectSamplingRecord dRec(bRec.map.P, bRec.map.sys.n);
	Spectrum value = light->sampleDirect(dRec, bRec.rng->randomFloat2());
	Spectrum retVal(0.0f);
	if(!value.isZero())
	{
		float3 oldWo = bRec.wo;
		bRec.wo = normalize(bRec.map.sys.toLocal(dRec.d));
		bRec.typeMask = flags;
		Spectrum bsdfVal = mat.bsdf.f(bRec);
		if (!bsdfVal.isZero() && !g_SceneData.Occluded(Ray(dRec.ref, dRec.d), 0, dRec.dist))
		{
			const float bsdfPdf = mat.bsdf.pdf(bRec);
			const float weight = MonteCarlo::PowerHeuristic(1, dRec.pdf, 1, bsdfPdf);
			retVal = value * bsdfVal * weight;
		}
		bRec.typeMask = EAll;
		bRec.wo = oldWo;
	}
	return retVal;
#else	
	Spectrum Ld = make_float3(0.0f);
	float lightPdf, bsdfPdf;
	DirectSamplingRecord dRec(bRec.map.P, bRec.map.sys.n);
	Spectrum Li = light->sampleDirect(dRec, bRec.rng->randomFloat2());
	lightPdf = dRec.pdf;
	if(lightPdf > 0.0f && !Li.isZero())
	{
		bRec.wo = bRec.map.sys.toLocal(dRec.d);
		Spectrum f = mat.bsdf.f(bRec);
		Ray r(dRec.ref, dRec.d);
		if(!f.isZero() && !g_SceneData.Occluded(r, 0, dRec.dist))
		{
			Li = Li * Transmittance(r, 0, dRec.dist);
			if(light->IsDeltaLight())
				Ld += f * Li * AbsDot(r.direction, bRec.map.sys.n);
			else
			{
				bRec.typeMask = flags;
				bsdfPdf = mat.bsdf.pdf(bRec);
				float weight = MonteCarlo::PowerHeuristic(1, lightPdf, 1, bsdfPdf);
				Ld += f * Li * AbsDot(r.direction, bRec.map.sys.n) * weight;
				bRec.typeMask = EAll;
			}
		}
	}
	
	if(!light->IsDeltaLight())
	{
		bRec.typeMask = flags;
		Spectrum f = mat.bsdf.sample(bRec, bRec.rng->randomFloat2());
		float3 wi = bRec.map.sys.toWorld(bRec.wo);
		if(!f.isZero() && bsdfPdf > 0.0f)
		{
			float weight = 1.0f;
			if (!(bRec.sampledType & EDelta))
			{
                if (lightPdf == 0.0f)
                    return Ld;
                weight = MonteCarlo::PowerHeuristic(1, bsdfPdf, 1, lightPdf);
            }
			Spectrum Li = make_float3(0.0f);
			TraceResult r2;
			r2.Init();
			if(k_TraceRay(wi, bRec.map.P, &r2) && r2.LightIndex() == li)
				Li = r2.Le(bRec.map.P, bRec.map.sys.n, -wi);
			else Li = light->eval(bRec.map.P, bRec.map.sys, wi);
			if(!Li.isZero())
			{
				Li = Li * Transmittance(Ray(bRec.map.P, wi), 0, r2.m_fDist);
				Ld += Li * f * AbsDot(wi, bRec.map.sys.n) * weight;
			}
		}
	}

	return Ld;
#endif
}

Spectrum UniformSampleAllLights(const BSDFSamplingRecord& bRec, const e_KernelMaterial& mat, int nSamples)
{
	//only sample the relevant lights and assume the others emit the same
	Spectrum L = Spectrum(0.0f);
	for(unsigned int i = 0; i < g_SceneData.m_uEmitterCount; i++)
	{
		unsigned int l = g_SceneData.m_uEmitterIndices[i];
		e_KernelLight* light = g_SceneData.m_sLightData.Data + l;
		if(light->As()->IsRemoved)
			continue;
		Spectrum Ld = Spectrum(0.0f);
		for(int j = 0; j < nSamples; j++)
		{
			Ld += EstimateDirect((BSDFSamplingRecord&)bRec, mat, light, l, EBSDFType(EAll & ~EDelta));
		}
		L += Ld / float(nSamples);
	}
	return L * float(g_SceneData.m_sLightData.UsedCount) / float(g_SceneData.m_uEmitterCount);
}

Spectrum UniformSampleOneLight(const BSDFSamplingRecord& bRec, const e_KernelMaterial& mat)
{
	if(!g_SceneData.m_uEmitterCount)
		return 0.0f;
	float emitpdf;
	unsigned int index = g_SceneData.m_uEmitterIndices[g_SceneData.m_emitterPDF.SampleDiscrete(bRec.rng->randomFloat(), &emitpdf)];
	return float(g_SceneData.m_sLightData.UsedCount) * EstimateDirect((BSDFSamplingRecord&)bRec, mat, g_SceneData.m_sLightData.Data + index, index, EBSDFType(EAll & ~EDelta));
}