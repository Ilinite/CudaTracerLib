#include "hip/hip_runtime.h"
#include "k_TraceHelper.h"
#include "../Math/Compression.h"
#include "../Math/half.h"
#include "hip/hip_runtime.h"
#include "../Engine/e_Sensor.h"
#include "../Engine/e_Mesh.h"
#include "../Engine/e_TriangleData.h"
#include "../Engine/e_Material.h"
#include "../Engine/e_IntersectorData.h"
#include "../Engine/e_Node.h"
#include "../Engine/e_DynamicScene.h"

//#define SKIP_OUTER_TREE

enum
{
	MaxBlockHeight = 6,            // Upper bound for blockDim.y.
	EntrypointSentinel = 0x76543210,   // Bottom-most stack entry, indicating the end of traversal.
};

e_KernelDynamicScene g_SceneDataDevice;
unsigned int g_RayTracedCounterDevice;
e_Sensor g_CameraDataDevice;
CudaRNGBuffer g_RNGDataDevice;

e_KernelDynamicScene g_SceneDataHost;
unsigned int g_RayTracedCounterHost;
e_Sensor g_CameraDataHost;
CudaRNGBuffer g_RNGDataHost;

texture<float4, 1>		t_nodesA;
texture<float4, 1>		t_tris;
texture<unsigned int,  1>		t_triIndices;
texture<float4, 1>		t_SceneNodes;
texture<float4, 1>		t_NodeTransforms;
texture<float4, 1>		t_NodeInvTransforms;

texture<int2, 1> t_TriDataA;
texture<float4, 1> t_TriDataB;

void traversalResult::toResult(TraceResult* tR, e_KernelDynamicScene& data)
{
	tR->m_fDist = dist;
	tR->m_fBaryCoords = ((half2*)&bCoords)->ToFloat2();
	tR->m_pNode = data.m_sNodeData.Data + nodeIdx;
	tR->m_pTri = data.m_sTriData.Data + triIdx;
}

CUDA_FUNC_IN void loadModl(int i, float4x4* o)
{
#ifdef ISCUDA
	float4* f = (float4*)o;
	f[0] = tex1Dfetch(t_NodeTransforms, i * 4 + 0);
	f[1] = tex1Dfetch(t_NodeTransforms, i * 4 + 1);
	f[2] = tex1Dfetch(t_NodeTransforms, i * 4 + 2);
	f[3] = tex1Dfetch(t_NodeTransforms, i * 4 + 3);
#else
	*o = g_SceneData.m_sSceneBVH.m_pNodeTransforms[i];
#endif
}

CUDA_FUNC_IN void loadInvModl(int i, float4x4* o)
{
#ifdef ISCUDA
	float4* f = (float4*)o;
	f[0] = tex1Dfetch(t_NodeInvTransforms, i * 4 + 0);
	f[1] = tex1Dfetch(t_NodeInvTransforms, i * 4 + 1);
	f[2] = tex1Dfetch(t_NodeInvTransforms, i * 4 + 2);
	f[3] = tex1Dfetch(t_NodeInvTransforms, i * 4 + 3);
#else
	*o = g_SceneData.m_sSceneBVH.m_pInvNodeTransforms[i];
#endif
}

CUDA_FUNC_IN bool k_TraceRayNode(const Vec3f& dir, const Vec3f& ori, TraceResult* a_Result, const e_Node* N)
{
	unsigned int mIndex = N->m_uMeshIndex;
	e_KernelMesh mesh = g_SceneData.m_sMeshData[mIndex];
	bool found = false;
	int traversalStack[64];
	traversalStack[0] = EntrypointSentinel;
	float   dirx = dir.x;
	float   diry = dir.y;
	float   dirz = dir.z;
	const float ooeps = math::exp2(-80.0f);
	float   idirx = 1.0f / (math::abs(dir.x) > ooeps ? dir.x : copysignf(ooeps, dir.x));
	float   idiry = 1.0f / (math::abs(dir.y) > ooeps ? dir.y : copysignf(ooeps, dir.y));
	float   idirz = 1.0f / (math::abs(dir.z) > ooeps ? dir.z : copysignf(ooeps, dir.z));
	float   origx = ori.x;
	float	origy = ori.y;
	float	origz = ori.z;						// Ray origin.
	float   oodx = origx * idirx;
	float   oody = origy * idiry;
	float   oodz = origz * idirz;
	char*   stackPtr;                       // Current position in traversal stack.
	int     leafAddr;                       // First postponed leaf, non-negative if none.
	int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf.
			stackPtr = (char*)&traversalStack[0];
			leafAddr = 0;   // No postponed leaf.
			nodeAddr = 0;   // Start from the root.
	while(nodeAddr != EntrypointSentinel)
	{
		while (unsigned int(nodeAddr) < unsigned int(EntrypointSentinel))
		{
#ifdef ISCUDA
			const float4 n0xy = tex1Dfetch(t_nodesA, mesh.m_uBVHNodeOffset + nodeAddr + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
			const float4 n1xy = tex1Dfetch(t_nodesA, mesh.m_uBVHNodeOffset + nodeAddr + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
			const float4 nz   = tex1Dfetch(t_nodesA, mesh.m_uBVHNodeOffset + nodeAddr + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
				  float4 tmp  = tex1Dfetch(t_nodesA, mesh.m_uBVHNodeOffset + nodeAddr + 3); // child_index0, child_index1
#else
			Vec4f* dat = (Vec4f*)g_SceneData.m_sBVHNodeData.Data;
			const Vec4f n0xy = dat[mesh.m_uBVHNodeOffset + nodeAddr + 0];
			const Vec4f n1xy = dat[mesh.m_uBVHNodeOffset + nodeAddr + 1];
			const Vec4f nz   = dat[mesh.m_uBVHNodeOffset + nodeAddr + 2];
				  Vec4f tmp  = dat[mesh.m_uBVHNodeOffset + nodeAddr + 3];
#endif
				  Vec2i  cnodes = *(Vec2i*)&tmp;
			const float c0lox = n0xy.x * idirx - oodx;
			const float c0hix = n0xy.y * idirx - oodx;
			const float c0loy = n0xy.z * idiry - oody;
			const float c0hiy = n0xy.w * idiry - oody;
			const float c0loz = nz.x   * idirz - oodz;
			const float c0hiz = nz.y   * idirz - oodz;
			const float c1loz = nz.z   * idirz - oodz;
			const float c1hiz = nz.w   * idirz - oodz;
			const float c0min = math::spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, 0);
			const float c0max = math::spanEndKepler  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, a_Result->m_fDist);
			const float c1lox = n1xy.x * idirx - oodx;
			const float c1hix = n1xy.y * idirx - oodx;
			const float c1loy = n1xy.z * idiry - oody;
			const float c1hiy = n1xy.w * idiry - oody;
			const float c1min = math::spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, 0);
			const float c1max = math::spanEndKepler  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, a_Result->m_fDist);
			bool swp = (c1min < c0min);
			bool traverseChild0 = (c0max >= c0min);
			bool traverseChild1 = (c1max >= c1min);
			if (!traverseChild0 && !traverseChild1)
			{
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}
			else
			{
				nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;
				if (traverseChild0 && traverseChild1)
				{
					if (swp)
						swapk(&nodeAddr, &cnodes.y);
					stackPtr += 4;
					*(int*)stackPtr = cnodes.y;
				}
			}

			if (nodeAddr < 0 && leafAddr  >= 0)     // Postpone max 1
			{
				leafAddr = nodeAddr;
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}

#ifdef ISCUDA
            unsigned int mask;
            asm("{\n"
                "   .reg .pred p;               \n"
                "setp.ge.s32        p, %1, 0;   \n"
                "vote.ballot.b32    %0,p;       \n"
                "}"
                : "=r"(mask)
                : "r"(leafAddr));
#else
			unsigned int mask = leafAddr >= 0;
#endif
			if(!mask)
				break;
		}
		while (leafAddr < 0)
		{
			if (leafAddr != -214783648)
			{
				for (int triAddr = ~leafAddr;; triAddr++)
				{
#ifdef ISCUDA
					const float4 v00 = tex1Dfetch(t_tris, mesh.m_uBVHTriangleOffset + triAddr * 3 + 0);
					const float4 v11 = tex1Dfetch(t_tris, mesh.m_uBVHTriangleOffset + triAddr * 3 + 1);
					const float4 v22 = tex1Dfetch(t_tris, mesh.m_uBVHTriangleOffset + triAddr * 3 + 2);
					unsigned int index = tex1Dfetch(t_triIndices, mesh.m_uBVHIndicesOffset + triAddr);
#else
					Vec4f* dat = (Vec4f*)g_SceneData.m_sBVHIntData.Data;
					const Vec4f v00 = dat[mesh.m_uBVHTriangleOffset + triAddr * 3 + 0];
					const Vec4f v11 = dat[mesh.m_uBVHTriangleOffset + triAddr * 3 + 1];
					const Vec4f v22 = dat[mesh.m_uBVHTriangleOffset + triAddr * 3 + 2];
					unsigned int index = g_SceneData.m_sBVHIndexData.Data[mesh.m_uBVHIndicesOffset + triAddr].index;
#endif

					float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;
					float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);
					float t = Oz * invDz;
					if (t > 1e-2f && t < a_Result->m_fDist)
					{
						float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;
						float Dx = dirx*v11.x + diry*v11.y + dirz*v11.z;
						float u = Ox + t*Dx;
						if (u >= 0.0f)
						{
							float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
							float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
							float v = Oy + t*Dy;
							if (v >= 0.0f && u + v <= 1.0f)
							{
								unsigned int ti = index >> 1;
								e_TriangleData* tri = g_SceneData.m_sTriData.Data + ti + mesh.m_uTriangleOffset;
								int q = 1;
								/*if (USE_ALPHA)
								{
									e_KernelMaterial* mat = g_SceneData.m_sMatData.Data + tri->getMatIndex(N->m_uMaterialOffset);
									DifferentialGeometry dg;
									dg.bary = make_float2(u, v);
									for (int i = 0; i < NUM_UV_SETS; i++)
										dg.uv[i] = tri->math::lerpUV(i, dg.bary);
									float a = mat->SampleAlphaMap(dg);
									q = a >= mat->m_fAlphaThreshold;

								}*/
								if (q)
								{
									a_Result->m_pNode = N;
									a_Result->m_pTri = tri;
									a_Result->m_fBaryCoords = Vec2f(u, v);
									a_Result->m_fDist = t;
									found = true;
								}
							}
						}
					}
					if (index & 1)
						break;
				}
			}
			leafAddr = nodeAddr;
			if (nodeAddr < 0)
			{
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}
		}
	}
	return found;
}

bool k_TraceRay(const Vec3f& dir, const Vec3f& ori, TraceResult* a_Result)
{
	Platform::Increment(&g_RayTracedCounter);
	if(!g_SceneData.m_sNodeData.UsedCount)
		return false;
#ifdef SKIP_OUTER_TREE
	const int node = 0;
	e_Node* N = g_SceneData.m_sNodeData.Data + node;
	//transform a_Result->m_fDist to local system
	float4x4 modl;
	loadInvModl(node, &modl);
	Vec3f d = modl.TransformDirection(dir), o = modl.TransformPoint(ori);
	k_TraceRayNode(d, o, a_Result, N);
#else
	int traversalStackOuter[64];
	int at = 1;
	traversalStackOuter[0] = g_SceneData.m_sSceneBVH.m_sStartNode;
	const float ooeps = math::exp2(-80.0f);
	Vec3f O, I;
	I.x = 1.0f / (math::abs(dir.x) > ooeps ? dir.x : copysignf(ooeps, dir.x));
	I.y = 1.0f / (math::abs(dir.y) > ooeps ? dir.y : copysignf(ooeps, dir.y));
	I.z = 1.0f / (math::abs(dir.z) > ooeps ? dir.z : copysignf(ooeps, dir.z));
	O = I * ori;
	while(at)
	{
		int nodeAddrOuter = traversalStackOuter[--at];
		while (nodeAddrOuter >= 0 && nodeAddrOuter != EntrypointSentinel)
		{
#ifdef ISCUDA
			const float4 n0xy = tex1Dfetch(t_SceneNodes, nodeAddrOuter + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
			const float4 n1xy = tex1Dfetch(t_SceneNodes, nodeAddrOuter + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
			const float4 nz   = tex1Dfetch(t_SceneNodes, nodeAddrOuter + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
				  float4 tmp  = tex1Dfetch(t_SceneNodes, nodeAddrOuter + 3); // child_index0, child_index1
#else
			const Vec4f n0xy = g_SceneData.m_sSceneBVH.m_pNodes[nodeAddrOuter / 4].a;
			const Vec4f n1xy = g_SceneData.m_sSceneBVH.m_pNodes[nodeAddrOuter / 4].b;
			const Vec4f nz   = g_SceneData.m_sSceneBVH.m_pNodes[nodeAddrOuter / 4].c;
				  Vec4f tmp  = g_SceneData.m_sSceneBVH.m_pNodes[nodeAddrOuter / 4].d;
#endif
			Vec2i  cnodesOuter = *(Vec2i*)&tmp;
			const float c0lox = n0xy.x * I.x - O.x;
			const float c0hix = n0xy.y * I.x - O.x;
			const float c0loy = n0xy.z * I.y - O.y;
			const float c0hiy = n0xy.w * I.y - O.y;
			const float c0loz = nz.x   * I.z - O.z;
			const float c0hiz = nz.y   * I.z - O.z;
			const float c1loz = nz.z   * I.z - O.z;
			const float c1hiz = nz.w   * I.z - O.z;
			const float c0min = math::spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, 0);
			const float c0max = math::spanEndKepler  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, a_Result->m_fDist);
			const float c1lox = n1xy.x * I.x - O.x;
			const float c1hix = n1xy.y * I.x - O.x;
			const float c1loy = n1xy.z * I.y - O.y;
			const float c1hiy = n1xy.w * I.y - O.y;
			const float c1min = math::spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, 0);
			const float c1max = math::spanEndKepler  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, a_Result->m_fDist);
			bool swpOuter = (c1min < c0min);
			bool traverseChild0Outer = (c0max >= c0min);
			bool traverseChild1Outer = (c1max >= c1min);
			if ((!traverseChild0Outer && !traverseChild1Outer) && at)
				nodeAddrOuter = traversalStackOuter[--at];
			else if(!traverseChild0Outer && !traverseChild1Outer)
			{//empty stack and nowhere to go...
				nodeAddrOuter = 0;
				break;
			}
			else
			{
				nodeAddrOuter = (traverseChild0Outer) ? cnodesOuter.x : cnodesOuter.y;
				if (traverseChild0Outer && traverseChild1Outer)
				{
					if (swpOuter)
						swapk(&nodeAddrOuter, &cnodesOuter.y);
					traversalStackOuter[at++] = cnodesOuter.y;
				}
			}
		}
		if(nodeAddrOuter < 0 && nodeAddrOuter != -214783648)
		{
			int node = ~nodeAddrOuter;
			e_Node* N = g_SceneData.m_sNodeData.Data + node;
			//transform a_Result->m_fDist to local system
			float4x4 modl, modl2;
			loadInvModl(node, &modl);
			loadModl(node, &modl2);
			Vec3f d = modl.TransformDirection(dir), o = modl.TransformPoint(ori);
			k_TraceRayNode(d, o, a_Result, N);
		}
	}
#endif
	return a_Result->hasHit();
}

void k_INITIALIZE(e_DynamicScene* a_Scene, const CudaRNGBuffer& a_RngBuf)
{
	if (!a_Scene)
		return;

	e_KernelDynamicScene a_Data = a_Scene->getKernelSceneData();

	size_t offset;
	hipChannelFormatDesc	cdf4 = hipCreateChannelDesc<float4>(),
							cdu1 = hipCreateChannelDesc<unsigned int>(),
							cdi2 = hipCreateChannelDesc<int2>(),
							cdh4 = cudaCreateChannelDescHalf4();
	hipError_t
	r = hipBindTexture(&offset, &t_nodesA, a_Data.m_sBVHNodeData.Data, &cdf4, a_Data.m_sBVHNodeData.UsedCount * sizeof(e_BVHNodeData));
	r = hipBindTexture(&offset, &t_tris, a_Data.m_sBVHIntData.Data, &cdf4, a_Data.m_sBVHIntData.UsedCount * sizeof(e_TriIntersectorData));
	r = hipBindTexture(&offset, &t_triIndices, a_Data.m_sBVHIndexData.Data, &cdu1, a_Data.m_sBVHIndexData.UsedCount * sizeof(e_TriIntersectorData2));
	r = hipBindTexture(&offset, &t_SceneNodes, a_Data.m_sSceneBVH.m_pNodes, &cdf4, a_Data.m_sSceneBVH.m_uNumNodes * sizeof(e_BVHNodeData));
	r = hipBindTexture(&offset, &t_NodeTransforms, a_Data.m_sSceneBVH.m_pNodeTransforms, &cdf4, a_Data.m_sNodeData.UsedCount * sizeof(float4x4));
	r = hipBindTexture(&offset, &t_NodeInvTransforms, a_Data.m_sSceneBVH.m_pInvNodeTransforms, &cdf4, a_Data.m_sNodeData.UsedCount * sizeof(float4x4));

	r = hipBindTexture(&offset, &t_TriDataA, a_Data.m_sTriData.Data, &cdi2, a_Data.m_sTriData.UsedCount * sizeof(e_TriangleData));
	r = hipBindTexture(&offset, &t_TriDataB, a_Data.m_sTriData.Data, &cdh4, a_Data.m_sTriData.UsedCount * sizeof(e_TriangleData));

	unsigned int b = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(g_RayTracedCounterDevice), &b, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(g_SceneDataDevice), &a_Data, sizeof(e_KernelDynamicScene));
	hipMemcpyToSymbol(HIP_SYMBOL(g_RNGDataDevice), &a_RngBuf, sizeof(CudaRNGBuffer));

	g_SceneDataHost = a_Scene->getKernelSceneData(false);
	g_RNGDataHost = a_RngBuf;
	g_RayTracedCounterHost = 0;
}

void fillDG(const Vec2f& bary, const e_TriangleData* tri, const e_Node* node, DifferentialGeometry& dg)
{
	float4x4 localToWorld, worldToLocal;
	loadModl(node - g_SceneData.m_sNodeData.Data, &localToWorld);
	loadInvModl(node - g_SceneData.m_sNodeData.Data, &worldToLocal);
	dg.bary = bary;
	dg.hasUVPartials = false;
#if defined(ISCUDA) && NUM_UV_SETS == 1
	unsigned int i = tri - g_SceneData.m_sTriData.Data;
	int2 nme = tex1Dfetch(t_TriDataA, i * 4 + 0);
	float4 rowB = tex1Dfetch(t_TriDataB, i * 4 + 1);
	float4 rowC = tex1Dfetch(t_TriDataB, i * 4 + 2);
	float4 rowD = tex1Dfetch(t_TriDataB, i * 4 + 3);
	Vec3f na = Uchar2ToNormalizedFloat3(nme.x), nb = Uchar2ToNormalizedFloat3(nme.x >> 16), nc = Uchar2ToNormalizedFloat3(nme.y);
	float w = 1.0f - dg.bary.x - dg.bary.y, u = dg.bary.x, v = dg.bary.y;
	dg.extraData = nme.y >> 24;
	dg.sys.n = u * na + v * nb + w * nc;
	Vec3f dpdu = Vec3f(rowB.x, rowB.y, rowB.z);
	Vec3f dpdv = Vec3f(rowB.z, rowC.x, rowC.y);
	dg.sys.s = dpdu - dg.sys.n * dot(dg.sys.n, dpdu);
	dg.sys.t = cross(dg.sys.s, dg.sys.n);
	dg.sys = dg.sys * localToWorld;
	dg.n = normalize(worldToLocal.TransformTranspose(Vec4f(na + nb + nc, 0.0f)).getXYZ());
	dg.dpdu = localToWorld.TransformDirection(dpdu);
	dg.dpdv = localToWorld.TransformDirection(dpdv);
	Vec2f ta = Vec2f(rowC.z, rowC.w), tb = Vec2f(rowD.x, rowD.y), tc = Vec2f(rowD.z, rowD.w);
	dg.uv[0] = u * ta + v * tb + w * tc;

	if (dot(dg.n, dg.sys.n) < 0.0f)
		dg.n = -dg.n;
#else
	tri->fillDG(localToWorld, worldToLocal, dg);
#endif
}

unsigned int k_getNumRaysTraced()
{
	unsigned int i;
	hipMemcpyFromSymbol(&i, HIP_SYMBOL(g_RayTracedCounterDevice), sizeof(unsigned int));
	return i + g_RayTracedCounterHost;
}

void k_setNumRaysTraced(unsigned int i)
{
	g_RayTracedCounterHost = i;
	hipMemcpyToSymbol(HIP_SYMBOL(g_RayTracedCounterDevice), &i, sizeof(unsigned int));
}

#define DYNAMIC_FETCH_THRESHOLD 20
#define STACK_SIZE 32
__device__ int g_warpCounter;
__device__ __inline__ int   min_min2   (int a, int b, int c) { int v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   min_max2   (int a, int b, int c) { int v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_min2   (int a, int b, int c) { int v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_max2   (int a, int b, int c) { int v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ float fmin_fmin2 (float a, float b, float c) { return __int_as_float(min_min2(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax2 (float a, float b, float c) { return __int_as_float(min_max2(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin2 (float a, float b, float c) { return __int_as_float(max_min2(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax2 (float a, float b, float c) { return __int_as_float(max_max2(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float spanBeginKepler2(float a0, float a1, float b0, float b1, float c0, float c1, float d){	return fmax_fmax2( min(a0,a1), min(b0,b1), fmin_fmax2(c0, c1, d)); }
__device__ __inline__ float spanEndKepler2(float a0, float a1, float b0, float b1, float c0, float c1, float d)	{	return fmin_fmin2( max(a0,a1), max(b0,b1), fmax_fmin2(c0, c1, d)); }

template<bool ANY_HIT> __global__ void intersectKernel_SKIPOUTER(int numRays, traversalRay* a_RayBuffer, traversalResult* a_ResBuffer)
{
    int traversalStack[STACK_SIZE];
    traversalStack[0] = EntrypointSentinel; // Bottom-most entry.

    // Live state during traversal, stored in registers.

    float   origx, origy, origz;            // Ray origin.
    char*   stackPtr;                       // Current position in traversal stack.
    int     leafAddr;                       // First postponed leaf, non-negative if none.
    int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf.
    int     hitIndex;                       // Triangle index of the closest intersection, -1 if none.
    float   hitT;                           // t-value of the closest intersection.
    float   tmin;
    int     rayidx;
    float   oodx;
    float   oody;
    float   oodz;
    float   dirx;
    float   diry;
    float   dirz;
    float   idirx;
    float   idiry;
    float   idirz;
	Vec2f bCoords;

    // Initialize persistent threads.

    __shared__ volatile int nextRayArray[MaxBlockHeight]; // Current ray index in global buffer.

    // Persistent threads: fetch and process rays in a loop.

    do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        // Fetch new rays from the global pool using lane 0.

        const bool          terminated     = nodeAddr==EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));

        if(terminated)
        {
            if (idxTerminated == 0)
                rayBase = atomicAdd(&g_warpCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
            if (rayidx >= numRays)
                break;

            // Fetch ray.

			float4 o1 = ((float4*)a_RayBuffer)[rayidx * 2 + 0];
			float4 d1 = ((float4*)a_RayBuffer)[rayidx * 2 + 1];
			//to local
			float4x4 modl;
			loadInvModl(0, &modl);
			float3 d = modl.TransformDirection(Vec3f(d1.x, d1.y, d1.z)), o = modl.TransformPoint(Vec3f(o1.x, o1.y, o1.z));

            origx = o.x;
            origy = o.y;
            origz = o.z;
			//tmin  = o1.w / length(d);
			tmin = o1.w;
            dirx  = d.x;
            diry  = d.y;
            dirz  = d.z;
            //hitT  = d1.w / length(d);
			hitT = d1.w;
			float ooeps = math::exp2(-80.0f); // Avoid div by zero.
            idirx = 1.0f / (math::abs(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
            idiry = 1.0f / (math::abs(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
            idirz = 1.0f / (math::abs(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
            oodx  = origx * idirx;
            oody  = origy * idiry;
            oodz  = origz * idirz;

            // Setup traversal.

            stackPtr = (char*)&traversalStack[0];
            leafAddr = 0;   // No postponed leaf.
            nodeAddr = 0;   // Start from the root.
            hitIndex = -1;  // No triangle intersected so far.
		}
		
		while(nodeAddr != EntrypointSentinel)
		{
			while (unsigned int(nodeAddr) < unsigned int(EntrypointSentinel))
			{
				const float4 n0xy = tex1Dfetch(t_nodesA, nodeAddr + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
				const float4 n1xy = tex1Dfetch(t_nodesA, nodeAddr + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
				const float4 nz   = tex1Dfetch(t_nodesA, nodeAddr + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
					  float4 tmp  = tex1Dfetch(t_nodesA, nodeAddr + 3); // child_index0, child_index1
						int2  cnodes= *(int2*)&tmp;

				// Intersect the ray against the child nodes.

                const float c0lox = n0xy.x * idirx - oodx;
                const float c0hix = n0xy.y * idirx - oodx;
                const float c0loy = n0xy.z * idiry - oody;
                const float c0hiy = n0xy.w * idiry - oody;
                const float c0loz = nz.x   * idirz - oodz;
                const float c0hiz = nz.y   * idirz - oodz;
                const float c1loz = nz.z   * idirz - oodz;
                const float c1hiz = nz.w   * idirz - oodz;
                const float c0min = spanBeginKepler2(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
                const float c0max = spanEndKepler2  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
                const float c1lox = n1xy.x * idirx - oodx;
                const float c1hix = n1xy.y * idirx - oodx;
                const float c1loy = n1xy.z * idiry - oody;
                const float c1hiy = n1xy.w * idiry - oody;
                const float c1min = spanBeginKepler2(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
                const float c1max = spanEndKepler2  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

                bool swp = (c1min < c0min);

                bool traverseChild0 = (c0max >= c0min);
                bool traverseChild1 = (c1max >= c1min);

                // Neither child was intersected => pop stack.

                if (!traverseChild0 && !traverseChild1)
                {
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // Otherwise => fetch child pointers.

                else
                {
                    nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

                    // Both children were intersected => push the farther one.

                    if (traverseChild0 && traverseChild1)
                    {
                        if (swp)
                            swapk(nodeAddr, cnodes.y);
                        stackPtr += 4;
                        *(int*)stackPtr = cnodes.y;
                    }
                }

                // First leaf => postpone and continue traversal.

                if (nodeAddr < 0 && leafAddr  >= 0)     // Postpone max 1
//              if (nodeAddr < 0 && leafAddr2 >= 0)     // Postpone max 2
                {
                    //leafAddr2= leafAddr;          // postpone 2
                    leafAddr = nodeAddr;
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // All SIMD lanes have found a leaf? => process them.

                // NOTE: inline PTX implementation of "if(!__any(leafAddr >= 0)) break;".
                // tried everything with CUDA 4.2 but always got several redundant instructions.

                unsigned int mask;
                asm("{\n"
                    "   .reg .pred p;               \n"
                    "setp.ge.s32        p, %1, 0;   \n"
                    "vote.ballot.b32    %0,p;       \n"
                    "}"
                    : "=r"(mask)
                    : "r"(leafAddr));
                if(!mask)
                    break;
			}
			while (leafAddr < 0)
			{
				for (int triAddr = ~leafAddr;; triAddr++)
				{
					// Tris in TEX (good to fetch as a single batch)
					const float4 v00 = tex1Dfetch(t_tris, triAddr * 3 + 0);
					const float4 v11 = tex1Dfetch(t_tris, triAddr * 3 + 1);
					const float4 v22 = tex1Dfetch(t_tris, triAddr * 3 + 2);
					unsigned int index = tex1Dfetch(t_triIndices, triAddr);

					float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;
					float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);
					float t = Oz * invDz;

					if (t > tmin && t < hitT)
					{
						// Compute and check barycentric u.

						float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;
						float Dx = dirx*v11.x + diry*v11.y + dirz*v11.z;
						float u = Ox + t*Dx;

						if (u >= 0.0f)
						{
							// Compute and check barycentric v.

							float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
							float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
							float v = Oy + t*Dy;

							if (v >= 0.0f && u + v <= 1.0f)
							{
								// Record intersection.
								// Closest intersection not required => terminate.

								hitT = t;
								hitIndex = index >> 1;
								bCoords = Vec2f(u,v);
								if (ANY_HIT)
								{
									nodeAddr = EntrypointSentinel;
									break;
								}
							}
						}
					}
					if(index & 1)
						break;
				} // triangle

				leafAddr = nodeAddr;
				if (nodeAddr < 0)
				{
					nodeAddr = *(int*)stackPtr;
					stackPtr -= 4;
				}
			}
			if( __popc(__ballot(true)) < DYNAMIC_FETCH_THRESHOLD )
				break;
		}
		Vec4i res = Vec4i(0, 0, 0, 0);
		if(hitIndex != -1)
		{
			//res.x = __float_as_int(hitT * math::sqrt(dirx * dirx + diry * diry + dirz * dirz));
			res.x = __float_as_int(hitT);
			res.y = 0;
			res.z = hitIndex;
			half2 h(bCoords);
			res.w = *(int*)&h;
		}
		((int4*)a_ResBuffer)[rayidx] = res;
	} while(true);
}

template<bool ANY_HIT> __global__ void intersectKernel(int numRays, traversalRay* a_RayBuffer, traversalResult* a_ResBuffer)
{
	// Traversal stack in CUDA thread-local memory.

    int traversalStack[STACK_SIZE];
    traversalStack[0] = EntrypointSentinel; // Bottom-most entry.

    // Live state during traversal, stored in registers.

    float   origx, origy, origz;            // Ray origin.
    char*   stackPtr;                       // Current position in traversal stack.
    int     leafAddr;                       // First postponed leaf, non-negative if none.
    int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf.
    int     hitIndex;                       // Triangle index of the closest intersection, -1 if none.
    float   hitT;                           // t-value of the closest intersection.
    float   tmin;
    int     rayidx;
    float   oodx;
    float   oody;
    float   oodz;
    float   dirx;
    float   diry;
    float   dirz;
    float   idirx;
    float   idiry;
    float   idirz;
	Vec2f bCorrds;
	int nodeIdx = 0;

	int ltraversalStack[STACK_SIZE];
	ltraversalStack[0] = EntrypointSentinel;
    float   lorigx, lorigy, lorigz;
    char*   lstackPtr;
    int     lleafAddr;
    int     lnodeAddr = EntrypointSentinel;
    float   lhitT;
    float   ltmin;
    float   loodx;
    float   loody;
    float   loodz;
    float   ldirx;
    float   ldiry;
    float   ldirz;
    float   lidirx;
    float   lidiry;
    float   lidirz;

    // Initialize persistent threads.

    __shared__ volatile int nextRayArray[MaxBlockHeight]; // Current ray index in global buffer.

    // Persistent threads: fetch and process rays in a loop.

    do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        // Fetch new rays from the global pool using lane 0.

        const bool          terminated     = nodeAddr==EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));

        if(terminated)
        {
            if (idxTerminated == 0)
                rayBase = atomicAdd(&g_warpCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
            if (rayidx >= numRays)
                break;

            // Fetch ray.

			float4 o = ((float4*)a_RayBuffer)[rayidx * 2 + 0];
			float4 d = ((float4*)a_RayBuffer)[rayidx * 2 + 1];
            origx = o.x;
            origy = o.y;
            origz = o.z;
            tmin  = o.w;
            dirx  = d.x;
            diry  = d.y;
            dirz  = d.z;
            hitT  = d.w;
			float ooeps = math::exp2(-80.0f); // Avoid div by zero.
            idirx = 1.0f / (math::abs(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
            idiry = 1.0f / (math::abs(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
            idirz = 1.0f / (math::abs(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
            oodx  = origx * idirx;
            oody  = origy * idiry;
            oodz  = origz * idirz;

            // Setup traversal.

            stackPtr = (char*)&traversalStack[0];
            leafAddr = 0;   // No postponed leaf.
            leafAddr = nodeAddr = g_SceneData.m_sSceneBVH.m_sStartNode;   // Start from the root. set the leafAddr to support scenes with one node
            hitIndex = -1;  // No triangle intersected so far.
        }

        // Traversal loop.
		TraceResult r2 = k_TraceRay(Ray(a_RayBuffer[rayidx].a.getXYZ(), a_RayBuffer[rayidx].b.getXYZ()));
		int4 res = make_int4(0, 0, 0, 0);
		if (r2.hasHit())
		{
			res.x = __float_as_int(r2.m_fDist);
			res.y = r2.getNodeIndex();
			res.z = r2.m_pTri - g_SceneData.m_sTriData.Data;
			half2 h(r2.m_fBaryCoords);
			res.w = *(int*)&h;
		}
		((int4*)a_ResBuffer)[rayidx] = res;
		nodeAddr = EntrypointSentinel;

		/*if (g_SceneData.m_sNodeData.UsedCount == 0)
			nodeAddr = EntrypointSentinel;

		while (nodeAddr != EntrypointSentinel)
        {
			//nodeAddr = nodeAddr == EntrypointSentinel - 1 ? EntrypointSentinel : nodeAddr;
            // Traverse internal nodes until all SIMD lanes have found a leaf.

            while (unsigned int(nodeAddr) < unsigned int(EntrypointSentinel))   // functionally equivalent, but faster
            {
                // Fetch AABBs of the two child nodes.

                const float4 n0xy = tex1Dfetch(t_SceneNodes, nodeAddr + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
                const float4 n1xy = tex1Dfetch(t_SceneNodes, nodeAddr + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
                const float4 nz   = tex1Dfetch(t_SceneNodes, nodeAddr + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
                      float4 tmp  = tex1Dfetch(t_SceneNodes, nodeAddr + 3); // child_index0, child_index1
                      int2  cnodes= *(int2*)&tmp;

                // Intersect the ray against the child nodes.

                const float c0lox = n0xy.x * idirx - oodx;
                const float c0hix = n0xy.y * idirx - oodx;
                const float c0loy = n0xy.z * idiry - oody;
                const float c0hiy = n0xy.w * idiry - oody;
                const float c0loz = nz.x   * idirz - oodz;
                const float c0hiz = nz.y   * idirz - oodz;
                const float c1loz = nz.z   * idirz - oodz;
                const float c1hiz = nz.w   * idirz - oodz;
                const float c0min = spanBeginKepler2(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
                const float c0max = spanEndKepler2  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
                const float c1lox = n1xy.x * idirx - oodx;
                const float c1hix = n1xy.y * idirx - oodx;
                const float c1loy = n1xy.z * idiry - oody;
                const float c1hiy = n1xy.w * idiry - oody;
                const float c1min = spanBeginKepler2(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
                const float c1max = spanEndKepler2  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

                bool swp = (c1min < c0min);

                bool traverseChild0 = (c0max >= c0min);
                bool traverseChild1 = (c1max >= c1min);

                // Neither child was intersected => pop stack.

                if (!traverseChild0 && !traverseChild1)
                {
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }
                else// Otherwise => fetch child pointers.
                {
                    nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

                    // Both children were intersected => push the farther one.

                    if (traverseChild0 && traverseChild1)
                    {
                        if (swp)
                            swapk(nodeAddr, cnodes.y);
                        stackPtr += 4;
                        *(int*)stackPtr = cnodes.y;
                    }
                }

                // First leaf => postpone and continue traversal.

                if (nodeAddr < 0 && leafAddr  >= 0)     // Postpone max 1
                {
                    leafAddr = nodeAddr;
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                unsigned int mask;
                asm("{\n"
                    "   .reg .pred p;               \n"
                    "setp.ge.s32        p, %1, 0;   \n"
                    "vote.ballot.b32    %0,p;       \n"
                    "}"
                    : "=r"(mask)
                    : "r"(leafAddr));
                if(!mask)
                    break;
            }

            // Process postponed leaf nodes.

            while (leafAddr < 0)
            {
				e_Node* N = g_SceneData.m_sNodeData.Data + (~leafAddr);
				if (terminated)
				{
					float4x4 modl;
					loadInvModl(~leafAddr, &modl);
					float3 d = modl.TransformDirection(Vec3f(dirx, diry, dirz)), o = modl.TransformPoint(Vec3f(origx, origy, origz));

					lorigx = o.x;
					lorigy = o.y;
					lorigz = o.z;
					ltmin = tmin;
					ldirx = d.x;
					ldiry = d.y;
					ldirz = d.z;
					lhitT = hitT;
					float ooeps = math::exp2(-80.0f); // Avoid div by zero.
					lidirx = 1.0f / (math::abs(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
					lidiry = 1.0f / (math::abs(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
					lidirz = 1.0f / (math::abs(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
					loodx = lorigx * lidirx;
					loody = lorigy * lidiry;
					loodz = lorigz * lidirz;
					lstackPtr = (char*)&ltraversalStack[0];
					lleafAddr = 0;   // No postponed leaf.
					lnodeAddr = 0;   // Start from the root.
				}

				unsigned int m_uBVHNodeOffset = g_SceneData.m_sMeshData[N->m_uMeshIndex].m_uBVHNodeOffset,
					m_uBVHTriangleOffset = g_SceneData.m_sMeshData[N->m_uMeshIndex].m_uBVHTriangleOffset,
					m_uBVHIndicesOffset = g_SceneData.m_sMeshData[N->m_uMeshIndex].m_uBVHIndicesOffset,
					m_uTriangleOffset = g_SceneData.m_sMeshData[N->m_uMeshIndex].m_uTriangleOffset;

				while (lnodeAddr != EntrypointSentinel)
				{
					while (unsigned int(lnodeAddr) < unsigned int(EntrypointSentinel))
					{
						const float4 n0xy = tex1Dfetch(t_nodesA, lnodeAddr + 0 + m_uBVHNodeOffset); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
						const float4 n1xy = tex1Dfetch(t_nodesA, lnodeAddr + 1 + m_uBVHNodeOffset); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
						const float4 nz = tex1Dfetch(t_nodesA, lnodeAddr + 2 + m_uBVHNodeOffset); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
						float4 tmp = tex1Dfetch(t_nodesA, lnodeAddr + 3 + m_uBVHNodeOffset); // child_index0, child_index1
						int2  cnodes = *(int2*)&tmp;

						// Intersect the ray against the child nodes.

						const float c0lox = n0xy.x * lidirx - loodx;
						const float c0hix = n0xy.y * lidirx - loodx;
						const float c0loy = n0xy.z * lidiry - loody;
						const float c0hiy = n0xy.w * lidiry - loody;
						const float c0loz = nz.x   * lidirz - loodz;
						const float c0hiz = nz.y   * lidirz - loodz;
						const float c1loz = nz.z   * lidirz - loodz;
						const float c1hiz = nz.w   * lidirz - loodz;
						const float c0min = spanBeginKepler2(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, ltmin);
						const float c0max = spanEndKepler2(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, lhitT);
						const float c1lox = n1xy.x * lidirx - loodx;
						const float c1hix = n1xy.y * lidirx - loodx;
						const float c1loy = n1xy.z * lidiry - loody;
						const float c1hiy = n1xy.w * lidiry - loody;
						const float c1min = spanBeginKepler2(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, ltmin);
						const float c1max = spanEndKepler2(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, lhitT);

						bool swp = (c1min < c0min);

						bool traverseChild0 = (c0max >= c0min);
						bool traverseChild1 = (c1max >= c1min);

						// Neither child was intersected => pop stack.

						if (!traverseChild0 && !traverseChild1)
						{
							lnodeAddr = *(int*)lstackPtr;
							lstackPtr -= 4;
						}
						else// Otherwise => fetch child pointers.
						{
							lnodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

							// Both children were intersected => push the farther one.

							if (traverseChild0 && traverseChild1)
							{
								if (swp)
									swapk(lnodeAddr, cnodes.y);
								lstackPtr += 4;
								*(int*)lstackPtr = cnodes.y;
							}
						}

						if (lnodeAddr < 0 && lleafAddr >= 0)     // Postpone max 1
						{
							lleafAddr = lnodeAddr;
							lnodeAddr = *(int*)lstackPtr;
							lstackPtr -= 4;
						}

						unsigned int mask;
						asm("{\n"
							"   .reg .pred p;               \n"
							"setp.ge.s32        p, %1, 0;   \n"
							"vote.ballot.b32    %0,p;       \n"
							"}"
							: "=r"(mask)
							: "r"(lleafAddr));
						if (!mask)
							break;
					}
					while (lleafAddr < 0)
					{
						for (int triAddr = ~lleafAddr;; triAddr++)
						{
							// Tris in TEX (good to fetch as a single batch)
							const float4 v00 = tex1Dfetch(t_tris, triAddr * 3 + 0 + m_uBVHTriangleOffset);
							const float4 v11 = tex1Dfetch(t_tris, triAddr * 3 + 1 + m_uBVHTriangleOffset);
							const float4 v22 = tex1Dfetch(t_tris, triAddr * 3 + 2 + m_uBVHTriangleOffset);
							unsigned int index = tex1Dfetch(t_triIndices, min(triAddr + m_uBVHIndicesOffset, 8135));

							float Oz = v00.w - lorigx*v00.x - lorigy*v00.y - lorigz*v00.z;
							float invDz = 1.0f / (ldirx*v00.x + ldiry*v00.y + ldirz*v00.z);
							float t = Oz * invDz;

							if (t > ltmin && t < lhitT)
							{
								// Compute and check barycentric u.

								float Ox = v11.w + lorigx*v11.x + lorigy*v11.y + lorigz*v11.z;
								float Dx = ldirx*v11.x + ldiry*v11.y + ldirz*v11.z;
								float u = Ox + t*Dx;

								if (u >= 0.0f)
								{
									// Compute and check barycentric v.

									float Oy = v22.w + lorigx*v22.x + lorigy*v22.y + lorigz*v22.z;
									float Dy = ldirx*v22.x + ldiry*v22.y + ldirz*v22.z;
									float v = Oy + t*Dy;

									if (v >= 0.0f && u + v <= 1.0f)
									{
										// Record intersection.
										// Closest intersection not required => terminate.

										nodeIdx = ~leafAddr;
										lhitT = t;
										hitIndex = (index >> 1) + m_uTriangleOffset;
										bCorrds = Vec2f(u, v);
										if (ANY_HIT)
										{
											nodeAddr = lnodeAddr = EntrypointSentinel;
											break;
										}
									}
								}
							}
							if (index & 1)
								break;
						} // triangle
						hitT = lhitT;

						lleafAddr = lnodeAddr;
						if (lnodeAddr < 0)
						{
							lnodeAddr = *(int*)lstackPtr;
							lstackPtr -= 4;
						}
					}
					//BUGGY
					//if( __popc(__ballot(true)) < DYNAMIC_FETCH_THRESHOLD / 2 )
					//{
					//	//we can't pop yet
					//	nodeAddr = EntrypointSentinel - 1;
					//	//can't break cause we don't want to pop postponed leaf
					//	goto outerlabel;//jump AFTER store cause we will do that later
					//}
				}
				// Another leaf was postponed => process it as well.		
				leafAddr = nodeAddr;
				if (nodeAddr < 0)
				{
					nodeAddr = *(int*)stackPtr;
					stackPtr -= 4;
				}
			} // leaf

            // DYNAMIC FETCH
			//BUGGY
            //if( __popc(__ballot(true)) < DYNAMIC_FETCH_THRESHOLD )
            //    break;
        } // traversal

        // Remap intersected triangle index, and store the result.

		int4 res = make_int4(0,0,0,0);
		if(hitIndex != -1)
		{
			res.x = __float_as_int(hitT);
			res.y = nodeIdx;
			res.z = hitIndex;
			half2 h(bCorrds);
			res.w = *(int*)&h;
		}
		((int4*)a_ResBuffer)[rayidx] = res;*/
//outerlabel: ;
    } while(true);
}

void __internal__IntersectBuffers(int N, traversalRay* a_RayBuffer, traversalResult* a_ResBuffer, bool SKIP_OUTER, bool ANY_HIT)
{
	ThrowCudaErrors(hipDeviceSetCacheConfig (hipFuncCachePreferL1));
	unsigned int zero = 0;
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter), &zero, sizeof(unsigned int)));
	/*if(SKIP_OUTER)
		if(ANY_HIT)
			intersectKernel_SKIPOUTER<true><<< 180, dim3(32, 4, 1)>>>(N, a_RayBuffer, a_ResBuffer);
		else intersectKernel_SKIPOUTER<false><<< 180, dim3(32, 4, 1)>>>(N, a_RayBuffer, a_ResBuffer);
	else*/
	{
		if(ANY_HIT)
			intersectKernel<true><<< 180, dim3(32, 4, 1)>>>(N, a_RayBuffer, a_ResBuffer);
		else intersectKernel<false><<< 180, dim3(32, 4, 1)>>>(N, a_RayBuffer, a_ResBuffer);
	}
	ThrowCudaErrors(hipDeviceSynchronize());
	g_RayTracedCounterHost += N;
}