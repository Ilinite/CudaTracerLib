#include "hip/hip_runtime.h"
#include "k_Tracer.h"
#include "k_TraceHelper.h"

__device__ TraceResult g_Res;

__global__ void trace(Ray r)
{
	k_TraceRay<true>(r.direction, r.origin, &g_Res);
}

TraceResult TraceSingleRay(Ray r, e_DynamicScene* s, e_Camera* c)
{
	k_TracerRNGBuffer tmp;
	s->UpdateInvalidated();
	k_INITIALIZE(s->getKernelSceneData());
	k_STARTPASS(s, c, tmp)
	TraceResult r2;
	r2.Init();
	hipMemcpyToSymbol(HIP_SYMBOL(g_Res), &r2, sizeof(TraceResult));
	trace<<<1,1>>>(r);
	hipDeviceSynchronize();
	TraceResult q;
	hipMemcpyFromSymbol(&q, HIP_SYMBOL(g_Res), sizeof(q));
	return q;
}


__global__ void genRNG2(hiprandState* states, unsigned int a_Spacing, unsigned int a_Offset, unsigned int a_Num)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < a_Num)
	{
		k_TracerRNG* r = (k_TracerRNG*)(states + i);
		r->Initialize(i, a_Spacing, a_Offset);
	}
}

void k_TracerRNGBuffer::createGenerators(unsigned int a_Spacing, unsigned int a_Offset)
{
	genRNG2<<< m_uNumGenerators / 1024 + 1, 1024 >>> (m_pGenerators, a_Spacing, a_Offset, m_uNumGenerators);
	hipDeviceSynchronize();
}

CUDA_FUNC_IN unsigned int FloatToUInt(float f2)
{
	unsigned int f = *(unsigned int*)&f2;
	unsigned int mask = -int(f >> 31) | 0x80000000;
	return f ^ mask;
}

CUDA_FUNC_IN float UIntToFloat(float f2)
{
	unsigned int f = *(unsigned int*)&f2;
	unsigned int mask = ((f >> 31) - 1) | 0x80000000;
	unsigned int i = f ^ mask;
	return *(float*)&i;
}

CUDA_DEVICE uint3 g_EyeHitBoxMin;
CUDA_DEVICE uint3 g_EyeHitBoxMax;
__global__ void k_GuessPass(int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y, N = y * w + x;
	CudaRNG localState = g_RNGData();
	if(x < w && y < h)
	{
		Ray r = g_CameraData.GenRay<false>(x, y, w, h, localState.randomFloat(), localState.randomFloat());
		TraceResult r2;
		r2.Init();
		int d = -1;
		while(k_TraceRay<true>(r.direction, r.origin, &r2) && ++d < 10)
		{
			e_KernelBSDF bsdf = r2.GetBSDF(g_SceneData.m_sMatData.Data);
			float3 inc;
			float pdf;
			float3 col = bsdf.Sample_f(-1.0f * r.direction, &inc, BSDFSample(localState), &pdf);
			float3 p = r(r2.m_fDist);
			r = Ray(r(r2.m_fDist), inc);
			uint3 pu = make_uint3(FloatToUInt(p.x), FloatToUInt(p.y), FloatToUInt(p.z));
			atomicMin(&g_EyeHitBoxMin.x, pu.x);
			atomicMin(&g_EyeHitBoxMin.y, pu.y);
			atomicMin(&g_EyeHitBoxMin.z, pu.z);
			atomicMax(&g_EyeHitBoxMax.x, pu.x);
			atomicMax(&g_EyeHitBoxMax.y, pu.y);
			atomicMax(&g_EyeHitBoxMax.z, pu.z);
			r2.Init();
		}
	}
	g_RNGData(localState);
}

AABB k_RandTracerBase::GetEyeHitPointBox()
{
	uint3 ma = make_uint3(FloatToUInt(-FLT_MAX)), mi = make_uint3(FloatToUInt(FLT_MAX));
	hipMemcpyToSymbol(HIP_SYMBOL(g_EyeHitBoxMin), &mi, 12);
	hipMemcpyToSymbol(HIP_SYMBOL(g_EyeHitBoxMax), &ma, 12);
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, m_sRngs);
	int qw = 128, qh = 128, p0 = 16;
	k_GuessPass<<<dim3( qw/p0, qh/p0, 1), dim3(p0, p0, 1)>>>(qw, qh);
	hipDeviceSynchronize();
	AABB m_sEyeBox;
	hipMemcpyFromSymbol(&m_sEyeBox.minV, HIP_SYMBOL(g_EyeHitBoxMin), 12);
	hipMemcpyFromSymbol(&m_sEyeBox.maxV, HIP_SYMBOL(g_EyeHitBoxMax), 12);
	m_sEyeBox.minV = make_float3(UIntToFloat(m_sEyeBox.minV.x), UIntToFloat(m_sEyeBox.minV.y), UIntToFloat(m_sEyeBox.minV.z));
	m_sEyeBox.maxV = make_float3(UIntToFloat(m_sEyeBox.maxV.x), UIntToFloat(m_sEyeBox.maxV.y), UIntToFloat(m_sEyeBox.maxV.z));
	return m_sEyeBox;
}