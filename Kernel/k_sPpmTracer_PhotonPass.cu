#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include "k_TraceHelper.h"

CUDA_DEVICE k_PhotonMapCollection g_Map;

template<bool DIRECT> CUDA_FUNC_IN bool TracePhoton(Ray& r, Spectrum Le, CudaRNG& rng)
{
	r.direction = normalize(r.direction);
	e_KernelAggregateVolume& V = g_SceneData.m_sVolume;
	TraceResult r2;
	r2.Init(true);
	int depth = -1;
	//bool inMesh = false;
	BSDFSamplingRecord bRec;
	while(++depth < 12 && k_TraceRay(r.direction, r.origin, &r2))
	{
		if(V.HasVolumes())
		{
			float minT, maxT;
			while(V.IntersectP(r, 0, r2.m_fDist, &minT, &maxT))
			{
				float3 x = r(minT), w = -r.direction;
				Spectrum sigma_s = V.sigma_s(x, w), sigma_t = V.sigma_t(x, w);
				float d = -logf(rng.randomFloat()) / sigma_t.average();
				bool cancel = d >= (maxT - minT) || d >= r2.m_fDist;
				d = clamp(d, minT, maxT);
				Le += V.Lve(x, w) * d;
				if(g_Map.StorePhoton<false>(r(minT + d * rng.randomFloat()), Le, w, make_float3(0,0,0)) == k_StoreResult::Full)
					return false;
				if(cancel)
					break;
				float A = (sigma_s / sigma_t).average();
				if(rng.randomFloat() <= A)
				{
					float3 wi;
					float pf = V.Sample(x, -r.direction, rng, &wi);
					Le /= A;
					Le *= pf;
					r.origin = r(minT + d);
					r.direction = wi;
					r2.Init();
					if(!k_TraceRay(r.direction, r.origin, &r2))
						return true;
				}
				else break;//Absorption
			}
		}
		float3 x = r(r2.m_fDist);
		r2.getBsdfSample(r, rng, &bRec);
		const e_KernelBSSRDF* bssrdf;
		Spectrum ac;
		if(r2.getMat().GetBSSRDF(bRec.map, &bssrdf))
		{
			//inMesh = false;
			ac = Le;
			while(true)
			{
				float3 w = -r.direction;
				TraceResult r3 = k_TraceRay(Ray(x, r.direction));
				Spectrum sigma_s = bssrdf->sigp_s, sigma_t = bssrdf->sigp_s + bssrdf->sig_a;
				float d = -logf(rng.randomFloat()) / sigma_t.average();
				bool cancel = d >= (r3.m_fDist);
				d = clamp(d, 0.0f, r3.m_fDist);
				if(g_Map.StorePhoton<false>(x + r.direction * (d * rng.randomFloat()), ac, w, make_float3(0,0,0)) == k_StoreResult::Full)
					return false;
				if(cancel)
				{
					x = x + r.direction * r3.m_fDist;
					Frame sys;
					r3.lerpFrame(sys);
					float3 wi = VectorMath::refract(r.direction, -sys.n, 1.0f/bssrdf->e);
					bRec.wo = bRec.map.sys.toLocal(wi);//ugly
					break;
				}
				float A = (sigma_s / sigma_t).average();
				if(rng.randomFloat() <= A)
				{
					ac /= A;
					float3 wo = Warp::squareToUniformSphere(rng.randomFloat2());
					ac *= 1.f / (4.f * PI);
					r.origin = x + r.direction * d;
					r.direction = wo;
				}
				else return true;
			}
		}
		else
		{
			float3 wo = -r.direction;
			if((DIRECT && depth > 0) || !DIRECT)
				if(r2.getMat().bsdf.hasComponent(EDiffuse))
					if(g_Map.StorePhoton<true>(x, Le, wo, bRec.map.sys.n) == k_StoreResult::Full)
						return false;
			Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
			if(!bRec.sampledType)
				break;
			//inMesh = dot(r.direction, bRec.map.sys.n) < 0;
			ac = Le * f;
		}
		if(depth > 5)
		{
			float prob = MIN(1.0f, ac.max() / Le.max());
			if(rng.randomFloat() > prob)
				break;
			Le = ac / prob;
		}
		else Le = ac;
		r = Ray(x, (bRec.getOutgoing()));
		r2.Init();
	}
	return true;
}

template<bool DIRECT> __global__ void k_PhotonPass(unsigned int spp)
{ 
	CudaRNG rng = g_RNGData();
	for(int _photonNum = 0; _photonNum < spp; _photonNum++)
	{
		Ray photonRay;
		const e_KernelLight* light;
		Spectrum Le = g_SceneData.sampleEmitterRay(photonRay, light, rng.randomFloat2(), rng.randomFloat2());
		if(Le.isZero())
			continue;
		if(TracePhoton<DIRECT>(photonRay, Le, rng))
			atomicInc(&g_Map.m_uPhotonNumEmitted, 0xffffffff);
		else break;
	}
	g_RNGData(rng);
}

void k_sPpmTracer::doPhotonPass()
{
	hipMemcpyToSymbol(HIP_SYMBOL(g_Map), &m_sMaps, sizeof(k_PhotonMapCollection));
	k_INITIALIZE(m_pScene->getKernelSceneData());
	k_STARTPASS(m_pScene, m_pCamera, g_sRngs);
	const unsigned long long p0 = 6 * 32, spp = 3, n = 180;
	if(m_bDirect)
		k_PhotonPass<true><<< n, p0 >>>(spp);
	else k_PhotonPass<false><<< n, p0 >>>(spp);
	hipDeviceSynchronize();
	hipMemcpyFromSymbol(&m_sMaps, HIP_SYMBOL(g_Map), sizeof(k_PhotonMapCollection));
}