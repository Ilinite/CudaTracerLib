#include "hip/hip_runtime.h"
#include "Spline.h"
#include "..\Base\STL.h"

float Spline::evalCubicInterp1D(float x, const float *values, size_t size, float min, float max, bool extrapolate)
{
	/* Give up when given an out-of-range or NaN argument */
	if (!(x >= min && x <= max) && !extrapolate)
		return 0.0f;

	/* Transform 'x' so that knots lie at integer positions */
	float t = ((x - min) * (size - 1)) / (max - min);

	/* Find the index of the left knot in the queried subinterval, be
	   robust to cases where 't' lies exactly on the right endpoint */
	size_t k = MAX((size_t) 0, MIN((size_t) t, size - 2));

	float f0  = values[k],
	      f1  = values[k+1],
	      d0, d1;

	/* Approximate the derivatives */
	if (k > 0)
		d0 = 0.5f * (values[k+1] - values[k-1]);
	else
		d0 = values[k+1] - values[k];

	if (k + 2 < size)
		d1 = 0.5f * (values[k+2] - values[k]);
	else
		d1 = values[k+1] - values[k];

	/* Compute the relative position within the interval */
	t = t - (float) k;

	float t2 = t*t, t3 = t2*t;

	return
		( 2*t3 - 3*t2 + 1) * f0 +
		(-2*t3 + 3*t2)     * f1 +
		(   t3 - 2*t2 + t) * d0 +
		(   t3 - t2)       * d1;
}

float Spline::evalCubicInterp1DN(float x, const float *nodes, const float *values, size_t size, bool extrapolate)
{
	/* Give up when given an out-of-range or NaN argument */
	if (!(x >= nodes[0] && x <= nodes[size-1]) && !extrapolate)
		return 0.0f;

	size_t k = (size_t) MAX((ptrdiff_t) 0, MIN((ptrdiff_t) size - 2,
		STL_lower_bound(nodes, nodes + size, x) - nodes - 1));

	float f0       = values[k],
	      f1       = values[k+1],
	      width    = nodes[k+1] - nodes[k],
	      d0, d1;

	/* Approximate the derivatives */
	if (k > 0)
		d0 = width * (f1 - values[k-1]) / (nodes[k+1] - nodes[k-1]);
	else
		d0 = f1 - f0;

	if (k + 2 < size)
		d1 = width * (values[k+2] - f0) / (nodes[k+2] - nodes[k]);
	else
		d1 = f1 - f0;

	float t = (x - nodes[k]) / width;
	float t2 = t*t, t3 = t2*t;

	return
	    ( 2*t3 - 3*t2 + 1) * f0 +
	    (-2*t3 + 3*t2)     * f1 +
	    (   t3 - 2*t2 + t) * d0 +
	    (   t3 - t2)       * d1;
}

float Spline::integrateCubicInterp1D(size_t idx, const float *values, size_t size, float min, float max)
{
	float f0 = values[idx], f1 = values[idx+1], d0, d1;

	/* Approximate the derivatives */
	if (idx > 0)
		d0 = 0.5f * (values[idx+1] - values[idx-1]);
	else
		d0 = values[idx+1] - values[idx];

	if (idx + 2 < size)
		d1 = 0.5f * (values[idx+2] - values[idx]);
	else
		d1 = values[idx+1] - values[idx];

	return ((d0-d1) * (float) (1.0 / 12.0) + (f0+f1) * 0.5f) * (max-min) / (size - 1);
}

float Spline::integrateCubicInterp1DN(size_t idx, const float *nodes, const float *values, size_t size)
{
	float f0       = values[idx],
	      f1       = values[idx+1],
	      width    = nodes[idx+1] - nodes[idx],
	      d0, d1;

	/* Approximate the derivatives */
	if (idx > 0)
		d0 = width * (f1 - values[idx-1]) / (nodes[idx+1] - nodes[idx-1]);
	else
		d0 = f1 - f0;

	if (idx + 2 < size)
		d1 = width * (values[idx+2] - f0) / (nodes[idx+2] - nodes[idx]);
	else
		d1 = f1 - f0;

	return ((d0-d1) * (float) (1.0 / 12.0) + (f0+f1) * 0.5f) * width;
}

float Spline::sampleCubicInterp1D(size_t idx, float *values, size_t size, float min, float max, float sample, float *fval)
{
	float f0 = values[idx], f1 = values[idx+1], d0, d1;

	/* Approximate the derivatives */
	if (idx > 0)
		d0 = 0.5f * (values[idx+1] - values[idx-1]);
	else
		d0 = values[idx+1] - values[idx];

	if (idx + 2 < size)
		d1 = 0.5f * (values[idx+2] - values[idx]);
	else
		d1 = values[idx+1] - values[idx];

	/* Bracketing interval and starting guess */
	float a = 0, c = 1, b;

	if (f0 != f1) /* Importance sample linear interpolant */
		b = (f0-math::safe_sqrt(f0*f0 + sample * (f1*f1-f0*f0))) / (f0-f1);
	else
		b = sample;

	sample *= ((d0-d1) * (float) (1.0 / 12.0) + (f0+f1) * 0.5f);

	/* Invert CDF using Newton-Bisection */
	while (true) {
		if (!(b >= a && b <= c))
			b = 0.5f * (a + c);

		/* CDF and PDF in Horner form */
		float value = b*(f0 + b*(.5f*d0 + b*((float) (1.0f/3.0f) * (-2*d0-d1)
			+ f1 - f0 + b*(0.25f*(d0 + d1) + 0.5f * (f0 - f1))))) - sample;
		float deriv = f0 + b*(d0 + b*(-2*d0 - d1 + 3*(f1-f0) + b*(d0 + d1 + 2*(f0 - f1))));

		if (abs(value) < 1e-6f) {
			if (fval)
				*fval = deriv;
			return min + (idx+b) * (max-min) / (size-1);
		}

		if (value > 0)
			c = b;
		else
			a = b;

		b -= value / deriv;
	}
}

float Spline::sampleCubicInterp1DN(size_t idx, float *nodes, float *values, size_t size, float sample, float *fval)
{
	float f0       = values[idx],
	      f1       = values[idx+1],
	      width    = nodes[idx+1] - nodes[idx],
	      d0, d1;

	/* Approximate the derivatives */
	if (idx > 0)
		d0 = width * (f1 - values[idx-1]) / (nodes[idx+1] - nodes[idx-1]);
	else
		d0 = f1 - f0;

	if (idx + 2 < size)
		d1 = width * (values[idx+2] - f0) / (nodes[idx+2] - nodes[idx]);
	else
		d1 = f1 - f0;

	/* Bracketing interval and starting guess */
	float a = 0, c = 1, b;

	if (f0 != f1) /* Importance sample linear interpolant */
		b = (f0-math::safe_sqrt(f0*f0 + sample * (f1*f1-f0*f0))) / (f0-f1);
	else
		b = sample;

	sample *= ((d0-d1) * (float) (1.0 / 12.0) + (f0+f1) * 0.5f);

	/* Invert CDF using Newton-Bisection */
	while (true) {
		if (!(b >= a && b <= c))
			b = 0.5f * (a + c);

		/* CDF and PDF in Horner form */
		float value = b*(f0 + b*(.5f*d0 + b*((float) (1.0f/3.0f) * (-2*d0-d1)
			+ f1 - f0 + b*(0.25f*(d0 + d1) + 0.5f * (f0 - f1))))) - sample;
		float deriv = f0 + b*(d0 + b*(-2*d0 - d1 + 3*(f1-f0) + b*(d0 + d1 + 2*(f0 - f1))));

		if (abs(value) < 1e-6f) {
			if (fval)
				*fval = deriv;
			return nodes[idx] + width*b;
		}

		if (value > 0)
			c = b;
		else
			a = b;

		b -= value / deriv;
	}
}

float Spline::evalCubicInterp2D(const float2 &_p, const float *values, const uint2 &_size, const float2 &_min, const float2 &_max, bool extrapolate)
{
	const float* p = (float*)&_p, *min = (float*)&_min, *max = (float*)&_max;
	const unsigned int* size = (unsigned int*)&_size;

	float knotWeights[2][4];
	unsigned int knot[2];

	/* Compute interpolation weights separately for each dimension */
	for (int dim=0; dim<2; ++dim) {
		float *weights = knotWeights[dim];
		/* Give up when given an out-of-range or NaN argument */
		if (!(p[dim] >= min[dim] && p[dim] <= max[dim]) && !extrapolate)
			return 0.0f;

		/* Transform 'p' so that knots lie at integer positions */
		float t = ((p[dim] - min[dim]) * (size[dim] - 1))
			/ (max[dim]-min[dim]);

		/* Find the index of the left knot in the queried subinterval, be
		   robust to cases where 't' lies exactly on the right endpoint */
		knot[dim] = MIN((unsigned int) t, size[dim] - 2);

		/* Compute the relative position within the interval */
		t = t - (float) knot[dim];

		/* Compute node weights */
		float t2 = t*t, t3 = t2*t;
		weights[0] = 0.0f;
		weights[1] = 2*t3 - 3*t2 + 1;
		weights[2] = -2*t3 + 3*t2;
		weights[3] = 0.0f;

		/* Derivative weights */
		float d0 = t3 - 2*t2 + t,
			  d1 = t3 - t2;

		/* Turn derivative weights into node weights using
		   an appropriate chosen finite differences stencil */
		if (knot[dim] > 0) {
			weights[2] +=  0.5f * d0;
			weights[0] -=  0.5f * d0;
		} else {
			weights[2] += d0;
			weights[1] -= d0;
		}

		if (knot[dim] + 2 < size[dim]) {
			weights[3] += 0.5f * d1;
			weights[1] -= 0.5f * d1;
		} else {
			weights[2] += d1;
			weights[1] -= d1;
		}
	}

	float result = 0.0f;
	for (int y=-1; y<=2; ++y) {
		float wy = knotWeights[1][y+1];
		for (int x=-1; x<=2; ++x) {
			float wxy = knotWeights[0][x+1] * wy;

			if (wxy == 0)
				continue;

			size_t pos = (knot[1] + y) * size[0] + knot[0] + x;

			result += values[pos] * wxy;
		}
	}
	return result;
}

float Spline::evalCubicInterp2DN(const float2 &_p, const float **nodes_, const float *values, const uint2 &_size, bool extrapolate)
{
	const float* p = (float*)&_p;
	const unsigned int* size = (unsigned int*)&_size;

	float knotWeights[2][4];
	unsigned int knot[2];

	/* Compute interpolation weights separately for each dimension */
	for (int dim=0; dim<2; ++dim) {
		const float *nodes = nodes_[dim];
		float *weights = knotWeights[dim];

		/* Give up when given an out-of-range or NaN argument */
		if (!(p[dim] >= nodes[0] && p[dim] <= nodes[size[dim]-1]) && !extrapolate)
			return 0.0f;

		/* Find the index of the left knot in the queried subinterval, be
		   robust to cases where 't' lies exactly on the right endpoint */
		int k = MAX((ptrdiff_t) 0, MIN((ptrdiff_t) size[dim] - 2,
			STL_lower_bound(nodes, nodes + size[dim], p[dim]) - nodes - 1));
		knot[dim] = k;

		float width = nodes[k+1] - nodes[k];

		/* Compute the relative position within the interval */
		float t = (p[dim] - nodes[k]) / width,
			  t2 = t*t, t3 = t2*t;

		/* Compute node weights */
		weights[0] = 0.0f;
		weights[1] = 2*t3 - 3*t2 + 1;
		weights[2] = -2*t3 + 3*t2;
		weights[3] = 0.0f;

		/* Derivative weights */
		float d0 = t3 - 2*t2 + t, d1 = t3 - t2;

		/* Turn derivative weights into node weights using
		   an appropriate chosen finite differences stencil */
		if (k > 0) {
			float factor = width / (nodes[k+1]-nodes[k-1]);
			weights[2] += d0 * factor;
			weights[0] -= d0 * factor;
		} else {
			weights[2] += d0;
			weights[1] -= d0;
		}

		if (k + 2 < (int)size[dim]) {
			float factor = width / (nodes[k+2]-nodes[k]);
			weights[3] += d1 * factor;
			weights[1] -= d1 * factor;
		} else {
			weights[2] += d1;
			weights[1] -= d1;
		}
	}

	float result = 0.0f;
	for (int y=-1; y<=2; ++y) {
		float wy = knotWeights[1][y+1];
		for (int x=-1; x<=2; ++x) {
			float wxy = knotWeights[0][x+1] * wy;

			if (wxy == 0)
				continue;

			size_t pos = (knot[1] + y) * size[0] + knot[0] + x;

			result += values[pos] * wxy;
		}
	}
	return result;
}

float Spline::evalCubicInterp3D(const float3 &_p, const float *values, const uint3 &_size, const float3 &_min, const float3 &_max, bool extrapolate)
{
	const float* p = (float*)&_p, *min = (float*)&_min, *max = (float*)&_max;
	const unsigned int* size = (unsigned int*)&_size;

	float knotWeights[3][4];
	unsigned int knot[3];

	/* Compute interpolation weights separately for each dimension */
	for (int dim=0; dim<3; ++dim) {
		float *weights = knotWeights[dim];
		/* Give up when given an out-of-range or NaN argument */
		if (!(p[dim] >= min[dim] && p[dim] <= max[dim]) && !extrapolate)
			return 0.0f;

		/* Transform 'p' so that knots lie at integer positions */
		float t = ((p[dim] - min[dim]) * (size[dim] - 1))
			/ (max[dim]-min[dim]);

		/* Find the index of the left knot in the queried subinterval, be
		   robust to cases where 't' lies exactly on the right endpoint */
		knot[dim] = MIN((unsigned int) t, size[dim] - 2);

		/* Compute the relative position within the interval */
		t = t - (float) knot[dim];

		/* Compute node weights */
		float t2 = t*t, t3 = t2*t;
		weights[0] = 0.0f;
		weights[1] = 2*t3 - 3*t2 + 1;
		weights[2] = -2*t3 + 3*t2;
		weights[3] = 0.0f;

		/* Derivative weights */
		float d0 = t3 - 2*t2 + t,
			  d1 = t3 - t2;

		/* Turn derivative weights into node weights using
		   an appropriate chosen finite differences stencil */
		if (knot[dim] > 0) {
			weights[2] +=  0.5f * d0;
			weights[0] -=  0.5f * d0;
		} else {
			weights[2] += d0;
			weights[1] -= d0;
		}

		if (knot[dim] + 2 < size[dim]) {
			weights[3] += 0.5f * d1;
			weights[1] -= 0.5f * d1;
		} else {
			weights[2] += d1;
			weights[1] -= d1;
		}
	}

	float result = 0.0f;
	for (int z=-1; z<=2; ++z) {
		float wz = knotWeights[2][z+1];
		for (int y=-1; y<=2; ++y) {
			float wyz = knotWeights[1][y+1] * wz;
			for (int x=-1; x<=2; ++x) {
				float wxyz = knotWeights[0][x+1] * wyz;

				if (wxyz == 0)
					continue;

				size_t pos = ((knot[2] + z) * size[1] + (knot[1] + y))
					* size[0] + knot[0] + x;

				result += values[pos] * wxyz;
			}
		}
	}
	return result;
}

float Spline::evalCubicInterp3DN(const float3 &_p, const float **nodes_, const float *values, const uint3 &_size, bool extrapolate)
{
	const float* p = (float*)&_p;
	const unsigned int* size = (unsigned int*)&_size;

	float knotWeights[3][4];
	unsigned int knot[3];

	/* Compute interpolation weights separately for each dimension */
	for (int dim=0; dim<3; ++dim) {
		const float *nodes = nodes_[dim];
		float *weights = knotWeights[dim];

		/* Give up when given an out-of-range or NaN argument */
		if (!(p[dim] >= nodes[0] && p[dim] <= nodes[size[dim]-1]) && !extrapolate)
			return 0.0f;

		/* Find the index of the left knot in the queried subinterval, be
		   robust to cases where 't' lies exactly on the right endpoint */
		int k = MAX((ptrdiff_t) 0, MIN((ptrdiff_t) size[dim] - 2,
			STL_lower_bound(nodes, nodes + size[dim], p[dim]) - nodes - 1));
		knot[dim] = k;

		float width = nodes[k+1] - nodes[k];

		/* Compute the relative position within the interval */
		float t = (p[dim] - nodes[k]) / width,
			  t2 = t*t, t3 = t2*t;

		/* Compute node weights */
		weights[0] = 0.0f;
		weights[1] = 2*t3 - 3*t2 + 1;
		weights[2] = -2*t3 + 3*t2;
		weights[3] = 0.0f;

		/* Derivative weights */
		float d0 = t3 - 2*t2 + t, d1 = t3 - t2;

		/* Turn derivative weights into node weights using
		   an appropriate chosen finite differences stencil */
		if (k > 0) {
			float factor = width / (nodes[k+1]-nodes[k-1]);
			weights[2] += d0 * factor;
			weights[0] -= d0 * factor;
		} else {
			weights[2] += d0;
			weights[1] -= d0;
		}

		if (k + 2 < (int)size[dim]) {
			float factor = width / (nodes[k+2]-nodes[k]);
			weights[3] += d1 * factor;
			weights[1] -= d1 * factor;
		} else {
			weights[2] += d1;
			weights[1] -= d1;
		}
	}

	float result = 0.0f;
	for (int z=-1; z<=2; ++z) {
		float wz = knotWeights[2][z+1];
		for (int y=-1; y<=2; ++y) {
			float wyz = knotWeights[1][y+1] * wz;
			for (int x=-1; x<=2; ++x) {
				float wxyz = knotWeights[0][x+1] * wyz;

				if (wxyz == 0)
					continue;

				size_t pos = ((knot[2] + z) * size[1] + (knot[1] + y))
					* size[0] + knot[0] + x;

				result += values[pos] * wxyz;
			}
		}
	}
	return result;
}