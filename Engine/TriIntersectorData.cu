#include "hip/hip_runtime.h"
#include "TriIntersectorData.h"
#include <MathTypes.h>

namespace CudaTracerLib {

void TriIntersectorData::setData(const Vec3f& a, const Vec3f& b, const Vec3f& c)
{
	//if(absdot(normalize(cross(a - c, b - c)), make_float3(1, 0, 0)) < 0.05f)
	/*{
	float l = length((a + b + c) / 3.0f - a) / 200.0f;
	//e += make_float3(0,0.02f,-0.02f);
	//p *= 1.02f;
	//q *= 1.02f;
	p = p + make_float3(l, -l, l);
	q = q + make_float3(-l, l, -l);
	e = -p - q;
	e = e + make_float3(0,l,-l)/200.0f;
	}*/
	float4x4 m;
	m.col(0, Vec4f(a - c, 0));
	m.col(1, Vec4f(b - c, 0));
	m.col(2, Vec4f(cross(a - c, b - c), 0));
	m.col(3, Vec4f(c, 1));
	m = m.inverse();
	this->a = Vec4f(m(2, 0), m(2, 1), m(2, 2), -m(2, 3));
	this->b = m.row(0);
	this->c = m.row(1);
	Vec3f v1, v2, v3;
	getData(v1, v2, v3);
	//*(float2*)t2 = make_float2(m[0].x, m[0].y);
	//*(half2*)(((int*)t2) + 2) = half2(m[0].z, m[0].w);
	//this->b = make_float4(m[1].x, m[1].y, m[1].z, m[1].w);
	//if(this->a.x == -0.0f)
	//	this->a.x = 0.0f;
	//this->a = make_float4(m[2].y, m[2].z, -m[2].w, m[0].y);
	//this->b = make_float4(m[0].z, m[0].w, m[1].y, m[1].z);
	//t2->c = m[1].w;
	//t2->setXs(m[2].x, m[0].x, m[1].x);
}

void TriIntersectorData::getData(Vec3f& v0, Vec3f& v1, Vec3f& v2) const
{
	float4x4 m = float4x4::Identity();
	m.row(0, b);
	m.row(1, c);
	m.row(2, a);
	m(2, 3) *= -1.0f;
	m = m.inverse();
	Vec3f e02 = m.col(0).getXYZ(), e12 = m.col(1).getXYZ();
	v2 = m.col(3).getXYZ();
	v0 = v2 + e02;
	v1 = v2 + e12;
}

bool TriIntersectorData::Intersect(const Ray& r, float* dist, Vec2f* bary) const
{
	float Oz = a.w - r.ori().x*a.x - r.ori().y*a.y - r.ori().z*a.z;
	float invDz = 1.0f / (r.dir().x*a.x + r.dir().y*a.y + r.dir().z*a.z);
	float t = Oz * invDz;
	float tmax = dist ? *dist : FLT_MAX;
	if (t > 0.0001f && t < tmax)
	{
		float Ox = b.w + r.ori().x*b.x + r.ori().y*b.y + r.ori().z*b.z;
		float Dx = r.dir().x*b.x + r.dir().y*b.y + r.dir().z*b.z;
		float u = Ox + t*Dx;
		if (u >= 0.0f)
		{
			float Oy = c.w + r.ori().x*c.x + r.ori().y*c.y + r.ori().z*c.z;
			float Dy = r.dir().x*c.x + r.dir().y*c.y + r.dir().z*c.z;
			float v = Oy + t*Dy;
			if (v >= 0.0f && u + v <= 1.0f)
			{
				if (dist)
					*dist = t;
				if (bary)
					*bary = Vec2f(u, v);
				return true;
			}
		}
	}
	return false;
}

}