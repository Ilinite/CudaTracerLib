#include "hip/hip_runtime.h"
#include "KernelDynamicScene.h"
#include <Kernel/TraceHelper.h>
#include "Light.h"

namespace CudaTracerLib {

const KernelLight* KernelDynamicScene::sampleEmitter(float& emPdf, Vec2f& sample) const
{
	if (m_sLightData.UsedCount == 0)
		return 0;
	unsigned int idx = (unsigned int)(m_sLightData.UsedCount * sample.x);
	if (idx >= m_sLightData.UsedCount)
	{
		printf("sampled incorrect light! N = %d, sample.x = %f, idx = %d\n", m_sLightData.UsedCount, sample.x, idx);
		idx = m_sLightData.UsedCount - 1;
	}
	sample.x = sample.x - idx / float(m_sLightData.UsedCount);
	emPdf = 1.0f / float(m_sLightData.UsedCount);
	return m_sLightData.Data + idx;
}

float KernelDynamicScene::pdfEmitterDiscrete(const KernelLight *emitter) const
{
	return 1.0f / float(m_sLightData.UsedCount);
}

bool KernelDynamicScene::Occluded(const Ray& r, float tmin, float tmax, TraceResult* res) const
{
	const float eps = 0.01f;//remember this is an occluded test, so we shrink the interval!
	TraceResult r2 = Traceray(r);
	if (r2.hasHit() && res)
		*res = r2;
	bool end = r2.m_fDist < tmax * (1.0f - eps);
	if (isinf(tmax) && !r2.hasHit())
		end = false;
	return r2.m_fDist > tmin * (1.0f + eps) && end;
	//return tmin < r2.m_fDist && r2.m_fDist < tmax;
}

Spectrum KernelDynamicScene::evalTransmittance(const Vec3f& p1, const Vec3f& p2) const
{
	Vec3f d = p2 - p1;
	float l = d.length();
	d /= l;
	return (-m_sVolume.tau(Ray(p1, d), 0, l)).exp();
}

Spectrum KernelDynamicScene::EvalEnvironment(const Ray& r) const
{
	if (m_uEnvMapIndex != UINT_MAX)
		return m_sLightData[m_uEnvMapIndex].As<InfiniteLight>()->evalEnvironment(r);
	else return Spectrum(0.0f);
}

Spectrum KernelDynamicScene::EvalEnvironment(const Ray& r, const Ray& rX, const Ray& rY) const
{
	if (m_uEnvMapIndex != UINT_MAX)
		return m_sLightData[m_uEnvMapIndex].As<InfiniteLight>()->evalEnvironment(r, rX, rY);
	else return Spectrum(0.0f);
}

Spectrum KernelDynamicScene::sampleEmitterDirect(DirectSamplingRecord &dRec, const Vec2f &_sample) const
{
	Vec2f sample = _sample;
	float emPdf;
	const KernelLight *emitter = sampleEmitter(emPdf, sample);
	if (emitter == 0)
	{
		dRec.pdf = 0;
		dRec.object = 0;
		return 0.0f;
	}
	Spectrum value = emitter->sampleDirect(dRec, sample);
	if (dRec.pdf != 0)
	{
		dRec.pdf *= emPdf;
		value /= emPdf;
		dRec.object = emitter;
		return value;
	}
	else
	{
		return Spectrum(0.0f);
	}
}

Spectrum KernelDynamicScene::sampleAttenuatedEmitterDirect(DirectSamplingRecord &dRec, const Vec2f &_sample) const
{
	Spectrum value = sampleEmitterDirect(dRec, _sample);
	return value * evalTransmittance(dRec.ref, dRec.p);
}

Spectrum KernelDynamicScene::sampleSensorDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	Spectrum value = m_Camera.sampleDirect(dRec, sample);
	if (dRec.pdf != 0)
	{
		dRec.object = &g_SceneData;
		return value;
	}
	else
	{
		return Spectrum(0.0f);
	}
}

Spectrum KernelDynamicScene::sampleAttenuatedSensorDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	Spectrum value = sampleSensorDirect(dRec, sample);
	return value * evalTransmittance(dRec.ref, dRec.p);
}

float KernelDynamicScene::pdfEmitterDirect(const DirectSamplingRecord &dRec) const
{
	const KernelLight *emitter = (KernelLight*)dRec.object;
	return emitter->pdfDirect(dRec) * pdfEmitterDiscrete(emitter);
}

float KernelDynamicScene::pdfSensorDirect(const DirectSamplingRecord &dRec) const
{
	return m_Camera.pdfDirect(dRec);
}

Spectrum KernelDynamicScene::sampleEmitterPosition(PositionSamplingRecord &pRec, const Vec2f &_sample) const
{
	Vec2f sample = _sample;
	float emPdf;
	const KernelLight *emitter = sampleEmitter(emPdf, sample);

	Spectrum value = emitter->samplePosition(pRec, sample);

	pRec.object = emitter;
	pRec.pdf *= emPdf;

	return value / emPdf;
}

Spectrum KernelDynamicScene::sampleSensorPosition(PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	pRec.object = &m_Camera;
	return m_Camera.samplePosition(pRec, sample, extra);
}

float KernelDynamicScene::pdfEmitterPosition(const PositionSamplingRecord &pRec) const
{
	const KernelLight *emitter = (const KernelLight*)pRec.object;
	return emitter->pdfPosition(pRec) * pdfEmitterDiscrete(emitter);
}

float KernelDynamicScene::pdfSensorPosition(const PositionSamplingRecord &pRec) const
{
	const Sensor *sensor = (const Sensor*)pRec.object;
	return sensor->pdfPosition(pRec);
}

Spectrum KernelDynamicScene::sampleEmitterRay(Ray& ray, const KernelLight*& emitter, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	Vec2f sample = spatialSample;
	float emPdf;
	emitter = sampleEmitter(emPdf, sample);

	return emitter->sampleRay(ray, sample, directionalSample) / emPdf;
}

Spectrum KernelDynamicScene::sampleSensorRay(Ray& ray, const Sensor*& sensor, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	sensor = &m_Camera;
	return sensor->sampleRay(ray, spatialSample, directionalSample);
}

}
