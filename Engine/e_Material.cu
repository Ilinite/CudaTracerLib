#include "hip/hip_runtime.h"
#include "e_Material.h"

e_KernelMaterial::e_KernelMaterial(const char* name)
{
	memset(Name, 0, sizeof(Name));
	if(name)
		memcpy(Name, name, strlen(name));
	HeightScale = 1.0f;
	NodeLightIndex = 0xffffffff;
	m_fAlphaThreshold = 1.0f;
	bsdf.type = 0;
	usedBssrdf = false;
}

CUDA_FUNC_IN float3 nor(float* D, int l, int t, int m, int r, int b, float HeightScale)
{
	//return normalize(cross(make_float3(0, -1, D[t] - D[m]), make_float3(-1, 0, D[l] - D[m])) + cross(make_float3(0, 1, D[b] - D[m]), make_float3(1, 0, D[r] - D[m])));
	return normalize(make_float3(D[m]-D[l], D[m]-D[t], HeightScale));
}

bool e_KernelMaterial::SampleNormalMap(const MapParameters& uv, float3* normal) const
{
	if(NormalMap.used)
	{
		float3 n;
		NormalMap.tex.Evaluate(uv).toLinearRGB(n.x,n.y,n.z);
		*normal = n * 2.0f - make_float3(1);
		return true;
	}
	else if(HeightMap.used)
	{
		float d = 1.0f / 256;//fucked up guess
		float m[16];
		for(int i = 0; i < 4; i++)
			for(int j = 0; j < 4; j++)
			{
				MapParameters mp = uv;
				*(float2*)&mp.uv = mp.uv + make_float2(i - 1, j - 1) * d;
				m[i * 4 + j] = HeightMap.tex.Evaluate(mp).average();
			}
		*normal = nor(m, 4, 1, 5, 6, 9, HeightScale); 
		return true;
	}
	else return false;
}

float e_KernelMaterial::SampleAlphaMap(const MapParameters& uv) const
{
	if(AlphaMap.used)
	{//return 1;
		if(AlphaMap.tex.type == e_KernelImageTexture_TYPE)
		{
			return AlphaMap.tex.As<e_KernelImageTexture>()->tex->SampleAlpha(uv.uv) != 1 ? 0 : 1;
		}
		Spectrum s = AlphaMap.tex.Evaluate(uv);
		if(s.isZero())
			return 0.0f;
		else return 1.0f;
	}
	else return 1.0f;
}

bool e_KernelMaterial::GetBSSRDF(const MapParameters& uv, const e_KernelBSSRDF** res) const
{
	*res = &bssrdf;
	return usedBssrdf;
}

void e_KernelMaterial::setBssrdf(const Spectrum& sig_a, const Spectrum& sigp_s, float e)
{
	usedBssrdf = true;
	bssrdf.e = e;
	bssrdf.sig_a = sig_a;
	bssrdf.sigp_s = sigp_s;
}
