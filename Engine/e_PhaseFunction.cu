#include "hip/hip_runtime.h"
#include "e_PhaseFunction.h"
#include <Base/CudaRandom.h>
#include "e_Samples.h"

float e_HGPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	float temp = 1.0f + m_g*m_g + 2.0f * m_g * dot(pRec.wi, pRec.wo);
	return (1.0f / (4.0f * PI)) * (1 - m_g*m_g) / (temp * math::sqrt(temp));
}

float e_HGPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	Vec2f sample = sampler.randomFloat2();

	float cosTheta;
	if (math::abs(m_g) < EPSILON)
	{
		cosTheta = 1 - 2*sample.x;
	}
	else
	{
		float sqrTerm = (1 - m_g * m_g) / (1 - m_g + 2 * m_g * sample.x);
		cosTheta = (1 + m_g * m_g - sqrTerm * sqrTerm) / (2 * m_g);
	}

	float sinTheta = math::sqrt(1.0f-cosTheta*cosTheta), sinPhi, cosPhi;

	sincos(2*PI*sample.y, &sinPhi, &cosPhi);

	pRec.wo = Frame(-pRec.wi).toWorld(Vec3f(
		sinTheta * cosPhi,
		sinTheta * sinPhi,
		cosTheta
	));

	return 1.0f;
}

float e_HGPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	e_HGPhaseFunction::Sample(pRec, sampler);
	pdf = e_HGPhaseFunction::Evaluate(pRec);
	return 1.0f;
}

float e_IsotropicPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	return Warp::squareToUniformSpherePdf();
}

float e_IsotropicPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	Vec2f sample = sampler.randomFloat2();
	pRec.wo = Warp::squareToUniformSphere(sample);
	return 1.0f;
}

float e_IsotropicPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	pRec.wo = Warp::squareToUniformSphere(sampler.randomFloat2());
	pdf = Warp::squareToUniformSpherePdf();
	return 1.0f;
}

e_KajiyaKayPhaseFunction::e_KajiyaKayPhaseFunction(float ks, float kd, float e, Vec3f o)
	: e_BasePhaseFunction(EPhaseFunctionType::pEAnisotropic), m_ks(ks), m_kd(kd), m_exponent(e), orientation(o)
{
	int nParts = 1000;
	float stepSize = PI / nParts, m=4, theta = stepSize;

	m_normalization = 0; /* 0 at the endpoints */
	for (int i=1; i<nParts; ++i) {
		float value = math::pow(cosf(theta - PI/2), m_exponent)
			* sinf(theta);
		m_normalization += value * m;
		theta += stepSize;
		m = 6-m;
	}

	m_normalization = 1/(m_normalization * stepSize/3 * 2 * PI);
}

void e_KajiyaKayPhaseFunction::Update()
{
	int nParts = 1000;
	float stepSize = PI / nParts, m=4, theta = stepSize;

	m_normalization = 0; /* 0 at the endpoints */
	for (int i=1; i<nParts; ++i) {
		float value = math::pow(cosf(theta - PI/2), m_exponent)
			* sinf(theta);
		m_normalization += value * m;
		theta += stepSize;
		m = 6-m;
	}

	m_normalization = 1/(m_normalization * stepSize/3 * 2 * PI);
}

float e_KajiyaKayPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	if (length(orientation) == 0)
		return m_kd / (4*PI);

	Frame frame(normalize(orientation));
	Vec3f reflectedLocal = frame.toLocal(pRec.wo);

	reflectedLocal.z = -dot(pRec.wi, frame.n);
	float a = math::sqrt((1-reflectedLocal.z*reflectedLocal.z) / (reflectedLocal.x*reflectedLocal.x + reflectedLocal.y*reflectedLocal.y));
	reflectedLocal.y *= a;
	reflectedLocal.x *= a;
	Vec3f R = frame.toWorld(reflectedLocal);

	return math::pow(max(0.0f, dot(R, pRec.wo)), m_exponent) * m_normalization * m_ks + m_kd / (4*PI);
}

float e_KajiyaKayPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	pRec.wo = Warp::squareToUniformSphere(sampler.randomFloat2());
	return Evaluate(pRec) * (4 * PI);
}

float e_KajiyaKayPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	pRec.wo = Warp::squareToUniformSphere(sampler.randomFloat2());
	pdf = Warp::squareToUniformSpherePdf();
	return Evaluate(pRec) * (4 * PI);
}

float e_RayleighPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	float mu = dot(pRec.wi, pRec.wo);
	return (3.0f/(16.0f*PI)) * (1+mu*mu);
}

float e_RayleighPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	Vec2f sample(sampler.randomFloat2());

	float z = 2 * (2*sample.x - 1),
			tmp = math::sqrt(z*z+1),
			A = math::pow(z+tmp, (float) (1.0f/3.0f)),
			B = math::pow(z-tmp, (float) (1.0f/3.0f)),
			cosTheta = A + B,
			sinTheta = math::sqrt(1.0f-cosTheta*cosTheta),
			phi = 2*PI*sample.y, cosPhi, sinPhi;
	sincos(phi, &sinPhi, &cosPhi);

	Vec3f dir = Vec3f(
		sinTheta * cosPhi,
		sinTheta * sinPhi,
		cosTheta);

	pRec.wo = Frame(-pRec.wi).toWorld(dir);
	return 1.0f;
}

float e_RayleighPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	e_RayleighPhaseFunction::Sample(pRec, sampler);
	pdf = e_RayleighPhaseFunction::Evaluate(pRec);
	return 1.0f;
}