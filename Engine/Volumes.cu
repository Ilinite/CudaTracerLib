#include "hip/hip_runtime.h"
#include "Buffer.h"
#include "Volumes.h"
#include <Base/CudaRandom.h>
#include "Samples.h"
#include "Grid.h"

namespace CudaTracerLib {

bool BaseVolumeRegion::IntersectP(const Ray &ray, const float minT, const float maxT, float *t0, float *t1) const
{
	Ray r = ray * WorldToVolume;
	bool b = AABB(Vec3f(0), Vec3f(1)).Intersect(r, t0, t1);
	if(b)
	{
		*t0 = math::clamp(*t0, minT, maxT);
		*t1 = math::clamp(*t1, minT, maxT);
	}
	return b && *t1 > *t0 && *t1 > 0;
}

Spectrum HomogeneousVolumeDensity::tau(const Ray &ray, const float minT, const float maxT) const
{
	float t0, t1;
	if (!IntersectP(ray, minT, maxT, &t0, &t1))
	{
		return Spectrum(0.0f);
	}
	return length(ray(t0) - ray(t1)) * (sig_a + sig_s);
}

bool HomogeneousVolumeDensity::sampleDistance(const Ray& ray, float minT, float maxT, float rand, MediumSamplingRecord& mRec) const
{
	float m_mediumSamplingWeight = -1;
	Spectrum sig_t = sig_a + sig_s, albedo = sig_s / sig_t;
	for (int i = 0; i < 2; i++)
		if (albedo[i] > m_mediumSamplingWeight && sig_t[i] != 0)
			m_mediumSamplingWeight = albedo[i];
	if (m_mediumSamplingWeight > 0)
		m_mediumSamplingWeight = max(m_mediumSamplingWeight, 0.5f);
	float sampledDistance = FLT_MAX;
	int channel = int(rand * SPECTRUM_SAMPLES);
	rand = (rand - channel * 1.0f / SPECTRUM_SAMPLES) * SPECTRUM_SAMPLES;
	if (rand < m_mediumSamplingWeight)
	{
		rand /= m_mediumSamplingWeight;
		float samplingDensity = sig_t[channel];
		sampledDistance = -logf(1 - rand) / samplingDensity;
	}
	bool success = true;
	if (sampledDistance < maxT - minT)
	{
		mRec.t = minT + sampledDistance;
		mRec.p = ray(mRec.t);
		mRec.sigmaA = sig_a;
		mRec.sigmaS = sig_s;
		if (mRec.p == ray.origin)
			success = false;
	}
	else
	{
		sampledDistance = maxT - minT;
		success = false;
	}

	Spectrum t = (-sig_t * sampledDistance).exp();
	mRec.pdfFailure = t.average();
	mRec.pdfSuccess = (sig_t * t).average();
	mRec.transmittance = (sig_t * (-sampledDistance)).exp();
	mRec.pdfSuccessRev = mRec.pdfSuccess = mRec.pdfSuccess * m_mediumSamplingWeight;
	mRec.pdfFailure = m_mediumSamplingWeight * mRec.pdfFailure + (1 - m_mediumSamplingWeight);
	if (mRec.transmittance.max() < 1e-10f)
		mRec.transmittance = Spectrum(0.0f);

	return success;
}

DenseVolGridBaseType::DenseVolGridBaseType(Stream<char>* a_Buffer, Vec3u dim, size_t sizePerElement, size_t alignment)
{
	StreamReference<char> streamRef = a_Buffer->malloc_aligned(dim.x * dim.y * dim.z * (unsigned int)sizePerElement, (unsigned int)alignment);
	data = streamRef.AsVar<char>();
}

void DenseVolGridBaseType::InvalidateDeviceData(Stream<char>* a_Buffer)
{
	a_Buffer->translate(data).Invalidate();
}

VolumeGrid::VolumeGrid(const PhaseFunction& func, const float4x4& ToWorld, Stream<char>* a_Buffer, Vec3u dim)
	: grid(a_Buffer, dim), singleGrid(true)
{
	VolumeToWorld = ToWorld;
	BaseVolumeRegion::Func = func;
	sigAMin = sigSMin = leMin = Spectrum(0.0f);
	sigAMax = sigSMax = leMax = Spectrum(1.0f);
	Update();
}

VolumeGrid::VolumeGrid(const PhaseFunction& func, const float4x4& ToWorld, Stream<char>* a_Buffer, Vec3u dimA, Vec3u dimS, Vec3u dimL)
	: gridA(a_Buffer, dimA), gridS(a_Buffer, dimS), gridL(a_Buffer, dimL), singleGrid(false)
{
	VolumeToWorld = ToWorld;
	BaseVolumeRegion::Func = func;
	sigAMin = sigSMin = leMin = Spectrum(0.0f);
	sigAMax = sigSMax = leMax = Spectrum(1.0f);
	Update();
}

void VolumeGrid::Update()
{
	BaseVolumeRegion::Update();
	float dimf[] = { (float)grid.dim.x - 1, (float)grid.dim.y - 1, (float)grid.dim.z - 1 };
	if (!singleGrid)
	{
		uint3 dims[] = {gridA.dim, gridS.dim, gridL.dim};
		dimf[0] = dimf[1] = dimf[2] = 0;
		for (int i = 0; i < 3; i++)
		{
			dimf[0] = max(dimf[0], float(dims[i].x - 1));
			dimf[1] = max(dimf[1], float(dims[i].y - 1));
			dimf[2] = max(dimf[2], float(dims[i].z - 1));
		}
	}
	m_stepSize = FLT_MAX;
	Vec3f size = VolumeToWorld.Scale();
	for (int i = 0; i < 3; i++)
		m_stepSize = min(m_stepSize, size[i] / dimf[i]);
	m_stepSize /= 2.0f;
}

Spectrum VolumeGrid::tau(const Ray &ray, const float minT, const float maxT) const
{
	float t0, t1;
	float length = CudaTracerLib::length(ray.direction);
	if (length == 0.f) return 0.f;
	Ray rn(ray.origin, ray.direction / length);
	if (!IntersectP(rn, minT * length, maxT * length, &t0, &t1)) return 0.;
	float f = integrateDensity(rn, t0, t1);
	return sigAMin + (sigAMax - sigAMin) * f + sigSMin + (sigSMax - sigSMin) * f;
	/*
	Spectrum tau(0.);
	t0 += u * stepSize;
	while (t0 < t1) {
		tau += sigma_t(rn(t0), -rn.direction);
		t0 += stepSize;
	}
	return tau * stepSize;*/
	/*float t0, t1;
	if (!IntersectP(ray, minT, maxT, &t0, &t1))
		return Spectrum(0.0f);
	Spectrum tau(0.0f);
	int N = 10;
	float step = (t1 - t0) / float(N);
	t0 += step / 2;
	int i = -1;
	while (t0 < t1 && i++ < N)
	{
		tau += sigma_t(ray(t0), -ray.direction);
		t0 += step;
	}
	return tau * step;*/
}

float VolumeGrid::integrateDensity(const Ray& ray, float t0, float t1) const
{
#if 0
	float length = t1 - t0, maxComp = 0;
	Vec3f p = ray(t0), pLast = ray(t1);
	float pf[] = { p.x, p.y, p.z };
	float pLastf[] = { pLast.x, pLast.y, pLast.z };
	for (int i = 0; i<3; ++i)
		maxComp = max(max(maxComp, math::abs(pf[i])), math::abs(pLastf[i]));
	if (length < 1e-6f * maxComp)
		return 0.0f;
	float m_scale = 1.0f;
	unsigned int nSteps = (unsigned int)ceilf(length / (2 * m_stepSize));
	nSteps += nSteps % 2;
	float stepSize = length / nSteps;
	const Vec3f increment = ray.direction * stepSize;
	float integratedDensity = densityT(p) + densityT(pLast);
	p += increment;
	float m = 4;
	for (unsigned int i = 1; i < nSteps; ++i)
	{
		integratedDensity += m * densityT(p);
		m = 6 - m;
		Vec3f next = p + increment;
		if (p == next)
		{
			printf("integrateDensity() not stepping forward, stepsize = %f.\n", stepSize);
			break;
		}
		p = next;
	}
	return integratedDensity * m_scale * stepSize * (1.0f / 3.0f);
#endif

	Ray rayL = ray * WorldToVolume;
	float Td = rayL.direction.length();
	float minTL = t0 * Td, maxTL = t1 * Td;
	rayL.direction.normalize();
	float integratedDensity = 0;
	TraverseGrid(rayL, minTL, maxTL, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
	{
		float d0 = 2 * grid.sampleTrilinear(grid.dimF * rayL(rayT)), d1 = 2 * grid.sampleTrilinear(grid.dimF * rayL(cellEndT)), d = (d0 + d1) / 2.0f;
		integratedDensity += d * (cellEndT - rayT);
		/*
		//Performs analytic integration over the cell, somewhat buggy
		unsigned char xFlag = 0x66, zFlag = 0xcc;
		float V[8];
		for (int i = 0; i < 8; i++)
			V[i] = grid.value(cell_pos.x + ((xFlag >> i) & 1), cell_pos.y + i / 4, cell_pos.z + ((zFlag >> i) & 1));
		Vec3f r = rayL(rayT) * grid.dimF - Vec3f(cell_pos.x, cell_pos.y, cell_pos.z);
		float t1 = cellEndT - rayT, t2 = t1 * t1, t3 = t1 * t1 * t1;
		Vec3f R = r * t1 + 0.5f * rayL.direction * t2;
		float Rxy = r.x * r.y * t1 + 0.5f * r.x * rayL.direction.y * t2 + 0.5f * r.y * rayL.direction.x * t2 + 1.0f / 3.0f * rayL.direction.x * rayL.direction.y * t3;
		float Ryz = r.y * r.z * t1 + 0.5f * r.y * rayL.direction.z * t2 + 0.5f * r.z * rayL.direction.y * t2 + 1.0f / 3.0f * rayL.direction.y * rayL.direction.z * t3;
		float Rxz = r.x * r.z * t1 + 0.5f * r.x * rayL.direction.z * t2 + 0.5f * r.z * rayL.direction.x * t2 + 1.0f / 3.0f * rayL.direction.x * rayL.direction.z * t3;
		integratedDensity += t1 * V[0] + (V[1] - V[0]) * R.x + (V[3] - V[0]) * R.z + (V[2] + V[0] - V[3] - V[1]) * Rxz;
		integratedDensity += (V[0] - V[4]) * R.y + (V[1] - V[0] - V[5] + V[4]) * Rxy + (V[3] - V[0] - V[7] + V[4]) * Ryz;*/
	}, AABB(Vec3f(0), Vec3f(1)), grid.dimF);
	return integratedDensity * (t1 - t0) / (maxTL - minTL);

}

bool VolumeGrid::invertDensityIntegral(const Ray& ray, float t0, float t1, float desiredDensity,
										 float &integratedDensity, float &t, float &densityAtMinT, float &densityAtT) const
{

	integratedDensity = densityAtMinT = densityAtT = 0.0f;
	Ray rayL = ray * WorldToVolume;
	float Td = rayL.direction.length();
	float minTL = t0 * Td, maxTL = t1 * Td;
	rayL.direction.normalize();
	bool found = false;
	densityAtMinT = sigma_t(ray(t0), Vec3f(0)).average();
	TraverseGrid(rayL, minTL, maxTL, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
	{
		float d0 = grid.sampleTrilinear(grid.dimF * rayL(rayT)) * (sigAMax + sigSMax).average(),
			  d1 = grid.sampleTrilinear(grid.dimF * rayL(cellEndT)) * (sigAMax + sigSMax).average(),
			  d = (d0 + d1) / 2.0f;
		float D = d * (cellEndT - rayT) * (t1 - t0) / (maxTL - minTL);
		if(integratedDensity + D > desiredDensity)
		{
			densityAtT = d;
			t = (desiredDensity - integratedDensity) / d + rayT * (t1 - t0) / (maxTL - minTL);
			integratedDensity = desiredDensity;
			found = true;
			cancelTraversal = true;
		}
		else integratedDensity += D;
	}, AABB(Vec3f(0), Vec3f(1)), grid.dimF);
	return found;

#if 0
	integratedDensity = densityAtMinT = densityAtT = 0.0f;
	float length = t1 - t0, maxComp = 0;
	Vec3f p = ray(t0), pLast = ray(t1);
	for (int i = 0; i<3; ++i)
		maxComp = max(max(maxComp, math::abs(p[i])), math::abs(pLast[i]));
	if (length < 1e-6f * maxComp)
		return 0.0f;
	float m_scale = 1;
	unsigned int nSteps = (unsigned int)ceilf(length / (2 * m_stepSize));
	float stepSize = length / nSteps, multiplier = (1.0f / 6.0f) * stepSize * m_scale;
	Vec3f fullStep = ray.direction * stepSize, halfStep = fullStep * .5f;
	float node1 = sigma_t(p, Vec3f(0)).average();
	densityAtMinT = node1 * m_scale;
	for (unsigned int i = 0; i < nSteps; ++i)
	{
		float node2 = sigma_t(p + halfStep, Vec3f(0.0f)).average(),
			  node3 = sigma_t(p + fullStep, Vec3f(0.0f)).average();
		float newDensity = integratedDensity + multiplier * (node1 + node2 * 4 + node3);
		if (newDensity >= desiredDensity)
		{
			/*float a = 0, b = stepSize, x = a,
				fx = integratedDensity - desiredDensity,
				stepSizeSqr = stepSize * stepSize,
				temp = m_scale / stepSizeSqr;
			int it = 1;
			while (true)
			{
				float dfx = temp * (node1 * stepSizeSqr
					- (3 * node1 - 4 * node2 + node3)*stepSize*x
					+ 2 * (node1 - 2 * node2 + node3)*x*x);
				x -= fx / dfx;
				if (x <= a || x >= b || dfx == 0)
					x = 0.5f * (b + a);
				float intval = integratedDensity + temp * (1.0f / 6.0f) * (x *
					(6 * node1*stepSizeSqr - 3 * (3 * node1 - 4 * node2 + node3)*stepSize*x
					+ 4 * (node1 - 2 * node2 + node3)*x*x));
				fx = intval - desiredDensity;

				if (math::abs(fx) < 1e-3f) {
					t = minT + stepSize * i + x;
					integratedDensity = intval;
					densityAtT = temp * (node1 * stepSizeSqr
						- (3 * node1 - 4 * node2 + node3)*stepSize*x
						+ 2 * (node1 - 2 * node2 + node3)*x*x);
					return true;
				}
				else if (++it > 30)
				{
					printf("invertDensityIntegral(): stuck in Newton-Bisection -- "
							"round-off error issues? The step size was %e, fx=%f, dfx=%f, "
							"a=%f, b=%f", stepSize, fx, dfx, a, b);
					return false;
				}
				if (fx > 0)
					b = x;
				else
					a = x;
			}*/
			float V = desiredDensity - integratedDensity, s = (node3 - node1) / (2 * stepSize);
			float r = (node1 * node1) / (4 * stepSize * stepSize) + V / stepSize;
			if (r < 0)
				printf("r = %f\n", r);
			float tl = -node1 / (2 * stepSize) + math::sqrt(r);
			t = t0 + stepSize * i + tl;
			integratedDensity = desiredDensity;
			densityAtT = s * tl + node1;
			return true;
		}
		Vec3f next = p + fullStep;
		if (p == next)
		{
			printf("invertDensityIntegral() not stepping forward, stepsize = %f.\n", stepSize);
			break;
		}
		integratedDensity = newDensity;
		node1 = node3;
		p = next;
	}
	return false;
#endif
}

bool VolumeGrid::sampleDistance(const Ray& ray, float minT, float maxT, float sample, MediumSamplingRecord& mRec) const
{
	float t0, t1;
	float length = CudaTracerLib::length(ray.direction);
	if (length == 0.f) return 0.f;
	Ray rn(ray.origin, ray.direction / length);
	if (!IntersectP(rn, minT * length, maxT * length, &t0, &t1)) return 0.;
	float integratedDensity, densityAtMinT, densityAtT;
	float desiredDensity = -logf(1 - sample);
	bool success = false;
	if (invertDensityIntegral(rn, t0, t1, desiredDensity, integratedDensity, mRec.t, densityAtMinT, densityAtT))
	{
		success = true;
		mRec.p = ray(mRec.t);
		mRec.sigmaS = sigma_s(mRec.p, -ray.direction);
		mRec.sigmaA = sigma_s(mRec.p, -ray.direction);
	}
	float expVal = math::exp(-integratedDensity);
	mRec.pdfFailure = expVal;
	mRec.pdfSuccess = expVal * densityAtT;
	mRec.pdfSuccessRev = expVal * densityAtMinT;
	mRec.transmittance = Spectrum(expVal);
	return success && mRec.pdfSuccess > 0;
}

bool KernelAggregateVolume::IntersectP(const Ray &ray, float minT, float maxT, float *t0, float *t1) const
{
	*t0 = FLT_MAX;
	*t1 = -FLT_MAX;
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
	{
		float a, b;
		if (m_pVolumes[i].IntersectP(ray, minT, maxT, &a, &b))
		{
			*t0 = min(*t0, a);
			*t1 = max(*t1, b);
		}
	}
	return (*t0 < *t1);
}

Spectrum KernelAggregateVolume::sigma_a(const Vec3f& p, const Vec3f& w) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].sigma_a(p, w);
	return s;
}

Spectrum KernelAggregateVolume::sigma_s(const Vec3f& p, const Vec3f& w) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].sigma_s(p, w);
	return s;
}

Spectrum KernelAggregateVolume::Lve(const Vec3f& p, const Vec3f& w) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].Lve(p, w);
	return s;
}

Spectrum KernelAggregateVolume::sigma_t(const Vec3f &p, const Vec3f &wo) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].sigma_t(p, wo);
	return s;
}

Spectrum KernelAggregateVolume::tau(const Ray &ray, float minT, float maxT) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].tau(ray, minT, maxT);
	return s;
}

float KernelAggregateVolume::Sample(const Vec3f& p, const Vec3f& wo, CudaRNG& rng, Vec3f* wi)
{
	PhaseFunctionSamplingRecord r2(wo);
	r2.wi = wo;
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
		if (m_pVolumes[i].WorldBound().Contains(p))
		{
			float pdf;
			float pf = m_pVolumes[i].As()->Func.Sample(r2, pdf, rng);
			*wi = r2.wo;
			return pf;
		}
		
	return 0.0f;
}

float KernelAggregateVolume::p(const Vec3f& p, const Vec3f& wo, const Vec3f& wi, CudaRNG& rng)
{
	PhaseFunctionSamplingRecord r2(wo, wi);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
		if (m_pVolumes[i].WorldBound().Contains(p))
			return m_pVolumes[i].As()->Func.Evaluate(r2);
	return 0.0f;
}

CUDA_FUNC_IN int ffsn(unsigned int v, int n) {
	for (int i = 0; i<n - 1; i++) {
		v &= v - 1; // remove the least significant bit
	}
	return v & ~(v - 1); // extract the least significant bit
}

bool KernelAggregateVolume::sampleDistance(const Ray& ray, float minT, float maxT, CudaRNG& rng, MediumSamplingRecord& mRec) const
{
	if (m_uVolumeCount == 1 && m_pVolumes[0].WorldBound().Intersect(ray))
		return m_pVolumes[0].sampleDistance(ray, minT, maxT, rng.randomFloat(), mRec);
	else if (m_uVolumeCount == 1)
		return false;

	float n = 0;
	unsigned int flag = 0;
	for (unsigned int i = 0; i < m_uVolumeCount; i++)
		if (m_pVolumes[i].WorldBound().Intersect(ray))
		{
			n++;
			flag |= 1 << i;
		}
	if (!n)
		return 0;
	float sample = rng.randomFloat();
	int nth = int(sample * n);
	int i = ffsn(flag, nth);
	return m_pVolumes[i].sampleDistance(ray, minT, maxT, rng.randomFloat(), mRec);
}

KernelAggregateVolume::KernelAggregateVolume(Stream<VolumeRegion>* D, bool devicePointer)
{
	m_uVolumeCount = 0;
	for (Stream<VolumeRegion>::iterator it = D->begin(); it != D->end(); ++it)
	{
		m_pVolumes[m_uVolumeCount] = *(*it);
		m_uVolumeCount++;
	}
	box = AABB::Identity();
	for (unsigned int i = 0; i < m_uVolumeCount; i++)
		box = box.Extend(D->operator()(i)->WorldBound());
}

}
