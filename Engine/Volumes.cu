#include "hip/hip_runtime.h"
#include "Volumes.h"
#include <Base/CudaRandom.h>
#include "Samples.h"
#include "SpatialGridTraversal.h"
#include <Math/MonteCarlo.h>

namespace CudaTracerLib {

bool BaseVolumeRegion::IntersectP(const Ray &ray, float minT, float maxT, float *t0, float *t1) const
{
	Ray r = ray * WorldToVolume;
	bool b = AABB(Vec3f(0), Vec3f(1)).Intersect<true>(r, &minT, &maxT);
	if(b)
	{
		if (t0) *t0 = minT;
		if (t1) *t1 = maxT;
	}
	return b;
}

Spectrum HomogeneousVolumeDensity::tau(const Ray &ray, float minT, float maxT) const
{
	float t0, t1;
	if (!IntersectP(ray, minT, maxT, &t0, &t1))
	{
		return Spectrum(0.0f);
	}
	return length(ray(t0) - ray(t1)) * (sig_a + sig_s);
}

bool HomogeneousVolumeDensity::sampleDistance(const Ray& ray, float minT, float maxT, float rand, MediumSamplingRecord& mRec) const
{
	//use balance heuristic
	float m_mediumSamplingWeight = -1;
	Spectrum sig_t = sig_a + sig_s, albedo = sig_s / sig_t;
	for (int i = 0; i < SPECTRUM_SAMPLES; i++)
		if (albedo[i] > m_mediumSamplingWeight && sig_t[i] != 0)
			m_mediumSamplingWeight = albedo[i];
	if (m_mediumSamplingWeight > 0)
		m_mediumSamplingWeight = max(m_mediumSamplingWeight, 0.5f);

	//sample distance for random channel
	float sampledDistance = FLT_MAX;
	if (rand < m_mediumSamplingWeight)
	{
		rand /= m_mediumSamplingWeight;

		unsigned int channel;
		MonteCarlo::sampleReuse(SPECTRUM_SAMPLES, rand, channel);

		float samplingDensity = sig_t[channel];
		sampledDistance = -math::log(1 - rand) / samplingDensity;
	}

	//fill info for sampled point
	bool success = true;
	if (sampledDistance < maxT - minT)
	{
		mRec.t = minT + sampledDistance;
		mRec.p = ray(mRec.t);
		mRec.sigmaA = sig_a;
		mRec.sigmaS = sig_s;
		if (mRec.p == ray.ori())
			success = false;
	}
	else
	{
		sampledDistance = maxT - minT;
		success = false;
	}

	//compute probabilities
	Spectrum tmp = (-sig_t * sampledDistance).exp();
	mRec.pdfFailure = tmp.avg();
	mRec.pdfSuccess = (sig_t * tmp).avg();

	mRec.transmittance = (sig_t * (-sampledDistance)).exp();
	mRec.pdfSuccessRev = mRec.pdfSuccess = mRec.pdfSuccess * m_mediumSamplingWeight;
	mRec.pdfFailure = m_mediumSamplingWeight * mRec.pdfFailure + (1 - m_mediumSamplingWeight);
	if (mRec.transmittance.max() < 1e-8f)
		mRec.transmittance = Spectrum(0.0f);

	return success;
}

VolumeGrid::VolumeGrid()
	: BaseVolumeRegion(CreateAggregate<PhaseFunction>(IsotropicPhaseFunction()), float4x4::Identity()), sigAMin(0.0f), sigSMin(0.0f), leMin(0.0f), sigAMax(0.0f), sigSMax(0.0f), leMax(0.0f),
	grid(), singleGrid(true)
{
	VolumeGrid::Update();
}

VolumeGrid::VolumeGrid(const PhaseFunction& func, const float4x4& ToWorld, Stream<char>* a_Buffer, Vec3u dim)
	: BaseVolumeRegion(func, ToWorld), sigAMin(0.0f), sigSMin(0.0f), leMin(0.0f), sigAMax(0.0f), sigSMax(0.0f), leMax(0.0f),
	  grid(a_Buffer, dim), singleGrid(true)
{
	VolumeGrid::Update();
}

VolumeGrid::VolumeGrid(const PhaseFunction& func, const float4x4& ToWorld, Stream<char>* a_Buffer, Vec3u dimA, Vec3u dimS, Vec3u dimL)
	: BaseVolumeRegion(func, ToWorld), sigAMin(0.0f), sigSMin(0.0f), leMin(0.0f), sigAMax(0.0f), sigSMax(0.0f), leMax(0.0f),
	  gridA(a_Buffer, dimA), gridS(a_Buffer, dimS), gridL(a_Buffer, dimL), singleGrid(false)
{
	VolumeGrid::Update();
}

void VolumeGrid::Update()
{
	BaseVolumeRegion::Update();
	float dimf[] = { (float)grid.dim.x - 1, (float)grid.dim.y - 1, (float)grid.dim.z - 1 };
	if (!singleGrid)
	{
		uint3 dims[] = {gridA.dim, gridS.dim, gridL.dim};
		dimf[0] = dimf[1] = dimf[2] = 0;
		for (int i = 0; i < 3; i++)
		{
			dimf[0] = max(dimf[0], float(dims[i].x - 1));
			dimf[1] = max(dimf[1], float(dims[i].y - 1));
			dimf[2] = max(dimf[2], float(dims[i].z - 1));
		}
	}
	m_stepSize = FLT_MAX;
	Vec3f size = VolumeToWorld.Scale();
	for (int i = 0; i < 3; i++)
		m_stepSize = min(m_stepSize, size[i] / dimf[i]);
	m_stepSize /= 2.0f;
}

Spectrum VolumeGrid::tau(const Ray &ray, const float minT, const float maxT) const
{
	float t0, t1;
	float length = CudaTracerLib::length(ray.dir());
	if (length == 0.f) return 0.f;
	Ray rn(ray.ori(), ray.dir() / length);
	if (!IntersectP(rn, minT * length, maxT * length, &t0, &t1)) return 0.0f;
	return integrateDensity(rn, t0, t1);
}

Spectrum VolumeGrid::integrateDensity(const Ray& ray, float t0, float t1) const
{
	Ray rayL = ray * WorldToVolume;
	float Td = rayL.dir().length();
	float minTL = t0 * Td, maxTL = t1 * Td;
	rayL.dir() = normalize(rayL.dir());
	float D_s = 0.0f, D_a = 0.0f;
	Vec3f cell_size = Vec3f(1) / grid.dimF, dir = rayL.dir() / cell_size;
	TraverseGridRay(rayL, minTL, maxTL, AABB(Vec3f(0), Vec3f(1)), grid.dimF, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
	{
		float d_s, d_a;
		if (singleGrid)
			d_s = d_a = grid.sampleTrilinear(grid.dimF * rayL(rayT)) + grid.sampleTrilinear(grid.dimF * rayL(cellEndT));
		else
		{
			d_s = gridS.sampleTrilinear(gridS.dimF * rayL(rayT)) + gridS.sampleTrilinear(gridS.dimF * rayL(cellEndT));
			d_a = gridA.sampleTrilinear(gridA.dimF * rayL(rayT)) + gridA.sampleTrilinear(gridA.dimF * rayL(cellEndT));
		}
		d_s /= 2; d_a /= 2;
		D_s += d_s * (cellEndT - rayT);
		D_a += d_a * (cellEndT - rayT);
	});
	float Lcl_To_World = (t1 - t0) / (maxTL - minTL);
	D_a *= Lcl_To_World;
	D_s *= Lcl_To_World;
	return sigAMin + (sigAMax - sigAMin) * D_s + sigSMin + (sigSMax - sigSMin) * D_a;
}

bool VolumeGrid::invertDensityIntegral(const Ray& ray, float t0, float t1, float desiredDensity,
									   float& integratedDensity, float &t, float &densityAtMinT, float &densityAtT) const
{
	integratedDensity = densityAtMinT = densityAtT = 0.0f;
	Ray rayL = ray * WorldToVolume;
	float Td = rayL.dir().length();
	float minTL = t0 * Td, maxTL = t1 * Td;
	rayL.dir() = normalize(rayL.dir());
	bool found = false;
	densityAtMinT = sigma_t(ray(t0), NormalizedT<Vec3f>(rayL.dir())).avg();
	float Lcl_To_World = (t1 - t0) / (maxTL - minTL);
	TraverseGridRay(rayL, minTL, maxTL, AABB(Vec3f(0), Vec3f(1)), grid.dimF, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
	{
		float d_s, d_a;
		if (singleGrid)
			d_s = d_a = grid.sampleTrilinear(grid.dimF * rayL(rayT)) + grid.sampleTrilinear(grid.dimF * rayL(cellEndT));
		else
		{
			d_s = gridS.sampleTrilinear(gridS.dimF * rayL(rayT)) + gridS.sampleTrilinear(gridS.dimF * rayL(cellEndT));
			d_a = gridA.sampleTrilinear(gridA.dimF * rayL(rayT)) + gridA.sampleTrilinear(gridA.dimF * rayL(cellEndT));
		}
		d_s /= 2; d_a /= 2;
		d_s = Spectrum(sigSMin + (sigSMax - sigSMin) * d_s).avg();
		d_a = Spectrum(sigAMin + (sigAMax - sigAMin) * d_s).avg();

		float D = (d_s + d_a) * (cellEndT - rayT) * Lcl_To_World;
		if (integratedDensity + D >= desiredDensity)
		{
			densityAtT = d_s + d_a;
			t = (desiredDensity - integratedDensity) / densityAtT + rayT * Lcl_To_World;
			integratedDensity = desiredDensity;
			found = true;
			cancelTraversal = true;
		}
		else
		{
			integratedDensity += D;
		}
	});
	return found;
}

bool VolumeGrid::sampleDistance(const Ray& ray, float minT, float maxT, float sample, MediumSamplingRecord& mRec) const
{
	float t0, t1;
	float length = CudaTracerLib::length(ray.dir());
	if (length == 0.f) return 0.f;
	Ray rn(ray.ori(), ray.dir() / length);
	if (!IntersectP(rn, minT * length, maxT * length, &t0, &t1)) return false;
	float integratedDensity, densityAtMinT, densityAtT;
	float desiredDensity = -logf(1 - sample);
	bool success = false;
	if (invertDensityIntegral(rn, t0, t1, desiredDensity, integratedDensity, mRec.t, densityAtMinT, densityAtT))
	{
		success = true;
		mRec.p = ray(mRec.t);
		mRec.sigmaS = sigma_s(mRec.p, NormalizedT<Vec3f>(-ray.dir()));
		mRec.sigmaA = sigma_s(mRec.p, NormalizedT<Vec3f>(-ray.dir()));
	}
	float expVal = math::exp(-integratedDensity);
	mRec.pdfFailure = expVal;
	mRec.pdfSuccess = expVal * densityAtT;
	mRec.pdfSuccessRev = expVal * densityAtMinT;
	mRec.transmittance = Spectrum(expVal);
	return success && mRec.pdfSuccess > 0;
}

bool KernelAggregateVolume::IntersectP(const Ray &ray, float minT, float maxT, float *t0, float *t1) const
{
	*t0 = FLT_MAX;
	*t1 = -FLT_MAX;
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
	{
		float a, b;
		if (m_pVolumes[i].IntersectP(ray, minT, maxT, &a, &b))
		{
			*t0 = min(*t0, a);
			*t1 = max(*t1, b);
		}
	}
	return (*t0 < *t1);
}

Spectrum KernelAggregateVolume::sigma_a(const Vec3f& p, const NormalizedT<Vec3f>& w) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].sigma_a(p, w);
	return s;
}

Spectrum KernelAggregateVolume::sigma_s(const Vec3f& p, const NormalizedT<Vec3f>& w) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].sigma_s(p, w);
	return s;
}

Spectrum KernelAggregateVolume::Lve(const Vec3f& p, const NormalizedT<Vec3f>& w) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].Lve(p, w);
	return s;
}

Spectrum KernelAggregateVolume::sigma_t(const Vec3f &p, const NormalizedT<Vec3f> &wo) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].sigma_t(p, wo);
	return s;
}

Spectrum KernelAggregateVolume::tau(const Ray &ray, float minT, float maxT) const
{
	Spectrum s = Spectrum(0.0f);
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
			s += m_pVolumes[i].tau(ray, minT, maxT);
	return s;
}

float KernelAggregateVolume::Sample(const Vec3f& p, PhaseFunctionSamplingRecord& pRec, float& pdf, const Vec2f& _sample) const
{
	Vec2f sample = _sample;
	float vol_sample_pdf = 0;
	const auto* vol = sampleVolume(Ray(p, pRec.wi), 0, FLT_MAX, sample.x, vol_sample_pdf);
	if (vol)
		return vol->As()->Func.Sample(pRec, pdf, sample);
	else return 0.0f;
}

float KernelAggregateVolume::p(const Vec3f& p, const PhaseFunctionSamplingRecord& pRec) const
{
	float ph = 0, sumWt = 0;
	for(unsigned int i = 0; i < m_uVolumeCount; i++)
		if (m_pVolumes[i].WorldBound().Contains(p))
		{
			float wt = m_pVolumes[i].sigma_s(p, pRec.wo).avg();
			sumWt += wt;
			ph += wt * m_pVolumes[i].As()->Func.Evaluate(pRec);
		}
	return sumWt != 0 ? ph / sumWt : 0.0f;
}

bool KernelAggregateVolume::sampleDistance(const Ray& ray, float minT, float maxT, float sample, MediumSamplingRecord& mRec) const
{
	float vol_sample_pdf = 0;
	const auto* vol = sampleVolume(ray, minT, maxT, sample, vol_sample_pdf);
	if(vol && vol->sampleDistance(ray, minT, maxT, sample, mRec))
	{
		//mRec.pdfSuccess *= vol_sample_pdf;
		//mRec.pdfSuccessRev *= vol_sample_pdf;
		//mRec.pdfFailure *= vol_sample_pdf;
		return true;
	}
	else return false;
}

//http://stackoverflow.com/questions/7669057/find-nth-set-bit-in-an-int
CUDA_FUNC_IN int ffsn(unsigned int v, int n) {
	for (int i = 0; i<n - 1; i++) {
		v &= v - 1; // remove the least significant bit
	}
	return v & ~(v - 1); // extract the least significant bit
}

const VolumeRegion* KernelAggregateVolume::sampleVolume(const Ray& ray, float minT, float maxT, float& sample, float& pdf) const
{
	if (m_uVolumeCount == 0)
		return 0;
	else if (m_uVolumeCount == 1)
		return m_pVolumes[0].WorldBound().Intersect<true>(ray, &minT, &maxT) ? m_pVolumes : 0;

	//find all intersecting volumes
	unsigned int n = 0;
	unsigned int flag = 0;
	for (unsigned int i = 0; i < m_uVolumeCount; i++)
	{
		float a = minT, b = maxT;
		if (m_pVolumes[i].WorldBound().Intersect<true>(ray, &a, &b))
		{
			n++;
			flag |= 1 << i;
		}
	}

	if (!n)
		return 0;

	//randomly (uniform) choose one
	unsigned int nth;
	MonteCarlo::sampleReuse(n, sample, nth);
	int i = ffsn(flag, nth);
	pdf = 1.0f / n;
	return m_pVolumes + i;
}

}
