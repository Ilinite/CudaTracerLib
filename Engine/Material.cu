#include "hip/hip_runtime.h"
#include "Material.h"

namespace CudaTracerLib {

void initbssrdf(VolumeRegion& reg)
{
	const float a = 1e10f;
	PhaseFunction func;
	func.SetData(IsotropicPhaseFunction());
	reg.SetData(HomogeneousVolumeDensity(func, (float4x4::Translate(Vec3f(0.5f)) % float4x4::Scale(Vec3f(0.5f / a))).inverse(), Spectrum(0.0f), Spectrum(0.0f), Spectrum(0.0f)));
	reg.As()->Update();
}

Material::Material()
{
	parallaxMinSamples = 10;
	parallaxMaxSamples = 50;
	enableParallaxOcclusion = false;
	Name = "NoNameMaterial";
	HeightScale = 1.0f;
	NodeLightIndex = UINT_MAX;
	m_fAlphaThreshold = 1.0f;
	bsdf.setTypeToken(0);
	usedBssrdf = 0;
	AlphaMap.used = NormalMap.used = HeightMap.used = 0;
	initbssrdf(bssrdf);
}

Material::Material(const std::string& name)
{
	parallaxMinSamples = 10;
	parallaxMaxSamples = 50;
	enableParallaxOcclusion = false;
	Name = name;
	HeightScale = 1.0f;
	NodeLightIndex = UINT_MAX;
	m_fAlphaThreshold = 1.0f;
	bsdf.setTypeToken(0);
	usedBssrdf = 0;
	AlphaMap.used = NormalMap.used = HeightMap.used = 0;
	initbssrdf(bssrdf);
}

CUDA_FUNC_IN void parallaxOcclusion(Vec2f& texCoord, KernelMIPMap* tex, const Vec3f& vViewTS, float HeightScale, int MinSamples, int MaxSamples)
{
	const Vec2f vParallaxDirection = normalize(vViewTS.getXY());
	float fLength = length(vViewTS);
	float fParallaxLength = sqrt(fLength * fLength - vViewTS.z * vViewTS.z) / vViewTS.z;
	const Vec2f vParallaxOffsetTS = vParallaxDirection * fParallaxLength * HeightScale;

	int nNumSteps = (int)math::lerp(MaxSamples, MinSamples, Frame::cosTheta(normalize(vViewTS)));
	float CurrHeight = 0.0f;
	float StepSize = 1.0 / (float)nNumSteps;
	float PrevHeight = 1.0;
	int    StepIndex = 0;
	Vec2f TexOffsetPerStep = StepSize * vParallaxOffsetTS;
	Vec2f TexCurrentOffset = texCoord;
	float  CurrentBound = 1.0;
	float  ParallaxAmount = 0.0;

	Vec2f pt1 = Vec2f(0);
	Vec2f pt2 = Vec2f(0);

	Vec2f texOffset2 = Vec2f(0);

	while (StepIndex < nNumSteps)
	{
		TexCurrentOffset -= TexOffsetPerStep;
		CurrHeight = tex->Sample(TexCurrentOffset).average();
		CurrentBound -= StepSize;
		if (CurrHeight > CurrentBound)
		{
			pt1 = Vec2f(CurrentBound, CurrHeight);
			pt2 = Vec2f(CurrentBound + StepSize, PrevHeight);

			texOffset2 = TexCurrentOffset - TexOffsetPerStep;

			StepIndex = nNumSteps + 1;
			PrevHeight = CurrHeight;
		}
		else
		{
			StepIndex++;
			PrevHeight = CurrHeight;
		}
	}
	float Delta2 = pt2.x - pt2.y;
	float Delta1 = pt1.x - pt1.y;
	float Denominator = Delta2 - Delta1;
	ParallaxAmount = Denominator != 0 ? (pt1.x * Delta2 - pt2.x * Delta1) / Denominator : 0;
	Vec2f ParallaxOffset = vParallaxOffsetTS * (1 - ParallaxAmount);
	texCoord -= ParallaxOffset;
}

bool Material::SampleNormalMap(DifferentialGeometry& dg, const Vec3f& wi) const
{
	if (NormalMap.used)
	{
		Vec3f n;
		NormalMap.tex.Evaluate(dg).toLinearRGB(n.x, n.y, n.z);
		Vec3f nWorld = dg.toWorld(n - Vec3f(0.5f));
		dg.sys.n = normalize(nWorld);
		dg.sys.t = normalize(cross(nWorld, dg.sys.s));
		dg.sys.s = normalize(cross(nWorld, dg.sys.t));
		return true;
	}
	else if (HeightMap.used && HeightMap.tex.Is<ImageTexture>())
	{
		TextureMapping2D& map = HeightMap.tex.As<ImageTexture>()->mapping;
		Vec2f uv = map.Map(dg);
		if (enableParallaxOcclusion)
		{
			parallaxOcclusion(uv, HeightMap.tex.As<ImageTexture>()->tex.operator->(), dg.toLocal(-wi), HeightScale, parallaxMinSamples, parallaxMaxSamples);
			dg.uv[map.setId] = map.TransformPointInverse(uv);
		}

		Spectrum grad[2];
		HeightMap.tex.As<ImageTexture>()->tex->evalGradient(uv, grad);
		float dDispDu = grad[0].getLuminance();
		float dDispDv = grad[1].getLuminance();
		Vec3f dpdu = dg.dpdu + dg.sys.n * (
			dDispDu - dot(dg.sys.n, dg.dpdu));
		Vec3f dpdv = dg.dpdv + dg.sys.n * (
			dDispDv - dot(dg.sys.n, dg.dpdv));

		dg.sys.n = normalize(cross(dpdu, dpdv));
		dg.sys.s = normalize(dpdu - dg.sys.n
			* dot(dg.sys.n, dpdu));
		dg.sys.t = cross(dg.sys.n, dg.sys.s);

		if (dot(dg.sys.n, dg.n) < 0)
			dg.sys.n *= -1;

		return true;
	}
	else return false;
}

float Material::SampleAlphaMap(const DifferentialGeometry& uv) const
{
	if (AlphaMap.used)
	{//return 1;
		if (AlphaMap.tex.Is<ImageTexture>())
		{
			Vec2f uv2 = AlphaMap.tex.As<ImageTexture>()->mapping.Map(uv);
			return AlphaMap.tex.As<ImageTexture>()->tex->SampleAlpha(uv2) != 1 ? 0 : 1;
		}
		Spectrum s = AlphaMap.tex.Evaluate(uv);
		if (s.isZero())
			return 0.0f;
		else return 1.0f;
	}
	else return 1.0f;
}

bool Material::GetBSSRDF(const DifferentialGeometry& uv, const VolumeRegion** res) const
{
	if (usedBssrdf)
		*res = &bssrdf;
	return !!usedBssrdf;
}

}