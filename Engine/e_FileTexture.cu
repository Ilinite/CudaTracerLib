#include "hip/hip_runtime.h"
#include "StdAfx.h"
#include "e_FileTexture.h"
#include "e_ErrorHandler.h"
#include "e_FileTextureHelper.h"

/// Integer floor function (single precision)
template <typename Scalar> CUDA_FUNC_IN int floorToInt(Scalar value) { return (int)floor(value); }

/// Integer ceil function (single precision)
template <typename Scalar> CUDA_FUNC_IN int ceilToInt(Scalar value) { return (int)ceil(value); }

CUDA_FUNC_IN float hypot2(float a, float b)
{
	return math::sqrt(a * a + b * b);
}

Spectrum e_KernelMIPMap::Texel(unsigned int level, const Vec2f& a_UV) const
{
	Vec2f l;
	if (!WrapCoordinates(a_UV, Vec2f(m_uWidth >> level, m_uHeight >> level), m_uWrapMode, &l))
		return Spectrum(0.0f);
	else
	{
		unsigned int x = (unsigned int)l.x, y = (unsigned int)l.y;
		void* data;
#ifdef ISCUDA
		data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
		data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
		Spectrum s;
		if(m_uType == vtRGBE)
			s.fromRGBE(*(RGBE*)data);
		else s.fromRGBCOL(*(RGBCOL*)data);
		return s;
	}
}

Spectrum e_KernelMIPMap::triangle(unsigned int level, const Vec2f& a_UV) const
{
	level = math::clamp(level, 0u, m_uLevels-1);
	Vec2f s = Vec2f(m_uWidth >> level, m_uHeight >> level), is = Vec2f(1) / s;
	Vec2f l = a_UV * s;// - make_float2(0.5f)
	float ds = math::frac(l.x), dt = math::frac(l.y);
	return (1.f-ds) * (1.f-dt) * Texel(level, a_UV) +
		(1.f - ds) * dt       * Texel(level, a_UV + Vec2f(0, is.y)) +
		ds       * (1.f - dt) * Texel(level, a_UV + Vec2f(is.x, 0)) +
		ds       * dt       * Texel(level, a_UV + Vec2f(is.x, is.y));
}

Spectrum e_KernelMIPMap::evalEWA(unsigned int level, const Vec2f &uv, float A, float B, float C) const
{
	if (level >= m_uLevels)
		return Texel(m_uLevels - 1, Vec2f(0));

	Vec2f size = Vec2f(m_uWidth >> level, m_uHeight >> level);
	float u = uv.x * size.x - 0.5f;
	float v = uv.y * size.y - 0.5f;

	/* Do the same to the ellipse coefficients */
	Vec2f ratio = size / m_fDim;
	A /= ratio.x * ratio.x;
	B /= ratio.x * ratio.y;
	C /= ratio.y * ratio.y;

	float invDet = 1.0f / (-B*B + 4.0f*A*C),
		deltaU = 2.0f * math::sqrt(C * invDet),
		deltaV = 2.0f * math::sqrt(A * invDet);
	int u0 = ceilToInt(u - deltaU), u1 = floorToInt(u + deltaU);
	int v0 = ceilToInt(v - deltaV), v1 = floorToInt(v + deltaV);

	float As = A * MTS_MIPMAP_LUT_SIZE,
		  Bs = B * MTS_MIPMAP_LUT_SIZE,
		  Cs = C * MTS_MIPMAP_LUT_SIZE;

	Spectrum result(0.0f);
	float denominator = 0.0f;
	float ddq = 2 * As, uu0 = u0 - u;
	int nSamples = 0;

	for (int vt = v0; vt <= v1; ++vt)
	{
		const float vv = vt - v;

		float q = As*uu0*uu0 + (Bs*uu0 + Cs*vv)*vv;
		float dq = As*(2 * uu0 + 1) + Bs*vv;

		for (int ut = u0; ut <= u1; ++ut)
		{
			if (q < MTS_MIPMAP_LUT_SIZE)
			{
				unsigned int qi = (unsigned int)q;
				if (qi < MTS_MIPMAP_LUT_SIZE)
				{
					const float weight = m_weightLut[(int)q];
					result += Texel(level, Vec2f(ut, vt) / size) * weight;
					denominator += weight;
					++nSamples;
				}
			}
			q += dq;
			dq += ddq;
		}
	}

	if (denominator == 0)
		return triangle(level, uv);
	return result / denominator;
}

Spectrum e_KernelMIPMap::Sample(const Vec2f& uv) const
{
	if (m_uFilterMode == TEXTURE_Point)
		return Texel(0, uv);
	else return triangle(0, uv);
}

float e_KernelMIPMap::SampleAlpha(const Vec2f& uv) const
{
	Vec2f l;
	if (!WrapCoordinates(uv, Vec2f(m_uWidth, m_uHeight), m_uWrapMode, &l))
		return 0.0f;
	unsigned int x = (unsigned int)l.x, y = (unsigned int)l.y, level = 0;
	void* data;
#ifdef ISCUDA
			data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
			data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
	if(m_uType == vtRGBE)
		return 1.0f;
	else return float(((RGBCOL*)data)->w) / 255.0f;
}

Spectrum e_KernelMIPMap::Sample(const Vec2f& a_UV, float width) const
{
	float level = m_uLevels - 1 + math::log2(max((float)width, 1e-8f));
	if (level < 0)
		return triangle(0, a_UV);
	else if (level >= m_uLevels - 1)
		return Texel(m_uLevels - 1, a_UV);
	else
	{
		int iLevel = math::Floor2Int(level);
		float delta = level - iLevel;
		return (1.f-delta) * triangle(iLevel, a_UV) + delta * triangle(iLevel+1, a_UV);
	}
}

Spectrum e_KernelMIPMap::Sample(float width, int x, int y) const
{
	float l = m_uLevels - 1 + math::log2(max((float)width, 1e-8f));
	int level = (int)math::clamp(l, 0.0f, float(m_uLevels - 1));
	void* data;
#ifdef ISCUDA
		data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
		data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
	Spectrum s;
	if(m_uType == vtRGBE)
		s.fromRGBE(*(RGBE*)data);
	else s.fromRGBCOL(*(RGBCOL*)data);
	return s;	
}

void e_KernelMIPMap::evalGradient(const Vec2f& uv, Spectrum* gradient) const
{
	const int level = 0;

	float u = uv.x * m_fDim.x - 0.5f, v = uv.y * m_fDim.y - 0.5f;

	int xPos = math::Float2Int(u), yPos = math::Float2Int(v);
	float dx = u - xPos, dy = v - yPos;

	const Spectrum p00 = Texel(level, Vec2f(xPos, yPos) / m_fDim);
	const Spectrum p10 = Texel(level, Vec2f(xPos + 1, yPos) / m_fDim);
	const Spectrum p01 = Texel(level, Vec2f(xPos, yPos + 1) / m_fDim);
	const Spectrum p11 = Texel(level, Vec2f(xPos + 1, yPos + 1) / m_fDim);
	Spectrum tmp = p01 + p10 - p11;

	gradient[0] = (p10 + p00*(dy - 1) - tmp*dy) * m_fDim.x;
	gradient[1] = (p01 + p00*(dx - 1) - tmp*dx) * m_fDim.y;
}

Spectrum e_KernelMIPMap::eval(const Vec2f& uv, const Vec2f& d0, const Vec2f& d1) const
{
	if (m_uFilterMode == TEXTURE_Point)
		return Texel(0, uv);
	else if (m_uFilterMode == TEXTURE_Bilinear)
		return triangle(0, uv);

	/* Convert into texel coordinates */
	float du0 = d0.x * m_fDim.x, dv0 = d0.y * m_fDim.y,
		  du1 = d1.x * m_fDim.x, dv1 = d1.y * m_fDim.y;

	/* Turn the texture-space Jacobian into the coefficients of an
	implicitly defined ellipse. */
	float A = dv0*dv0 + dv1*dv1,
		B = -2.0f * (du0*dv0 + du1*dv1),
		C = du0*du0 + du1*du1,
		F = A*C - B*B*0.25f;

	float root = hypot2(A - C, B),
		Aprime = 0.5f * (A + C - root),
		Cprime = 0.5f * (A + C + root),
		majorRadius = Aprime != 0 ? math::sqrt(F / Aprime) : 0,
		minorRadius = Cprime != 0 ? math::sqrt(F / Cprime) : 0;

	if (!(minorRadius > 0) || !(majorRadius > 0) || F < 0)
	{
		float level = log2f(max(majorRadius, 1e-4f));
		int ilevel = math::Floor2Int(level);
		if (ilevel < 0)
			return triangle(0, uv);
		else
		{
			float a = level - ilevel;
			return triangle(ilevel, uv) * (1.0f - a)
				 + triangle(ilevel + 1, uv) * a;
		}
	}
	else
	{
		const float m_maxAnisotropy = 16;
		if (minorRadius * m_maxAnisotropy < majorRadius)
		{
			minorRadius = majorRadius / m_maxAnisotropy;
			float theta = 0.5f * std::atan(B / (A - C)), sinTheta, cosTheta;
			sincos(theta, &sinTheta, &cosTheta);
			float a2 = majorRadius*majorRadius,
				b2 = minorRadius*minorRadius,
				sinTheta2 = sinTheta*sinTheta,
				cosTheta2 = cosTheta*cosTheta,
				sin2Theta = 2 * sinTheta*cosTheta;

			A = a2*cosTheta2 + b2*sinTheta2;
			B = (a2 - b2) * sin2Theta;
			C = a2*sinTheta2 + b2*cosTheta2;
			F = a2*b2;
		}
		/* Switch to normalized coefficients */
		float scale = 1.0f / F;
		A *= scale; B *= scale; C *= scale;
		/* Determine a suitable MIP map level, such that the filter
		covers a reasonable amount of pixels */
		float level = max(0.0f, log2f(minorRadius));
		int ilevel = (int)level;
		float a = level - ilevel;

		/* Switch to bilinear interpolation, be wary of round-off errors */
		if (majorRadius < 1 || !(A > 0 && C > 0))
			return triangle(ilevel, uv);
		else
			return evalEWA(ilevel, uv, A, B, C) * (1.0f - a) +
				   evalEWA(ilevel + 1, uv, A, B, C) * a;
	}
}

struct MapPoint
{
	CUDA_FUNC_IN Vec2f cubizePoint4(Vec3f& position, int& face)
	{
		Vec3f q = position.abs();
		if(q.x > q.y && q.x > q.z)
			face = 0;
		else if(q.y > q.z)
			face = 1;
		else face = 2;
		int f = face;
		float* val = (float*)&position;
		face = 2 * face + (val[face] > 0 ? 0 : 1);

		int2 uvIdxs[3] = {make_int2(2, 1), make_int2(0, 2), make_int2(0, 1)};
		float sc = val[uvIdxs[f].x], tc = val[uvIdxs[f].y], w = abs(val[f]);
		float sign1 = (face == 0 || face == 5) ? -1 : 1, sign2 = face == 2 ? 1 : -1;
		return (Vec2f(sc * sign1, tc * sign2) / w + Vec2f(1)) / 2.0f;
	}

	CUDA_FUNC_IN Vec3f operator()(float w, float h, unsigned int& x, unsigned int y, imgData* maps)
	{
		float sinPhi, cosPhi, sinTheta, cosTheta;
		sincos((1.0f - x / w) * 2 * PI, &sinPhi, &cosPhi);
		sincos((1.0f - y / h) * PI, &sinTheta, &cosTheta);
		Vec3f d = Vec3f(sinPhi*sinTheta, cosTheta, -cosPhi*sinTheta);
		int face;
		Vec2f uv = cubizePoint4(d, face);
		if(face == 2 || face == 3)
			x = (x + int(w) / 4) % int(w);
		Spectrum s = maps[face].Load(int(uv.x * (maps[face].w - 1)), int((1.0f - uv.y) * (maps[face].h - 1)));
		float r, g, b;
		s.toLinearRGB(r, g, b);
		return Vec3f(r, g, b);
	}
};

CUDA_CONST imgData mapsCuda[6];
__global__ void generateSkydome(unsigned int w, unsigned int h, Vec3f* Target)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x, y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x < w && y < h)
	{
		unsigned int xp = x;
		Vec3f c = MapPoint()(w, h, xp, y, mapsCuda);
		Target[y * w + xp] = c;
	}
}

void e_MIPMap::CreateSphericalSkydomeTexture(const char* front, const char* back, const char* left, const char* right, const char* top, const char* bottom, const char* outFile)
{
	imgData maps[6];
	parseImage(front, maps + 5);
	parseImage(back, maps + 4);
	parseImage(left, maps + 1);
	parseImage(right, maps + 0);
	parseImage(top, maps + 2);
	parseImage(bottom, maps + 3);
	MapPoint M;
	unsigned int w = maps[0].w * 2, h = maps[0].h;
	FIBITMAP* bitmap = FreeImage_AllocateT(FIT_RGBF, w, h, 32);
	Vec3f* B = (Vec3f*)FreeImage_GetBits(bitmap);
	const bool useCuda = true;
	if(useCuda)
	{
		imgData mapsC[6];
		for(int i = 0; i < 6; i++)
		{
			mapsC[i] = maps[i];
			CUDA_MALLOC(&mapsC[i].data, 4 * maps[i].w * maps[i].h);
			hipMemcpy(mapsC[i].data, maps[i].data, 4 * maps[i].w * maps[i].h, hipMemcpyHostToDevice); 
		}
		hipMemcpyToSymbol(HIP_SYMBOL(mapsCuda), &mapsC[0], sizeof(mapsCuda));
		void* T;
		CUDA_MALLOC(&T, sizeof(Vec3f) * w * h);
		generateSkydome << <dim3((w + 31) / 32, (h + 31) / 32, 1), dim3(32, 32, 1) >> >(w, h, (Vec3f*)T);
		hipDeviceSynchronize();
		hipMemcpy(B, T, sizeof(Vec3f) * w * h, hipMemcpyDeviceToHost);
		CUDA_FREE(T);
		for(int i = 0; i < 6; i++)
			CUDA_FREE(mapsC[i].data);
	}
	else
	{
		for(unsigned int x = 0; x < w; x++)
			for(unsigned int y = 0; y < h; y++)
			{
				unsigned int xp = x;
				Vec3f c = M(w, h, xp, y, maps);
				B[y * w + xp] = c;
			}
	}
	bool b = FreeImage_Save(FIF_EXR, bitmap, outFile);
	FreeImage_Unload(bitmap);
	for(int i = 0; i < 6; i++)
		free(maps[i].data);
}