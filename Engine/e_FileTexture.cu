#include "hip/hip_runtime.h"
#include "StdAfx.h"
#include "e_FileTexture.h"
#include "e_ErrorHandler.h"
#include "e_FileTextureHelper.h"

Spectrum e_KernelMIPMap::Texel(unsigned int level, const float2& a_UV) const
{
	float2 l;
	if(!WrapCoordinates(a_UV, make_float2(m_uWidth >> level, m_uHeight >> level), m_uWrapMode, &l))
		return Spectrum(0.0f);
	else
	{
		unsigned int x = (unsigned int)l.x, y = (unsigned int)l.y;
		void* data;
#ifdef ISCUDA
		data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
		data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
		Spectrum s;
		if(m_uType == vtRGBE)
			s.fromRGBE(*(RGBE*)data);
		else s.fromRGBCOL(*(RGBCOL*)data);
		return s;
	}
}

Spectrum e_KernelMIPMap::triangle(unsigned int level, const float2& a_UV) const
{
	level = clamp(level, 0u, m_uLevels-1);
	float2 s = make_float2(m_uWidth >> level, m_uHeight >> level), is = make_float2(1) / s;
	float2 l = a_UV * s;// - make_float2(0.5f)
	float ds = frac(l.x), dt = frac(l.y);
	return (1.f-ds) * (1.f-dt) * Texel(level, a_UV) +
			(1.f-ds) * dt       * Texel(level, a_UV + make_float2(0, is.y)) +
			ds       * (1.f-dt) * Texel(level, a_UV + make_float2(is.x, 0)) +
			ds       * dt       * Texel(level, a_UV + make_float2(is.x, is.y));
}

Spectrum e_KernelMIPMap::Sample(const float2& uv) const
{
	return triangle(0, uv);
}

float e_KernelMIPMap::SampleAlpha(const float2& uv) const
{
	float2 l;
	if(!WrapCoordinates(uv, make_float2(m_uWidth, m_uHeight), m_uWrapMode, &l))
		return 0.0f;
	unsigned int x = (unsigned int)l.x, y = (unsigned int)l.y, level = 0;
	void* data;
#ifdef ISCUDA
			data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
			data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
	if(m_uType == vtRGBE)
		return 1.0f;
	else return float(((RGBCOL*)data)->w) / 255.0f;
}

Spectrum e_KernelMIPMap::Sample(const float2& a_UV, float width) const
{
	float level = m_uLevels - 1 + Log2(MAX((float)width, 1e-8f));
	if (level < 0)
		return triangle(0, a_UV);
	else if (level >= m_uLevels - 1)
		return Texel(m_uLevels - 1, a_UV);
	else
	{
		int iLevel = Floor2Int(level);
		float delta = level - iLevel;
		return (1.f-delta) * triangle(iLevel, a_UV) + delta * triangle(iLevel+1, a_UV);
	}
}

Spectrum e_KernelMIPMap::Sample(float width, int x, int y) const
{
	float l = m_uLevels - 1 + Log2(MAX((float)width, 1e-8f));
	int level = (int)clamp(l, 0.0f, float(m_uLevels - 1));
	void* data;
#ifdef ISCUDA
		data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
		data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
	Spectrum s;
	if(m_uType == vtRGBE)
		s.fromRGBE(*(RGBE*)data);
	else s.fromRGBCOL(*(RGBCOL*)data);
	return s;	
}

struct MapPoint
{
	CUDA_FUNC_IN float2 cubizePoint4(float3& position, int& face)
	{
		float3 q = fabsf(position);
		if(q.x > q.y && q.x > q.z)
			face = 0;
		else if(q.y > q.z)
			face = 1;
		else face = 2;
		int f = face;
		float* val = (float*)&position;
		face = 2 * face + (val[face] > 0 ? 0 : 1);

		int2 uvIdxs[3] = {make_int2(2, 1), make_int2(0, 2), make_int2(0, 1)};
		float sc = val[uvIdxs[f].x], tc = val[uvIdxs[f].y], w = abs(val[f]);
		float sign1 = (face == 0 || face == 5) ? -1 : 1, sign2 = face == 2 ? 1 : -1;
		return (make_float2(sc * sign1, tc * sign2) / w + make_float2(1)) / 2.0f;
	}

	CUDA_FUNC_IN float3 operator()(float w, float h, unsigned int& x, unsigned int y, imgData* maps)
	{
		float sinPhi, cosPhi, sinTheta, cosTheta;
		sincos((1.0f - x / w) * 2 * PI, &sinPhi, &cosPhi);
		sincos((1.0f - y / h) * PI, &sinTheta, &cosTheta);
		float3 d = make_float3(sinPhi*sinTheta, cosTheta, -cosPhi*sinTheta);
		int face;
		float2 uv = cubizePoint4(d, face);
		if(face == 2 || face == 3)
			x = (x + int(w) / 4) % int(w);
		Spectrum s = maps[face].Load(int(uv.x * (maps[face].w - 1)), int((1.0f - uv.y) * (maps[face].h - 1)));
		float r, g, b;
		s.toLinearRGB(r, g, b);
		return make_float3(r, g, b);
	}
};

CUDA_CONST imgData mapsCuda[6];
__global__ void generateSkydome(unsigned int w, unsigned int h, float3* Target)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x, y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x < w && y < h)
	{
		unsigned int xp = x;
		float3 c = MapPoint()(w, h, xp, y, mapsCuda);
		Target[y * w + xp] = c;
	}
}

void e_MIPMap::CreateSphericalSkydomeTexture(const char* front, const char* back, const char* left, const char* right, const char* top, const char* bottom, const char* outFile)
{
	imgData maps[6];
	parseImage(front, maps + 5);
	parseImage(back, maps + 4);
	parseImage(left, maps + 1);
	parseImage(right, maps + 0);
	parseImage(top, maps + 2);
	parseImage(bottom, maps + 3);
	MapPoint M;
	unsigned int w = maps[0].w * 2, h = maps[0].h;
	FIBITMAP* bitmap = FreeImage_AllocateT(FIT_RGBF, w, h, 32);
	float3* B = (float3*)FreeImage_GetBits(bitmap);
	const bool useCuda = true;
	if(useCuda)
	{
		imgData mapsC[6];
		for(int i = 0; i < 6; i++)
		{
			mapsC[i] = maps[i];
			CUDA_MALLOC(&mapsC[i].data, 4 * maps[i].w * maps[i].h);
			hipMemcpy(mapsC[i].data, maps[i].data, 4 * maps[i].w * maps[i].h, hipMemcpyHostToDevice); 
		}
		hipMemcpyToSymbol(HIP_SYMBOL(mapsCuda), &mapsC[0], sizeof(mapsCuda));
		void* T;
		CUDA_MALLOC(&T, sizeof(float3) * w * h);
		generateSkydome<<<dim3((w+31)/32,(h+31)/32,1), dim3(32, 32, 1)>>>(w,h,(float3*)T);
		hipDeviceSynchronize();
		hipMemcpy(B, T, sizeof(float3) * w * h, hipMemcpyDeviceToHost);
		CUDA_FREE(T);
		for(int i = 0; i < 6; i++)
			CUDA_FREE(mapsC[i].data);
	}
	else
	{
		for(unsigned int x = 0; x < w; x++)
			for(unsigned int y = 0; y < h; y++)
			{
				unsigned int xp = x;
				float3 c = M(w, h, xp, y, maps);
				B[y * w + xp] = c;
			}
	}
	bool b = FreeImage_Save(FIF_EXR, bitmap, outFile);
	FreeImage_Unload(bitmap);
	for(int i = 0; i < 6; i++)
		free(maps[i].data);
}