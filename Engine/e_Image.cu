#include "hip/hip_runtime.h"
#include "e_Image.h"
#include <cuda_surface_types.h>

#define BASE 5

//#define FAST_ADD_SAMPLE

Spectrum e_Image::Pixel::toSpectrum(float splat)
{
	Spectrum s;
	s.fromXYZ(xyz[0], xyz[1], xyz[2]);

	if(weightSum != 0.0f)
		s = fmaxf(Spectrum(0.0f), s / weightSum);
	Spectrum s2;
	s2.fromXYZ(xyzSplat[0], xyzSplat[1], xyzSplat[2]);
	return s + s2 * splat;
}

#ifndef FAST_ADD_SAMPLE
void e_Image::AddSample(int sx, int sy, const Spectrum &_L)
{
	Spectrum L = _L;
	L.clampNegative();
	if(L.isNaN())
		return;
	float xyz[3];
	L.toXYZ(xyz[0], xyz[1], xyz[2]);
	float avg = L.average();
	float dimageX = sx - 0.5f;
	float dimageY = sy - 0.5f;
	int x0 = Ceil2Int (dimageX - filter.As<e_KernelFilterBase>()->xWidth);
	int x1 = Floor2Int(dimageX + filter.As<e_KernelFilterBase>()->xWidth);
	int y0 = Ceil2Int (dimageY - filter.As<e_KernelFilterBase>()->yWidth);
	int y1 = Floor2Int(dimageY + filter.As<e_KernelFilterBase>()->yWidth);
	x0 = MAX(x0, 0);
	x1 = MIN(x1, 0 + xResolution - 1);
	y0 = MAX(y0, 0);
	y1 = MIN(y1, 0 + yResolution - 1);
	if ((x1-x0) < 0 || (y1-y0) < 0)
		return;
	float invX = filter.As<e_KernelFilterBase>()->invXWidth, invY = filter.As<e_KernelFilterBase>()->invYWidth;
	for (int y = y0; y <= y1; ++y)
	{
		for (int x = x0; x <= x1; ++x)
		{
			// Evaluate filter value at $(x,y)$ pixel
			float fx = fabsf((x - dimageX) * invX * FILTER_TABLE_SIZE);
			float fy = fabsf((y - dimageY) * invY * FILTER_TABLE_SIZE);
			int ify = MIN(Floor2Int(fx), FILTER_TABLE_SIZE-1);
			int ifx = MIN(Floor2Int(fy), FILTER_TABLE_SIZE-1);
			float filterWt = filterTable[ifx][ify];
			//filterWt = filter.Evaluate(x - dimageX, y - dimageY);

			// Update pixel values with filtered sample contribution
			Pixel* pixel = getPixel((y - 0) * xResolution + (x - 0));
			float wh = filterWt * avg;
#ifdef ISCUDA
			for(int i = 0; i < 3; i++)
				atomicAdd(pixel->xyz + i, filterWt * xyz[i]);
			atomicAdd(&pixel->weightSum, filterWt);
			atomicAdd(&pixel->I, wh);
			atomicAdd(&pixel->I2, wh * wh);
#else
			for(int i = 0; i < 3; i++)
				pixel->xyz[i] += filterWt * xyz[i];
			pixel->weightSum += filterWt;
			pixel->I += wh;
			pixel->I2 += wh * wh;
#endif
		}
	}
}
#else
void e_Image::AddSample(int x, int y, const Spectrum &L)
{
	float xyz[3];
	L.toXYZ(xyz[0], xyz[1], xyz[2]);
	Pixel* pixel = getPixel(y * xResolution + x);
	const float filterWt = 1.0f, wh = filterWt * L.average();
#ifdef ISCUDA
		for(int i = 0; i < 3; i++)
			atomicAdd(pixel->xyz + i, filterWt * xyz[i]);
		atomicAdd(&pixel->weightSum, filterWt);
		atomicAdd(&pixel->I, wh);
		atomicAdd(&pixel->I2, wh * wh);
#else
		for(int i = 0; i < 3; i++)
			pixel->xyz[i] += filterWt * xyz[i];
		pixel->weightSum += filterWt;
		pixel->I += wh;
		pixel->I2 += wh * wh;
#endif
}
#endif

void e_Image::Splat(int sx, int sy, const Spectrum &L)
{
	if (sx >= xResolution || sy >= yResolution)
		return;
	Pixel* pixel = getPixel(sy * xResolution + sx);
	float xyz[3];
	L.toXYZ(xyz[0], xyz[1], xyz[2]);
/*#ifdef ISCUDA
	for(int i = 0; i < 3; i++)
		atomicAdd(pixel->xyzSplat + i, xyz[i]);
#else*/
	for(int i = 0; i < 3; i++)
		pixel->xyzSplat[i] += xyz[i];
//#endif
}

CUDA_FUNC_IN unsigned int FloatToUInt(float f)
{
	//int mask = -int(*(unsigned int*)&f >> 31) | 0x80000000;
	//return (*(unsigned int*)&f) ^ mask;
	return unsigned int(clamp(f, 0.0f, 100.0f) * 1000000);
}

CUDA_FUNC_IN float UIntToFloat(unsigned int f)
{
	//unsigned int mask = ((f >> 31) - 1) | 0x80000000, q = f ^ mask;
	//return *(float*)&q;
	return float(f) / 1000000.0f;
}

///Reinhard Tone Mapping Operator
CUDA_ALIGN(16) CUDA_DEVICE float g_LogLum;
CUDA_ALIGN(16) CUDA_DEVICE unsigned int g_MaxLum;
CUDA_ALIGN(16) CUDA_SHARED float s_LogLum;
CUDA_ALIGN(16) CUDA_SHARED unsigned int s_MaxLum;
CUDA_GLOBAL void rtm_SumLogLum(e_Image::Pixel* P, unsigned int w, unsigned int h, float splatScale, unsigned int NumFrame)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x < w && y < h)
	{
		s_LogLum = s_MaxLum = 0;
		__syncthreads();
		Spectrum L_w = P[y * w + x].toSpectrum(splatScale);
		float avg = L_w.average();
		float& E = P[y * w + x].E, &E2 = P[y * w + x].E2;
		if(NumFrame > BASE)
		{
			E += avg;
			E2 += avg * avg;
		}
		float f2 = L_w.getLuminance();
		float logLum = logf(0.0001f + f2);
		atomicAdd(&g_LogLum, logLum);
		atomicMax(&g_MaxLum, FloatToUInt(f2));
		__syncthreads();
		if(!threadIdx.x && !threadIdx.y)
		{
			atomicAdd(&g_LogLum, s_LogLum);
			atomicMax(&g_MaxLum, s_MaxLum);
		}
	}
}

struct memTarget
{
	RGBCOL* viewTarget;
	int w;

	CUDA_FUNC_IN  void operator()(int x, int y, RGBCOL c)
	{
		viewTarget[y * w + x] = c;
	}
};

struct texTarget
{
	hipSurfaceObject_t viewCudaSurfaceObject;
	int w;

	CUDA_ONLY_FUNC void operator()(int x, int y, RGBCOL c)
	{
		surf2Dwrite(c, viewCudaSurfaceObject, x * 4, y);
	}
};

void e_Image::SetSample(int x, int y, RGBCOL c)
{
	if(outState == 1)
#ifdef ISCUDA
		surf2Dwrite(c, viewCudaSurfaceObject, x * 4, y);
#else
		;
#endif
	else viewTarget[y * xResolution + x] = c;
}

template<typename TARGET> CUDA_GLOBAL void rtm_Scale(e_Image::Pixel* P, TARGET T, unsigned int w, unsigned int h, float splatScale, float L_w, float alpha, float L_white2)
{
	unsigned int x = threadId % w, y = threadId / w;
	if(x < w && y < h)
	{
		float3 yxy;
		P[y * w + x].toSpectrum(splatScale).toYxy(yxy.x, yxy.y, yxy.z);
		float L = alpha / L_w * yxy.x;
		float L_d = (L * (1.0f + L / L_white2)) / (1.0f + L);
		yxy.x = L_d;
		Spectrum c;
		c.fromYxy(yxy.x, yxy.y, yxy.z);	
		T(x, y, c.toRGBCOL());
	}
}

template<typename TARGET> CUDA_GLOBAL void rtm_Copy(e_Image::Pixel* P, TARGET T, unsigned int w, unsigned int h, float splatScale, ImageDrawType TYPE, unsigned int NumFrame)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x, y = threadIdx.y + blockDim.y * blockIdx.y;
	CUDA_SHARED float sumI, sumI2;
	sumI = sumI2 = 0.0f;
	__syncthreads();
	if(x < w && y < h)
	{
		Spectrum c = P[y * w + x].toSpectrum(splatScale);
		float avg = c.average();

		float& E = P[y * w + x].E, &E2 = P[y * w + x].E2;
		if(NumFrame > BASE)
		{
			E += avg;
			E2 += avg * avg;
		}

		if(TYPE == ImageDrawType::Normal)
		{
			T(x, y, c.toRGBCOL());
		}
		else if(TYPE == ImageDrawType::BlockVariance)
		{
			Spectrum c = P[y * w + x].toSpectrum(splatScale);
			atomicAdd(&sumI, avg);
			atomicAdd(&sumI2, avg * avg);
			float N = blockDim.x * blockDim.y;
			__syncthreads();
			float i = sumI / N, i2 = sumI2 / N;
			float var = i2 - i * i;
			T(x, y, SpectrumConverter::Float3ToCOLORREF(make_float3(abs(var))));
		}
		else if(TYPE == ImageDrawType::PixelVariance)
		{
			float W = P[y * w + x].weightSum, i = P[y * w + x].I / W, i2 = P[y * w + x].I2 / W;
			float var = i2 - i * i;
			T(x, y, SpectrumConverter::Float3ToCOLORREF(make_float3(abs(var))));
		}
		else if(TYPE == ImageDrawType::BlockPixelVariance)
		{
			float W = P[y * w + x].weightSum, i = P[y * w + x].I / W, i2 = P[y * w + x].I2 / W;
			float var = i2 - i * i;
			atomicAdd(&sumI, var);
			__syncthreads();
			float f = sumI / float(blockDim.x * blockDim.y);
			T(x, y, SpectrumConverter::Float3ToCOLORREF(make_float3(abs(f))));
		}
		else if(TYPE == ImageDrawType::AverageVariance)
		{
			float e = E / float(NumFrame - BASE), e2 = E2 / float(NumFrame - BASE);
			float var = e2 - e * e;
			T(x, y, SpectrumConverter::Float3ToCOLORREF(make_float3(sqrtf(var))));
		}
		else if(TYPE == ImageDrawType::BlockAverageVariance)
		{
			float e = E / float(NumFrame - BASE), e2 = E2 / float(NumFrame - BASE);
			float var = e2 - e * e;
			atomicAdd(&sumI, var);
			__syncthreads();
			float f = sumI / float(blockDim.x * blockDim.y);
			T(x, y, SpectrumConverter::Float3ToCOLORREF(make_float3(sqrtf(f))));
		}
	}

}

void e_Image::InternalUpdateDisplay(float splatScale)
{
	if(outState > 2)
		return;
	if(usedHostPixels)
	{
		hipMemcpy(cudaPixels, hostPixels, sizeof(Pixel) * xResolution * yResolution, hipMemcpyHostToDevice);
	}
	memTarget T1;
	texTarget T2;
	T1.w = T2.w = xResolution;
	T1.viewTarget = viewTarget;
	T2.viewCudaSurfaceObject = viewCudaSurfaceObject;
	int block = 32;
	NumFrame++;
	if(drawStyle == ImageDrawType::HDR)
	{
		CUDA_ALIGN(16) float Lum_avg = 0;
		unsigned int val = FloatToUInt(0);
		hipError_t r = hipMemcpyToSymbol(HIP_SYMBOL(g_LogLum), &Lum_avg, sizeof(Lum_avg));
		r = hipMemcpyToSymbol(HIP_SYMBOL(g_MaxLum), &val, sizeof(unsigned int));
		rtm_SumLogLum<<<dim3(xResolution / 32 + 1, yResolution / 32 + 1), dim3(32, 32)>>>(cudaPixels, xResolution, yResolution, splatScale, NumFrame);
		r = hipDeviceSynchronize();
		r = hipMemcpyFromSymbol(&Lum_avg, HIP_SYMBOL(g_LogLum), sizeof(Lum_avg));
		unsigned int mLum;
		r = hipMemcpyFromSymbol(&mLum, HIP_SYMBOL(g_MaxLum), sizeof(unsigned int));
		float maxLum = UIntToFloat(mLum);
		float L_w = exp(Lum_avg / float(xResolution * yResolution));
		//float middleGrey = 1.03f - 2.0f / (2.0f + log10(L_w + 1.0f));
		float alpha = 0.18, lumWhite2 = MAX(maxLum * maxLum, 0.1f);
		if(outState == 1)
			rtm_Scale<<<dim3(xResolution / block + 1, yResolution / block + 1), dim3(block, block)>>>(cudaPixels, T2, xResolution, yResolution, splatScale, L_w, alpha, lumWhite2);
		else rtm_Scale<<<dim3(xResolution / block + 1, yResolution / block + 1), dim3(block, block)>>>(cudaPixels, T1, xResolution, yResolution, splatScale, L_w, alpha, lumWhite2);
	}
	else
	{
		if(outState == 1)
			rtm_Copy<<<dim3(xResolution / block + 1, yResolution / block + 1), dim3(block, block)>>>(cudaPixels, T2, xResolution, yResolution, splatScale, drawStyle, NumFrame);
		else rtm_Copy<<<dim3(xResolution / block + 1, yResolution / block + 1), dim3(block, block)>>>(cudaPixels, T1, xResolution, yResolution, splatScale, drawStyle, NumFrame);
	}
}

void e_Image::Clear()
{
	NumFrame = 0;
	usedHostPixels = false;
	Platform::SetMemory(hostPixels, sizeof(Pixel) * xResolution * yResolution);
	hipMemset(cudaPixels, 0, sizeof(Pixel) * xResolution * yResolution);
	if(outState == 2)
		hipMemset(viewTarget, 0, sizeof(RGBCOL) * xResolution * yResolution);
	else if(outState == 1)
		hipMemcpyToArray(viewCudaArray, 0, 0, viewTarget, sizeof(RGBCOL) * xResolution * yResolution, hipMemcpyDeviceToDevice);
}

CUDA_GLOBAL void rtm_VarBuffer(e_Image::Pixel* P, float* T, unsigned int w, unsigned int h, float splatScale, unsigned int NumFrame)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x, y = threadIdx.y + blockDim.y * blockIdx.y;
	CUDA_SHARED float sumI, sumI2;
	sumI = sumI2 = 0.0f;
	if(x < w && y < h)
	{
		Spectrum c = P[y * w + x].toSpectrum(splatScale);
		float avg = c.average();

		float& E = P[y * w + x].E, &E2 = P[y * w + x].E2;
		float e = E / float(NumFrame - BASE), e2 = E2 / float(NumFrame - BASE);
		float var = e2 - e * e;
		atomicAdd(&sumI, var);
		__syncthreads();
		float f = sumI / float(blockDim.x * blockDim.y);
		T[blockIdx.y * blockDim.x + blockIdx.x] = sqrt(f);
	}
}

void e_Image::calculateBlockVariance(int block, float splatScale, float* deviceBuffer)
{
	rtm_VarBuffer<<<dim3(xResolution / block + 1, yResolution / block + 1), dim3(block, block)>>>(cudaPixels, deviceBuffer, xResolution, yResolution, splatScale, NumFrame);
}