#include "hip/hip_runtime.h"
#include "e_Sensor.h"

Spectrum e_SphericalCamera::sampleRay(Ray &ray, const float2 &pixelSample, const float2 &apertureSample) const
{
	float sinPhi, cosPhi, sinTheta, cosTheta;
	sincos((1.0f - pixelSample.x * m_invResolution.x) * 2 * PI, &sinPhi, &cosPhi);
	sincos((1.0f - pixelSample.y * m_invResolution.y) * PI, &sinTheta, &cosTheta);

	float3 d = make_float3(sinPhi*sinTheta, cosTheta, -cosPhi*sinTheta);
	ray = Ray(toWorld.Translation(), toWorld.TransformNormal(d));

	return Spectrum(1.0f);
}

Spectrum e_SphericalCamera::sampleDirect(DirectSamplingRecord &dRec, const float2 &sample) const
{
	float3 refP = toWorldInverse * dRec.ref;
	float3 d(refP);
	float dist = length(d), invDist = 1.0f / dist;
	d *= invDist;

	dRec.uv = make_float2(
		math::modulo(atan2f(d.x, -d.z) * INV_TWOPI, 1.0f) * m_resolution.x,
		math::safe_acos(d.y) * INV_PI * m_resolution.y
	);

	float sinTheta = math::safe_sqrt(1-d.y*d.y);

	dRec.p = toWorld.Translation();
	dRec.d = (dRec.p - dRec.ref) * invDist;
	dRec.dist = dist;
	dRec.n = make_float3(0.0f);
	dRec.pdf = 1;
	dRec.measure = EDiscrete;

	return Spectrum((1/(2 * PI * PI * MAX(sinTheta, EPSILON))) * invDist * invDist);
}

float e_SphericalCamera::pdfDirection(const DirectionSamplingRecord &dRec, const PositionSamplingRecord &pRec) const
{
	if (dRec.measure != ESolidAngle)
		return 0.0f;

	float3 d = toWorldInverse.TransformNormal(dRec.d);
	float sinTheta = math::safe_sqrt(1-d.y*d.y);

	return 1 / (2 * PI * PI * MAX(sinTheta, EPSILON));
}

Spectrum e_SphericalCamera::evalDirection(const DirectionSamplingRecord &dRec, const PositionSamplingRecord &pRec) const
{
	if (dRec.measure != ESolidAngle)
		return Spectrum(0.0f);

	float3 d = toWorldInverse.TransformNormal(dRec.d);
	float sinTheta = math::safe_sqrt(1-d.y*d.y);

	return Spectrum(1 / (2 * PI * PI * MAX(sinTheta, EPSILON)));
}

bool e_SphericalCamera::getSamplePosition(const PositionSamplingRecord &pRec, const DirectionSamplingRecord &dRec, float2 &samplePosition) const
{
	float3 d = normalize(toWorldInverse.TransformNormal(dRec.d));

	samplePosition = make_float2(
		math::modulo(atan2(d.x, -d.z) * INV_TWOPI, (float) 1) * m_resolution.x,
		math::safe_acos(d.y) * INV_PI * m_resolution.y
	);

	return true;
}

float4x4 persp(float fov, float clipNear, float clipFar)
{
	float recip = 1.0f / (clipFar - clipNear);

	/* Perform a scale so that the field of view is mapped
	 * to the interval [-1, 1] */
	float cot = 1.0f / tanf(fov/2.0f);

	float4x4 trafo(
		cot,  0,    0,   0,
		0,    cot,  0,   0,
		0,    0,    clipFar * recip, -clipNear * clipFar * recip,
		0,    0,    1,   0
	);
	return trafo.Transpose();
}
void e_PerspectiveCamera::Update()
{
	e_SensorBase::Update();
	float2 relSize = make_float2(1), relOffset = make_float2(0);
	m_cameraToSample =
			  persp(fov, m_fNearFarDepths.x, m_fNearFarDepths.y)
			* float4x4::Translate(make_float3(1.0f, 1.0f/aspect, 0.0f))
			* float4x4::Scale(make_float3(0.5f, 0.5f*aspect, 1.0f))
			* float4x4::Translate(make_float3(-relOffset.x, -relOffset.y, 0.0f))
			* float4x4::Scale(make_float3(1.0f / relSize.x, 1.0f / relSize.y, 1.0f));
	m_sampleToCamera = m_cameraToSample.Inverse();

	m_dx = m_sampleToCamera * make_float3(m_invResolution.x, 0.0f, 0.0f)
			- m_sampleToCamera * make_float3(0.0f);
	m_dy = m_sampleToCamera * make_float3(0.0f, m_invResolution.y, 0.0f)
			- m_sampleToCamera * make_float3(0.0f);

	float3 min = m_sampleToCamera * make_float3(0, 0, 0),
		   max = m_sampleToCamera * make_float3(1, 1, 0);
	m_imageRect = AABB(min / min.z, max / max.z);
	m_imageRect.minV.z = -FLT_MAX; m_imageRect.maxV.z = FLT_MAX;
	m_normalization = 1.0f / (m_imageRect.Size().x * m_imageRect.Size().y);

//	DirectSamplingRecord dRec(make_float3(301.48853f,398.27206f,559.20007f),make_float3(0),make_float2(0));
	//sampleDirect(dRec, make_float2(0));
}

float e_PerspectiveCamera::importance(const float3 &d) const
{
	float cosTheta = Frame::cosTheta(d);

	/* Check if the direction points behind the camera */
	if (cosTheta <= 0)
		return 0.0f;

	/* Compute the position on the plane at distance 1 */
	float invCosTheta = 1.0f / cosTheta;
	float2 p = make_float2(d.x * invCosTheta, d.y * invCosTheta);

		/* Check if the point lies inside the chosen crop rectangle */
	if (!m_imageRect.Contains(make_float3(p,0)))
		return 0.0f;
	return invCosTheta * invCosTheta * invCosTheta * m_normalization;
}

Spectrum e_PerspectiveCamera::sampleRay(Ray &ray, const float2 &pixelSample, const float2 &apertureSample) const
{
	float3 nearP = m_sampleToCamera * make_float3(
		pixelSample.x * m_invResolution.x,
		pixelSample.y * m_invResolution.y, 0.0f);

	/* Turn that into a normalized ray direction, and
		adjust the ray interval accordingly */
	float3 d = normalize(nearP);
	ray = Ray(toWorld.Translation(), toWorld.TransformNormal(d));

	return Spectrum(1.0f);
}

CUDA_FUNC_IN float PdfWtoA(const float pdfW, const float dist, const float cosThere)
{
    return pdfW * fabs(cosThere) / (dist * dist);
}

Spectrum e_PerspectiveCamera::sampleDirect(DirectSamplingRecord &dRec, const float2 &sample) const
{
	float3 refP = toWorldInverse * dRec.ref;

	/* Check if it is outside of the clip range */
	if (refP.z < m_fNearFarDepths.x || refP.z > m_fNearFarDepths.y) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	float3 screenSample = m_cameraToSample * refP;
	dRec.uv = make_float2(screenSample.x, screenSample.y);
	if (dRec.uv.x < 0 || dRec.uv.x  > 1 ||
		dRec.uv.y < 0 || dRec.uv.y > 1) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	dRec.uv.x *= m_resolution.x;
	dRec.uv.y *= m_resolution.y;

	float3 localD = refP;
	float dist	  = length(localD),
		  invDist = 1.0f / dist;
	localD *= invDist;

	dRec.p = toWorld.Translation();
	dRec.d = invDist * (dRec.p - dRec.ref);
	dRec.dist = dist;
	dRec.n = toWorld.Forward();
	dRec.pdf = 1;
	dRec.measure = EDiscrete;

	const float cosToCamera = dot(dRec.refN, dRec.d);
	const float cosAtCamera = Frame::cosTheta(-dRec.d);
	const float cameraPdfW = 1.f / (cosAtCamera * cosAtCamera * cosAtCamera) * m_normalization;
	const float cameraPdfA = PdfWtoA(cameraPdfW, dist, cosToCamera);
	//return cameraPdfA;
	return Spectrum(importance(localD)*invDist*invDist);
}

Spectrum e_PerspectiveCamera::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const float2 &sample, const float2 *extra) const
{
	float3 samplePos = make_float3(sample.x, sample.y, 0.0f);

	if (extra) {
		/* The caller wants to condition on a specific pixel position */
		samplePos.x = (extra->x + sample.x) * m_invResolution.x;
		samplePos.y = (extra->y + sample.y) * m_invResolution.y;
	}

	pRec.uv = make_float2(samplePos.x * m_resolution.x,
		samplePos.y * m_resolution.y);

	/* Compute the corresponding position on the
		near plane (in local camera space) */
	float3 nearP = m_sampleToCamera * samplePos;

	/* Turn that into a normalized ray direction */
	float3 d = normalize(nearP);
	dRec.d = toWorld.TransformNormal(d);
	dRec.measure = ESolidAngle;
	dRec.pdf = m_normalization / (d.z * d.z * d.z);

	return Spectrum(1.0f);
}

bool e_PerspectiveCamera::getSamplePosition(const PositionSamplingRecord &pRec, const DirectionSamplingRecord &dRec, float2 &samplePosition) const
{
	float3 local = toWorldInverse.TransformNormal(dRec.d);

	if (local.z <= 0)
		return false;

	float3 screenSample = m_cameraToSample * local;
	if (screenSample.x < 0 || screenSample.x > 1 ||
		screenSample.y < 0 || screenSample.y > 1)
		return false;

	samplePosition = make_float2(
			screenSample.x * m_resolution.x,
			screenSample.y * m_resolution.y);

	return true;
}

void e_ThinLensCamera::Update()
{
	e_SensorBase::Update();
	m_sampleToCamera = float4x4::Scale(make_float3(2, 2, 1)) * float4x4::Translate(make_float3(-1.0f, -1.0f, 0.0f)) * float4x4::Perspective(fov, aspect, m_fNearFarDepths.x, m_fNearFarDepths.y).Inverse();
	m_cameraToSample = m_sampleToCamera.Inverse();

	m_dx = m_sampleToCamera * make_float3(m_invResolution.x, 0.0f, 0.0f)
			- m_sampleToCamera * make_float3(0.0f);
	m_dy = m_sampleToCamera * make_float3(0.0f, m_invResolution.y, 0.0f)
			- m_sampleToCamera * make_float3(0.0f);

	m_aperturePdf = 1 / (PI * m_apertureRadius * m_apertureRadius);

	float3 min = m_sampleToCamera * make_float3(0, 0, 0),
		   max = m_sampleToCamera * make_float3(1, 1, 0);
	AABB m_imageRect = AABB(min / min.z, max / max.z);
	m_normalization = 1.0f / (m_imageRect.Size().x * m_imageRect.Size().y);
}

float e_ThinLensCamera::importance(const float3 &p, const float3 &d, float2* sample) const
{
	float cosTheta = Frame::cosTheta(d);
	if (cosTheta <= 0)
		return 0.0f;
	float invCosTheta = 1.0f / cosTheta;
	float3 scr = m_cameraToSample * (p + d * (m_focusDistance*invCosTheta));
	if (scr.x < 0 || scr.x > 1 ||
		scr.y < 0 || scr.y > 1)
		return 0.0f;

	if (sample) {
		sample->x = scr.x * m_resolution.x;
		sample->y = scr.y * m_resolution.y;
	}

	return m_normalization * invCosTheta * invCosTheta * invCosTheta;
}

Spectrum e_ThinLensCamera::sampleRay(Ray &ray, const float2 &pixelSample, const float2 &apertureSample) const
{
	float2 tmp = Warp::squareToUniformDiskConcentric(apertureSample) * m_apertureRadius;

	/* Compute the corresponding position on the
		near plane (in local camera space) */
	float3 nearP = m_sampleToCamera * make_float3(
		pixelSample.x * m_invResolution.x,
		pixelSample.y * m_invResolution.y, 0.0f);

	/* Aperture position */
	float3 apertureP = make_float3(tmp.x, tmp.y, 0.0f);

	/* Sampled position on the focal plane */
	float3 focusP = nearP * (m_focusDistance / nearP.z);

	/* Turn these into a normalized ray direction, and
		adjust the ray interval accordingly */
	float3 d = normalize(focusP - apertureP);
		
	ray = Ray(toWorld * apertureP, toWorld.TransformNormal(d));

	return Spectrum(1.0f);
}

Spectrum e_ThinLensCamera::sampleDirect(DirectSamplingRecord &dRec, const float2 &sample) const
{
	float3 refP = toWorldInverse * dRec.ref;

	/* Check if it is outside of the clip range */
	if (refP.z < m_fNearFarDepths.x || refP.z > m_fNearFarDepths.y) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	/* Sample a position on the aperture (in local coordinates) */
	float2 tmp = Warp::squareToUniformDiskConcentric(sample) * m_apertureRadius;
	float3 apertureP = make_float3(tmp.x, tmp.y, 0);

	/* Compute the normalized direction vector from the
		aperture position to the reference point */
	float3 localD = (refP - apertureP);
	float dist = length(localD),
			invDist = 1.0f / dist;
	localD *= invDist;

	float value = importance(apertureP, localD, &dRec.uv);
	if (value == 0.0f) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	dRec.p = toWorld * apertureP;
	dRec.d = (dRec.p - dRec.ref) * invDist;
	dRec.dist = dist;
	dRec.n = toWorld.Forward();
	dRec.pdf = m_aperturePdf * dist*dist/(Frame::cosTheta(localD));
	dRec.measure = ESolidAngle;

	/* intentionally missing a cosine factor wrt. the aperture
		disk (it is already accounted for in importance()) */
	return Spectrum(value * invDist * invDist);
}

float e_ThinLensCamera::pdfDirect(const DirectSamplingRecord &dRec) const
{
	float dp = -dot(dRec.n, dRec.d);
	if (dp < 0)
		return 0.0f;

	if (dRec.measure == ESolidAngle)
		return m_aperturePdf * dRec.dist*dRec.dist / dp;
	else if (dRec.measure == EArea)
		return m_aperturePdf;
	else
		return 0.0f;
}

Spectrum e_ThinLensCamera::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const float2 &sample, const float2 *extra) const
{
	float3 samplePos = make_float3(sample.x, sample.y, 0.0f);

	if (extra) {
		/* The caller wants to condition on a specific pixel position */
		samplePos.x = (extra->x + sample.x) * m_invResolution.x;
		samplePos.y = (extra->y + sample.y) * m_invResolution.y;
	}

	pRec.uv = make_float2(samplePos.x * m_resolution.x,
		samplePos.y * m_resolution.y);

	/* Compute the corresponding position on the
		near plane (in local camera space) */
	float3 nearP = m_sampleToCamera * samplePos;
	nearP.x = nearP.x * (m_focusDistance / nearP.z);
	nearP.y = nearP.y * (m_focusDistance / nearP.z);
	nearP.z = m_focusDistance;

	float3 apertureP = toWorldInverse * pRec.p;

	/* Turn that into a normalized ray direction */
	float3 d = normalize(nearP - apertureP);
	dRec.d = toWorld.TransformNormal(d);
	dRec.measure = ESolidAngle;
	dRec.pdf = m_normalization / (d.z * d.z * d.z);

	return Spectrum(1.0f);
}

void e_OrthographicCamera::Update()
{
	e_SensorBase::Update();
	m_sampleToCamera = float4x4::Scale(make_float3(2, 2, 1)) * float4x4::Translate(make_float3(-1.0f, -1.0f, 0.0f)) * float4x4::Orthographic(screenScale.x, screenScale.y, m_fNearFarDepths.x, m_fNearFarDepths.y).Inverse();
	m_cameraToSample = m_sampleToCamera.Inverse();

	m_invSurfaceArea = 1.0f / (
		length(toWorld * m_sampleToCamera.Right()) *
		length(toWorld * m_sampleToCamera.Up()) );
	m_scale = length(toWorld.Forward());
}

Spectrum e_OrthographicCamera::sampleRay(Ray &ray, const float2 &pixelSample, const float2 &apertureSample) const
{
	float3 nearP = m_sampleToCamera * make_float3(
		pixelSample.x * m_invResolution.x,
		pixelSample.y * m_invResolution.y, 0.0f);

	ray = Ray(toWorld * make_float3(nearP.x, nearP.y, 0.0f), toWorld.Forward());

	return Spectrum(1.0f);
}

Spectrum e_OrthographicCamera::sampleDirect(DirectSamplingRecord &dRec, const float2 &) const
{
	dRec.n = toWorld.Forward();
	float scale = length(dRec.n);

	float3 localP = toWorldInverse * dRec.ref;
	localP.z *= scale;

	float3 sample = m_cameraToSample * localP;

	if (sample.x < 0 || sample.x > 1 || sample.y < 0 ||
		sample.y > 1 || sample.z < 0 || sample.z > 1) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	dRec.p = toWorld * make_float3(localP.x, localP.y, 0.0f);
	dRec.n /= scale;
	dRec.d = -dRec.n;
	dRec.dist = localP.z;
	dRec.uv = make_float2(sample.x * m_resolution.x,
						  sample.y * m_resolution.y);
	dRec.pdf = 1.0f;
	dRec.measure = EDiscrete;

	return Spectrum(m_invSurfaceArea);
}

Spectrum e_OrthographicCamera::samplePosition(PositionSamplingRecord &pRec, const float2 &sample, const float2 *extra) const
{
	float3 samplePos = make_float3(sample.x, sample.y, 0.0f);

	if (extra) {
		/* The caller wants to condition on a specific pixel position */
		samplePos.x = (extra->x + sample.x) * m_invResolution.x;
		samplePos.y = (extra->y + sample.y) * m_invResolution.y;
	}

	pRec.uv = make_float2(samplePos.x * m_resolution.x,	samplePos.y * m_resolution.y);

	float3 nearP = m_sampleToCamera * samplePos;

	nearP.z = 0.0f;
	pRec.p = toWorld * nearP;
	pRec.n = toWorld.Forward();
	pRec.pdf = m_invSurfaceArea;
	pRec.measure = EArea;
	return Spectrum(1.0f);
}

bool e_OrthographicCamera::getSamplePosition(const PositionSamplingRecord &pRec, const DirectionSamplingRecord &dRec, float2 &samplePosition) const
{
	float3 localP = toWorldInverse * pRec.p;
	float3 sample = m_cameraToSample * localP;

	if (sample.x < 0 || sample.x > 1 || sample.y < 0 || sample.y > 1)
		return false;

	samplePosition = make_float2(sample.x * m_resolution.x,
		                    sample.y * m_resolution.y);
	return true;
}

void e_TelecentricCamera::Update()
{
	e_SensorBase::Update();
	m_sampleToCamera = float4x4::Scale(make_float3(2, 2, 1)) * float4x4::Translate(make_float3(-1.0f, -1.0f, 0.0f)) * float4x4::Orthographic(screenScale.x, screenScale.y, m_fNearFarDepths.x, m_fNearFarDepths.y).Inverse();
	m_cameraToSample = m_sampleToCamera.Inverse();

	m_normalization = 1.0f / (
		length(toWorld * m_sampleToCamera.Right()) *
		length(toWorld * m_sampleToCamera.Up()) );

	m_aperturePdf = 1.0f / (PI * m_apertureRadius * m_apertureRadius);
}

Spectrum e_TelecentricCamera::sampleRay(Ray &ray, const float2 &pixelSample, const float2 &apertureSample) const
{
	float2 diskSample = Warp::squareToUniformDiskConcentric(apertureSample)
		* (m_apertureRadius / screenScale.x);

	/* Compute the corresponding position on the
		near plane (in local camera space) */
	float3 focusP = m_sampleToCamera * make_float3(
		pixelSample.x * m_invResolution.x,
		pixelSample.y * m_invResolution.y, 0.0f);
	focusP.z = m_focusDistance;

	/* Compute the ray origin */
	float3 orig = make_float3(diskSample.x+focusP.x,
		diskSample.y+focusP.y, 0.0f);

	ray = Ray(toWorld * orig, toWorld.TransformNormal(focusP - orig));

	return Spectrum(1.0f);
}

Spectrum e_TelecentricCamera::sampleDirect(DirectSamplingRecord &dRec, const float2 &sample) const
{
	float f = m_focusDistance, apertureRadius = m_apertureRadius / screenScale.x;

	float3 localP = toWorldInverse * dRec.ref;

	float dist = localP.z;
	if (dist < m_fNearFarDepths.x || dist > m_fNearFarDepths.y) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	/* Circle of confusion */
	float radius = abs(localP.z - f) * apertureRadius/f;
	radius += apertureRadius;

	/* Sample the ray origin */
	float2 disk = Warp::squareToUniformDiskConcentric(sample);
	float3 diskP = make_float3(disk.x*radius+localP.x, disk.y*radius+localP.y, 0.0f);

	/* Compute the intersection with the focal plane */
	float3 localD = localP - diskP;
	float3 intersection = diskP + localD * (f/localD.z);

	/* Determine the associated sample coordinates */
	float3 uv = m_cameraToSample * intersection;
	if (uv.x < 0 || uv.x > 1 || uv.y < 0 || uv.y > 1) {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}

	dRec.uv = make_float2(uv.x, uv.y);
	dRec.p = toWorld * diskP;
	dRec.n = toWorld.Forward();
	dRec.d = dRec.p - dRec.ref;
	dRec.dist = length(dRec.d);
	dRec.d /= dRec.dist;
	dRec.measure = ESolidAngle;

	dRec.pdf = dist*dist / (-dot(dRec.n, dRec.d)* PI * radius*radius);

	return Spectrum(m_normalization);
}

Spectrum e_TelecentricCamera::samplePosition(PositionSamplingRecord &pRec, const float2 &sample, const float2 *extra) const
{
	float a = sample.x + 1.0f, b = sample.y + 1.0f;
	unsigned int tmp1 = *(unsigned int*)&a & 0x7FFFFF;
	unsigned int tmp2 = *(unsigned int*)&b & 0x7FFFFF;

	float rand1 = (tmp1 >> 11)   * (1.0f / 0xFFF);
	float rand2 = (tmp2 >> 11)   * (1.0f / 0xFFF);
	float rand3 = (tmp1 & 0x7FF) * (1.0f / 0x7FF);
	float rand4 = (tmp2 & 0x7FF) * (1.0f / 0x7FF);

	float2 aperturePos = Warp::squareToUniformDiskConcentric(make_float2(rand1, rand2))
		* (m_apertureRadius / screenScale.x);
	float2 samplePos = make_float2(rand3, rand4);

	if (extra) {
		/* The caller wants to condition on a specific pixel position */
		pRec.uv = *extra + samplePos;
		samplePos.x = pRec.uv.x * m_invResolution.x;
		samplePos.y = pRec.uv.y * m_invResolution.y;
	}

	float3 p = m_sampleToCamera * make_float3(
		aperturePos.x + samplePos.x, aperturePos.y + samplePos.y, 0.0f);

	pRec.p = toWorld * make_float3(p.x, p.y, 0.0f);
	pRec.n = toWorld.Forward();
	pRec.pdf = m_aperturePdf;
	pRec.measure = EArea;
	return Spectrum(1.0f);
}

Spectrum e_TelecentricCamera::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const float2 &sample, const float2 *extra) const
{
	float3 nearP = m_sampleToCamera * make_float3(sample.x, sample.y, 0.0f);

	/* Turn that into a normalized ray direction */
	float3 d = normalize(nearP);
	dRec.d = toWorld.TransformNormal(d);
	dRec.measure = ESolidAngle;
	dRec.pdf = m_normalization / (d.z * d.z * d.z);

	return Spectrum(1.0f);
}

float4x4 e_Sensor::View() const
{
	float4x4 m_mView = As<e_SensorBase>()->getWorld();
	float3 pos = m_mView.Translation();
	m_mView = m_mView * float4x4::Translate(-pos);
	return m_mView;
}

float3 e_Sensor::Position() const
{
	return As<e_SensorBase>()->getWorld().Translation();
}

void e_Sensor::SetToWorld(const float3& pos, const float4x4& rot)
{
	SetToWorld(rot * float4x4::Translate(pos));
}

void e_Sensor::SetToWorld(const float3& pos, const float3& _f)
{
	float3 f = normalize(_f);
	float3 r = normalize(cross(f, make_float3(0,1,0)));
	float3 u = normalize(cross(r, f));
	float4x4 m_mView = float4x4::Identity();
	m_mView.X = make_float4(r.x, r.y, r.z, 0.0f);
	m_mView.Y = make_float4(u.x, u.y, u.z, 0.0f);
	m_mView.Z = make_float4(f.x, f.y, f.z, 0.0f);
	SetToWorld(pos, m_mView);
}

void e_Sensor::SetFilmData(int w, int h)
{
	As<e_SensorBase>()->SetFilmData(w, h);
}

void e_Sensor::SetToWorld(const float4x4& w)
{
	As()->SetToWorld(w);
}

float4x4 e_Sensor::getGLViewProjection() const
{
	float4x4 proj = getProjectionMatrix();
	return (float4x4::Translate(-1.0f * Position()) * (View().Inverse() )) * proj;
}