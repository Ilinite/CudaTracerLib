#include "hip/hip_runtime.h"
#include "PhaseFunction.h"
#include <Base/CudaRandom.h>
#include "Samples.h"
#include <Math/Frame.h>
#include <Math/Warp.h>

namespace CudaTracerLib {

float HGPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	float temp = 1.0f + m_g*m_g + 2.0f * m_g * dot(pRec.wi, pRec.wo);
	return (1.0f / (4.0f * PI)) * (1 - m_g*m_g) / (temp * math::sqrt(temp));
}

float HGPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	Vec2f sample = sampler.randomFloat2();

	float cosTheta;
	if (math::abs(m_g) < EPSILON)
	{
		cosTheta = 1 - 2 * sample.x;
	}
	else
	{
		float sqrTerm = (1 - m_g * m_g) / (1 - m_g + 2 * m_g * sample.x);
		cosTheta = (1 + m_g * m_g - sqrTerm * sqrTerm) / (2 * m_g);
	}

	float sinTheta = math::sqrt(1.0f - cosTheta*cosTheta), sinPhi, cosPhi;

	sincos(2 * PI*sample.y, &sinPhi, &cosPhi);

	pRec.wo = Frame(-pRec.wi).toWorld(NormalizedT<Vec3f>(
		sinTheta * cosPhi,
		sinTheta * sinPhi,
		cosTheta
		));

	return 1.0f;
}

float HGPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	HGPhaseFunction::Sample(pRec, sampler);
	pdf = HGPhaseFunction::Evaluate(pRec);
	return 1.0f;
}

float IsotropicPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	return Warp::squareToUniformSpherePdf();
}

float IsotropicPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	Vec2f sample = sampler.randomFloat2();
	pRec.wo = Warp::squareToUniformSphere(sample);
	return 1.0f;
}

float IsotropicPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	pRec.wo = Warp::squareToUniformSphere(sampler.randomFloat2());
	pdf = Warp::squareToUniformSpherePdf();
	return 1.0f;
}

KajiyaKayPhaseFunction::KajiyaKayPhaseFunction(float ks, float kd, float e)
	: BasePhaseFunction(EPhaseFunctionType::pEAnisotropic), m_ks(ks), m_kd(kd), m_exponent(e)
{
	Update();
}

KajiyaKayPhaseFunction::KajiyaKayPhaseFunction()
	: BasePhaseFunction(EPhaseFunctionType::pEAnisotropic), m_ks(0.4f), m_kd(0.2f), m_exponent(4.0f)
{
	Update();
}

void KajiyaKayPhaseFunction::Update()
{
	int nParts = 1000;
	float stepSize = PI / nParts, m = 4, theta = stepSize;

	m_normalization = 0; /* 0 at the endpoints */
	for (int i = 1; i < nParts; ++i) {
		float value = math::pow(cosf(theta - PI / 2), m_exponent)
			* sinf(theta);
		m_normalization += value * m;
		theta += stepSize;
		m = 6 - m;
	}

	m_normalization = 1 / (m_normalization * stepSize / 3 * 2 * PI);
}

float KajiyaKayPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	if (pRec.wi.max() == 0)
		return m_kd / (4 * PI);

	Frame frame(pRec.wi);
	Vec3f reflectedLocal = frame.toLocal(pRec.wo);

	reflectedLocal.z = -dot(pRec.wi, frame.n);
	float a = math::sqrt((1 - reflectedLocal.z*reflectedLocal.z) / (reflectedLocal.x*reflectedLocal.x + reflectedLocal.y*reflectedLocal.y));
	reflectedLocal.y *= a;
	reflectedLocal.x *= a;
	Vec3f R = frame.toWorld(reflectedLocal);

	return math::pow(max(0.0f, dot(R, pRec.wo)), m_exponent) * m_normalization * m_ks + m_kd / (4 * PI);
}

float KajiyaKayPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	pRec.wo = Warp::squareToUniformSphere(sampler.randomFloat2());
	return Evaluate(pRec) * (4 * PI);
}

float KajiyaKayPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	pRec.wo = Warp::squareToUniformSphere(sampler.randomFloat2());
	pdf = Warp::squareToUniformSpherePdf();
	return Evaluate(pRec) * (4 * PI);
}

float RayleighPhaseFunction::Evaluate(const PhaseFunctionSamplingRecord &pRec) const
{
	float mu = dot(pRec.wi, pRec.wo);
	return (3.0f / (16.0f*PI)) * (1 + mu*mu);
}

float RayleighPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, CudaRNG& sampler) const
{
	Vec2f sample(sampler.randomFloat2());

	float z = 2 * (2 * sample.x - 1),
		tmp = math::sqrt(z*z + 1),
		A = math::pow(z + tmp, (float)(1.0f / 3.0f)),
		B = math::pow(z - tmp, (float)(1.0f / 3.0f)),
		cosTheta = A + B,
		sinTheta = math::sqrt(1.0f - cosTheta*cosTheta),
		phi = 2 * PI*sample.y, cosPhi, sinPhi;
	sincos(phi, &sinPhi, &cosPhi);

	auto dir = NormalizedT<Vec3f>(
		sinTheta * cosPhi,
		sinTheta * sinPhi,
		cosTheta);

	pRec.wo = Frame(-pRec.wi).toWorld(dir);
	return 1.0f;
}

float RayleighPhaseFunction::Sample(PhaseFunctionSamplingRecord &pRec, float &pdf, CudaRNG& sampler) const
{
	RayleighPhaseFunction::Sample(pRec, sampler);
	pdf = RayleighPhaseFunction::Evaluate(pRec);
	return 1.0f;
}

}