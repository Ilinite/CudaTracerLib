#include "hip/hip_runtime.h"
#include "StdAfx.h"
#include "MIPMap.h"
#include "MIPMapHelper.h"
#include <Base/FileStream.h>
#include <CudaMemoryManager.h>
#define FREEIMAGE_LIB
#include <FreeImage.h>

//MipMap evaluation copied from Mitsuba.

namespace CudaTracerLib {

template <typename Scalar> CUDA_FUNC_IN int floorToInt(Scalar value) { return (int)math::floor(value); }
template <typename Scalar> CUDA_FUNC_IN int ceilToInt(Scalar value) { return (int)math::ceil(value); }

CUDA_FUNC_IN float hypot2(float a, float b)
{
	return math::sqrt(a * a + b * b);
}

Spectrum KernelMIPMap::Texel(unsigned int level, const Vec2f& a_UV) const
{
	Vec2f l;
	if (!WrapCoordinates(a_UV, Vec2f((float)(m_uWidth >> level), (float)(m_uHeight >> level)), m_uWrapMode, &l))
		return Spectrum(0.0f);
	else
	{
		unsigned int x = (unsigned int)l.x, y = (unsigned int)l.y;
		void* data;
		int i = max(0, ((int)m_uWidth >> (int)level));
#ifdef ISCUDA
		data = m_pDeviceData + (m_sOffsets[level] + y * i + x);
#else
		data = m_pHostData + (m_sOffsets[level] + y * i + x);
#endif
		Spectrum s;
		if (m_uType == vtRGBE)
			s.fromRGBE(*(RGBE*)data);
		else s.fromRGBCOL(*(RGBCOL*)data);
		return s;
	}
}

Spectrum KernelMIPMap::triangle(unsigned int level, const Vec2f& a_UV) const
{
	level = math::clamp(level, 0u, m_uLevels - 1);
	Vec2f s = Vec2f((float)(m_uWidth >> level), (float)(m_uHeight >> level)), is = Vec2f(1.0f) / s;
	Vec2f l = a_UV * s;// - make_float2(0.5f)
	float ds = math::frac(l.x), dt = math::frac(l.y);
	return (1.f - ds) * (1.f - dt) * Texel(level, a_UV) +
		(1.f - ds) * dt       * Texel(level, a_UV + Vec2f(0, is.y)) +
		ds       * (1.f - dt) * Texel(level, a_UV + Vec2f(is.x, 0)) +
		ds       * dt       * Texel(level, a_UV + Vec2f(is.x, is.y));
}

Spectrum KernelMIPMap::evalEWA(unsigned int level, const Vec2f &uv, float A, float B, float C) const
{
	if (level >= m_uLevels)
		return Texel(m_uLevels - 1, Vec2f(0));

	Vec2f size = Vec2f((float)(m_uWidth >> level), (float)(m_uHeight >> level));
	float u = uv.x * size.x - 0.5f;
	float v = uv.y * size.y - 0.5f;

	/* Do the same to the ellipse coefficients */
	Vec2f ratio = size / m_fDim;
	A /= ratio.x * ratio.x;
	B /= ratio.x * ratio.y;
	C /= ratio.y * ratio.y;

	float invDet = 1.0f / (-B*B + 4.0f*A*C),
		deltaU = 2.0f * math::sqrt(C * invDet),
		deltaV = 2.0f * math::sqrt(A * invDet);
	int u0 = ceilToInt(u - deltaU), u1 = floorToInt(u + deltaU);
	int v0 = ceilToInt(v - deltaV), v1 = floorToInt(v + deltaV);

	float As = A * MTS_MIPMAP_LUT_SIZE,
		Bs = B * MTS_MIPMAP_LUT_SIZE,
		Cs = C * MTS_MIPMAP_LUT_SIZE;

	Spectrum result(0.0f);
	float denominator = 0.0f;
	float ddq = 2 * As, uu0 = u0 - u;
	int nSamples = 0;

	for (int vt = v0; vt <= v1; ++vt)
	{
		const float vv = vt - v;

		float q = As*uu0*uu0 + (Bs*uu0 + Cs*vv)*vv;
		float dq = As*(2 * uu0 + 1) + Bs*vv;

		for (int ut = u0; ut <= u1; ++ut)
		{
			if (q < MTS_MIPMAP_LUT_SIZE)
			{
				unsigned int qi = (unsigned int)q;
				if (qi < MTS_MIPMAP_LUT_SIZE)
				{
					const float weight = m_weightLut[(int)q];
					result += Texel(level, Vec2f((float)ut, (float)vt) / size) * weight;
					denominator += weight;
					++nSamples;
				}
			}
			q += dq;
			dq += ddq;
		}
	}

	if (denominator == 0)
		return triangle(level, uv);
	return result / denominator;
}

Spectrum KernelMIPMap::Sample(const Vec2f& uv) const
{
	if (m_uFilterMode == TEXTURE_Point)
		return Texel(0, uv);
	else return triangle(0, uv);
}

float KernelMIPMap::SampleAlpha(const Vec2f& uv) const
{
	Vec2f l;
	if (!WrapCoordinates(uv, Vec2f((float)m_uWidth, (float)m_uHeight), m_uWrapMode, &l))
		return 0.0f;
	unsigned int x = (unsigned int)l.x, y = (unsigned int)l.y, level = 0;
	void* data;
#ifdef ISCUDA
	data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
	data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
	if (m_uType == vtRGBE)
		return 1.0f;
	else return float(((RGBCOL*)data)->w) / 255.0f;
}

Spectrum KernelMIPMap::Sample(const Vec2f& a_UV, float width) const
{
	float level = m_uLevels - 1 + math::log2(max((float)width, 1e-8f));
	if (level < 0)
		return triangle(0, a_UV);
	else if (level >= m_uLevels - 1)
		return Texel(m_uLevels - 1, a_UV);
	else
	{
		int iLevel = math::Floor2Int(level);
		float delta = level - iLevel;
		return (1.f - delta) * triangle(iLevel, a_UV) + delta * triangle(iLevel + 1, a_UV);
	}
}

Spectrum KernelMIPMap::Sample(float width, int x, int y) const
{
	float l = m_uLevels - 1 + math::log2(max((float)width, 1e-8f));
	int level = (int)math::clamp(l, 0.0f, float(m_uLevels - 1));
	void* data;
#ifdef ISCUDA
	data = m_pDeviceData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#else
	data = m_pHostData + (m_sOffsets[level] + y * (m_uWidth >> level) + x);
#endif
	Spectrum s;
	if (m_uType == vtRGBE)
		s.fromRGBE(*(RGBE*)data);
	else s.fromRGBCOL(*(RGBCOL*)data);
	return s;
}

void KernelMIPMap::evalGradient(const Vec2f& uv, Spectrum* gradient) const
{
	const int level = 0;

	float u = uv.x * m_fDim.x - 0.5f, v = uv.y * m_fDim.y - 0.5f;

	int xPos = math::Float2Int(u), yPos = math::Float2Int(v);
	float dx = u - xPos, dy = v - yPos;

	const Spectrum p00 = Texel(level, Vec2f((float)xPos, (float)yPos) / m_fDim);
	const Spectrum p10 = Texel(level, Vec2f((float)xPos + 1, (float)yPos) / m_fDim);
	const Spectrum p01 = Texel(level, Vec2f((float)xPos, (float)yPos + 1) / m_fDim);
	const Spectrum p11 = Texel(level, Vec2f((float)xPos + 1, (float)yPos + 1) / m_fDim);
	Spectrum tmp = p01 + p10 - p11;

	gradient[0] = (p10 + p00*(dy - 1) - tmp*dy) * m_fDim.x;
	gradient[1] = (p01 + p00*(dx - 1) - tmp*dx) * m_fDim.y;
}

Spectrum KernelMIPMap::eval(const Vec2f& uv, const Vec2f& d0, const Vec2f& d1) const
{
	/* Convert into texel coordinates */
	float du0 = d0.x * m_fDim.x, dv0 = d0.y * m_fDim.y,
		  du1 = d1.x * m_fDim.x, dv1 = d1.y * m_fDim.y,
		  du = (du0 + du1) / 2.0f, dv = (dv0 + dv1) / 2.0f;

	if (m_uFilterMode == TEXTURE_Point)
		return Texel(0, uv);
	else if (m_uFilterMode == TEXTURE_Bilinear)
		return triangle(0, uv);
	else if (m_uFilterMode == TEXTURE_Trilinear)
	{
		float levela = math::log2(m_fDim.x / math::abs(du)),
			  levelb = math::log2(m_fDim.y / math::abs(dv)),
			  level = m_uLevels - math::clamp((levela + levelb) / 2.0f, 1.0f, (float)m_uLevels);
		int iLevel = math::Floor2Int(level), iLevel2 = math::clamp(iLevel + 1, 0, (int)m_uLevels - 1);
		float p = level - iLevel;
		Spectrum texelA = triangle(iLevel, uv), texelB = triangle(iLevel2, uv);
		return p * texelA + (1 - p) * texelB;
	}


	/* Turn the texture-space Jacobian into the coefficients of an
	implicitly defined ellipse. */
	float A = dv0*dv0 + dv1*dv1,
		B = -2.0f * (du0*dv0 + du1*dv1),
		C = du0*du0 + du1*du1,
		F = A*C - B*B*0.25f;

	float root = hypot2(A - C, B),
		Aprime = 0.5f * (A + C - root),
		Cprime = 0.5f * (A + C + root),
		majorRadius = Aprime != 0 ? math::sqrt(F / Aprime) : 0,
		minorRadius = Cprime != 0 ? math::sqrt(F / Cprime) : 0;

	if (!(minorRadius > 0) || !(majorRadius > 0) || F < 0)
	{
		float level = log2f(max(majorRadius, 1e-4f));
		int ilevel = math::Floor2Int(level);
		if (ilevel < 0)
			return triangle(0, uv);
		else
		{
			float a = level - ilevel;
			return triangle(ilevel, uv) * (1.0f - a)
				+ triangle(ilevel + 1, uv) * a;
		}
	}
	else
	{
		const float m_maxAnisotropy = 16;
		if (minorRadius * m_maxAnisotropy < majorRadius)
		{
			minorRadius = majorRadius / m_maxAnisotropy;
			float theta = 0.5f * std::atan(B / (A - C)), sinTheta, cosTheta;
			sincos(theta, &sinTheta, &cosTheta);
			float a2 = majorRadius*majorRadius,
				b2 = minorRadius*minorRadius,
				sinTheta2 = sinTheta*sinTheta,
				cosTheta2 = cosTheta*cosTheta,
				sin2Theta = 2 * sinTheta*cosTheta;

			A = a2*cosTheta2 + b2*sinTheta2;
			B = (a2 - b2) * sin2Theta;
			C = a2*sinTheta2 + b2*cosTheta2;
			F = a2*b2;
		}
		/* Switch to normalized coefficients */
		float scale = 1.0f / F;
		A *= scale; B *= scale; C *= scale;
		/* Determine a suitable MIP map level, such that the filter
		covers a reasonable amount of pixels */
		float level = max(0.0f, log2f(minorRadius));
		int ilevel = (int)level;
		float a = level - ilevel;

		/* Switch to bilinear interpolation, be wary of round-off errors */
		if (majorRadius < 1 || !(A > 0 && C > 0))
			return triangle(ilevel, uv);
		else
			return evalEWA(ilevel, uv, A, B, C) * (1.0f - a) +
			evalEWA(ilevel + 1, uv, A, B, C) * a;
	}
}

struct MapPoint
{
	CUDA_FUNC_IN Vec2f cubizePoint4(Vec3f& position, int& face)
	{
		Vec3f q = position.abs();
		if (q.x > q.y && q.x > q.z)
			face = 0;
		else if (q.y > q.z)
			face = 1;
		else face = 2;
		int f = face;
		float* val = (float*)&position;
		face = 2 * face + (val[face] > 0 ? 0 : 1);

		int2 uvIdxs[3] = { make_int2(2, 1), make_int2(0, 2), make_int2(0, 1) };
		float sc = val[uvIdxs[f].x], tc = val[uvIdxs[f].y], w = math::abs(val[f]);
		float sign1 = (face == 0 || face == 5) ? -1.0f : 1.0f, sign2 = face == 2 ? 1.0f : -1.0f;
		return (Vec2f(sc * sign1, tc * sign2) / w + Vec2f(1)) / 2.0f;
	}

	CUDA_FUNC_IN Vec3f operator()(float w, float h, unsigned int& x, unsigned int y, imgData* maps)
	{
		float sinPhi, cosPhi, sinTheta, cosTheta;
		sincos((1.0f - x / w) * 2 * PI, &sinPhi, &cosPhi);
		sincos((1.0f - y / h) * PI, &sinTheta, &cosTheta);
		Vec3f d = Vec3f(sinPhi*sinTheta, cosTheta, -cosPhi*sinTheta);
		int face;
		Vec2f uv = cubizePoint4(d, face);
		if (face == 2 || face == 3)
			x = (x + int(w) / 4) % int(w);
		Spectrum s = maps[face].Load(int(uv.x * (maps[face].w() - 1)), int((1.0f - uv.y) * (maps[face].h() - 1)));
		float r, g, b;
		s.toLinearRGB(r, g, b);
		return Vec3f(r, g, b);
	}
};

CUDA_CONST imgData mapsCuda[6];
__global__ void generateSkydome(unsigned int w, unsigned int h, Vec3f* Target)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x, y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < w && y < h)
	{
		unsigned int xp = x;
		Vec3f c = MapPoint()(w, h, xp, y, mapsCuda);
		Target[y * w + xp] = c;
	}
}

void MIPMap::CreateSphericalSkydomeTexture(const std::string& front, const std::string& back, const std::string& left, const std::string& right, const std::string& top, const std::string& bottom, const std::string& outFile)
{
	imgData maps[6];
	parseImage(front, maps[5]);
	parseImage(back, maps[4]);
	parseImage(left, maps[1]);
	parseImage(right, maps[0]);
	parseImage(top, maps[2]);
	parseImage(bottom, maps[3]);
	MapPoint M;
	unsigned int w = maps[0].w() * 2, h = maps[0].h();
	FIBITMAP* bitmap = FreeImage_AllocateT(FIT_RGBF, w, h, 32);
	Vec3f* B = (Vec3f*)FreeImage_GetBits(bitmap);
	const bool useCuda = true;
	if (useCuda)
	{
		imgData mapsC[6];
		for (int i = 0; i < 6; i++)
		{
			mapsC[i] = maps[i];
			void* deviceData;
			CUDA_MALLOC(&deviceData, 4 * maps[i].w() * maps[i].h());
			mapsC[i].d(deviceData);
			hipMemcpy(mapsC[i].d(), maps[i].d(), 4 * maps[i].w() * maps[i].h(), hipMemcpyHostToDevice);
		}
		ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mapsCuda), &mapsC[0], sizeof(mapsCuda)));
		void* T;
		CUDA_MALLOC(&T, sizeof(Vec3f) * w * h);
		generateSkydome << <dim3(w / 32 + 1, h / 32 + 1, 1), dim3(32, 32, 1) >> >(w, h, (Vec3f*)T);
		ThrowCudaErrors(hipDeviceSynchronize());
		ThrowCudaErrors(hipMemcpy(B, T, sizeof(Vec3f) * w * h, hipMemcpyDeviceToHost));
		CUDA_FREE(T);
		for (int i = 0; i < 6; i++)
			CUDA_FREE(mapsC[i].d());
	}
	else
	{
		for (unsigned int x = 0; x < w; x++)
			for (unsigned int y = 0; y < h; y++)
			{
				unsigned int xp = x;
				Vec3f c = M((float)w, (float)h, xp, y, maps);
				B[y * w + xp] = c;
			}
	}
	FREE_IMAGE_FORMAT ff = FreeImage_GetFIFFromFilename(outFile.c_str());
	if (!FreeImage_Save(ff, bitmap, outFile.c_str()))
		throw std::runtime_error(std::string(__FUNCTION__) + " :: FreeImage_Save");
	FreeImage_Unload(bitmap);
	for (int i = 0; i < 6; i++)
		maps[i].Free();
}

CUDA_FUNC_IN float sample(imgData& img, const Vec3f& p)
{
	Vec2f q = clamp01(p.getXY()) * Vec2f((float)img.w(), (float)img.h());
	return img.Load((int)q.x, (int)q.y).avg();
}
template<int SEARCH_STEPS> CUDA_FUNC_IN bool text_cords_next_intersection(imgData& img, const Vec3f& pos, const Vec3f& dir, float& height, Vec2f& kw)
{
	Vec3f vec = dir;
	Vec3f step_fwd = vec / SEARCH_STEPS;
	Vec3f ray_pos = pos + step_fwd;
	for (int i = 1; i < SEARCH_STEPS; i++)
	{
		height = sample(img, ray_pos);
		if (height <= ray_pos.z)
			ray_pos += step_fwd;
		else
		{
			kw = ray_pos.getXY();
			return true;
		}
	}
	return false;
}
template<int SEARCH_STEPS, int LOOKUP_WIDTH> __global__ void generateRelaxedConeMap(imgData img, float* coneData)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < img.w() && y < img.h())
	{
		float src_texel_depth = img.Load(x, y).avg();
		float* radius_cone = coneData + (y * img.w() + x);
		const float MAX_CONE_RATIO = ((float)LOOKUP_WIDTH / (float)max(img.w(), img.h())) / (1 - src_texel_depth);
		*radius_cone = MAX_CONE_RATIO;
		Vec3f src = Vec3f(float(x) / img.w(), float(y) / img.h(), src_texel_depth);
		int xmin = math::clamp(x - LOOKUP_WIDTH, 0, (int)img.w() - 1), xmax = math::clamp(x + LOOKUP_WIDTH, 0, (int)img.w() - 1);
		int ymin = math::clamp(y - LOOKUP_WIDTH, 0, (int)img.h() - 1), ymax = math::clamp(y + LOOKUP_WIDTH, 0, (int)img.h() - 1);
		for (int ti = xmin; ti <= xmax; ti++)
			for (int tj = ymin; tj <= ymax; tj++)
			{
				float tj_depth = img.Load(ti, tj).avg();
				if ((ti == x && tj == y) || tj_depth <= src_texel_depth)
					continue;
				Vec3f dst = Vec3f(float(ti) / img.w(), float(tj) / img.h(), tj_depth);
				float d;
				Vec2f kw;
				if (!text_cords_next_intersection<SEARCH_STEPS>(img, src, dst - src, d, kw))
					continue;
				float cone_ratio = d <= src_texel_depth ? 1.0f : length(src.getXY() - kw) / (d - src_texel_depth);
				if (*radius_cone > cone_ratio)
					*radius_cone = cone_ratio;
			}
	}
}

template<int SEARCH_STEPS> __global__ void generateRelaxedConeMap(imgData img, float* coneData, Vec3f Offset)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < img.w() && y < img.h())
	{
		Vec3f src = Vec3f(float(x) / img.w(), float(y) / img.h(), 0);
		Vec3f dst = src + Offset;
		dst.z = sample(img, dst);
		Vec3f vec = dst - src;
		vec /= vec.z;
		vec *= 1 - dst.z;
		Vec3f step_fwd = vec / SEARCH_STEPS;
		Vec3f ray_pos = dst + step_fwd;
		for (int i = 1; i < SEARCH_STEPS; i++)
		{
			float current_depth = sample(img, ray_pos);
			if (current_depth <= ray_pos.z)
				ray_pos += step_fwd;
		}
		float src_texel_depth = img.Load(x, y).avg();
		float cone_ratio = (ray_pos.z >= src_texel_depth) ? 1.0 : length(ray_pos.getXY() - src.getXY()) / (src_texel_depth - ray_pos.z);
		float best_ratio = coneData[y * img.w() + x];
		if (cone_ratio > best_ratio)
			cone_ratio = best_ratio;
		coneData[y * img.w() + x] = cone_ratio;
	}
}

void MIPMap::CreateRelaxedConeMap(const std::string& a_InputFile, FileOutputStream& a_Out)
{	imgData data;
	if (!parseImage(a_InputFile, data))
		throw std::runtime_error("Impossible to load texture file!");
	RGBCOL* hostData = (RGBCOL*)data.d(), *deviceData;
	CUDA_MALLOC(&deviceData, data.w() * data.h() * 4);
	data.d(deviceData);
	ThrowCudaErrors(hipMemcpy(deviceData, hostData, data.w() * data.h() * 4, hipMemcpyHostToDevice));
	float* deviceDepthData;
	CUDA_MALLOC(&deviceDepthData, data.w() * data.h() * 4);
	generateRelaxedConeMap<32, 64> << < dim3(data.w() / 16 + 1, data.h() / 16 + 1), dim3(16, 16) >> >(data, deviceDepthData);

	CUDA_FREE(deviceData);
	data.d(hostData);

	float* hostConeData = new float[data.w() * data.h()];
	ThrowCudaErrors(hipMemcpy(hostConeData, deviceDepthData, data.w() * data.h() * 4, hipMemcpyDeviceToHost));

	data.d(hostConeData); //do NOT free the actual image data, this will be done later
	data.RescaleToPowerOf2();
	hostConeData = (float*)data.d();

	FREE_IMAGE_FORMAT ff = FIF_PNG;
	FIBITMAP* bitmap = FreeImage_Allocate(data.w(), data.h(), 24, 0x000000ff, 0x0000ff00, 0x00ff0000);
	BYTE* A = FreeImage_GetBits(bitmap);
	unsigned int pitch = FreeImage_GetPitch(bitmap);
	int off = 0;
	for ( int y = 0; y < data.h(); y++)
	{
		for ( int x = 0; x < data.w(); x++)
		{
			float d = hostConeData[y * data.w() + x];
			unsigned char col = (unsigned char)(d * 255.0f);
			A[off + x * 3 + 0] = col;
			A[off + x * 3 + 1] = col;
			A[off + x * 3 + 2] = col;
		}
		off += pitch;
	}
	if (!FreeImage_Save(ff, bitmap, "../Data/conemap.png"))
		throw std::runtime_error(__FUNCTION__);
	FreeImage_Unload(bitmap);

	a_Out << data.w();
	a_Out << data.h();
	a_Out << (unsigned int)4;
	a_Out << (int)data.t();
	a_Out << (int)TEXTURE_REPEAT;
	a_Out << (int)TEXTURE_Anisotropic;
	a_Out << (unsigned int)1;
	a_Out << data.w() * data.h() * 8;

	float oldw = (float)data.w(), oldh = (float)data.h();
	float2* imgData = new float2[data.w() * data.h()];
	for ( int i = 0; i < data.w(); i++)
		for (int j = 0; j < data.h(); j++)
		{
			float xs = float(i) / data.w() * oldw, ys = float(j) / data.h() * oldh;
			Spectrum s;
			s.fromRGBCOL(hostData[(int)ys * (int)oldw + (int)xs]);
			float d = s.avg();
			float c = hostConeData[j * data.w() + i];
			imgData[j * data.w() + i] = make_float2(d, c);
		}
	a_Out.Write(imgData, data.w() * data.h() * sizeof(float2));
	delete[] imgData;

	unsigned int m_sOffsets[MAX_MIPS];
	a_Out.Write(m_sOffsets, sizeof(m_sOffsets));
	for (int i = 0; i < MTS_MIPMAP_LUT_SIZE; ++i)
	{
		float r2 = (float)i / (float)(MTS_MIPMAP_LUT_SIZE - 1);
		float val = math::exp(-2.0f * r2) - math::exp(-2.0f);
		a_Out << val;
	}

	data.Free();
	free(hostData);
}

bool parseImage(const std::string& a_InputFile, imgData& data)
{
	FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(a_InputFile.c_str(), 0);
	if (fif == FIF_UNKNOWN)
	{
		fif = FreeImage_GetFIFFromFilename(a_InputFile.c_str());
	}
	if ((fif != FIF_UNKNOWN) && FreeImage_FIFSupportsReading(fif))
	{
		FIBITMAP *dib = FreeImage_Load(fif, a_InputFile.c_str(), 0);
		if (!dib)
			return false;
		unsigned int w = FreeImage_GetWidth(dib);
		unsigned int h = FreeImage_GetHeight(dib);
		unsigned int scan_width = FreeImage_GetPitch(dib);
		unsigned int pitch = FreeImage_GetPitch(dib);
		FREE_IMAGE_TYPE imageType = FreeImage_GetImageType(dib);
		unsigned int bpp = FreeImage_GetBPP(dib);
		BYTE *bits = (BYTE *)FreeImage_GetBits(dib);
		Texture_DataType type = Texture_DataType::vtRGBCOL;
		if (((imageType == FIT_RGBAF) && (bpp == 128)) || ((imageType == FIT_RGBF) && (bpp == 96)))
			type = Texture_DataType::vtRGBE;
		data.Allocate(w, h, type);
		const auto* palette = FreeImage_GetPalette(dib);
		for (unsigned int y = 0; y < h; ++y)
		{
			for (unsigned int x = 0; x < w; ++x)
			{
				FIRGBAF *pixel = (FIRGBAF *)(bits + bpp / 8 * x);
				BYTE* pixel2 = bits + bpp / 8 * x;
				BYTE pixel3;
				if (type == Texture_DataType::vtRGBE)
					data.SetRGBE(SpectrumConverter::Float3ToRGBE(Vec3f(pixel->red, pixel->green, pixel->blue)), x, y);
				else if (bpp <= 8)
				{
					if (!FreeImage_GetPixelIndex(dib, x, y, &pixel3) || !palette)
						throw std::runtime_error("Error while reading image!");
					auto color = palette[pixel3];
					data.SetRGBCOL(make_uchar4(color.rgbRed, color.rgbGreen, color.rgbBlue, 255), x, y);
				}
				else data.SetRGBCOL(make_uchar4(pixel2[FI_RGBA_RED], pixel2[FI_RGBA_GREEN], pixel2[FI_RGBA_BLUE], bpp == 32 ? pixel2[FI_RGBA_ALPHA] : 255), x, y);
			}
			bits += pitch;
		}
		FreeImage_Unload(dib);
		return true;
	}
	else
	{
		return false;
	}
}

}
