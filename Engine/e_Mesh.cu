#include "hip/hip_runtime.h"
#include "e_Mesh.h"
#include "..\Base\StringUtils.h"
#include "e_TraceResult.h"

void e_TriIntersectorData::setData(const float3& a, const float3& b, const float3& c)
{
	//if(AbsDot(normalize(cross(a - c, b - c)), make_float3(1, 0, 0)) < 0.05f)
	/*{
		float l = length((a + b + c) / 3.0f - a) / 200.0f;
		//e += make_float3(0,0.02f,-0.02f);
		//p *= 1.02f;
		//q *= 1.02f;
		p = p + make_float3(l, -l, l);
		q = q + make_float3(-l, l, -l);
		e = -p - q;
		e = e + make_float3(0,l,-l)/200.0f;
	}*/
	float4x4 m;
	m.col(0, make_float4(a - c, 0));
	m.col(1, make_float4(b - c, 0));
	m.col(2, make_float4(cross(a - c, b - c), 0));
	m.col(3, make_float4(c, 1));
	m = m.inverse();
	this->a = make_float4(m(2, 0), m(2, 1), m(2, 2), -m(2, 3));
	this->b = m.row(0);
	this->c = m.row(1);
	float3 v1, v2, v3;
	getData(v1, v2, v3);
	//*(float2*)t2 = make_float2(m[0].x, m[0].y);
	//*(half2*)(((int*)t2) + 2) = half2(m[0].z, m[0].w);
	//this->b = make_float4(m[1].x, m[1].y, m[1].z, m[1].w);
	//if(this->a.x == -0.0f)
	//	this->a.x = 0.0f;
	//this->a = make_float4(m[2].y, m[2].z, -m[2].w, m[0].y);
	//this->b = make_float4(m[0].z, m[0].w, m[1].y, m[1].z);
	//t2->c = m[1].w;
	//t2->setXs(m[2].x, m[0].x, m[1].x);
}

void e_TriIntersectorData::getData(float3& v0, float3& v1, float3& v2) const
{
	float4x4 m = float4x4::Identity();
	m.row(0, b);
	m.row(1, c);
	m.row(2, a);
	m(2, 3) *= -1.0f;
	m = m.inverse();
	float3 e02 = !m.col(0), e12 = !m.col(1);
	v2 = !m.col(3);
	v0 = v2 + e02;
	v1 = v2 + e12;
}

bool e_TriIntersectorData::Intersect(const Ray& r, TraceResult* a_Result) const
{
	float Oz = a.w - r.origin.x*a.x - r.origin.y*a.y - r.origin.z*a.z;
	float invDz = 1.0f / (r.direction.x*a.x + r.direction.y*a.y + r.direction.z*a.z);
	float t = Oz * invDz;
	if (t > 0.0001f && t < a_Result->m_fDist)
	{
		float Ox = b.w + r.origin.x*b.x + r.origin.y*b.y + r.origin.z*b.z;
		float Dx = r.direction.x*b.x + r.direction.y*b.y + r.direction.z*b.z;
		float u = Ox + t*Dx;
		if (u >= 0.0f)
		{
			float Oy = c.w + r.origin.x*c.x + r.origin.y*c.y + r.origin.z*c.z;
			float Dy = r.direction.x*c.x + r.direction.y*c.y + r.direction.z*c.z;
			float v = Oy + t*Dy;
			if (v >= 0.0f && u + v <= 1.0f)
			{
				a_Result->m_fDist = t;
				a_Result->m_fUV = make_float2(u, v);
				return true;
			}
		}
	}
	return false;
}