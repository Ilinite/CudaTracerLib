#include "hip/hip_runtime.h"
#include "e_Light.h"
#include "e_Mesh.h"

Spectrum e_PointLight::sampleRay(Ray &ray, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	ray = Ray(lightPos, Warp::squareToUniformSphere(directionalSample));
	return m_intensity * (4 * PI);
}

Spectrum e_PointLight::sampleDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	dRec.p = lightPos;
	dRec.pdf = 1.0f;
	dRec.measure = EDiscrete;
	dRec.uv = Vec2f(0.5f);
	dRec.d = dRec.p - dRec.ref;
	dRec.dist = length(dRec.d);
	float invDist = 1.0f / dRec.dist;
	dRec.d *= invDist;
	dRec.n = Vec3f(0.0f);
	dRec.pdf = 1;
	dRec.measure = EDiscrete;

	return m_intensity * (invDist * invDist);
}

Spectrum e_PointLight::samplePosition(PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	pRec.p = lightPos;
	pRec.n = Vec3f(0);
	pRec.pdf = 1.0f;
	pRec.measure = EDiscrete;
	return m_intensity * (4 * PI);
}

Spectrum e_PointLight::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	dRec.d = Warp::squareToUniformSphere(sample);
	dRec.pdf = INV_FOURPI;
	dRec.measure = ESolidAngle;
	return Spectrum(1.0f);
}

void e_DiffuseLight::setEmit(const Spectrum& L)
{
	m_radiance = L;
	m_power = L * PI * shapeSet.Area();
}

Spectrum e_DiffuseLight::sampleRay(Ray &ray, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	PositionSamplingRecord pRec;
	shapeSet.SamplePosition(pRec, spatialSample);
	Vec3f local = m_bOrthogonal ? Vec3f(0, 0, 1) : Warp::squareToCosineHemisphere(directionalSample);
	ray = Ray(pRec.p, Frame(pRec.n).toWorld(local));
	return m_power;
}

Spectrum e_DiffuseLight::eval(const Vec3f& p, const Frame& sys, const Vec3f &d) const
{
	if (dot(sys.n, d) <= 0)
		return Spectrum(0.0f);
	else
	{
		if (m_bOrthogonal && dot(d, sys.n) < 1 - DeltaEpsilon)
			return 0.0f;
		else return m_radiance;
	}
}

Spectrum e_DiffuseLight::sampleDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	shapeSet.SamplePosition(dRec, sample);
	dRec.d = dRec.p - dRec.ref;
	float distSquared = dot(dRec.d, dRec.d);
	dRec.dist = math::sqrt(distSquared);
	dRec.d /= dRec.dist;
	float dp = absdot(dRec.d, dRec.n);
	dRec.pdf *= dp != 0 ? (distSquared / dp) : 0.0f;
	dRec.measure = ESolidAngle;
	if (dot(dRec.d, dRec.refN) >= 0 && dot(dRec.d, dRec.n) < 0 && dRec.pdf != 0 && ((dot(dRec.d, dRec.n) > 1 - DeltaEpsilon && m_bOrthogonal) || !m_bOrthogonal)) {
		return m_radiance / dRec.pdf;
	} else {
		dRec.pdf = 0.0f;
		return Spectrum(0.0f);
	}
}

float e_DiffuseLight::pdfDirect(const DirectSamplingRecord &dRec) const
{
	if (dot(dRec.d, dRec.refN) >= 0 && dot(dRec.d, dRec.n) < 0) {
		float pdfPos = shapeSet.Pdf(dRec);

		if (dRec.measure == ESolidAngle)
			return pdfPos * (dRec.dist * dRec.dist) / absdot(dRec.d, dRec.n);
		else if (dRec.measure == EArea)
			return pdfPos;
		else
			return 0.0f;
	} else {
		return 0.0f;
	}
}

Spectrum e_DiffuseLight::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	Vec3f local = Warp::squareToCosineHemisphere(sample);
	dRec.d = m_bOrthogonal ? pRec.n : Frame(pRec.n).toWorld(local);
	dRec.pdf = m_bOrthogonal ? 1 : Warp::squareToCosineHemispherePdf(local);
	dRec.measure = ESolidAngle;
	return Spectrum(1.0f);
}

float e_DiffuseLight::pdfDirection(const DirectionSamplingRecord &dRec, const PositionSamplingRecord &pRec) const
{
	float dp = dot(dRec.d, pRec.n);

	if (dRec.measure != ESolidAngle || dp < 0)
		dp = 0.0f;

	return m_bOrthogonal ? 1 : INV_PI * dp;
}

Spectrum e_DiffuseLight::evalDirection(const DirectionSamplingRecord &dRec, const PositionSamplingRecord &pRec) const
{
	float dp = dot(dRec.d, pRec.n);

	if (dRec.measure != ESolidAngle || dp < 0)
		dp = 0.0f;

	return m_bOrthogonal ? Spectrum(1.0f) : Spectrum(INV_PI * dp);
}

Spectrum e_DiffuseLight::evalPosition(const PositionSamplingRecord &pRec) const
{
	return m_radiance * PI;
}

void e_DistantLight::setEmit(const Spectrum& L)
{
	m_normalIrradiance = L;
	m_power = m_normalIrradiance / m_invSurfaceArea;
}

Spectrum e_DistantLight::sampleRay(Ray &ray, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	Vec2f p = Warp::squareToUniformDiskConcentric(spatialSample);
	Vec3f perpOffset = ToWorld.toWorld(Vec3f(p.x, p.y, 0) * radius);
	Vec3f d = ToWorld.toWorld(Vec3f(0, 0, 1));
	ray = Ray(d * radius + perpOffset, d);
	return m_power;
}

Spectrum e_DistantLight::sampleDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	Vec3f d = ToWorld.toWorld(Vec3f(0, 0, 1));
	Vec3f diskCenter = d * radius;

	float distance = dot(dRec.ref - diskCenter, d);
	if (distance < 0) {
		/* This can happen when doing bidirectional renderings
			involving environment maps and directional sources. Just
			return zero */
		return Spectrum(0.0f);
	}

	dRec.p = dRec.ref - distance * d;
	dRec.d = -d;
	dRec.n = d;
	dRec.dist = distance;

	dRec.pdf = 1.0f;
	dRec.measure = EDiscrete;
	return m_normalIrradiance;
}

Spectrum e_DistantLight::samplePosition(PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	Vec2f p = Warp::squareToUniformDiskConcentric(sample);

	Vec3f perpOffset = ToWorld.toWorld(Vec3f(p.x, p.y, 0) * radius);
	Vec3f d = ToWorld.toWorld(Vec3f(0, 0, 1));

	pRec.p = d * radius + perpOffset;
	pRec.n = d;
	pRec.pdf = m_invSurfaceArea;
	pRec.measure = EArea;
	return m_power;
}

Spectrum e_DistantLight::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	dRec.d = pRec.n;
	dRec.pdf = 1.0f;
	dRec.measure = EDiscrete;
	return Spectrum(1.0f);
}

e_SpotLight::e_SpotLight(Vec3f p, Vec3f t, Spectrum L, float width, float fall)
	: e_LightBase(true), m_intensity(L), m_cutoffAngle(math::Radians(width)), m_beamWidth(math::Radians(fall))
{
	m_cosBeamWidth = cosf(m_beamWidth);
	m_cosCutoffAngle = cosf(m_cutoffAngle);
	m_invTransitionWidth = 1.0f / (m_cutoffAngle - m_beamWidth);
	Position = p;
	Target = t;
	ToWorld = Frame(t - p);
}

Spectrum e_SpotLight::sampleRay(Ray &ray, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	Vec3f local = Warp::squareToUniformCone( m_cosCutoffAngle, directionalSample);
	ray = Ray(Position, ToWorld.toWorld(local));
	float dirPdf = Warp::squareToUniformConePdf(m_cosCutoffAngle);
	return m_intensity * falloffCurve(local) / dirPdf;
}

Spectrum e_SpotLight::sampleDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	dRec.p = Position;
	dRec.pdf = 1.0f;
	dRec.measure = EDiscrete;
	dRec.uv = Vec2f(0.5f);
	dRec.d = dRec.p - dRec.ref;
	dRec.dist = length(dRec.d);
	float invDist = 1.0f / dRec.dist;
	dRec.d *= invDist;
	dRec.n = Vec3f(0.0f);
	dRec.pdf = 1;
	dRec.measure = EDiscrete;

	return m_intensity * falloffCurve(ToWorld.toLocal(-dRec.d)) * (invDist * invDist);
}

Spectrum e_SpotLight::samplePosition(PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	pRec.p = Position;
	pRec.n = Vec3f(0.0f);
	pRec.pdf = 1.0f;
	pRec.measure = EDiscrete;
	return m_intensity * (4 * PI);
}

Spectrum e_SpotLight::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	Vec3f d = Warp::squareToUniformCone(m_cosCutoffAngle, sample);
	dRec.d = ToWorld.toWorld(d);
	dRec.pdf = Warp::squareToUniformConePdf(m_cosCutoffAngle);
	dRec.measure = ESolidAngle;
	return evalDirection(dRec, pRec)/dRec.pdf;
}

Spectrum e_SpotLight::falloffCurve(const Vec3f &d) const
{
	const float cosTheta = Frame::cosTheta(normalize(d));

	if (cosTheta <= m_cosCutoffAngle)
		return Spectrum(0.0f);
	if (cosTheta >= m_cosBeamWidth)
		return 1.0f;
	return ((m_cutoffAngle - acosf(cosTheta)) * m_invTransitionWidth);
}

e_InfiniteLight::e_InfiniteLight(e_Stream<char>* a_Buffer, e_BufferReference<e_MIPMap, e_KernelMIPMap>& mip, const Spectrum& scale, const AABB& scenBox)
	: e_LightBase(false), radianceMap(mip->getKernelData()), m_SceneCenter(scenBox.Center()), m_SceneRadius(length(scenBox.Size()) / 1.5f), m_scale(scale)
{
	float surfaceArea = 4 * PI * m_SceneRadius * m_SceneRadius;
	m_invSurfaceArea = 1 / surfaceArea;

	m_size = Vec2f(radianceMap.m_uWidth, radianceMap.m_uHeight);
	unsigned int nEntries = (unsigned int) (m_size.x + 1) * (unsigned int) m_size.y;
	e_StreamReference(char) m1 = a_Buffer->malloc(nEntries * sizeof(float)), m2 = a_Buffer->malloc((m_size.y + 1) * sizeof(float)), m3 = a_Buffer->malloc(m_size.y * sizeof(float));
	m_cdfCols = m1.AsVar<float>();
	m_cdfRows = m2.AsVar<float>();
	m_rowWeights = m3.AsVar<float>();
	unsigned int colPos = 0, rowPos = 0;
	float rowSum = 0.0f;
	m_cdfRows[rowPos++] = 0;
	for (int y=0; y<m_size.y; ++y)
	{
		float colSum = 0;

		m_cdfCols[colPos++] = 0;
		for (int x=0; x<m_size.x; ++x)
		{
			Spectrum value = radianceMap.Sample(0, x, y);

			colSum += value.getLuminance();
			m_cdfCols[colPos++] = (float) colSum;
		}

		float normalization = 1.0f / (float) colSum;
		for (int x=1; x<m_size.x; ++x)
			m_cdfCols[colPos-x-1] *= normalization;
		m_cdfCols[colPos-1] = 1.0f;

		float weight = sinf((y + 0.5f) * PI / float(m_size.y));
		m_rowWeights[y] = weight;
		rowSum += colSum * weight;
		m_cdfRows[rowPos++] = (float) rowSum;
	}
	float normalization = 1.0f / (float) rowSum;
	for (int y=1; y<m_size.y; ++y)
		m_cdfRows[rowPos-y-1] *= normalization;
	m_cdfRows[rowPos-1] = 1.0f;
	m_normalization = 1.0f / (rowSum * (2 * PI / m_size.x) * (PI / m_size.y));
	m_pixelSize = Vec2f(2 * PI / m_size.x, PI / m_size.y);
	m1.Invalidate(); m2.Invalidate(); m3.Invalidate();

	float lvl = 0.65f;
	unsigned int INDEX = sampleReuse(m_cdfRows.operator->(), m_size.y, lvl);

	m_power = (surfaceArea * m_scale / m_normalization).average();

	m_worldTransform = m_worldTransformInverse = float4x4::Identity();
}

Spectrum e_InfiniteLight::sampleRay(Ray &ray, const Vec2f &spatialSample, const Vec2f &directionalSample) const
{
	Vec3f d; Spectrum value; float pdf;
	internalSampleDirection(directionalSample, d, value, pdf);
	d = m_worldTransform.TransformDirection(-d);
	Vec2f offset = Warp::squareToUniformDiskConcentric(spatialSample);
	Vec3f perpOffset = Frame(d).toWorld(Vec3f(offset.x, offset.y, 0));
	ray = Ray(m_SceneCenter + (perpOffset - d) * m_SceneRadius, d);

	return value * PI * m_SceneRadius * m_SceneRadius / pdf;
}

Spectrum e_InfiniteLight::sampleDirect(DirectSamplingRecord &dRec, const Vec2f &sample) const
{
	/* Sample a direction from the environment map */
	Spectrum value; Vec3f d; float pdf;
	internalSampleDirection(sample, d, value, pdf);
	d = m_worldTransform.TransformDirection(d);

	dRec.pdf = pdf;
	dRec.p = m_SceneCenter + d * m_SceneRadius;
	dRec.n = -normalize(d);
	dRec.dist = m_SceneRadius;
	dRec.d = d;
	dRec.measure = ESolidAngle;

	return value / pdf;
}

float e_InfiniteLight::pdfDirect(const DirectSamplingRecord &dRec) const
{
	float pdfSA = internalPdfDirection(m_worldTransformInverse.TransformDirection(dRec.d));

	if (dRec.measure == ESolidAngle)
		return pdfSA;
	else if (dRec.measure == EArea)
		return pdfSA * absdot(dRec.d, dRec.n) / (dRec.dist * dRec.dist);
	else
		return 0.0f;
}

Spectrum e_InfiniteLight::samplePosition(PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	Vec3f d = Warp::squareToUniformSphere(sample);

	pRec.p = m_SceneCenter + d * m_SceneRadius;
	pRec.n = -d;
	pRec.measure = EArea;
	pRec.pdf = m_invSurfaceArea;

	return Spectrum(m_power);
}

Spectrum e_InfiniteLight::sampleDirection(DirectionSamplingRecord &dRec, PositionSamplingRecord &pRec, const Vec2f &sample, const Vec2f *extra) const
{
	Spectrum value; Vec3f d; float pdf;
	internalSampleDirection(sample, d, value, pdf);

	dRec.measure = ESolidAngle;
	dRec.pdf = pdf;
	dRec.d = m_worldTransform .TransformDirection(-d);

	/* Be wary of roundoff errors */
	if (value.isZero() || pdf == 0)
		return Spectrum(0.0f);
	else
		return (value * m_normalization) / (pdf * m_scale);
}

Spectrum e_InfiniteLight::evalDirection(const DirectionSamplingRecord &dRec, const PositionSamplingRecord &pRec) const
{
	Vec3f v = m_worldTransformInverse.TransformDirection(-1.0f * dRec.d);

	/* Convert to latitude-longitude texture coordinates */
	Vec2f uv = Vec2f(
		atan2f(v.x, -v.z) * INV_TWOPI,
		math::safe_acos(v.y) * INV_PI
	);

	return radianceMap.Sample(uv, 0) * m_normalization;
}

void e_InfiniteLight::internalSampleDirection(Vec2f sample, Vec3f &d, Spectrum &value, float &pdf) const
{
	unsigned int	row = sampleReuse(m_cdfRows.operator->(), m_size.y, sample.y),
					col = sampleReuse(m_cdfCols.operator->() + row * unsigned int(m_size.x+1), m_size.x, sample.x);

	/* Using the remaining bits of precision to shift the sample by an offset
		drawn from a tent function. This effectively creates a sampling strategy
		for a linearly interpolated environment map */
		
	Vec2f pos = Vec2f(col, row) + Warp::squareToTent(sample);
	//Vec2f pos = sample * m_size;

	/* Bilinearly interpolate colors from the adjacent four neighbors */
	int xPos = math::clamp(math::Floor2Int(pos.x), 0, int(m_size.x - 1)), yPos = math::clamp(math::Floor2Int(pos.y), 0, int(m_size.y - 1));
	float dx1 = pos.x - xPos, dx2 = 1.0f - dx1,
		  dy1 = pos.y - yPos, dy2 = 1.0f - dy1;

	Spectrum value1 = radianceMap.Sample(0, xPos, yPos) * dx2 * dy2
		            + radianceMap.Sample(0, xPos + 1, yPos) * dx1 * dy2;
	Spectrum value2 = radianceMap.Sample(0, xPos, yPos + 1) * dx2 * dy1
		            + radianceMap.Sample(0, xPos + 1, yPos + 1) * dx1 * dy1;

	/* Compute the final color and probability density of the sample */
	value = (value1 + value2) * m_scale;
	pdf = (value1.getLuminance() * m_rowWeights[(int)math::clamp(float(yPos),   0.0f, m_size.y-1.0f)] +
		    value2.getLuminance() * m_rowWeights[(int)math::clamp(float(yPos+1), 0.0f, m_size.y-1.0f)]) * m_normalization;

	/* Turn into a proper direction on the sphere */
	float sinPhi, cosPhi, sinTheta, cosTheta;
	sincos(m_pixelSize.x * (pos.x + 0.5f), &sinPhi, &cosPhi);
	sincos(m_pixelSize.y * (pos.y + 0.5f), &sinTheta, &cosTheta);

	d = Vec3f(sinPhi*sinTheta, cosTheta, -cosPhi*sinTheta);
	pdf /= max(math::abs(sinTheta), EPSILON);
}

float e_InfiniteLight::internalPdfDirection(const Vec3f &d) const
{
	Vec2f uv = Vec2f(
		atan2f(d.x, -d.z) * INV_TWOPI,
		math::safe_acos(d.y) * INV_PI
	);
	float u = uv.x * m_size.x - 0.5f, v = uv.y * m_size.y - 0.5f;
	int xPos = math::Floor2Int(u), yPos = math::Floor2Int(v);
	float dx1 = u - xPos, dx2 = 1.0f - dx1,
		    dy1 = v - yPos, dy2 = 1.0f - dy1;
	Spectrum value1 = radianceMap.Sample(0, xPos, yPos) * dx2 * dy2
		            + radianceMap.Sample(0, xPos + 1, yPos) * dx1 * dy2;
	Spectrum value2 = radianceMap.Sample(0, xPos, yPos + 1) * dx2 * dy1
		            + radianceMap.Sample(0, xPos + 1, yPos + 1) * dx1 * dy1;
	float sinTheta = math::safe_sqrt(1-d.y*d.y);
	return (value1.getLuminance() * m_rowWeights[math::clamp(yPos,   0, (int)m_size.y-1)] +
		    value2.getLuminance() * m_rowWeights[math::clamp(yPos+1, 0, (int)m_size.y-1)])
			* m_normalization / max(math::abs(sinTheta), EPSILON);
}

unsigned int e_InfiniteLight::sampleReuse(float *cdf, unsigned int size, float &sample) const
{
	const float *entry = STL_lower_bound(cdf, cdf+size, sample);
	//unsigned int index = min(unsigned int(size - 2U), max(0U, unsigned int(entry - cdf - 1)));
	unsigned int index = min(max(0u, unsigned int(entry - cdf - 1)), unsigned int(size - 1));
	sample = (sample - cdf[index]) / (cdf[index+1] - cdf[index]);
	return index;
}

Spectrum e_InfiniteLight::evalEnvironment(const Ray &ray) const
{
	Vec3f v = normalize(m_worldTransformInverse.TransformDirection(ray.direction));

	/* Convert to latitude-longitude texture coordinates */
	Vec2f uv = Vec2f(
		atan2f(v.x, -v.z) * INV_TWOPI,
		math::safe_acos(v.y) * INV_PI
	);

	Spectrum value = radianceMap.Sample(uv, 0);

	return value * m_scale;
}

Spectrum e_InfiniteLight::evalEnvironment(const Ray &ray, const Ray& rX, const Ray& rY) const
{
	Vec3f v = normalize(m_worldTransformInverse.TransformDirection(ray.direction));

	/* Convert to latitude-longitude texture coordinates */
	Vec2f uv = Vec2f(
		atan2f(v.x, -v.z) * INV_TWOPI,
		math::safe_acos(v.y) * INV_PI
		);

	Vec3f  dvdx = rX.direction - v,
			dvdy = rY.direction - v;

	float	t1 = INV_TWOPI / (v.x*v.x + v.z*v.z),
			t2 = -INV_PI / max(math::safe_sqrt(1.0f - v.y*v.y), 1e-4f);

	Vec2f	dudx = Vec2f(t1 * (dvdx.z*v.x - dvdx.x*v.z), t2 * dvdx.y),
			dudy = Vec2f(t1 * (dvdy.z*v.x - dvdy.x*v.z), t2 * dvdy.y);

	Spectrum value = radianceMap.eval(uv, dudx, dudy);

	return value * m_scale;
}