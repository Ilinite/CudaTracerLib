#include "hip/hip_runtime.h"
#include "FastTracer.h"
#include <Kernel/TraceHelper.h>
#include <Kernel/TraceAlgorithms.h>
#include <Engine/DynamicScene.h>

namespace CudaTracerLib {

CUDA_DEVICE CudaStaticWrapper<FastTracerBuffer> g_primary_ray_buffer;
CUDA_DEVICE int g_NextRayCounterFT;

enum
{
	MaxBlockHeight = 6,
};

__global__ void pathCreateKernelFT(unsigned int w, unsigned int h)
{
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	const int tidx = threadIdx.x;
	volatile int& rayBase = nextRayArray[threadIdx.y];
	do
	{
		if (tidx == 0)
			rayBase = atomicAdd(&g_NextRayCounterFT, blockDim.x);

		int rayidx = rayBase + tidx;
		if (rayidx >= w * h)
			break;

		int x = rayidx % w, y = rayidx / w;
		NormalizedT<Ray> r;
		Spectrum W = g_SceneData.sampleSensorRay(r, Vec2f(x, y), Vec2f(0, 0));
		g_primary_ray_buffer->insertPayloadElement({(unsigned short)x, (unsigned short)y}, r);
	} while (true);
}

__global__ void doDirectKernel(unsigned int w, unsigned int h, Image I, float SCALE, bool depthImage, DeviceDepthImage dImg)
{
	EmptyRayData payload;
	NormalizedT<Ray> ray;
	TraceResult res;
	unsigned int rayIdx;
	while (g_primary_ray_buffer->tryFetchPayloadElement(payload, ray, res))
	{
		Spectrum s = 0.0f;
		if (res.hasHit())
			s = Spectrum(res.m_fDist / SCALE);
		I.AddSample(payload.x, payload.y, s);
		if (depthImage)
			dImg.Store(rayIdx % w, rayIdx / w, res.m_fDist);
	}
}

void FastTracer::DoRender(Image* I)
{
	bufA->StartFrame(g_SceneData.m_rayTraceEps);

	ZeroSymbol(g_NextRayCounterFT);
	CopyToSymbol(g_primary_ray_buffer, *bufA);
	pathCreateKernelFT << < dim3(180, 1, 1), dim3(32, 6, 1) >> >(w, h);
	CopyFromSymbol(*bufA, g_primary_ray_buffer);

	bufA->FinishIteration();

	I->Clear();
	ZeroSymbol(g_NextRayCounterFT);
	CopyToSymbol(g_primary_ray_buffer, *bufA);
	float scl = length(g_SceneData.m_sBox.Size());
	doDirectKernel << < dim3(180, 1, 1), dim3(32, 6, 1) >> >(w, h, *I, scl, hasDepthBuffer(), getDeviceDepthBuffer());
}

}