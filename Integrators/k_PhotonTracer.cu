#include "hip/hip_runtime.h"
#include "k_PhotonTracer.h"
#include "../Kernel/k_TraceHelper.h"
#include "../Kernel/k_TraceAlgorithms.h"
#include "../Engine/e_Light.h"
#include "../Engine/e_Sensor.h"

enum
{
	MaxBlockHeight = 6,
};

CUDA_DEVICE unsigned int g_NextRayCounter3;

CUDA_FUNC_IN void handleEmission(const Spectrum& weight, const PositionSamplingRecord& pRec, e_Image& g_Image, CudaRNG& rng)
{
	DirectSamplingRecord dRec(pRec.p, pRec.n);
	Spectrum value = weight * g_SceneData.sampleSensorDirect(dRec, rng.randomFloat2());
	if (!value.isZero() && ::V(dRec.p, dRec.ref))
	{
		const e_KernelLight* emitter = (const e_KernelLight*)pRec.object;
		value *= emitter->evalDirection(DirectionSamplingRecord(dRec.d), pRec);
		g_Image.Splat(dRec.uv.x, dRec.uv.y, value);
	}
}

CUDA_FUNC_IN void handleSurfaceInteraction(const Spectrum& weight, BSDFSamplingRecord& bRec, const TraceResult& r2, e_Image& g_Image, CudaRNG& rng)
{
	DirectSamplingRecord dRec(bRec.dg.P, bRec.dg.sys.n);
	Spectrum value = weight * g_SceneData.sampleSensorDirect(dRec, rng.randomFloat2());
	if(!value.isZero() && ::V(dRec.p, dRec.ref))
	{
		bRec.wo = bRec.dg.toLocal(dRec.d);
		value *= r2.getMat().bsdf.f(bRec);
		g_Image.Splat(dRec.uv.x, dRec.uv.y,  value);
	}
}

CUDA_FUNC_IN void doWork(e_Image& g_Image, CudaRNG& rng)
{
	PositionSamplingRecord pRec;
	Spectrum power = g_SceneData.sampleEmitterPosition(pRec, rng.randomFloat2()), throughput = Spectrum(1.0f);

	handleEmission(power, pRec, g_Image, rng);
	
	DirectionSamplingRecord dRec;
	power *= ((const e_KernelLight*)pRec.object)->sampleDirection(dRec, pRec, rng.randomFloat2());

	Ray r(pRec.p, dRec.d);
	TraceResult r2;
	r2.Init();
	int depth = -1;
	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	while(++depth < 12 && k_TraceRay(r.direction, r.origin, &r2))
	{
		r2.getBsdfSample(r, bRec, ETransportMode::EImportance, &rng);
		
		handleSurfaceInteraction(power * throughput, bRec, r2, g_Image, rng);

		Spectrum bsdfWeight = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());

		r = Ray(bRec.dg.P, bRec.getOutgoing());
		r2.Init();
		if(bsdfWeight.isZero())
			break;
		throughput *= bsdfWeight;
		if(depth > 5)
		{
			float q = min(throughput.max(), 0.95f);
			if(rng.randomFloat() >= q)
				break;
			throughput /= q;
		}
	}
}

__global__ void pathKernel(unsigned int N, e_Image g_Image)
{
	CudaRNG rng = g_RNGData();
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	volatile int& rayBase = nextRayArray[threadIdx.y];
	do
	{
		if (threadIdx.x == 0)
			rayBase = atomicAdd(&g_NextRayCounter3, blockDim.x);

		int rayidx = rayBase + threadIdx.x;
		if (rayidx >= N)
			break;

		doWork(g_Image, rng);
	} while (true);
	g_RNGData(rng);
}

void k_PhotonTracer::DoRender(e_Image* I)
{
	unsigned int zero = 0;
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_NextRayCounter3), &zero, sizeof(unsigned int)));
	k_INITIALIZE(m_pScene, g_sRngs);
	pathKernel << < 180, dim3(32, MaxBlockHeight, 1) >> >(w * h, *I);
	ThrowCudaErrors(hipDeviceSynchronize());
}

void k_PhotonTracer::Debug(e_Image* I, const Vec2i& pixel)
{
	k_INITIALIZE(m_pScene, g_sRngs);
	CudaRNG rng = g_RNGData();
	doWork(*I, rng);
	g_RNGData(rng);
}