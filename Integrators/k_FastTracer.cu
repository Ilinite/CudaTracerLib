#include "hip/hip_runtime.h"
#include "k_FastTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include "..\Kernel\k_TraceAlgorithms.h"

__global__ void pathCreateKernel(unsigned int w, unsigned int h, k_RayBuffer<k_FastTracer::rayData, 1> g_Intersector)
{
	int idx = threadId;
	if(idx >= w * h)
		return;
	int x = idx % w, y = idx / w;
	Ray r;
	g_SceneData.sampleSensorRay(r, make_float2(x,y), make_float2(0,0));
	traversalRay& ray = g_Intersector(idx, 0);
	ray.a = make_float4(r.origin, 0.0f);
	ray.b = make_float4(r.direction, FLT_MAX);
	k_FastTracer::rayData& dat = g_Intersector(idx);
	dat.x = x;
	dat.y = y;
	dat.throughput = Spectrum(1.0f);
	dat.L = Spectrum(0.0f);
}

__global__ void doDirectKernel(unsigned int w, unsigned int h, k_RayBuffer<k_FastTracer::rayData, 1> g_Intersector, e_Image I, float SCALE)
{/*
	CudaRNG rng = g_RNGData();
	int rayidx;
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        const bool          terminated     = 1;//nodeAddr == EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));	

        if(terminated)
        {			
            if (idxTerminated == 0)
				rayBase = atomicAdd(&g_NextRayCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
			if (rayidx >= N)
                break;
		}

		k_FastTracer::rayData r = a_RayBuffer[rayidx];
		TraceResult r2 = a_ResBuffer[rayidx];
		if(r2.hasHit())
		{
			g_Image.SetPixel(r.x, r.y, Spectrum(r2.m_fDist/length(g_SceneData.m_sBox.Size())));
			continue;
		}else continue;

		BSDFSamplingRecord bRec;
		r2.getBsdfSample(r.r, rng, &bRec);
		Spectrum bsdfWeight = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
		//DirectSamplingRecord dRec(bRec.map.P, bRec.ng, bRec.map.uv);
		//g_Image.SetPixel(r.x,r.y, bsdfWeight * g_SceneData.sampleEmitterDirect(dRec, rng.randomFloat2()));
		
		r.L += r2.Le(bRec.map.P, bRec.map.sys, -r.r.direction) * r.throughput;
		r.throughput *= bsdfWeight;
		a_ResBuffer[rayidx].Init();
		r.throughput *= bsdfWeight;
		r.r.origin = bRec.map.P;
		r.r.direction = bRec.getOutgoing();
		unsigned int id = atomicInc(&g_NextInsertCounter, -1);
		//r.L += UniformSampleAllLights(bRec, r2.getMat(), 1);
		a_RayBuffer[id] = r;
		if(lastPass)
		{
			g_Image.SetPixel(r.x, r.y, r.L);
		}

	}
	while(true);
	g_RNGData(rng);*/ 
	unsigned int idx = threadId;
	if(idx >= w * h)
		return;
	traversalResult& res = g_Intersector.res(idx, 0);
	RGBCOL col;
	col.x = col.w = 255;
	col.y = col.z = 0;
	if(res.dist)
	{
		//tar[rayidx] = Spectrum(a_ResBuffer[rayidx].m_fDist/SCALE).toRGBCOL();
		float f = res.dist/SCALE * 255.0f;
		unsigned char c = (unsigned char)f;
		unsigned int i = (255 << 24) | (c << 16) | (c << 8) | c;
		col = *(RGBCOL*)&i;
	}
	I.SetSample(idx % w, idx / w, *(RGBCOL*)&col);
}

#define MAX_PASS 5
__global__ void pathIterateKernel(unsigned int N, e_Image I, int pass, k_RayBuffer<k_FastTracer::rayData, 1> g_Intersector, k_RayBuffer<k_FastTracer::rayData, 1> g_Intersector2)//template
{
    unsigned int idx = threadId;
	if(idx >= N)
		return;
		CudaRNG rng = g_RNGData();
	traversalResult& res = g_Intersector.res(idx, 0);
	traversalRay& ray = g_Intersector(idx, 0);
	k_FastTracer::rayData dat = g_Intersector(idx);
	if(res.dist)
	{
		Ray r(!ray.a, !ray.b);
		TraceResult r2;
		res.toResult(&r2, g_SceneData);
		DifferentialGeometry dg;
		BSDFSamplingRecord bRec(dg);
		r2.getBsdfSample(r, rng, &bRec);
		
		//traversalResult* tr;
	//	g_Intersector[1].FetchRay(d.dIndex, &tr);
		//if(pass && ((tr->dist >= d.dDist) || (tr->dist == 0.0f)))
		//	d.L += d.D;
		//d.D = Spectrum(0.0f);
		
		dat.L += r2.Le(bRec.dg.P, bRec.dg.sys, -r.direction) * dat.throughput;
		Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
		dat.throughput *= f;
		unsigned int idx2 = g_Intersector2.insertRay();
		traversalRay& ray2 = g_Intersector2(idx2, 0);
		ray.a = make_float4(bRec.dg.P, 1e-2f);
		ray.b = make_float4(bRec.getOutgoing(), FLT_MAX);
		g_Intersector2(idx2) = dat;
		if(pass + 1 == MAX_PASS)
			I.AddSample(dat.x, dat.y, dat.L);
		/*
		if(pass != MAX_PASS)
		{
			DirectSamplingRecord dRec(bRec.map.P, bRec.map.sys.n, bRec.map.uv);
			Spectrum value = g_SceneData.sampleEmitterDirect(dRec, rng.randomFloat2());
			bRec.wo = bRec.map.sys.toLocal(dRec.d);
			Spectrum bsdfVal = r2.getMat().bsdf.f(bRec);
			const float bsdfPdf = r2.getMat().bsdf.pdf(bRec);
			const float weight = MonteCarlo::PowerHeuristic(1, dRec.pdf, 1, bsdfPdf);
			d.D = value * weight * bsdfVal * d.throughput;
			traversalRay* ray2 = g_Intersector[1].InsertRay(payloadIdx, &d.dIndex);
			ray2->a = make_float4(dRec.ref, 0);
			ray2->b = make_float4(dRec.d, FLT_MAX);
			d.dDist = dRec.dist;
		}*/
	}
	else I.AddSample(dat.x, dat.y, dat.L);
	g_RNGData(rng);
}

#include "..\Base\Timer.h"
static cTimer TT;
void k_FastTracer::doDirect(e_Image* I)
{
	k_RayBuffer<rayData, 1>* buf = bufA;
	k_ProgressiveTracer::DoRender(I);
	k_INITIALIZE(m_pScene, g_sRngs);
	float scl = length(g_SceneData.m_sBox.Size());
	pathCreateKernel<<< dim3((w*h)/(32*8)+1,1,1), dim3(32, 8, 1)>>>(w, h, *buf);
	buf->setGeneratedRayCount(w * h);
	/*
	TT.StartTimer();
	Ray r;
	float2 ps, at = make_float2(0);
	for(int i = 0; i < w; i++)
		for(int j = 0; j < h; j++)
		{
			float4* t = (float4*)(hostRays + (j * w + i));
			ps.x = i;
			ps.y = j;
			m_pCamera->sampleRay(r, ps, at);
			t[0] = make_float4(r.origin, 0);
			t[1] = make_float4(r.direction, FLT_MAX);
		}
	hipMemcpy(intersector->m_pRayBuffer, hostRays, sizeof(traversalRay) * w * h, hipMemcpyHostToDevice);
	m_fTimeSpentRendering = (float)TT.EndTimer();*/
	
	hipEventRecord(start, 0);
	buf->IntersectBuffers<false>(m_pScene->getNodeCount() == 1);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	m_fTimeSpentRendering = elapsedTime * 1e-3f;
		/*
	hipEventRecord(start, 0);
	I->StartNewRendering();
	pathIterateKernel<<< dim3((w*h)/(32*8)+1,1,1), dim3(32, 8, 1)>>>(w * h, w, *intersector, *I, 1, 1);
	I->UpdateDisplay();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	m_fTimeSpentRendering = elapsedTime * 1e-3f;

	TT.StartTimer();
	I->StartNewRendering();
	CudaRNG rng = g_RNGData();
	TraceResult r2;
	BSDFSamplingRecord bRec;
	hipMemcpy(hostResults, intersector->m_pResultBuffer, sizeof(traversalResult) * w * h, hipMemcpyDeviceToHost);
	for(int i = 0; i < w; i++)
		for(int j = 0; j < h; j++)
		{
			int id = j * w + i;
			if(hostResults[id].dist)
			{
				r.origin = !hostRays[id].a;
				r.direction = !hostRays[id].b;
				hostResults[id].toResult(&r2, g_SceneData);
				//r2.getBsdfSample(r, rng, &bRec);
				I->AddSample(i,j,Spectrum(-dot(bRec.ng, r.direction)));
			}
		}
	g_RNGData(rng);
	I->UpdateDisplay();
	m_fTimeSpentRendering = (float)TT.EndTimer();*/

	doDirectKernel<<< dim3((w*h)/(32*8)+1,1,1), dim3(32, 8, 1)>>>(w, h, *buf, *I, scl);
	
	m_uPassesDone++;
	m_uNumRaysTraced = w * h;
	hipEventRecord(start, 0);
}

void k_FastTracer::doPath(e_Image* I)
{
	k_ProgressiveTracer::DoRender(I);
	k_INITIALIZE(m_pScene, g_sRngs);
	pathCreateKernel<<< dim3((w*h)/(32*8)+1,1,1), dim3(32, 8, 1)>>>(w, h, *bufA);
	bufA->setGeneratedRayCount(w * h);
	int pass = 0;
	k_RayBuffer<rayData, 1>* srcBuf = bufA, *destBuf = bufB;
	m_uNumRaysTraced = 0;
	do
	{
		m_uNumRaysTraced += srcBuf->IntersectBuffers<false>(m_pScene->getNodeCount() == 1);
		unsigned int n = srcBuf->getCreatedRayCount();
		destBuf->setGeneratedRayCount(0);
		pathIterateKernel<<< dim3(n/(32*8)+1,1,1), dim3(32, 8, 1)>>>(n, *I, pass, *srcBuf, *destBuf);
		swapk(srcBuf, destBuf);
	}
	while(srcBuf->getCreatedRayCount() && ++pass < MAX_PASS);
	m_uPassesDone++;
	I->DoUpdateDisplay(m_uPassesDone);
}

void k_FastTracer::DoRender(e_Image* I)
{
	//doPath(I);
	doDirect(I);
}

void k_FastTracer::Debug(e_Image* I, int2 pixel)
{
	std::cout << "x : " << pixel.x << ", y : " << pixel.y << "\n";
}