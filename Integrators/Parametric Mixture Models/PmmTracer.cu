#include "hip/hip_runtime.h"
#include "PmmTracer.h"
#include <Kernel/TraceHelper.h>
#include <Kernel/TraceAlgorithms.h>

namespace CudaTracerLib {

CUDA_DEVICE SpatialLinkedMap<SpatialEntry> g_sMap;
CUDA_DEVICE SpatialSet<DirectionModel> g_dMap;

__global__ void tracePhotons()
{
	CudaRNG rng = g_RNGData();
	TraceResult r2;
	Ray r;
	g_SceneData.sampleEmitterRay(r, rng.randomFloat2(), rng.randomFloat2());
	int depth = 0;
	while ((r2 = traceRay(r)).hasHit() && depth++ < 7)
	{
		DifferentialGeometry dg;
		BSDFSamplingRecord bRec(dg);
		r2.getBsdfSample(r, bRec, ETransportMode::EImportance, &rng);
		Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
		if (f.isZero())
			break;
		Vec3f p = r(r2.m_fDist);
		g_sMap.store(p, bRec.wi);
		if (depth > 5)
			if (rng.randomFloat() >= f.max())
				break;
		r = Ray(p, bRec.getOutgoing());
		r2.Init();
	}
	g_RNGData(rng);
}

template<int max_SAMPLES> __global__ void updateCache(float ny)
{
	Vec3u i = Vec3u(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y,
		blockIdx.z * blockDim.z + threadIdx.z);
	if (i.x < g_dMap.gridSize && i.y < g_dMap.gridSize && i.z < g_dMap.gridSize)
	{
		Vec3f mi = g_dMap.hashMap.InverseTransform(i), ma = g_dMap.hashMap.InverseTransform(i + Vec3u(1));
		unsigned int idx = g_dMap.hashMap.Hash(i);
		g_dMap(idx).Update<max_SAMPLES>(g_sMap, mi, ma, ny);
	}
}

__global__ void visualize(Image I, int w, int h, float scale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < w && y < h)
	{
		Ray r = g_SceneData.GenerateSensorRay(x, y);
		TraceResult r2 = traceRay(r);
		float num = 0;
		if (r2.hasHit())
		{
			Vec3f p = r(r2.m_fDist);
			num = g_dMap(p).numSamples;
			//uint3 i = g_dMap.hashMap.Transform(p);
			//float3 mi = g_dMap.hashMap.InverseTransform(i), ma = g_dMap.hashMap.InverseTransform(i + make_uint3(1));
			//for(SpatialLinkedMap<SpatialEntry>::iterator it = g_sMap.begin(mi, ma); it != g_sMap.end(mi, ma); ++it)
			//	num++;
		}
		I.AddSample(x, y, Spectrum(num / scale));
	}
}

__global__ void visualizePdf(Image I, int w, int h, int xoff, int yoff, DirectionModel model)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < w && y < h)
	{
		float a = float(x) / float(w), b = float(y) / float(h);
		float pdf = model.gmm.p(VEC<float, 2>() % a % b);
		Spectrum c(pdf);
		I.ClearSample(x + xoff, y + yoff);
		I.AddSample(x + xoff, y + yoff, c);
	}
}

static unsigned int* modelToShow = 0;

void PmmTracer::DoRender(Image* I)
{
	I->Clear();

	sMap.ResetBuffer();
	hipMemcpyToSymbol(HIP_SYMBOL(g_sMap), &sMap, sizeof(sMap));
	while (!sMap.isFull())
	{
		tracePhotons << < 20, 256 >> >();
		hipMemcpyFromSymbol(&sMap, HIP_SYMBOL(g_sMap), sizeof(sMap));
	}
	hipMemcpyToSymbol(HIP_SYMBOL(g_dMap), &dMap, sizeof(dMap));
	int l = 6, L = dMap.gridSize / l + 1;
	updateCache<16> << <dim3(L, L, L), dim3(l, l, l) >> >(ny(passIteration++));

	unsigned int p = 16, w, h;
	I->getExtent(w, h);
	visualize << <dim3(w / p + 1, h / p + 1, 1), dim3(p, p, 1) >> >(*I, w, h, 20 * passIteration);
	int rectWidth = 64;
	if (modelToShow)
	{
		DirectionModel model;
		hipMemcpy(&model, dMap.deviceData + *modelToShow, sizeof(model), hipMemcpyDeviceToHost);
		visualizePdf << <dim3(8, 8, 1), dim3(8, 8, 1) >> >(*I, rectWidth, rectWidth, w - rectWidth, h - rectWidth, model);
	}

	hipError_t r = hipDeviceSynchronize();
}

void PmmTracer::StartNewTrace(Image* I)
{
	passIteration = 1;
	AABB box = this->GetEyeHitPointBox(m_pScene, true);
	//AABB box = m_pScene->getBox(m_pScene->getNodes());
	sMap.SetSceneDimensions(box);
	dMap.ResetBuffer();
	dMap.SetSceneDimensions(box);
	CudaRNG rng = g_RNGData();
	DirectionModel* models = new DirectionModel[dMap.NumEntries()];
	for (unsigned int i = 0; i < dMap.NumEntries(); i++)
		models[i].Initialze(rng);
	hipMemcpy(dMap.deviceData, models, dMap.NumEntries() * sizeof(DirectionModel), hipMemcpyHostToDevice);
	delete[] models;
	g_RNGData(rng);
}

void PmmTracer::Debug(Image* I, const Vec2i& p)
{
	/*k_INITIALIZE(m_pScene, g_sRngs);
	float3* deviceDirs;
	unsigned int* deviceNum;
	CUDA_MALLOC(&deviceDirs, sizeof(float3) * 10000);
	CUDA_MALLOC(&deviceNum, sizeof(unsigned int));
	copyDirections<<<1,1>>>(p.x, p.y, deviceDirs, deviceNum);
	unsigned int N;
	hipMemcpy(&N, deviceNum, sizeof(unsigned int), hipMemcpyDeviceToHost);
	float3* directions = (float3*)alloca(sizeof(float3) * N);
	hipMemcpy(directions, deviceDirs, sizeof(float3) * N, hipMemcpyDeviceToHost);
	CUDA_FREE(deviceDirs);
	CUDA_FREE(deviceNum);
	plotPoints(directions, N);*/

	k_INITIALIZE(m_pScene, g_sRngs);
	Ray r = g_SceneData.GenerateSensorRay(p.x, p.y);
	TraceResult r2 = traceRay(r);
	Vec3f pa = r(r2.m_fDist);
	unsigned int idx = dMap.hashMap.Hash(pa);
	modelToShow = new unsigned int(idx);
	DirectionModel model;
	hipMemcpy(&model, dMap.deviceData + idx, sizeof(model), hipMemcpyDeviceToHost);
	plotModel(model);
}

}