#include "hip/hip_runtime.h"
#include "PPPMTracer.h"
#include <Kernel/TraceHelper.h>
#include <Kernel/TraceAlgorithms.h>
#include <Math/half.h>

namespace CudaTracerLib {

CUDA_FUNC_IN float SH(float f)
{
	return f >= 0 ? 1 : 0;
}

CUDA_ONLY_FUNC void BeamBeamGrid::StoreBeam(const Beam& b, bool firstStore)
{
	unsigned int beam_idx = atomicInc(&m_uBeamIdx, (unsigned int)-1);
	if (beam_idx < m_uBeamLength)
	{
		m_pDeviceBeams[beam_idx] = b;
		bool storedAll = true;
#ifdef ISCUDA
		/*TraverseGrid(Ray(b.pos, b.dir), m_sStorage.hashMap, 0.0f, b.t, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
		{
		if (!m_sStorage.store(cell_pos, beam_idx))
		{
		storedAll = false;
		cancelTraversal = true;
		}
		});*/
#endif

		/*AABB box = m_sStorage.hashMap.getAABB();
		Ray r(b.pos, b.dir);
		float m_fGridSize = m_sStorage.hashMap.m_fGridSize;
		float tEnd = b.t;

		Vec3f m_vCellSize = box.Size() / (m_fGridSize - 1);
		Vec3i Step(sign<int>(r.direction.x), sign<int>(r.direction.y), sign<int>(r.direction.z));
		Vec3f inv_d = r.direction;
		const float ooeps = math::exp2(-40.0f);//80 is too small, will create underflow on GPU
		inv_d.x = 1.0f / (math::abs(inv_d.x) > ooeps ? inv_d.x : copysignf(ooeps, inv_d.x));
		inv_d.y = 1.0f / (math::abs(inv_d.y) > ooeps ? inv_d.y : copysignf(ooeps, inv_d.y));
		inv_d.z = 1.0f / (math::abs(inv_d.z) > ooeps ? inv_d.z : copysignf(ooeps, inv_d.z));
		Vec3f DeltaT = abs(m_vCellSize * inv_d);

		Vec3f NextCrossingT[5];
		Vec3u Pos[5];
		float rayT[5];
		float maxT[5];
		//coordinate system which has left axis pointing towards (-1, 0, 0) and up to (0, 1, 0)
		Frame T(Vec3f(-math::abs(r.direction.z), 0, math::sign(r.direction.z) * r.direction.x),
		Vec3f(-r.direction.x * r.direction.y, math::sqr(r.direction.x) + math::sqr(r.direction.z), -r.direction.y * r.direction.z),
		r.direction);

		int nRaysTerminated = 0;
		float r_ = m_fCurrentRadiusVol;
		for (int i = 0; i < 5; i++)
		{
		Vec3f pos = i == 0 ? r.origin : (r.origin + T.toWorld(Vec3f(-r_ + ((i - 1) / 2) * 2 * r_, -r_ + ((i - 1) % 2) * 2 * r_, 0)));
		if (!box.Intersect(Ray(pos, r.direction), rayT + i, maxT + i))
		{
		rayT[i] = -1;
		nRaysTerminated++;
		continue;
		}
		rayT[i] = math::clamp(rayT[i], 0.0f, tEnd);
		maxT[i] = math::clamp(maxT[i], 0.0f, tEnd);
		Vec3f q = (r.direction * rayT[i] + pos - box.minV) / box.Size() * (m_fGridSize - 1);
		Pos[i] = clamp(Vec3u(unsigned int(q.x), unsigned int(q.y), unsigned int(q.z)), Vec3u(0), Vec3u(m_fGridSize - 1));
		auto A = box.minV + (Vec3f(Pos[i].x, Pos[i].y, Pos[i].z) + Vec3f(SH(r.direction.x), SH(r.direction.y), SH(r.direction.z))) * m_vCellSize,
		B = pos - r.direction * rayT[i];
		NextCrossingT[i] = max(Vec3f(0.0f), Vec3f(rayT[i]) + (A - B) * inv_d);
		}
		int N = 0;
		Vec3u lastMin(UINT_MAX), lastMax(UINT_MAX);
		while (nRaysTerminated != 5)
		{
		N++;
		Vec3u minG(UINT_MAX), maxG(0);
		for (int i = 0; i < 5; i++)
		if (rayT[i] >= 0)
		{
		minG = min(minG, Pos[i]);
		maxG = max(maxG, Pos[i]);
		}
		for (unsigned int a = minG.x; a <= maxG.x; a++)
		for (unsigned int b = minG.y; b <= maxG.y; b++)
		for (unsigned int c = minG.z; c <= maxG.z; c++)
		{
		if (lastMin.x <= a && a <= lastMax.x && lastMin.y <= b && b <= lastMax.y && lastMin.z <= c && c <= lastMax.z)
		continue;
		m_sStorage.store(Vec3u(a, b, c), beam_idx);
		}
		lastMin = minG; lastMax = maxG;

		for (int i = 0; i < 5; i++)
		{
		if (rayT[i] < 0)
		continue;
		int bits = ((NextCrossingT[i][0] < NextCrossingT[i][1]) << 2) + ((NextCrossingT[i][0] < NextCrossingT[i][2]) << 1) + ((NextCrossingT[i][1] < NextCrossingT[i][2]));
		int stepAxis = (0x00000a66 >> (2 * bits)) & 3;
		Pos[i][stepAxis] += Step[stepAxis];
		if (Pos[i][stepAxis] >= m_fGridSize || NextCrossingT[i][stepAxis] > maxT[i])
		{
		nRaysTerminated++;
		rayT[i] = -1;
		continue;
		}
		rayT[i] = NextCrossingT[i][stepAxis];
		NextCrossingT[i][stepAxis] += DeltaT[stepAxis];
		}
		}*/

#ifdef ISCUDA
		const AABB objaabb = b.getAABB(m_fCurrentRadiusVol);
		const int maxAxis = b.dir.abs().arg_max();
		const int chopCount = (int)(objaabb.Size()[maxAxis] * m_sStorage.hashMap.m_vInvSize[maxAxis]) + 1;
		const float invChopCount = 1.0f / (float)chopCount;

		for (int chop = 0; chop < chopCount; ++chop)
		{
			AABB aabb = b.getSegmentAABB((chop)* invChopCount, (chop + 1) * invChopCount, m_fCurrentRadiusVol);

			m_sStorage.ForAllCells(aabb.minV, aabb.maxV, [&](const Vec3u& pos)
			{
				/*bool found_duplicate = false;
				m_sStorage.ForAll(pos, [&](unsigned int loc_idx, unsigned int b_idx)
				{
					if (found_duplicate) return;
					if (beam_idx == b_idx)
						found_duplicate = true;
				});
				if (!found_duplicate)*/
					storedAll &= m_sStorage.store(pos, beam_idx);
			});
		}

		//auto aabb = b.getAABB(m_fCurrentRadiusVol);
		//m_sStorage.ForAllCells(aabb.minV, aabb.maxV, [&](const Vec3u& pos)
		//{
		//	storedAll &= m_sStorage.store(pos, beam_idx);
		//});

		if (firstStore && storedAll)
			atomicInc(&m_uNumEmitted, (unsigned int)-1);
#endif
	}
}

CUDA_CONST unsigned int g_PassIdx;
CUDA_DEVICE unsigned int g_NumPhotonEmitted;
CUDA_DEVICE SpatialLinkedMap<PPPMPhoton> g_SurfaceMap;
CUDA_DEVICE CUDA_ALIGN(16) unsigned char g_VolEstimator[Dmax4(sizeof(PointStorage), sizeof(BeamGrid), sizeof(BeamBeamGrid), sizeof(BeamBVHStorage))];

template<typename VolEstimator> __global__ void k_PhotonPass(int photons_per_thread, bool DIRECT)
{
	CudaRNG rng = g_RNGData();
	CUDA_SHARED unsigned int local_Counter;
	local_Counter = 0;
	unsigned int local_Todo = photons_per_thread * blockDim.x * blockDim.y;

	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	KernelAggregateVolume& V = g_SceneData.m_sVolume;
	CUDA_SHARED unsigned int numStoredSurface;
	numStoredSurface = 0;
	__syncthreads();

	while (atomicInc(&local_Counter, (unsigned int)-1) < local_Todo)// && !g_SurfaceMap.isFull() && !((VolEstimator*)g_VolEstimator)->isFullK()
	{
		Ray r;
		const KernelLight* light;
		Vec2f sps = rng.randomFloat2(), sds = rng.randomFloat2();
		Spectrum Le = g_SceneData.sampleEmitterRay(r, light, sps, sds),
			throughput(1.0f);
		int depth = -1;
		bool wasStoredSurface = false, wasStoredVolume = false;
		bool delta = false;
		MediumSamplingRecord mRec;
		bool medium = false;
		const VolumeRegion* bssrdf = 0;

		while (++depth < PPM_MaxRecursion && !Le.isZero())// && !g_SurfaceMap.isFull() && !((VolEstimator*)g_VolEstimator)->isFullK()
		{
			TraceResult r2 = Traceray(r);
			float minT, maxT;
			bool inMedium = (!bssrdf && V.HasVolumes() && V.IntersectP(r, 0, r2.m_fDist, &minT, &maxT)) || bssrdf;
			if (inMedium)
				((VolEstimator*)g_VolEstimator)->StoreBeam(Beam(r.origin, r.direction, r2.m_fDist, throughput * Le), !wasStoredVolume);//store the beam even if sampled distance is to far ahead!
			if ((!bssrdf && inMedium && V.sampleDistance(r, 0, r2.m_fDist, rng, mRec))
				|| (bssrdf && bssrdf->sampleDistance(r, 0, r2.m_fDist, rng.randomFloat(), mRec)))
			{//mRec.t
				throughput *= mRec.transmittance / mRec.pdfSuccess;
				((VolEstimator*)g_VolEstimator)->StorePhoton(mRec.p, -r.direction, throughput * Le, !wasStoredVolume);
				throughput *= mRec.sigmaS;
				wasStoredVolume = true;
				if (bssrdf)
				{
					PhaseFunctionSamplingRecord mRec(-r.direction);
					throughput *= bssrdf->As()->Func.Sample(mRec, rng);
					r.direction = mRec.wi;
				}
				else throughput *= V.Sample(mRec.p, -r.direction, rng, &r.direction);
				r.origin = mRec.p;
				delta = false;
				medium = true;
			}
			else if (!r2.hasHit())
				break;
			else
			{
				if (medium)
					throughput *= mRec.transmittance / mRec.pdfFailure;
				Vec3f wo = bssrdf ? r.direction : -r.direction;
				r2.getBsdfSample(-wo, r(r2.m_fDist), bRec, ETransportMode::EImportance, &rng);
				if ((DIRECT && depth > 0) || !DIRECT)
					if (r2.getMat().bsdf.hasComponent(ESmooth) && dot(bRec.dg.sys.n, wo) > 0.0f)
					{
						auto ph = PPPMPhoton(throughput * Le, wo, bRec.dg.n, delta ? PhotonType::pt_Caustic : PhotonType::pt_Diffuse);
						ph.Pos = dg.P;
						bool b = g_SurfaceMap.store(dg.P, ph);
						if (b && !wasStoredSurface)
							atomicInc(&numStoredSurface, (unsigned int)-1);
						wasStoredSurface = true;
					}
				Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
				delta = bRec.sampledType & ETypeCombinations::EDelta;
				if (!bssrdf && r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
					bRec.wo.z *= -1.0f;
				else
				{
					if (!bssrdf)
						throughput *= f;
					bssrdf = 0;
					medium = false;
				}

				r = Ray(bRec.dg.P, bRec.getOutgoing());
			}
		}
	}

	__syncthreads();
	if (threadIdx.x == 0 && threadIdx.y == 0)
		atomicAdd(&g_NumPhotonEmitted, numStoredSurface);

	g_RNGData(rng);
}

void PPPMTracer::doPhotonPass()
{
	m_sSurfaceMap.ResetBuffer();
	m_pVolumeEstimator->StartNewPass(this, m_pScene);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_SurfaceMap), &m_sSurfaceMap, sizeof(m_sSurfaceMap)));
	ZeroSymbol(g_NumPhotonEmitted);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator, m_pVolumeEstimator->getSize()));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_PassIdx), &m_uPassesDone, sizeof(m_uPassesDone)));

	while (!m_sSurfaceMap.isFull() && !m_pVolumeEstimator->isFull())
	{
		if (dynamic_cast<PointStorage*>(m_pVolumeEstimator))
			k_PhotonPass<PointStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting);
		else if (dynamic_cast<BeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<BeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting);
		else if (dynamic_cast<BeamBeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<BeamBeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting);
		else if (dynamic_cast<BeamBVHStorage*>(m_pVolumeEstimator))
			k_PhotonPass<BeamBVHStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting);
		ThrowCudaErrors(hipMemcpyFromSymbol(&m_sSurfaceMap, HIP_SYMBOL(g_SurfaceMap), sizeof(m_sSurfaceMap)));
		ThrowCudaErrors(hipMemcpyFromSymbol(m_pVolumeEstimator, HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator->getSize()));
	}
	ThrowCudaErrors(hipMemcpyFromSymbol(&m_uPhotonEmittedPass, HIP_SYMBOL(g_NumPhotonEmitted), sizeof(m_uPhotonEmittedPass)));
	m_pVolumeEstimator->PrepareForRendering();
	m_uPhotonEmittedPass = max(m_uPhotonEmittedPass, m_pVolumeEstimator->getNumEmitted());
	if (m_uTotalPhotonsEmitted == 0)
		doPerPixelRadiusEstimation();
}

}