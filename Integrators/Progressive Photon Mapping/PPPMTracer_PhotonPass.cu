#include "hip/hip_runtime.h"
#include "PPPMTracer.h"
#include <Kernel/TraceHelper.h>
#include <Kernel/TraceAlgorithms.h>
#include <Math/half.h>
#include <Base/Timer.h>

namespace CudaTracerLib {

CUDA_ONLY_FUNC void BeamBeamGrid::StoreBeam(const Beam& b, bool firstStore)
{
	unsigned int beam_idx = atomicInc(&m_uBeamIdx, (unsigned int)-1);
	if (beam_idx < m_uBeamLength)
	{
		m_pDeviceBeams[beam_idx] = b;
#ifdef ISCUDA
		bool storedAll = true;
		const AABB objaabb = b.getAABB(m_fCurrentRadiusVol);
		const int maxAxis = b.getDir().abs().arg_max();
		const int chopCount = (int)(objaabb.Size()[maxAxis] * m_sStorage.getHashGrid().m_vInvSize[maxAxis]) + 1;
		const float invChopCount = 1.0f / (float)chopCount;

		for (int chop = 0; chop < chopCount; ++chop)
		{
			AABB aabb = b.getSegmentAABB((chop)* invChopCount, (chop + 1) * invChopCount, m_fCurrentRadiusVol);

			m_sStorage.ForAllCells(aabb.minV, aabb.maxV, [&](const Vec3u& pos)
			{
				/*bool found_duplicate = false;
				m_sStorage.ForAll(pos, [&](unsigned int loc_idx, unsigned int b_idx)
				{
				if (found_duplicate) return;
				if (beam_idx == b_idx)
				found_duplicate = true;
				});
				if (!found_duplicate)*/
				storedAll &= m_sStorage.store(pos, beam_idx);
			});
		}

		//auto aabb = b.getAABB(m_fCurrentRadiusVol);
		//m_sStorage.ForAllCells(aabb.minV, aabb.maxV, [&](const Vec3u& pos)
		//{
		//	storedAll &= m_sStorage.store(pos, beam_idx);
		//});

		if (firstStore && storedAll)
			atomicInc(&m_uNumEmitted, (unsigned int)-1);
#endif
	}
}

CUDA_CONST unsigned int g_PassIdx;
CUDA_DEVICE unsigned int g_NumPhotonEmitted;
CUDA_DEVICE SurfaceMapT g_SurfaceMap;
CUDA_DEVICE SurfaceMapT g_SurfaceMapCaustic;
CUDA_DEVICE CUDA_ALIGN(16) unsigned char g_VolEstimator[Dmax4(sizeof(PointStorage), sizeof(BeamGrid), sizeof(BeamBeamGrid), sizeof(BeamBVHStorage))];

template<typename VolEstimator> __global__ void k_PhotonPass(int photons_per_thread, bool DIRECT, bool finalGathering)
{
	CudaRNG rng = g_RNGData();
	CUDA_SHARED unsigned int local_Counter;
	local_Counter = 0;
	unsigned int local_Todo = photons_per_thread * blockDim.x * blockDim.y;

	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	KernelAggregateVolume& V = g_SceneData.m_sVolume;
	CUDA_SHARED unsigned int numStoredSurface;
	numStoredSurface = 0;
	__syncthreads();

	while (atomicInc(&local_Counter, (unsigned int)-1) < local_Todo)// && !g_SurfaceMap.isFull() && !((VolEstimator*)g_VolEstimator)->isFullK()
	{
		Ray r;
		const KernelLight* light;
		Vec2f sps = rng.randomFloat2(), sds = rng.randomFloat2();
		Spectrum Le = g_SceneData.sampleEmitterRay(r, light, sps, sds),
			throughput(1.0f);
		int depth = -1;
		bool wasStoredSurface = false, wasStoredVolume = false;
		bool delta = false;
		MediumSamplingRecord mRec;
		bool medium = false;
		const VolumeRegion* bssrdf = 0;

		while (++depth < PPM_MaxRecursion && !Le.isZero())// && !g_SurfaceMap.isFull() && !((VolEstimator*)g_VolEstimator)->isFullK()
		{
			TraceResult r2 = traceRay(r);
			float minT, maxT;
			bool inMedium = (!bssrdf && V.HasVolumes() && V.IntersectP(r, 0, r2.m_fDist, &minT, &maxT)) || bssrdf;
			((VolEstimator*)g_VolEstimator)->StoreBeam(Beam(r.origin, r.direction, r2.m_fDist, throughput * Le), !wasStoredVolume);//store the beam even if sampled distance is to far ahead!
			//wasStoredVolume = true;
			if ((!bssrdf && inMedium && V.sampleDistance(r, 0, r2.m_fDist, rng, mRec))
				|| (bssrdf && bssrdf->sampleDistance(r, 0, r2.m_fDist, rng.randomFloat(), mRec)))
			{//mRec.t
				throughput *= mRec.transmittance / mRec.pdfSuccess;
				throughput *= mRec.sigmaS;
				((VolEstimator*)g_VolEstimator)->StorePhoton(mRec.p, -r.direction, throughput * Le, !wasStoredVolume);
				wasStoredVolume = true;
				if (bssrdf)
				{
					PhaseFunctionSamplingRecord pRec(-r.direction);
					throughput *= bssrdf->As()->Func.Sample(pRec, rng);
					r.direction = pRec.wi;
				}
				else throughput *= V.Sample(mRec.p, -r.direction, rng, &r.direction);
				r.origin = mRec.p;
				delta = false;
				medium = true;
			}
			else if (!r2.hasHit())
				break;
			else
			{
				if (medium)
					throughput *= mRec.transmittance / mRec.pdfFailure;
				Vec3f wo = bssrdf ? r.direction : -r.direction;
				Spectrum f_i = throughput * Le;
				r2.getBsdfSample(-wo, r(r2.m_fDist), bRec, ETransportMode::EImportance, &rng, &f_i);
				if (r2.getMat().bsdf.hasComponent(ESmooth) && dot(bRec.dg.sys.n, wo) > 0.0f)
				{
					auto ph = PPPMPhoton(throughput * Le, wo, bRec.dg.n, delta ? PhotonType::pt_Caustic : PhotonType::pt_Diffuse);
					Vec3u cell_idx = g_SurfaceMap.getHashGrid().Transform(dg.P);
					ph.setPos(g_SurfaceMap.getHashGrid(), cell_idx, dg.P);
					bool b = false;
					if ((DIRECT && depth > 0) || !DIRECT)
						b |= g_SurfaceMap.store(cell_idx, ph);
					if (finalGathering && delta)
						b |= g_SurfaceMapCaustic.store(cell_idx, ph);
					if (b && !wasStoredSurface)
					{
						atomicInc(&numStoredSurface, (unsigned int)-1);
						wasStoredSurface = true;
					}
				}
				Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
				delta = bRec.sampledType & ETypeCombinations::EDelta;
				if (!bssrdf && r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
					bRec.wo.z *= -1.0f;
				else
				{
					if (!bssrdf)
						throughput *= f;
					bssrdf = 0;
					medium = false;
				}

				r = Ray(bRec.dg.P, bRec.getOutgoing());
			}
		}
	}

	__syncthreads();
	if (threadIdx.x == 0 && threadIdx.y == 0)
		atomicAdd(&g_NumPhotonEmitted, numStoredSurface);

	g_RNGData(rng);
}

void PPPMTracer::doPhotonPass()
{
	bool finalGathering = m_sParameters.getValue(KEY_FinalGathering());

	m_sSurfaceMap.ResetBuffer();
	if (finalGathering)
		m_sSurfaceMapCaustic.ResetBuffer();
	m_pVolumeEstimator->StartNewPass(this, m_pScene);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_SurfaceMap), &m_sSurfaceMap, sizeof(m_sSurfaceMap)));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_SurfaceMapCaustic), &m_sSurfaceMapCaustic, sizeof(m_sSurfaceMapCaustic)));
	ZeroSymbol(g_NumPhotonEmitted);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator, m_pVolumeEstimator->getSize()));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_PassIdx), &m_uPassesDone, sizeof(m_uPassesDone)));

	while (!m_sSurfaceMap.isFull() && !m_pVolumeEstimator->isFull())
	{
		if (dynamic_cast<BeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<BeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting, finalGathering);
		else if(dynamic_cast<PointStorage*>(m_pVolumeEstimator))
			k_PhotonPass<PointStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting, finalGathering);
		else if (dynamic_cast<BeamBeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<BeamBeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting, finalGathering);
		else if (dynamic_cast<BeamBVHStorage*>(m_pVolumeEstimator))
			k_PhotonPass<BeamBVHStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_useDirectLighting, finalGathering);
		ThrowCudaErrors(hipMemcpyFromSymbol(&m_sSurfaceMap, HIP_SYMBOL(g_SurfaceMap), sizeof(m_sSurfaceMap)));
		ThrowCudaErrors(hipMemcpyFromSymbol(&m_sSurfaceMapCaustic, HIP_SYMBOL(g_SurfaceMapCaustic), sizeof(m_sSurfaceMapCaustic)));
		ThrowCudaErrors(hipMemcpyFromSymbol(m_pVolumeEstimator, HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator->getSize()));
	}
	ThrowCudaErrors(hipMemcpyFromSymbol(&m_uPhotonEmittedPass, HIP_SYMBOL(g_NumPhotonEmitted), sizeof(m_uPhotonEmittedPass)));
	m_pVolumeEstimator->PrepareForRendering();
	m_uPhotonEmittedPass = max(m_uPhotonEmittedPass, m_pVolumeEstimator->getNumEmitted());
	m_sSurfaceMap.PrepareForUse();
	if (finalGathering)
		m_sSurfaceMapCaustic.PrepareForUse();
	if (m_uTotalPhotonsEmitted == 0)
		doPerPixelRadiusEstimation();
}

}