#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include <Kernel/k_TraceHelper.h>
#include <Kernel/k_TraceAlgorithms.h>
#include <Math/half.h>

void k_BeamBeamGrid::StoreBeam(const k_Beam& b, bool firstStore)
{
	unsigned int beam_idx = atomicInc(&m_uBeamIdx, (unsigned int)-1);
	if (beam_idx < m_uBeamLength)
	{
		m_pDeviceBeams[beam_idx] = b;
		bool storedAll = true;
		//complex variant which pre allocates sufficient storage
		/*Vec3u start_cell = m_sStorage.hashMap.Transform(b.pos);
		unsigned int n_buf_idx = 0;
		for(int i = 0; i < 3; i++)
			n_buf_idx += b.dir[i] < 0 ? start_cell[i] + 1 : m_sStorage.hashMap.m_fGridSize - start_cell[i];
		n_buf_idx = n_buf_idx / 2;
		unsigned int buf_idx = m_sStorage.allocStorage(n_buf_idx), i = 0;
		if(buf_idx >= m_sStorage.numData - n_buf_idx)
		{
			printf("buf_idx = %d, n = %d\n", buf_idx, n_buf_idx);
			return;
		}
#ifdef ISCUDA
		TraverseGrid(Ray(b.pos, b.dir), m_sStorage.hashMap, 0.0f, b.t, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
		{
			m_sStorage.store(cell_pos, beam_idx, buf_idx + i);
		});
#endif*/
		
#ifdef ISCUDA
		TraverseGrid(Ray(b.pos, b.dir), m_sStorage.hashMap, 0.0f, b.t, [&](float minT, float rayT, float maxT, float cellEndT, Vec3u& cell_pos, bool& cancelTraversal)
		{
			if (!m_sStorage.store(cell_pos, beam_idx))
			{
				storedAll = false;
				cancelTraversal = true;
			}
		});
#endif
		if (firstStore&&storedAll)
			atomicInc(&m_uNumEmitted, (unsigned int)-1);
	}
}

CUDA_CONST unsigned int g_PassIdx;
CUDA_DEVICE unsigned int g_NumPhotonEmitted;
CUDA_DEVICE e_SpatialLinkedMap<k_pPpmPhoton> g_SurfaceMap;
CUDA_DEVICE CUDA_ALIGN(16) unsigned char g_VolEstimator[Dmax4(sizeof(k_PointStorage), sizeof(k_BeamGrid), sizeof(k_BeamBeamGrid), sizeof(k_BeamBVHStorage))];

template<typename VolEstimator> __global__ void k_PhotonPass(int photons_per_thread,  bool DIRECT)
{
	CudaRNG rng = g_RNGData();
	CUDA_SHARED unsigned int local_Counter;
	local_Counter = 0;
	unsigned int local_Todo = photons_per_thread * blockDim.x * blockDim.y;

	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	e_KernelAggregateVolume& V = g_SceneData.m_sVolume;
	CUDA_SHARED unsigned int numStoredSurface;
	numStoredSurface = 0;
	__syncthreads();

	while (atomicInc(&local_Counter, (unsigned int)-1) < local_Todo)// && !g_SurfaceMap.isFull() && !((VolEstimator*)g_VolEstimator)->isFullK()
	{
		Ray r;
		const e_KernelLight* light;
		Vec2f sps = rng.randomFloat2(), sds = rng.randomFloat2();
		Spectrum Le = g_SceneData.sampleEmitterRay(r, light, sps, sds),
			throughput(1.0f);
		int depth = -1;
		bool wasStoredSurface = false, wasStoredVolume = false;
		bool delta = false;
		MediumSamplingRecord mRec;
		bool medium = false;
		const e_VolumeRegion* bssrdf = 0;

		while (++depth < PPM_MaxRecursion && !Le.isZero())// && !g_SurfaceMap.isFull() && !((VolEstimator*)g_VolEstimator)->isFullK()
		{
			TraceResult r2 = k_TraceRay(r);
			float minT, maxT;
			if ((!bssrdf && V.HasVolumes() && V.IntersectP(r, 0, r2.m_fDist, &minT, &maxT) && V.sampleDistance(r, 0, r2.m_fDist, rng, mRec))
				|| (bssrdf && bssrdf->sampleDistance(r, 0, r2.m_fDist, rng.randomFloat(), mRec)))
			{
				((VolEstimator*)g_VolEstimator)->StoreBeam(k_Beam(r.origin, r.direction, mRec.t, throughput * Le), !wasStoredVolume);
				throughput *= mRec.sigmaS * mRec.transmittance / mRec.pdfSuccess;
				((VolEstimator*)g_VolEstimator)->StorePhoton(mRec.p, -r.direction, throughput * Le, !wasStoredVolume);
				wasStoredVolume = true;
				if (bssrdf)
				{
					PhaseFunctionSamplingRecord mRec(-r.direction);
					throughput *= bssrdf->As()->Func.Sample(mRec, rng);
					r.direction = mRec.wi;
				}
				else throughput *= V.Sample(mRec.p, -r.direction, rng, &r.direction);
				r.origin = mRec.p;
				delta = false;
				medium = true;
			}
			else if (!r2.hasHit())
				break;
			else
			{
				if (medium)
					throughput *= mRec.transmittance / mRec.pdfFailure;
				Vec3f wo = bssrdf ? r.direction : -r.direction;
				r2.getBsdfSample(-wo, r(r2.m_fDist), bRec, ETransportMode::EImportance, &rng);
				if ((DIRECT && depth > 0) || !DIRECT)
					if (r2.getMat().bsdf.hasComponent(ESmooth) && dot(bRec.dg.sys.n, wo) > 0.0f)
					{
						auto ph = k_pPpmPhoton(throughput * Le, wo, bRec.dg.sys.n, delta ? PhotonType::pt_Caustic : PhotonType::pt_Diffuse);
						ph.Pos = dg.P;
						bool b = g_SurfaceMap.store(dg.P, ph);
						if (b && !wasStoredSurface)
							atomicInc(&numStoredSurface, (unsigned int)-1);
						wasStoredSurface = true;
					}
				Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
				delta = bRec.sampledType & ETypeCombinations::EDelta;
				if (!bssrdf && r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
					bRec.wo.z *= -1.0f;
				else
				{
					if (!bssrdf)
						throughput *= f;
					bssrdf = 0;
					medium = false;
				}

				r = Ray(bRec.dg.P, bRec.getOutgoing());
			}
		}
	}

	__syncthreads();
	if (threadIdx.x == 0 && threadIdx.y == 0)
		atomicAdd(&g_NumPhotonEmitted, numStoredSurface);

	g_RNGData(rng);
}

void k_sPpmTracer::doPhotonPass()
{
	m_sSurfaceMap.ResetBuffer();
	m_pVolumeEstimator->StartNewPass(this, m_pScene);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_SurfaceMap), &m_sSurfaceMap, sizeof(m_sSurfaceMap)));
	ZeroSymbol(g_NumPhotonEmitted);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator, m_pVolumeEstimator->getSize()));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_PassIdx), &m_uPassesDone, sizeof(m_uPassesDone)));

	while (!m_sSurfaceMap.isFull() && !m_pVolumeEstimator->isFull())
	{
		if (dynamic_cast<k_PointStorage*>(m_pVolumeEstimator))
			k_PhotonPass<k_PointStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		else if (dynamic_cast<k_BeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<k_BeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		else if (dynamic_cast<k_BeamBeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<k_BeamBeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		else if (dynamic_cast<k_BeamBVHStorage*>(m_pVolumeEstimator))
			k_PhotonPass<k_BeamBVHStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		ThrowCudaErrors(hipMemcpyFromSymbol(&m_sSurfaceMap, HIP_SYMBOL(g_SurfaceMap), sizeof(m_sSurfaceMap)));
		ThrowCudaErrors(hipMemcpyFromSymbol(m_pVolumeEstimator, HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator->getSize()));
	}
	ThrowCudaErrors(hipMemcpyFromSymbol(&m_uPhotonEmittedPass, HIP_SYMBOL(g_NumPhotonEmitted), sizeof(m_uPhotonEmittedPass)));
	m_pVolumeEstimator->PrepareForRendering();
	m_uPhotonEmittedPass = max(m_uPhotonEmittedPass, m_pVolumeEstimator->getNumEmitted());
	if (m_uTotalPhotonsEmitted == 0)
		doPerPixelRadiusEstimation();
}