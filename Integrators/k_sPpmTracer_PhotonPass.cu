#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include "..\Kernel\k_TraceAlgorithms.h"
#include <Math/half.h>

CUDA_DEVICE unsigned int g_NumPhotonEmitted;
CUDA_DEVICE e_SpatialLinkedMap<k_pPpmPhoton> g_SurfaceMap;
CUDA_DEVICE CUDA_ALIGN(16) unsigned char g_VolEstimator[Dmax4(sizeof(k_PointStorage), sizeof(k_BeamGrid), sizeof(k_BeamBeamGrid), sizeof(k_BeamBVHStorage))];

template<typename VolEstimator> __global__ void k_PhotonPass(int photons_per_thread,  bool DIRECT)
{
	CudaRNG rng = g_RNGData();
	CUDA_SHARED unsigned int local_Counter;
	local_Counter = 0;
	unsigned int local_Todo = photons_per_thread * blockDim.x * blockDim.y;

	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	e_KernelAggregateVolume& V = g_SceneData.m_sVolume;
	CUDA_SHARED unsigned int numStoredSurface;
	numStoredSurface = 0;
	__syncthreads();

	while (atomicInc(&local_Counter, (unsigned int)-1) < local_Todo && !g_SurfaceMap.isFull())
	{
		Ray r;
		const e_KernelLight* light;
		Vec2f sps = rng.randomFloat2(), sds = rng.randomFloat2();
		Spectrum Le = g_SceneData.sampleEmitterRay(r, light, sps, sds),
			throughput(1.0f);
		int depth = -1;
		bool wasStoredSurface = false, wasStoredVolume = false;
		bool delta = false;
		MediumSamplingRecord mRec;
		bool medium = false;
		const e_KernelBSSRDF* bssrdf = 0;

		while (++depth < PPM_MaxRecursion && !g_SurfaceMap.isFull() && !Le.isZero())
		{
			TraceResult r2 = k_TraceRay(r);
			float minT, maxT;
			if ((!bssrdf && V.HasVolumes() && V.IntersectP(r, 0, r2.m_fDist, &minT, &maxT) && V.sampleDistance(r, 0, r2.m_fDist, rng, mRec))
				|| (bssrdf && sampleDistanceHomogenous(r, 0, r2.m_fDist, rng.randomFloat(), mRec, bssrdf->sig_a, bssrdf->sigp_s)))
			{
				((VolEstimator*)g_VolEstimator)->StoreBeam(k_Beam(r.origin, r.direction, mRec.t, throughput * Le), !wasStoredVolume);
				throughput *= mRec.sigmaS * mRec.transmittance / mRec.pdfSuccess;
				((VolEstimator*)g_VolEstimator)->StorePhoton(mRec.p, -r.direction, throughput * Le, !wasStoredVolume);
				wasStoredVolume = true;
				if (bssrdf)
					r.direction = Warp::squareToUniformSphere(rng.randomFloat2());
				else throughput *= V.Sample(mRec.p, -r.direction, rng, &r.direction);
				r.origin = mRec.p;
				delta = false;
				medium = true;
			}
			else if (!r2.hasHit())
				break;
			else
			{
				if (medium)
					throughput *= mRec.transmittance / mRec.pdfFailure;
				Vec3f wo = bssrdf ? r.direction : -r.direction;
				r2.getBsdfSample(-wo, r(r2.m_fDist), bRec, ETransportMode::EImportance, &rng);
				if ((DIRECT && depth > 0) || !DIRECT)
					if (r2.getMat().bsdf.hasComponent(ESmooth) && dot(bRec.dg.sys.n, wo) > 0.0f)
					{
						auto ph = k_pPpmPhoton(throughput * Le, wo, bRec.dg.sys.n, delta ? PhotonType::pt_Caustic : PhotonType::pt_Diffuse);
						ph.Pos = dg.P;
						bool b = g_SurfaceMap.store(dg.P, ph);
						if (b && !wasStoredSurface)
							atomicInc(&numStoredSurface, (unsigned int)-1);
						wasStoredSurface = true;
					}
				Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
				delta = bRec.sampledType & ETypeCombinations::EDelta;
				if (!bssrdf && r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
					bRec.wo.z *= -1.0f;
				else
				{
					if (!bssrdf)
						throughput *= f;
					bssrdf = 0;
					medium = false;
				}

				r = Ray(bRec.dg.P, bRec.getOutgoing());
			}
		}
	}

	__syncthreads();
	if (threadIdx.x == 0 && threadIdx.y == 0)
		atomicAdd(&g_NumPhotonEmitted, numStoredSurface);

	g_RNGData(rng);
}

void k_sPpmTracer::doPhotonPass()
{
	m_sSurfaceMap.ResetBuffer();
	m_pVolumeEstimator->StartNewPass(this, m_pScene);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_SurfaceMap), &m_sSurfaceMap, sizeof(m_sSurfaceMap)));
	ZeroSymbol(g_NumPhotonEmitted);
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator, m_pVolumeEstimator->getSize()));

	while (!m_sSurfaceMap.isFull() && !m_pVolumeEstimator->isFull())
	{
		if (dynamic_cast<k_PointStorage*>(m_pVolumeEstimator))
			k_PhotonPass<k_PointStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		else if (dynamic_cast<k_BeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<k_BeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		else if (dynamic_cast<k_BeamBeamGrid*>(m_pVolumeEstimator))
			k_PhotonPass<k_BeamBeamGrid> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		else if (dynamic_cast<k_BeamBVHStorage*>(m_pVolumeEstimator))
			k_PhotonPass<k_BeamBVHStorage> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >(PPM_Photons_Per_Thread, m_bDirect);
		ThrowCudaErrors(hipMemcpyFromSymbol(&m_sSurfaceMap, HIP_SYMBOL(g_SurfaceMap), sizeof(m_sSurfaceMap)));
	}
	ThrowCudaErrors(hipMemcpyFromSymbol(&m_uPhotonEmittedPass, HIP_SYMBOL(g_NumPhotonEmitted), sizeof(m_uPhotonEmittedPass)));
	ThrowCudaErrors(hipMemcpyFromSymbol(m_pVolumeEstimator, HIP_SYMBOL(g_VolEstimator), m_pVolumeEstimator->getSize()));
	m_pVolumeEstimator->PrepareForRendering();
}