#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include "..\Kernel\k_TraceAlgorithms.h"

CUDA_DEVICE k_PhotonMapCollection<true> g_Map;

template<bool DIRECT> __global__ void k_PhotonPass()
{ 
	CudaRNG rng = g_RNGData();
	CUDA_SHARED unsigned int local_Counter;
	local_Counter = 0;
	unsigned int local_Todo = PPM_Photons_Per_Thread * blockDim.x * blockDim.y;

	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	e_KernelAggregateVolume& V = g_SceneData.m_sVolume;

	while (local_Counter < local_Todo && g_Map.m_uPhotonNumStored < g_Map.m_uPhotonBufferLength)
	{
		Ray r;
		const e_KernelLight* light;
		float2 sps = rng.randomFloat2(), sds = rng.randomFloat2();
		Spectrum Le = g_SceneData.sampleEmitterRay(r, light, sps, sds),
				 throughput(1.0f);
		int depth = -1;
		atomicInc(&local_Counter, (unsigned int)-1);
		bool wasStored = false;
		bool delta = false;
		MediumSamplingRecord mRec;
		bool medium = false;
		const e_KernelBSSRDF* bssrdf = 0;

		while (++depth < PPM_MaxRecursion && g_Map.m_uPhotonNumStored < g_Map.m_uPhotonBufferLength && !Le.isZero())
		{
			TraceResult r2 = k_TraceRay(r);
			float minT, maxT;
			if ((!bssrdf && V.HasVolumes() && V.IntersectP(r, 0, r2.m_fDist, -1, &minT, &maxT) && V.sampleDistance(r, 0, r2.m_fDist, -1, rng, mRec))
				|| (bssrdf && sampleDistanceHomogenous(r, 0, r2.m_fDist, rng.randomFloat(), mRec, bssrdf->sig_a, bssrdf->sigp_s)))
			{
				throughput *= mRec.sigmaS * mRec.transmittance / mRec.pdfSuccess;
				wasStored |= storePhoton(mRec.p, throughput * Le, -r.direction, make_float3(0, 0, 0), PhotonType::pt_Volume, g_Map);
				if (bssrdf)
					r.direction = Warp::squareToUniformSphere(rng.randomFloat2());
				else throughput *= V.Sample(mRec.p, -r.direction, r2.getNodeIndex(), rng, &r.direction);
				r.origin = mRec.p;
				delta = false;
				medium = true;
			}
			else if (!r2.hasHit())
				break;
			else
			{
				if (medium)
					throughput *= mRec.transmittance / mRec.pdfFailure;
				float3 wo = bssrdf ? r.direction : -r.direction;
				r2.getBsdfSample(-wo, r(r2.m_fDist), &bRec, &rng);
				bRec.mode = EImportance;
				if ((DIRECT && depth > 0) || !DIRECT)
					if (r2.getMat().bsdf.hasComponent(ESmooth) && dot(bRec.dg.sys.n, wo) > 0.0f)
						wasStored |= storePhoton(dg.P, throughput * Le, wo, bRec.dg.sys.n, delta ? PhotonType::pt_Caustic : PhotonType::pt_Diffuse, g_Map);
				Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
				delta = bRec.sampledType & ETypeCombinations::EDelta;
				if (!bssrdf && r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
					bRec.wo.z *= -1.0f;
				else
				{
					if (!bssrdf)
						throughput *= f;
					bssrdf = 0;
					medium = false;
				}

				r = Ray(bRec.dg.P, bRec.getOutgoing());
			}
		}
		if (wasStored)
			atomicInc(&g_Map.m_uPhotonNumEmitted, 0xffffffff);
	}

	g_RNGData(rng);
}

__global__ void buildHashGrid()
{
	unsigned int idx = threadId;
	if (idx < g_Map.m_uPhotonBufferLength)
	{
		k_pPpmPhoton& e = g_Map.m_pPhotons[idx];
		const k_PhotonMap<k_HashGrid_Reg>& map = (&g_Map.m_sSurfaceMap)[e.getType()];
		unsigned int i = map.m_sHash.Hash(e.getPos());
		unsigned int k = atomicExch(map.m_pDeviceHashGrid + i, idx);
		e.setNext(k);
	}
}

/*__global__ void buildHashGridLinkedList(float a_Radius)
{
	const float r2 = a_Radius * a_Radius;
	unsigned int idx = threadId;
	if (idx < g_Map.m_uPhotonBufferLength)
	{
		k_pPpmPhoton& e = g_Map.m_pPhotons[idx];
		k_PhotonMap<k_HashGrid_Reg>& map = (&g_Map.m_sSurfaceMap)[e.getType()];
		if (e.getType() == PhotonType::pt_Caustic || e.getType() == PhotonType::pt_Diffuse)
		{
			Frame f = Frame(e.getNormal());
			f.t *= a_Radius;
			f.s *= a_Radius;
			f.n *= a_Radius;
			float3 a = -1.0f * f.t - f.s, b = f.t - f.s, c = -1.0f * f.t + f.s, d = f.t + f.s;
			float3 low = fminf(fminf(a, b), fminf(c, d)) + e.getPos(), high = fmaxf(fmaxf(a, b), fmaxf(c, d)) + e.getPos();
			uint3 lo = map.m_sHash.Transform(low), hi = map.m_sHash.Transform(high);
			for (unsigned int a = lo.x; a <= hi.x; a++)
			for (unsigned int b = lo.y; b <= hi.y; b++)
			for (unsigned int c = lo.z; c <= hi.z; c++)
			{
				unsigned int hash_idx = map.m_sHash.Hash(make_uint3(a, b, c));
				unsigned int list_idx = atomicInc(&map.m_uLinkedListUsed, 0xffffffff);
				if (list_idx < map.m_uLinkedListLength)
				{
					unsigned int prev_list_idx = atomicExch(map.m_pDeviceHashGrid + hash_idx, list_idx);
					map.m_pDeviceLinkedList[list_idx] = make_uint2(idx, prev_list_idx);
				}
				else printf("list_idx = %d, length = %d", list_idx, map.m_uLinkedListLength);
			}
		}
	}
}*/

void k_sPpmTracer::doPhotonPass()
{
	hipMemcpyToSymbol(HIP_SYMBOL(g_Map), &m_sMaps, sizeof(k_PhotonMapCollection<true>));
	k_INITIALIZE(m_pScene, g_sRngs);
	while (!m_sMaps.PassFinished())
	{
		if (m_bDirect)
			k_PhotonPass<true> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >();
		else k_PhotonPass<false> << < m_uBlocksPerLaunch, dim3(PPM_BlockX, PPM_BlockY, 1) >> >();
		hipMemcpyFromSymbol(&m_sMaps, HIP_SYMBOL(g_Map), sizeof(k_PhotonMapCollection<true>));
	}
	buildHashGrid<< <m_sMaps.m_uPhotonBufferLength / (32 * 6) + 1, dim3(32, 6, 1) >> >();
	hipMemcpyFromSymbol(&m_sMaps, HIP_SYMBOL(g_Map), sizeof(k_PhotonMapCollection<true>));
}