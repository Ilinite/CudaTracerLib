#include "hip/hip_runtime.h"
#include "k_VCM.h"
#include "k_VCMHelper.h"

CUDA_DEVICE k_PhotonMapCollection<false> g_CurrentMap, g_NextMap;

CUDA_DEVICE Spectrum L_Surface(BPTSubPathState& aCameraState, BSDFSamplingRecord& bRec, float a_rSurfaceUNUSED, const e_KernelMaterial* mat, float mMisVcWeightFactor)
{
	Spectrum Lp = Spectrum(0.0f);
	const float r2 = a_rSurfaceUNUSED * a_rSurfaceUNUSED;
	Frame sys = bRec.dg.sys;
	sys.t *= a_rSurfaceUNUSED;
	sys.s *= a_rSurfaceUNUSED;
	sys.n *= a_rSurfaceUNUSED;
	Vec3f a = -1.0f * sys.t - sys.s, b = sys.t - sys.s, c = -1.0f * sys.t + sys.s, d = sys.t + sys.s;
	Vec3f low = min(min(a, b), min(c, d)) + bRec.dg.P, high = max(max(a, b), max(c, d)) + bRec.dg.P;
	uint3 lo = g_CurrentMap.m_sSurfaceMap.m_sHash.Transform(low), hi = g_CurrentMap.m_sSurfaceMap.m_sHash.Transform(high);
	for (unsigned int a = lo.x; a <= hi.x; a++)
	for (unsigned int b = lo.y; b <= hi.y; b++)
	for (unsigned int c = lo.z; c <= hi.z; c++)
	{
		unsigned int i0 = g_CurrentMap.m_sSurfaceMap.m_sHash.Hash(make_uint3(a, b, c)), i = g_CurrentMap.m_sSurfaceMap.m_pDeviceHashGrid[i0];
		while (i != 0xffffffff && i != 0xffffff)
		{
			k_pPpmPhoton e = g_CurrentMap.m_pPhotons[i];
			Vec3f n = e.getNormal(), wi = e.getWi(), P = e.getPos();
			Spectrum l = e.getL();
			float dist2 = distanceSquared(P, bRec.dg.P);
			if (dist2 < r2 && dot(n, bRec.dg.sys.n) > 0.8f)
			{
				bRec.wo = bRec.dg.toLocal(wi);
				const float cameraBsdfDirPdfW = pdf(*mat, bRec);
				Spectrum bsdfFactor = mat->bsdf.f(bRec);
				const float cameraBsdfRevPdfW = revPdf(*mat, bRec);
				const float wLight = e.dVCM * mMisVcWeightFactor + e.dVM * cameraBsdfDirPdfW;
				const float wCamera = aCameraState.dVCM * mMisVcWeightFactor + aCameraState.dVM * cameraBsdfRevPdfW;
				const float misWeight = 1.f / (wLight + 1.f + wCamera);

				float ke = k_tr(a_rSurfaceUNUSED, math::sqrt(dist2));
				Lp += misWeight * PI * ke * l * bsdfFactor / Frame::cosTheta(bRec.wo);
			}
			i = e.getNext();
		}
	}
	return Lp / float(g_CurrentMap.m_uPhotonNumEmitted);
}

CUDA_FUNC_IN void VCM(const Vec2f& pixelPosition, k_BlockSampleImage& img, CudaRNG& rng, int w, int h, float a_Radius, int a_NumIteration)
{
	float mLightSubPathCount = 1;
	const float etaVCM = (PI * a_Radius * a_Radius) * w * h;
	float mMisVmWeightFactor = 1;
	float mMisVcWeightFactor = 1.0f / etaVCM;

	const int NUM_V_PER_PATH = 5;
	BPTVertex lightPath[NUM_V_PER_PATH];
	BPTSubPathState lightPathState;
	sampleEmitter(lightPathState, rng, mMisVcWeightFactor);
	int emitterPathLength = 1, emitterVerticesStored = 0;
	for (; emitterVerticesStored < NUM_V_PER_PATH; emitterPathLength++)
	{
		TraceResult r2 = k_TraceRay(lightPathState.r);
		if (!r2.hasHit())
			break;

		BPTVertex& v = lightPath[emitterVerticesStored];
		r2.getBsdfSample(lightPathState.r, rng, &v.bRec);

		if (emitterPathLength > 1 || true)
			lightPathState.dVCM *= r2.m_fDist * r2.m_fDist;
		lightPathState.dVCM /= fabsf(Frame::cosTheta(v.bRec.wi));
		lightPathState.dVC /= fabsf(Frame::cosTheta(v.bRec.wi));
		lightPathState.dVM /= fabsf(Frame::cosTheta(v.bRec.wi));

		//store in list
		if (r2.getMat().bsdf.hasComponent(ESmooth))
		{
			v.dVCM = lightPathState.dVCM;
			v.dVC = lightPathState.dVC;
			v.dVM = lightPathState.dVM;
			v.throughput = lightPathState.throughput;
			v.mat = &r2.getMat();
			v.subPathLength = emitterPathLength + 1;
			emitterVerticesStored++;

#ifdef ISCUDA
			k_pPpmPhoton* photon;
			if (emitterPathLength > 1 && storePhoton(v.bRec.dg.P, v.throughput, -lightPathState.r.direction, v.bRec.dg.sys.n, PhotonType::pt_Diffuse, g_NextMap, &photon))
			{
				photon->dVC = v.dVC;
				photon->dVCM = v.dVCM;
				photon->dVM = v.dVM;
			}
#endif
		}

		//connect to camera
		if (r2.getMat().bsdf.hasComponent(ESmooth))
			connectToCamera(lightPathState, v.bRec, r2.getMat(), img.img, rng, mLightSubPathCount, mMisVmWeightFactor, 1, true);

		if (!sampleScattering(lightPathState, v.bRec, r2.getMat(), rng, mMisVcWeightFactor, mMisVmWeightFactor))
			break;
	}

	BPTSubPathState cameraState;
	sampleCamera(cameraState, rng, pixelPosition, mLightSubPathCount);
	Spectrum acc(0.0f);
	for (int camPathLength = 1; camPathLength <= NUM_V_PER_PATH; camPathLength++)
	{
		TraceResult r2 = k_TraceRay(cameraState.r);
		if (!r2.hasHit())
		{
			//sample environment map

			break;
		}

		DifferentialGeometry dg;
		BSDFSamplingRecord bRec(dg);
		r2.getBsdfSample(cameraState.r, rng, &bRec);

		cameraState.dVCM *= r2.m_fDist * r2.m_fDist;
		cameraState.dVCM /= fabsf(Frame::cosTheta(bRec.wi));
		cameraState.dVC /= fabsf(Frame::cosTheta(bRec.wi));
		cameraState.dVM /= fabsf(Frame::cosTheta(bRec.wi));

		if (r2.LightIndex() != 0xffffffff)
		{
			acc += cameraState.throughput * gatherLight(cameraState, bRec, r2, rng, camPathLength, true);
			break;
		}

		if (r2.getMat().bsdf.hasComponent(ESmooth))
		{
			acc += cameraState.throughput * connectToLight(cameraState, bRec, r2.getMat(), rng, mMisVmWeightFactor, true);

			for (int emitterVertexIdx = 0; emitterVertexIdx < emitterVerticesStored; emitterVertexIdx++)
			{
				BPTVertex lv = lightPath[emitterVertexIdx];
				acc += cameraState.throughput * lv.throughput * connectVertices(lv, cameraState, bRec, r2.getMat(), mMisVcWeightFactor, mMisVmWeightFactor, true);
			}

			//scale by 2 to account for no merging in the first iteration
#ifdef ISCUDA
			acc += cameraState.throughput * (a_NumIteration == 2 ? 2 : 1) * L_Surface(cameraState, bRec, a_Radius, &r2.getMat(), mMisVcWeightFactor);
#endif
		}

		if (!sampleScattering(cameraState, bRec, r2.getMat(), rng, mMisVcWeightFactor, mMisVmWeightFactor))
			break;
	}

	img.Add(pixelPosition.x, pixelPosition.y, acc);
}

__global__ void pathKernel(unsigned int w, unsigned int h, int xoff, int yoff, k_BlockSampleImage img, float a_Radius, int a_NumIteration)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x + xoff, y = blockIdx.y * blockDim.y + threadIdx.y + yoff;
	CudaRNG rng = g_RNGData();
	if (x < w && y < h)
		VCM(Vec2f(x, y), img, rng, w, h, a_Radius, a_NumIteration);
	g_RNGData(rng);
}

__global__ void buildHashGrid2()
{
	unsigned int idx = threadId;
	if (idx < g_NextMap.m_uPhotonNumEmitted)
	{
		k_pPpmPhoton& e = g_NextMap.m_pPhotons[idx];
		const k_PhotonMap<k_HashGrid_Reg>& map = (&g_NextMap.m_sSurfaceMap)[e.getType()];
		unsigned int i = map.m_sHash.Hash(e.getPos());
		unsigned int k = atomicExch(map.m_pDeviceHashGrid + i, idx);
		e.setNext(k);
	}
}

void k_VCM::RenderBlock(e_Image* I, int x, int y, int blockW, int blockH)
{
	float radius = getCurrentRadius(2);
	pathKernel << < numBlocks, threadsPerBlock >> >(w, h, x, y, m_pBlockSampler->getBlockImage(), radius, m_uPassesDone);
}

void k_VCM::DoRender(e_Image* I)
{
	m_sPhotonMapsNext.m_uPhotonNumEmitted = w * h;
	hipMemcpyToSymbol(HIP_SYMBOL(g_CurrentMap), &m_sPhotonMapsCurrent, sizeof(k_PhotonMapCollection<false>));
	hipMemcpyToSymbol(HIP_SYMBOL(g_NextMap), &m_sPhotonMapsNext, sizeof(k_PhotonMapCollection<false>));

	k_Tracer<true, true>::DoRender(I);
	hipMemcpyFromSymbol(&m_sPhotonMapsNext, HIP_SYMBOL(g_NextMap), sizeof(k_PhotonMapCollection<false>));
	buildHashGrid2 << <m_sPhotonMapsNext.m_uPhotonBufferLength / (32 * 6) + 1, dim3(32, 6, 1) >> >();
	hipMemcpyFromSymbol(&m_sPhotonMapsCurrent, HIP_SYMBOL(g_CurrentMap), sizeof(k_PhotonMapCollection<false>));
	hipMemcpyFromSymbol(&m_sPhotonMapsNext, HIP_SYMBOL(g_NextMap), sizeof(k_PhotonMapCollection<false>));

	swapk(m_sPhotonMapsNext, m_sPhotonMapsCurrent);
	m_uPhotonsEmitted += m_sPhotonMapsCurrent.m_uPhotonNumEmitted;

	m_sPhotonMapsNext.StartNewPass();
}

void k_VCM::StartNewTrace(e_Image* I)
{
	k_Tracer<true, true>::StartNewTrace(I);
	m_uPhotonsEmitted = 0;
	AABB m_sEyeBox = GetEyeHitPointBox(m_pScene, true);
	m_sEyeBox.Enlarge(0.1f);
	float r = (m_sEyeBox.maxV - m_sEyeBox.minV).sum() / float(w);
	m_sEyeBox.minV -= Vec3f(r);
	m_sEyeBox.maxV += Vec3f(r);
	m_fInitialRadius = r;
	m_sPhotonMapsCurrent.StartNewRendering(m_sEyeBox, m_sEyeBox, r);
	m_sPhotonMapsCurrent.StartNewPass();
	m_sPhotonMapsNext.StartNewRendering(m_sEyeBox, m_sEyeBox, r);
	m_sPhotonMapsNext.StartNewPass();
}

k_VCM::k_VCM()
{
	int gridLength = 100;
	int numPhotons = 1024 * 1024 * 5;
	m_sPhotonMapsCurrent = k_PhotonMapCollection<false>(numPhotons, gridLength*gridLength*gridLength, 0xffffffff);
	m_sPhotonMapsNext = k_PhotonMapCollection<false>(numPhotons, gridLength*gridLength*gridLength, 0xffffffff);
}