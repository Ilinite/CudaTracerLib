#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include "..\Kernel\k_TraceAlgorithms.h"

CUDA_FUNC_IN float k(float t)
{
	//float t2 = t * t;
	//return 1.0f + t2 * t * (-6.0f * t2 + 15.0f * t - 10.0f);
	return clamp01(1.0f + t * t * t * (-6.0f * t * t + 15.0f * t - 10.0f));
}

CUDA_FUNC_IN float k_tr(float r , float t)
{
	//if (t > r)
	//	printf("t : %f, r : %f", t, r);
	return k(t / r) / (PI * r * r);
}

CUDA_FUNC_IN float k_tr(float r, const float3& t)
{
	return k_tr(r, length(t));
}

template<bool VOL> CUDA_FUNC_IN Spectrum L_Volume(float a_r, CudaRNG& rng, const Ray& r, float tmin, float tmax, const Spectrum& sigt, const Spectrum& sigs, Spectrum& Tr, const k_PhotonMapCollection& photonMap)
{
	const k_PhotonMapReg& map = photonMap.m_sVolumeMap;
	Spectrum Tau = Spectrum(0.0f);
	float Vs = 1.0f / ((4.0f / 3.0f) * PI * a_r * a_r * a_r * photonMap.m_uPhotonNumEmitted), r2 = a_r * a_r;
	Spectrum L_n = Spectrum(0.0f);
	float a,b;
	if (!map.m_sHash.getAABB().Intersect(r, &a, &b))
		return L_n;//that would be dumb
	a = clamp(a, tmin, tmax);
	b = clamp(b, tmin, tmax);
	float d = 2.0f * a_r;
	b -= d / 2.0f;
	while(b > a)
	{
		Spectrum L = Spectrum(0.0f);
		float3 x = r(b);
		uint3 lo = map.m_sHash.Transform(x - make_float3(a_r)), hi = map.m_sHash.Transform(x + make_float3(a_r));
		for(unsigned int ac = lo.x; ac <= hi.x; ac++)
			for(unsigned int bc = lo.y; bc <= hi.y; bc++)
				for(unsigned int cc = lo.z; cc <= hi.z; cc++)
				{
					unsigned int i0 = map.m_sHash.Hash(make_uint3(ac, bc, cc)), i = map.m_pDeviceHashGrid[i0];
					while(i != 0xffffffff && i != 0xffffff)
					{
						k_pPpmPhoton e = photonMap.m_pPhotons[i];
						float3 wi = e.getWi(), P = e.getPos();
						Spectrum l = e.getL();
						if(DistanceSquared(P, x) < r2)
						{
							float p = VOL ? g_SceneData.m_sVolume.p(x, r.direction, wi, rng) : Warp::squareToUniformSpherePdf();
							L += p * l * Vs;
						}
						i = e.getNext();
					}
				}
		Spectrum tauDelta = VOL ? g_SceneData.m_sVolume.tau(r, b - d, b) : sigt * d;
		Tau += tauDelta;
		Spectrum o_s = VOL ? g_SceneData.m_sVolume.sigma_s(x, -r.direction) : sigs;
		L_n = L * d + L_n * (-tauDelta).exp() + (VOL ? g_SceneData.m_sVolume.Lve(x, -1.0f * r.direction) * d : Spectrum(0.0f));
		b -= d;
	}
	Tr = (-Tau).exp();
	return L_n;
}

CUDA_FUNC_IN Spectrum L_Surface(BSDFSamplingRecord& bRec, float a_rSurfaceUNUSED, const e_KernelMaterial* mat, const k_PhotonMapCollection& photonMap, const k_PhotonMapReg& map)
{
	Spectrum Lp = Spectrum(0.0f);
	const float r2 = a_rSurfaceUNUSED * a_rSurfaceUNUSED;
	Frame sys = bRec.dg.sys;
	sys.t *= a_rSurfaceUNUSED;
	sys.s *= a_rSurfaceUNUSED;
	sys.n *= a_rSurfaceUNUSED;
	float3 a = -1.0f * sys.t - sys.s, b = sys.t - sys.s, c = -1.0f * sys.t + sys.s, d = sys.t + sys.s;
	float3 low = fminf(fminf(a, b), fminf(c, d)) + bRec.dg.P, high = fmaxf(fmaxf(a, b), fmaxf(c, d)) + bRec.dg.P;
	uint3 lo = map.m_sHash.Transform(low), hi = map.m_sHash.Transform(high);
	for(unsigned int a = lo.x; a <= hi.x; a++)
		for(unsigned int b = lo.y; b <= hi.y; b++)
			for(unsigned int c = lo.z; c <= hi.z; c++)
			{
				unsigned int i0 = map.m_sHash.Hash(make_uint3(a, b, c)), i = map.m_pDeviceHashGrid[i0];
				while (i != 0xffffffff && i != 0xffffff)
				{
					k_pPpmPhoton e = photonMap.m_pPhotons[i];
					float3 n = e.getNormal(), wi = e.getWi(), P = e.getPos();
					Spectrum l = e.getL();
					float dist2 = DistanceSquared(P, bRec.dg.P);
					if (dist2 < r2 && dot(n, bRec.dg.sys.n) > 0.8f)
					{
						float ke = k_tr(a_rSurfaceUNUSED, sqrtf(dist2));
						Lp += PI * ke * l;
					}
					i = e.getNext();
				}
			}
	/*unsigned int hash_idx = map.m_sHash.Hash(bRec.dg.P);
	unsigned int list_idx = map.m_pDeviceHashGrid[hash_idx];
	while (list_idx != 0xffffffff)
	{
		uint2 list_entry = map.m_pDeviceLinkedList[list_idx];
		k_pPpmPhoton e = photonMap.m_pPhotons[list_entry.x];
		list_idx = list_entry.y;

		float3 n = e.getNormal(), wi = e.getWi(), P = e.getPos();
		Spectrum l = e.getL();
		float dist2 = DistanceSquared(P, bRec.dg.P);
		if (dist2 < r2 && dot(n, bRec.dg.sys.n) > 0.8f)
		{
			float ke = k_tr(a_rSurfaceUNUSED, sqrtf(dist2));
			float dA = PI * r2;
			Lp += PI * ke * l / dA;
		}
	}*/
	return Lp / float(photonMap.m_uPhotonNumEmitted) * mat->bsdf.getDiffuseReflectance(bRec) * INV_PI;
}

CUDA_FUNC_IN Spectrum L_Surface(BSDFSamplingRecord& bRec, float a_rSurfaceUNUSED, const e_KernelMaterial* mat, k_AdaptiveStruct& A, int idx,
	const Spectrum& importance, int a_PassIndex, float scale0, float scale1, const k_PhotonMapCollection& photonMap)
{
	//Adaptive Progressive Photon Mapping Implementation
	k_AdaptiveEntry ent = A.E[idx];
	float r2 = ent.r * ent.r, maxr = MAX(ent.r, ent.rd), rd2 = ent.rd * ent.rd, rd = ent.rd, r = ent.r;
	Frame sys = bRec.dg.sys;
	sys.t *= maxr;
	sys.s *= maxr;
	sys.n *= maxr;
	float3 ur = bRec.dg.sys.t * rd, vr = bRec.dg.sys.s * rd;
	float3 a = -1.0f * sys.t - sys.s, b = sys.t - sys.s, c = -1.0f * sys.t + sys.s, d = sys.t + sys.s;
	float3 low = fminf(fminf(a, b), fminf(c, d)) + bRec.dg.P, high = fmaxf(fmaxf(a, b), fmaxf(c, d)) + bRec.dg.P;
	const k_PhotonMapReg& map = photonMap.m_sSurfaceMap;
	uint3 lo = map.m_sHash.Transform(low), hi = map.m_sHash.Transform(high);
	Spectrum Lp = make_float3(0), gamma = mat->bsdf.getDiffuseReflectance(bRec)	* INV_PI;//only diffuse
	for (int a = lo.x; a <= hi.x; a++)
	for (int b = lo.y; b <= hi.y; b++)
	for (int c = lo.z; c <= hi.z; c++)
	{
		unsigned int i0 = map.m_sHash.Hash(make_uint3(a, b, c)), i = map.m_pDeviceHashGrid[i0];
		while (i != 0xffffffff && i != 0xffffff)
		{
			k_pPpmPhoton e = photonMap.m_pPhotons[i];
			float3 nor = e.getNormal(), wi = e.getWi(), P = e.getPos();
			Spectrum l = e.getL();
			float dist2 = DistanceSquared(P, bRec.dg.P);
			if (dot(nor, bRec.dg.sys.n) > 0.95f)
			{
				bRec.wo = bRec.dg.toLocal(wi);
				float psi = Spectrum(importance * gamma * l).getLuminance();
				if (dist2 < rd2)
				{
					const float3 e_l = bRec.dg.P - P;
					float aa = k_tr(rd, e_l + ur), ab = k_tr(rd, e_l - ur);
					float ba = k_tr(rd, e_l + vr), bb = k_tr(rd, e_l - vr);
					float cc = k_tr(rd, e_l);
					float laplu = psi / rd2 * (aa + ab - 2.0f * cc);
					float laplv = psi / rd2 * (ba + bb - 2.0f * cc);
					ent.I += laplu + laplv;
					ent.I2 += (laplu + laplv) * (laplu + laplv);
				}
				if (dist2 < r2)
				{
					float kri = k_tr(r, sqrtf(dist2));
					Lp += kri * l * PI;
					ent.psi += psi;
					ent.psi2 += psi * psi;
					ent.pl += kri;
				}
			}
			i = e.getNext();
		}
	}
	/*
#define UPD(tar, val, pow) tar = scale0 * tar + scale1 * (pow == 1 ? val : val * val);
	UPD(ent.I, I_tmp, 1)
	UPD(ent.I2, I_tmp, 2)
	UPD(ent.psi, psi_tmp, 1)
	UPD(ent.psi2, psi_tmp, 2)
	UPD(ent.pl, pl_tmp, 1)
#undef UPD
	float VAR_Lapl = ent.I2 - ent.I * ent.I;
	float VAR_Phi = ent.psi2 - ent.psi * ent.psi;*/
	float NJ = a_PassIndex * photonMap.m_uPhotonNumEmitted;
	float VAR_Lapl = ent.I2 / NJ - ent.I / NJ * ent.I / NJ;
	float VAR_Phi = ent.psi2 / NJ - ent.psi / NJ * ent.psi / NJ;
	float E_I = ent.I / NJ;
	float E_pl = ent.pl / NJ;

	if (VAR_Lapl)
	{
		ent.rd = 1.9635f * sqrtf(VAR_Lapl) * powf(a_PassIndex, -1.0f / 8.0f);
		ent.rd = clamp(ent.rd, A.r_min, A.r_max);
	}

	if (VAR_Lapl && VAR_Phi)
	{
		float k_2 = 10.0f * PI / 168.0f, k_22 = k_2 * k_2;
		float ta = (2.0f * sqrtf(VAR_Phi / float(photonMap.m_uPhotonNumEmitted))) / (PI * float(photonMap.m_uPhotonNumEmitted) * E_pl * k_22 * E_I * E_I);
		ent.r = powf(ta, 1.0f / 6.0f) * powf(a_PassIndex, -1.0f / 6.0f);
		ent.r = clamp(ent.r, A.r_min, A.r_max);
	}
	A.E[idx] = ent;

	//return Lp / (a_rSurfaceUNUSED * a_rSurfaceUNUSED);
	return L_Surface(bRec, ent.r, mat, photonMap, photonMap.m_sSurfaceMap);
}

template<bool DIRECT> CUDA_FUNC_IN Spectrum L_FinalGathering(TraceResult& r2, BSDFSamplingRecord& bRec, CudaRNG& rng, float a_rSurfaceUNUSED, const k_PhotonMapCollection& photonMap)
{
	Spectrum LCaustic = L_Surface(bRec, a_rSurfaceUNUSED, &r2.getMat(), photonMap, photonMap.m_sCausticMap);
	Spectrum L(0.0f);
	const int N = 10;
	for (int i = 0; i < N; i++)
	{
		Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
		Ray r(bRec.dg.P, bRec.getOutgoing());
		TraceResult r3 = k_TraceRay(r);
		if (r3.hasHit())
		{
			DifferentialGeometry dg;
			BSDFSamplingRecord bRec2(dg);
			r3.getBsdfSample(r, rng, &bRec2);
			L += f * L_Surface(bRec2, a_rSurfaceUNUSED, &r3.getMat(), photonMap, photonMap.m_sSurfaceMap);
			if (DIRECT)
				L += f * UniformSampleAllLights(bRec2, r3.getMat(), 1);
			else L += f * r3.Le(bRec2.dg.P, bRec2.dg.sys, -r.direction);
		}
	}
	return L / float(N) + LCaustic;
}

template<bool DIRECT, bool FINAL_GATHER> CUDA_FUNC_IN void k_EyePassF(int x, int y, int w, int h, float a_PassIndex, float a_rSurfaceUNUSED, float a_rVolume, k_AdaptiveStruct A, float scale0, float scale1, e_Image g_Image, const k_PhotonMapCollection& photonMap)
{
	CudaRNG rng = g_RNGData();
	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	float2 screenPos = make_float2(x, y) + rng.randomFloat2();
	Ray r, rX, rY;
	Spectrum importance = g_SceneData.sampleSensorRay(r, rX, rY, screenPos, rng.randomFloat2());
	TraceResult r2;
	r2.Init();
	int depth = -1;
	Spectrum L(0.0f), throughput(1.0f);
	while(k_TraceRay(r.direction, r.origin, &r2) && depth++ < 5)
	{
		r2.getBsdfSample(r, rng, &bRec);
		if (depth == 0)
			dg.computePartials(r, rX, rY);
		if(g_SceneData.m_sVolume.HasVolumes())
		{
			float tmin, tmax;
			if (g_SceneData.m_sVolume.IntersectP(r, 0, r2.m_fDist, &tmin, &tmax))
			{
				Spectrum Tr;
				L += throughput * L_Volume<true>(a_rVolume, rng, r, tmin, tmax, Spectrum(0.0f), Spectrum(0.0f), Tr, photonMap);
				throughput = throughput * Tr;
			}
		}
		if(DIRECT)
			L += throughput * UniformSampleAllLights(bRec, r2.getMat(), 1);
		L += throughput * r2.Le(bRec.dg.P, bRec.dg.sys, -r.direction);//either it's the first bounce -> account or it's a specular reflection -> ...
		const e_KernelBSSRDF* bssrdf;
		if(r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
		{
			Spectrum t_f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
			bRec.wo.z *= -1.0f;
			Ray rTrans = Ray(bRec.dg.P, bRec.getOutgoing());
			TraceResult r3 = k_TraceRay(rTrans);
			Spectrum Tr;
			L += throughput * L_Volume<false>(a_rVolume, rng, rTrans, 0, r3.m_fDist, bssrdf->sigp_s + bssrdf->sig_a, bssrdf->sigp_s, Tr, photonMap);
			break;
		}
		bool hasDiffuse = r2.getMat().bsdf.hasComponent(EDiffuse), hasSpecGlossy = r2.getMat().bsdf.hasComponent(EDelta | EGlossy);
		if(hasDiffuse)
		{
			if (FINAL_GATHER)
				L += throughput * L_FinalGathering<DIRECT>(r2, bRec, rng, a_rSurfaceUNUSED, photonMap);
			else L += throughput * L_Surface(bRec, a_rSurfaceUNUSED, &r2.getMat(), photonMap, photonMap.m_sSurfaceMap);
			//L += throughput * L_Surface(bRec, a_rSurfaceUNUSED, &r2.getMat(), A, y * w + x, throughput, a_PassIndex, scale0, scale1, photonMap);
			if(!hasSpecGlossy)
				break;
		}
		if(hasSpecGlossy)
		{
			bRec.sampledType = 0;
			bRec.typeMask = EDelta | EGlossy;
			Spectrum t_f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
			if(!bRec.sampledType)
				break;
			throughput = throughput * t_f;
			r = Ray(bRec.dg.P, bRec.getOutgoing());
			r2.Init();
		}
		//else break;
	}
	if(!r2.hasHit())
	{
		if(g_SceneData.m_sVolume.HasVolumes())
		{
			float tmin, tmax;
			g_SceneData.m_sVolume.IntersectP(r, 0, r2.m_fDist, &tmin, &tmax);
			Spectrum Tr;
			L += throughput * L_Volume<true>(a_rVolume, rng, r, tmin, tmax, Spectrum(0.0f), Spectrum(0.0f), Tr, photonMap);
		}
		L += throughput * g_SceneData.EvalEnvironment(r);
	}
	g_Image.AddSample(screenPos.x, screenPos.y, importance * L);
	//Spectrum qs;
	//float t = A.E[y * w + x].r / a_rSurfaceUNUSED;
	//t = (A.E[y * w + x].r - A.r_min) / (A.r_max - A.r_min);
	//qs.fromHSL(1.0f / 3.0f - t / 3.0f, 1, 0.5f);
	//g_Image.AddSample(screenPos.x, screenPos.y, qs);
	/*auto ent = A.E[y * w + x];
	float NJ = a_PassIndex * photonMap.m_uPhotonNumEmitted;
	float VAR_Lapl = ent.I2 / NJ - ent.I / NJ * ent.I / NJ;
	float VAR_Phi = ent.psi2 / NJ - ent.psi / NJ * ent.psi / NJ;
	float E_I = ent.I / NJ;
	float E_pl = ent.pl / NJ;
	g_Image.AddSample(screenPos.x, screenPos.y, Spectrum(VAR_Phi*100));*/
	g_RNGData(rng);
}

template<bool DIRECT, bool FINAL_GATHER> __global__ void k_EyePass(int2 off, int w, int h, float a_PassIndex, float a_rSurfaceUNUSED, float a_rVolume, k_AdaptiveStruct A, float scale0, float scale1, e_Image g_Image, k_PhotonMapCollection photonMap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	x += off.x;
	y += off.y;
	if (x < w && y < h)
		k_EyePassF<DIRECT, FINAL_GATHER>(x, y, w, h, a_PassIndex, a_rSurfaceUNUSED, a_rVolume, A, scale0, scale1, g_Image, photonMap);
}

#define TN(r) (r * powf(float(m_uPassesDone), -1.0f/6.0f))
void k_sPpmTracer::doEyePass(e_Image* I)
{
	//I->Clear();
	k_INITIALIZE(m_pScene, g_sRngs);
	float s1 = float(m_uPassesDone - 1) / float(m_uPassesDone), s2 = 1.0f / float(m_uPassesDone);
	k_AdaptiveStruct A(TN(r_min), TN(r_max), m_pEntries);
	unsigned int p = 16;
	bool blocks = m_pScene->getVolumes().getLength() || m_bLongRunning || w * h > 1024 * 1024;
	dim3 bls = blocks ? dim3(8, 8, 1) : dim3(w / p + 1, h / p + 1, 1);
	int nx = blocks ? w / (bls.x * p) + 1 : 1, ny = blocks ? h / (bls.y * p) + 1 : 1;
	for(int i = 0; i < nx; i++)
	for (int j = 0; j < ny; j++)
	{
		int2 off = make_int2(bls.x * p * i, bls.y * p * j);
		if (m_bDirect)
		{
			if (m_bFinalGather)
				k_EyePass<true, true> << <bls, dim3(p, p, 1) >> >(off, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, m_sMaps);
			else k_EyePass<true, false> << <bls, dim3(p, p, 1) >> >(off, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, m_sMaps);
		}
		else
		{
			if (m_bFinalGather)
				k_EyePass<false, true> << <bls, dim3(p, p, 1) >> >(off, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, m_sMaps);
			else k_EyePass<false, false> << <bls, dim3(p, p, 1) >> >(off, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, m_sMaps);
		}
	}
	//Debug(I, make_int2(269, 158));
	I->DoUpdateDisplay(0);
}

void k_sPpmTracer::Debug(e_Image* I, int2 pixel)
{
	if(m_uPhotonsEmitted == (unsigned long long)-1)
		return;
	static k_AdaptiveEntry* hostEntries = 0;
	if (hostEntries == 0)
		hostEntries = new k_AdaptiveEntry[w * h];
	hipMemcpy(hostEntries, m_pEntries, w * h * sizeof(k_AdaptiveEntry), hipMemcpyDeviceToHost);
	k_AdaptiveStruct A(TN(r_min), TN(r_max), hostEntries);
	float s1 = float(m_uPassesDone - 1) / float(m_uPassesDone), s2 = 1.0f / float(m_uPassesDone);
	k_INITIALIZE(m_pScene, g_sRngs);
	k_PhotonMapCollection map = m_sMaps;
	k_PhotonMapReg& map2 = map.m_sSurfaceMap;
	static k_pPpmPhoton* hostPhotons = 0;
	static unsigned int* hostGrid = 0;
	if (hostPhotons == 0)
	{
		hostPhotons = new k_pPpmPhoton[map.m_uPhotonBufferLength];
		hostGrid = new unsigned int[map2.m_uGridLength];
	}
	hipMemcpy(hostPhotons, map.m_pPhotons, sizeof(k_pPpmPhoton)* map.m_uPhotonBufferLength, hipMemcpyDeviceToHost);
	hipMemcpy(hostGrid, map2.m_pDeviceHashGrid, sizeof(unsigned int)* map2.m_uGridLength, hipMemcpyDeviceToHost);
	map.m_pPhotons = hostPhotons;
	map2.m_pDeviceHashGrid = hostGrid;
	if (m_bDirect)
	{
		if (m_bFinalGather)
			k_EyePassF<true, true>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
		else k_EyePassF<true, false>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
	}
	else
	{
		if (m_bFinalGather)
			k_EyePassF<false, true>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
		else k_EyePassF<false, false>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
	}
}

__global__ void k_StartPass(int w, int h, float r, float rd, k_AdaptiveEntry* E)
{
	int i = threadId, x = i % w, y = i / w;
	if(x < w && y < h)
	{
		E[i].r = r;
		E[i].rd = rd;
		E[i].psi = E[i].psi2 = E[i].I = E[i].I2 = E[i].pl = 0.0f;
	}
}

void k_sPpmTracer::doStartPass(float r, float rd)
{
	int p = 32;
	k_StartPass<<<dim3(w / p + 1, h / p + 1, 1), dim3(p,p,1)>>>(w, h, r, rd, m_pEntries);
}