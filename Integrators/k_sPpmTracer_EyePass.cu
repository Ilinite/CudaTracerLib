#include "hip/hip_runtime.h"
#include "k_sPpmTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include "..\Kernel\k_TraceAlgorithms.h"

template<bool VOL> CUDA_FUNC_IN Spectrum L_Volume(float a_r, CudaRNG& rng, const Ray& r, float tmin, float tmax, const Spectrum& sigt, const Spectrum& sigs, Spectrum& Tr, const k_PhotonMapCollection<true>& photonMap, unsigned int a_NodeIndex = 0xffffffff)
{
	const k_PhotonMapReg& map = photonMap.m_sVolumeMap;
	Spectrum Tau = Spectrum(0.0f);
	float Vs = 1.0f / ((4.0f / 3.0f) * PI * a_r * a_r * a_r * photonMap.m_uPhotonNumEmitted), r2 = a_r * a_r;
	Spectrum L_n = Spectrum(0.0f);
	float a,b;
	if (!map.m_sHash.getAABB().Intersect(r, &a, &b))
		return L_n;//that would be dumb
	a = math::clamp(a, tmin, tmax);
	b = math::clamp(b, tmin, tmax);
	float d = 2.0f * a_r;
	b -= d / 2.0f;
	while(b > a)
	{
		Spectrum L = Spectrum(0.0f);
		Vec3f x = r(b);
		uint3 lo = map.m_sHash.Transform(x - Vec3f(a_r)), hi = map.m_sHash.Transform(x + Vec3f(a_r));
		for(unsigned int ac = lo.x; ac <= hi.x; ac++)
			for(unsigned int bc = lo.y; bc <= hi.y; bc++)
				for(unsigned int cc = lo.z; cc <= hi.z; cc++)
				{
					unsigned int i0 = map.m_sHash.Hash(make_uint3(ac, bc, cc)), i = map.m_pDeviceHashGrid[i0], NUM = 0;
					while(i != 0xffffffff && i != 0xffffff && NUM++ < 100)
					{
						k_pPpmPhoton e = photonMap.m_pPhotons[i];
						Vec3f wi = e.getWi(), P = e.getPos();
						Spectrum l = e.getL();
						if(distanceSquared(P, x) < r2)
						{
							float p = VOL ? g_SceneData.m_sVolume.p(x, r.direction, wi, rng, a_NodeIndex) : Warp::squareToUniformSpherePdf();
							L += p * l * Vs;
						}
						i = e.getNext();
					}
				}
		Spectrum tauDelta = VOL ? g_SceneData.m_sVolume.tau(r, b - d, b, a_NodeIndex) : sigt * d;
		Tau += tauDelta;
		Spectrum o_s = VOL ? g_SceneData.m_sVolume.sigma_s(x, -r.direction, a_NodeIndex) : sigs;
		L_n = L * d + L_n * (-tauDelta).exp() + (VOL ? g_SceneData.m_sVolume.Lve(x, -1.0f * r.direction, a_NodeIndex) * d : Spectrum(0.0f));
		b -= d;
	}
	Tr = (-Tau).exp();
	return L_n;
}

CUDA_FUNC_IN Spectrum L_Surface(BSDFSamplingRecord& bRec, float a_rSurfaceUNUSED, const e_KernelMaterial* mat, const k_PhotonMapCollection<true>& photonMap, const k_PhotonMapReg& map)
{
	Spectrum Lp = Spectrum(0.0f);
	const float r2 = a_rSurfaceUNUSED * a_rSurfaceUNUSED;
	Frame sys = bRec.dg.sys;
	sys.t *= a_rSurfaceUNUSED;
	sys.s *= a_rSurfaceUNUSED;
	sys.n *= a_rSurfaceUNUSED;
	Vec3f a = -1.0f * sys.t - sys.s, b = sys.t - sys.s, c = -1.0f * sys.t + sys.s, d = sys.t + sys.s;
	Vec3f low = min(min(a, b), min(c, d)) + bRec.dg.P, high = max(max(a, b), max(c, d)) + bRec.dg.P;
	Vec3u lo = map.m_sHash.Transform(low), hi = map.m_sHash.Transform(high);
	for(unsigned int a = lo.x; a <= hi.x; a++)
		for(unsigned int b = lo.y; b <= hi.y; b++)
			for(unsigned int c = lo.z; c <= hi.z; c++)
			{
				unsigned int i0 = map.m_sHash.Hash(make_uint3(a, b, c)), i = map.m_pDeviceHashGrid[i0];
				while (i != 0xffffffff && i != 0xffffff)
				{
					k_pPpmPhoton e = photonMap.m_pPhotons[i];
					Vec3f n = e.getNormal(), wi = e.getWi(), P = e.getPos();
					Spectrum l = e.getL();
					float dist2 = distanceSquared(P, bRec.dg.P);
					if (dist2 < r2 )//&& dot(n, bRec.dg.sys.n) > 0.8f
					{
						bRec.wo = bRec.dg.toLocal(wi);
						Spectrum bsdfFactor = mat->bsdf.f(bRec);
						float ke = k_tr(a_rSurfaceUNUSED, math::sqrt(dist2));
						Lp += PI * ke * l * bsdfFactor / Frame::cosTheta(bRec.wo);
					}
					i = e.getNext();
				}
			}
	/*unsigned int hash_idx = map.m_sHash.Hash(bRec.dg.P);
	unsigned int list_idx = map.m_pDeviceHashGrid[hash_idx];
	while (list_idx != 0xffffffff)
	{
		uint2 list_entry = map.m_pDeviceLinkedList[list_idx];
		k_pPpmPhoton e = photonMap.m_pPhotons[list_entry.x];
		list_idx = list_entry.y;

		float3 n = e.getNormal(), wi = e.getWi(), P = e.getPos();
		Spectrum l = e.getL();
		float dist2 = DistanceSquared(P, bRec.dg.P);
		if (dist2 < r2 && dot(n, bRec.dg.sys.n) > 0.8f)
		{
			float ke = k_tr(a_rSurfaceUNUSED, math::sqrt(dist2));
			float dA = PI * r2;
			Lp += PI * ke * l / dA;
		}
	}*/
	return Lp / float(photonMap.m_uPhotonNumEmitted);
}

CUDA_FUNC_IN Spectrum L_Surface(BSDFSamplingRecord& bRec, float a_rSurfaceUNUSED, const e_KernelMaterial* mat, k_AdaptiveStruct& A, int idx,
	const Spectrum& importance, int a_PassIndex, float scale0, float scale1, const k_PhotonMapCollection<true>& photonMap)
{
	//Adaptive Progressive Photon Mapping Implementation
	k_AdaptiveEntry ent = A.E[idx];
	float r2 = ent.r * ent.r, maxr = max(ent.r, ent.rd), rd2 = ent.rd * ent.rd, rd = ent.rd, r = ent.r;
	Frame sys = bRec.dg.sys;
	sys.t *= maxr;
	sys.s *= maxr;
	sys.n *= maxr;
	Vec3f ur = bRec.dg.sys.t * rd, vr = bRec.dg.sys.s * rd;
	Vec3f a = -1.0f * sys.t - sys.s, b = sys.t - sys.s, c = -1.0f * sys.t + sys.s, d = sys.t + sys.s;
	Vec3f low = min(min(a, b), min(c, d)) + bRec.dg.P, high = max(max(a, b), max(c, d)) + bRec.dg.P;
	const k_PhotonMapReg& map = photonMap.m_sSurfaceMap;
	Vec3u lo = map.m_sHash.Transform(low), hi = map.m_sHash.Transform(high);
	Spectrum Lp = 0.0f, gamma = mat->bsdf.f(bRec)	* INV_PI;//only diffuse //BUG f
	for (unsigned int a = lo.x; a <= hi.x; a++)
	for (unsigned int b = lo.y; b <= hi.y; b++)
	for (unsigned int c = lo.z; c <= hi.z; c++)
	{
		unsigned int i0 = map.m_sHash.Hash(make_uint3(a, b, c)), i = map.m_pDeviceHashGrid[i0];
		while (i != 0xffffffff && i != 0xffffff)
		{
			k_pPpmPhoton e = photonMap.m_pPhotons[i];
			Vec3f nor = e.getNormal(), wi = e.getWi(), P = e.getPos();
			Spectrum l = e.getL();
			float dist2 = distanceSquared(P, bRec.dg.P);
			if (dot(nor, bRec.dg.sys.n) > 0.95f)
			{
				bRec.wo = bRec.dg.toLocal(wi);
				float psi = Spectrum(importance * gamma * l).getLuminance();
				if (dist2 < rd2)
				{
					const Vec3f e_l = bRec.dg.P - P;
					float aa = k_tr(rd, e_l + ur), ab = k_tr(rd, e_l - ur);
					float ba = k_tr(rd, e_l + vr), bb = k_tr(rd, e_l - vr);
					float cc = k_tr(rd, e_l);
					float laplu = psi / rd2 * (aa + ab - 2.0f * cc);
					float laplv = psi / rd2 * (ba + bb - 2.0f * cc);
					ent.I += laplu + laplv;
					ent.I2 += (laplu + laplv) * (laplu + laplv);
				}
				if (dist2 < r2)
				{
					float kri = k_tr(r, math::sqrt(dist2));
					Lp += kri * l * PI;
					ent.psi += psi;
					ent.psi2 += psi * psi;
					ent.pl += kri;
				}
			}
			i = e.getNext();
		}
	}
	/*
#define UPD(tar, val, pow) tar = scale0 * tar + scale1 * (pow == 1 ? val : val * val);
	UPD(ent.I, I_tmp, 1)
	UPD(ent.I2, I_tmp, 2)
	UPD(ent.psi, psi_tmp, 1)
	UPD(ent.psi2, psi_tmp, 2)
	UPD(ent.pl, pl_tmp, 1)
#undef UPD
	float VAR_Lapl = ent.I2 - ent.I * ent.I;
	float VAR_Phi = ent.psi2 - ent.psi * ent.psi;*/
	float NJ = a_PassIndex * photonMap.m_uPhotonNumEmitted;
	float VAR_Lapl = ent.I2 / NJ - ent.I / NJ * ent.I / NJ;
	float VAR_Phi = ent.psi2 / NJ - ent.psi / NJ * ent.psi / NJ;
	float E_I = ent.I / NJ;
	float E_pl = ent.pl / NJ;

	if (VAR_Lapl)
	{
		ent.rd = 1.9635f * math::sqrt(VAR_Lapl) * powf(a_PassIndex, -1.0f / 8.0f);
		ent.rd = math::clamp(ent.rd, A.r_min, A.r_max);
	}

	if (VAR_Lapl && VAR_Phi)
	{
		float k_2 = 10.0f * PI / 168.0f, k_22 = k_2 * k_2;
		float ta = (2.0f * math::sqrt(VAR_Phi / float(photonMap.m_uPhotonNumEmitted))) / (PI * float(photonMap.m_uPhotonNumEmitted) * E_pl * k_22 * E_I * E_I);
		ent.r = powf(ta, 1.0f / 6.0f) * powf(a_PassIndex, -1.0f / 6.0f);
		ent.r = math::clamp(ent.r, A.r_min, A.r_max);
	}
	A.E[idx] = ent;

	//return Lp / (a_rSurfaceUNUSED * a_rSurfaceUNUSED);
	return L_Surface(bRec, ent.r, mat, photonMap, photonMap.m_sSurfaceMap);
}

template<bool DIRECT> CUDA_FUNC_IN Spectrum L_FinalGathering(TraceResult& r2, BSDFSamplingRecord& bRec, CudaRNG& rng, float a_rSurfaceUNUSED, const k_PhotonMapCollection<true>& photonMap)
{
	Spectrum LCaustic = L_Surface(bRec, a_rSurfaceUNUSED, &r2.getMat(), photonMap, photonMap.m_sCausticMap);
	Spectrum L(0.0f);
	const int N = 10;
	for (int i = 0; i < N; i++)
	{
		Spectrum f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
		Ray r(bRec.dg.P, bRec.getOutgoing());
		TraceResult r3 = k_TraceRay(r);
		if (r3.hasHit())
		{
			DifferentialGeometry dg;
			BSDFSamplingRecord bRec2(dg);
			r3.getBsdfSample(r, bRec2, ETransportMode::ERadiance);
			L += f * L_Surface(bRec2, a_rSurfaceUNUSED, &r3.getMat(), photonMap, photonMap.m_sSurfaceMap);
			if (DIRECT)
				L += f * UniformSampleAllLights(bRec2, r3.getMat(), 1, rng);
			else L += f * r3.Le(bRec2.dg.P, bRec2.dg.sys, -r.direction);
		}
	}
	return L / float(N) + LCaustic;
}

template<bool DIRECT, bool FINAL_GATHER> CUDA_FUNC_IN void k_EyePassF(int x, int y, int w, int h, float a_PassIndex, float a_rSurfaceUNUSED, float a_rVolume, k_AdaptiveStruct A, k_BlockSampleImage& img, const k_PhotonMapCollection<true>& photonMap)
{
	CudaRNG rng = g_RNGData();
	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	Vec2f screenPos = Vec2f(x, y) + rng.randomFloat2();
	Ray r, rX, rY;
	Spectrum throughput = g_SceneData.sampleSensorRay(r, rX, rY, screenPos, rng.randomFloat2());
	TraceResult r2;
	r2.Init();
	int depth = -1;
	Spectrum L(0.0f);
	while(k_TraceRay(r.direction, r.origin, &r2) && depth++ < 5)
	{
		r2.getBsdfSample(r, bRec, ETransportMode::ERadiance);
		if (depth == 0)
			dg.computePartials(r, rX, rY);
		if(g_SceneData.m_sVolume.HasVolumes())
		{
			float tmin, tmax;
			if (g_SceneData.m_sVolume.IntersectP(r, 0, r2.m_fDist, &tmin, &tmax))
			{
				Spectrum Tr;
				L += throughput * L_Volume<true>(a_rVolume, rng, r, tmin, tmax, Spectrum(0.0f), Spectrum(0.0f), Tr, photonMap);
				throughput = throughput * Tr;
			}
		}
		if(DIRECT)
			L += throughput * UniformSampleAllLights(bRec, r2.getMat(), 1, rng);
		L += throughput * r2.Le(bRec.dg.P, bRec.dg.sys, -r.direction);//either it's the first bounce -> account or it's a specular reflection -> ...
		const e_KernelBSSRDF* bssrdf;
		if (r2.getMat().GetBSSRDF(bRec.dg, &bssrdf))
		{
			Spectrum t_f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
			bRec.wo.z *= -1.0f;
			Ray rTrans = Ray(bRec.dg.P, bRec.getOutgoing());
			TraceResult r3 = k_TraceRay(rTrans);
			Spectrum Tr;
			/*if (r2.getMat().bsdf.m_bReflectDirAtSurface)
				L += throughput * L_Volume<true>(a_rVolume, rng, rTrans, 0, r3.m_fDist, r2.getNodeIndex(), Spectrum(0.0f), Spectrum(0.0f), Tr, photonMap);
				else */L += throughput * L_Volume<false>(a_rVolume, rng, rTrans, 0, r3.m_fDist, bssrdf->sigp_s + bssrdf->sig_a, bssrdf->sigp_s, Tr, photonMap, r2.getNodeIndex());
			//break;
		}
		bool hasSmooth = r2.getMat().bsdf.hasComponent(ESmooth),
			 hasSpecGlossy = r2.getMat().bsdf.hasComponent(EDelta | EGlossy),
			 hasGlossy = r2.getMat().bsdf.hasComponent(EGlossy);
		if (hasSmooth)
		{
			if (FINAL_GATHER)
				L += throughput * (hasGlossy ? 0.5f : 1) * L_FinalGathering<DIRECT>(r2, bRec, rng, a_rSurfaceUNUSED, photonMap);
			else L += throughput * (hasGlossy ? 0.5f : 1) * L_Surface(bRec, a_rSurfaceUNUSED, &r2.getMat(), photonMap, photonMap.m_sSurfaceMap);
			//L += throughput * L_Surface(bRec, a_rSurfaceUNUSED, &r2.getMat(), A, y * w + x, throughput, a_PassIndex, scale0, scale1, photonMap);
			if(!hasSpecGlossy)
				break;
		}
		if (hasSpecGlossy)
		{
			bRec.sampledType = 0;
			bRec.typeMask = EDelta | EGlossy;
			Spectrum t_f = r2.getMat().bsdf.sample(bRec, rng.randomFloat2());
			if(!bRec.sampledType)
				break;
			throughput = throughput * t_f * (hasGlossy ? 0.5f : 1);
			r = Ray(bRec.dg.P, bRec.getOutgoing());
			r2.Init();
		}
		else break;
	}
	if(!r2.hasHit())
	{
		if(g_SceneData.m_sVolume.HasVolumes())
		{
			float tmin, tmax;
			g_SceneData.m_sVolume.IntersectP(r, 0, r2.m_fDist, &tmin, &tmax);
			Spectrum Tr;
			L += throughput * L_Volume<true>(a_rVolume, rng, r, tmin, tmax, Spectrum(0.0f), Spectrum(0.0f), Tr, photonMap);
		}
		L += throughput * g_SceneData.EvalEnvironment(r);
	}
	img.Add(screenPos.x, screenPos.y, L);
	//Spectrum qs;
	//float t = A.E[y * w + x].r / a_rSurfaceUNUSED;
	//t = (A.E[y * w + x].r - A.r_min) / (A.r_max - A.r_min);
	//qs.fromHSL(1.0f / 3.0f - t / 3.0f, 1, 0.5f);
	//g_Image.AddSample(screenPos.x, screenPos.y, qs);
	/*auto ent = A.E[y * w + x];
	float NJ = a_PassIndex * photonMap.m_uPhotonNumEmitted;
	float VAR_Lapl = ent.I2 / NJ - ent.I / NJ * ent.I / NJ;
	float VAR_Phi = ent.psi2 / NJ - ent.psi / NJ * ent.psi / NJ;
	float E_I = ent.I / NJ;
	float E_pl = ent.pl / NJ;
	g_Image.AddSample(screenPos.x, screenPos.y, Spectrum(VAR_Phi*100));*/
	g_RNGData(rng);
}

template<bool DIRECT, bool FINAL_GATHER> __global__ void k_EyePass(Vec2i off, int w, int h, float a_PassIndex, float a_rSurfaceUNUSED, float a_rVolume, k_AdaptiveStruct A, k_BlockSampleImage img, k_PhotonMapCollection<true> photonMap)
{
	Vec2i pixel = k_TracerBase::getPixelPos(off.x, off.y);
	if (pixel.x < w && pixel.y < h)
		k_EyePassF<DIRECT, FINAL_GATHER>(pixel.x, pixel.y, w, h, a_PassIndex, a_rSurfaceUNUSED, a_rVolume, A, img, photonMap);
}

#define TN(r) (r * powf(float(m_uPassesDone), -1.0f/6.0f))
void k_sPpmTracer::RenderBlock(e_Image* I, int x, int y, int blockW, int blockH)
{
	float radius2 = powf(powf(m_fInitialRadius, float(2)) / std::pow(float(m_uPassesDone), 0.5f * (1 - ALPHA)), 1.0f / 2.0f);
	float radius3 = powf(powf(m_fInitialRadius, float(3)) / std::pow(float(m_uPassesDone), 0.5f * (1 - ALPHA)), 1.0f / 3.0f);
	k_AdaptiveStruct A(TN(r_min), TN(r_max), m_pEntries);
	Vec2i off = Vec2i(x, y);
	k_BlockSampleImage img = m_pBlockSampler->getBlockImage();
	if (m_bDirect)
	{
		if (m_bFinalGather)
			k_EyePass<true, true> << <numBlocks, threadsPerBlock >> >(off, w, h, m_uPassesDone, radius2, radius3, A, img, m_sMaps);
		else k_EyePass<true, false> << <numBlocks, threadsPerBlock >> >(off, w, h, m_uPassesDone, radius2, radius3, A, img, m_sMaps);
	}
	else
	{
		if (m_bFinalGather)
			k_EyePass<false, true> << <numBlocks, threadsPerBlock >> >(off, w, h, m_uPassesDone, radius2, radius3, A, img, m_sMaps);
		else k_EyePass<false, false> << <numBlocks, threadsPerBlock >> >(off, w, h, m_uPassesDone, radius2, radius3, A, img, m_sMaps);
	}
}

void k_sPpmTracer::Debug(e_Image* I, const Vec2i& pixel, ITracerDebugger* debugger)
{
	/*if(m_uPhotonsEmitted == (unsigned long long)-1)
		return;
	static k_AdaptiveEntry* hostEntries = 0;
	if (hostEntries == 0)
		hostEntries = new k_AdaptiveEntry[w * h];
	hipMemcpy(hostEntries, m_pEntries, w * h * sizeof(k_AdaptiveEntry), hipMemcpyDeviceToHost);
	k_AdaptiveStruct A(TN(r_min), TN(r_max), hostEntries);
	k_INITIALIZE(m_pScene, g_sRngs);
	k_PhotonMapCollection<true> map = m_sMaps;
	k_PhotonMapReg& map2 = map.m_sSurfaceMap;
	static k_pPpmPhoton* hostPhotons = 0;
	static unsigned int* hostGrid = 0;
	if (hostPhotons == 0)
	{
		hostPhotons = new k_pPpmPhoton[map.m_uPhotonBufferLength];
		hostGrid = new unsigned int[map2.m_uGridLength];
	}
	hipMemcpy(hostPhotons, map.m_pPhotons, sizeof(k_pPpmPhoton)* map.m_uPhotonBufferLength, hipMemcpyDeviceToHost);
	hipMemcpy(hostGrid, map2.m_pDeviceHashGrid, sizeof(unsigned int)* map2.m_uGridLength, hipMemcpyDeviceToHost);
	map.m_pPhotons = hostPhotons;
	map2.m_pDeviceHashGrid = hostGrid;
	if (m_bDirect)
	{
		if (m_bFinalGather)
			k_EyePassF<true, true>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
		else k_EyePassF<true, false>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
	}
	else
	{
		if (m_bFinalGather)
			k_EyePassF<false, true>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
		else k_EyePassF<false, false>(pixel.x, pixel.y, w, h, m_uPassesDone, getCurrentRadius(2), getCurrentRadius(3), A, s1, s2, *I, map);
	}*/
}

__global__ void k_StartPass(int w, int h, float r, float rd, k_AdaptiveEntry* E)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
	int i = y * w + x;
	if(x < w && y < h)
	{
		E[i].r = r;
		E[i].rd = rd;
		E[i].psi = E[i].psi2 = E[i].I = E[i].I2 = E[i].pl = 0.0f;
	}
}

void k_sPpmTracer::doStartPass(float r, float rd)
{
	int p = 32;
	k_StartPass<<<dim3(w / p + 1, h / p + 1, 1), dim3(p,p,1)>>>(w, h, r, rd, m_pEntries);
}