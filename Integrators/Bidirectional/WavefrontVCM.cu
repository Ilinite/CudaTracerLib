#include "hip/hip_runtime.h"
#include "WavefrontVCM.h"

namespace CudaTracerLib {

WavefrontVCM::WavefrontVCM(unsigned int a_NumLightRays)
	: m_uNumLightRays(a_NumLightRays), m_sLightBufA(a_NumLightRays), m_sLightBufB(a_NumLightRays), m_sCamBufA(BLOCK_SAMPLER_BlockSize * BLOCK_SAMPLER_BlockSize), m_sCamBufB(BLOCK_SAMPLER_BlockSize * BLOCK_SAMPLER_BlockSize)
{
	ThrowCudaErrors(CUDA_MALLOC(&m_pDeviceLightVertices, sizeof(BPTVertex) * MAX_LIGHT_SUB_PATH_LENGTH * a_NumLightRays));

	int gridLength = 200;
	int numPhotons = a_NumLightRays * MAX_LIGHT_SUB_PATH_LENGTH;
	m_sPhotonMapsNext = VCMSurfMap(Vec3u(gridLength), numPhotons);
}

WavefrontVCM::~WavefrontVCM()
{
	m_sLightBufA.Free();
	m_sLightBufB.Free();
	m_sCamBufA.Free();
	m_sCamBufB.Free();
}

CUDA_CONST float mMisVcWeightFactor;
CUDA_CONST float mMisVmWeightFactor;
CUDA_CONST float mLightSubPathCount;

CUDA_DEVICE k_WVCM_LightBuffer g_sLightBufA, g_sLightBufB;

CUDA_DEVICE VCMSurfMap g_NextMap2;

CUDA_DEVICE k_WVCM_CamBuffer g_sCamBufA, g_sCamBufB;

CUDA_GLOBAL void createLightRays(unsigned int g_DeviceNumLightPaths)
{
	CudaRNG rng = g_RNGData();
	unsigned int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx < g_DeviceNumLightPaths)
	{
		BPTSubPathState state;
		sampleEmitter(state, rng, mMisVcWeightFactor);
		auto& ent = g_sLightBufA(idx);
		ent.m_uVertexStart = idx;
		ent.state = state;
		g_sLightBufA.operator()(idx, 0).a = Vec4f(state.r.ori(), 0);
		g_sLightBufA.operator()(idx, 0).b = Vec4f(state.r.dir(), FLT_MAX);
	}
	g_RNGData(rng);
}

CUDA_GLOBAL void extendLighRays(unsigned int N, BPTVertex* g_pLightVertices, Image I, int iteration)
{
	CudaRNG rng = g_RNGData();
	unsigned int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		auto ent = g_sLightBufA(idx);
		unsigned int vIdx = ent.m_uVertexStart & 0x00ffffff, vOff = (ent.m_uVertexStart & 0xff000000) >> 24;
		auto res = g_sLightBufA.res(idx, 0);
		if (res.dist)
		{
			TraceResult r2;
			res.toResult(&r2, g_SceneData);
			BPTVertex v;
			v.mat = 0;
			r2.getBsdfSample(ent.state.r, v.bRec, ETransportMode::EImportance, &rng, &ent.state.throughput);

			if (vOff > 1 || true)
				ent.state.dVCM *= r2.m_fDist * r2.m_fDist;
			ent.state.dVCM /= math::abs(Frame::cosTheta(v.bRec.wi));
			ent.state.dVC /= math::abs(Frame::cosTheta(v.bRec.wi));

			if (r2.getMat().bsdf.hasComponent(ESmooth))
			{
				v.dVCM = ent.state.dVCM;
				v.dVC = ent.state.dVC;
				v.throughput = ent.state.throughput;
				v.mat = &r2.getMat();
				v.subPathLength = vOff + 1;
			}
			g_pLightVertices[vIdx * MAX_LIGHT_SUB_PATH_LENGTH + vOff] = v;

			auto ph = k_MISPhoton(v.throughput, -ent.state.r.dir(), v.bRec.dg.sys.n, PhotonType::pt_Diffuse, v.dVC, v.dVCM, v.dVM);
			Vec3u cell_idx = g_NextMap2.getHashGrid().Transform(v.bRec.dg.P);
			ph.setPos(g_NextMap2.getHashGrid(), cell_idx, v.bRec.dg.P);
			if (!g_NextMap2.store(cell_idx, ph))
				printf("WVCM : not enough photon storage allocated!\n");

			if (r2.getMat().bsdf.hasComponent(ESmooth))
				connectToCamera(ent.state, v.bRec, r2.getMat(), I, rng, mLightSubPathCount, mMisVmWeightFactor, 1, true);

			if (vOff < MAX_LIGHT_SUB_PATH_LENGTH - 1 && sampleScattering(ent.state, v.bRec, r2.getMat(), rng, mMisVcWeightFactor, mMisVmWeightFactor))
			{
				ent.m_uVertexStart = ((vOff + 1) << 24) | vIdx;
				unsigned int newRayIdx = g_sLightBufB.insertRay(0);
				g_sLightBufB(newRayIdx) = ent;
				g_sLightBufB(newRayIdx, 0).a = Vec4f(ent.state.r.ori(), 0);
				g_sLightBufB(newRayIdx, 0).b = Vec4f(ent.state.r.dir(), FLT_MAX);
			}
		}
	}
	g_RNGData(rng);
}

void WavefrontVCM::DoRender(Image* I)
{
	m_uLightOff = 0;
	k_INITIALIZE(m_pScene, g_sRngs);
	float a_Radius = this->getCurrentRadius(2);
	const float etaVCM = (PI * a_Radius * a_Radius) * m_uNumLightRays;
	float MisVmWeightFactor = 1;
	float MisVcWeightFactor = 1.0f / etaVCM;
	float one = 1;
	ThrowCudaErrors(hipMemset(m_pDeviceLightVertices, 0, sizeof(BPTVertex) * MAX_LIGHT_SUB_PATH_LENGTH * m_uNumLightRays));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mMisVcWeightFactor), &MisVcWeightFactor, sizeof(MisVcWeightFactor)));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mMisVmWeightFactor), &MisVmWeightFactor, sizeof(MisVmWeightFactor)));
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mLightSubPathCount), &one, sizeof(one)));

	m_sLightBufA.Clear();
	CopyToSymbol(g_sLightBufA, m_sLightBufA);
	createLightRays << <m_uNumLightRays / (32 * 6) + 1, dim3(32, 6) >> >(m_uNumLightRays);
	ThrowCudaErrors(hipDeviceSynchronize());
	m_sLightBufA.setNumRays(m_uNumLightRays, 0);

	m_sPhotonMapsNext.ResetBuffer();
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_NextMap2), &m_sPhotonMapsNext, sizeof(m_sPhotonMapsNext)));

	k_WVCM_LightBuffer* srcBuf = &m_sLightBufA, *destBuf = &m_sLightBufB;
	int i = 0;
	do
	{
		destBuf->Clear();
		srcBuf->IntersectBuffers<false>(false);
		ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_sLightBufA), srcBuf, sizeof(*srcBuf)));
		ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_sLightBufB), destBuf, sizeof(*destBuf)));
		extendLighRays << <srcBuf->getNumRays(0) / (32 * 6) + 1, dim3(32, 6) >> >(srcBuf->getNumRays(0), m_pDeviceLightVertices, *I, i++);
		ThrowCudaErrors(hipDeviceSynchronize());
		ThrowCudaErrors(hipMemcpyFromSymbol(srcBuf, HIP_SYMBOL(g_sLightBufA), sizeof(*srcBuf)));
		ThrowCudaErrors(hipMemcpyFromSymbol(destBuf, HIP_SYMBOL(g_sLightBufB), sizeof(*destBuf)));
		swapk(srcBuf, destBuf);

	} while (srcBuf->getNumRays(0));

	ThrowCudaErrors(hipMemcpyFromSymbol(&m_sPhotonMapsNext, HIP_SYMBOL(g_NextMap2), sizeof(m_sPhotonMapsNext)));

	Tracer<true, true>::DoRender(I);
}

void WavefrontVCM::StartNewTrace(Image* I)
{
	Tracer<true, true>::StartNewTrace(I);
	m_uPhotonsEmitted = 0;
	AABB m_sEyeBox = GetEyeHitPointBox(m_pScene, true);
	m_sEyeBox = m_sEyeBox.Extend(0.1f);
	float r = (m_sEyeBox.maxV - m_sEyeBox.minV).sum() / float(w);
	m_sEyeBox.minV -= Vec3f(r);
	m_sEyeBox.maxV += Vec3f(r);
	m_fInitialRadius = r;
	m_sPhotonMapsNext.SetSceneDimensions(m_sEyeBox);
}

CUDA_GLOBAL void createCameraRays(int xoff, int yoff, int blockW, int blockH, int w, int h)
{
	CudaRNG rng = g_RNGData();
	Vec2i pixel = TracerBase::getPixelPos(xoff, yoff);
	unsigned int idx = (pixel.y - yoff) * blockW + (pixel.x - xoff);
	if (pixel.x < w && pixel.y < h)
	{
		BPTSubPathState state;
		sampleCamera(state, rng, Vec2f(pixel.x, pixel.y), mLightSubPathCount);
		auto& ent = g_sCamBufA(idx);
		ent.x = pixel.x;
		ent.y = pixel.y;
		ent.acc = Spectrum(0.0f);
		ent.state = state;
		g_sCamBufA.operator()(idx, 0).a = Vec4f(state.r.ori(), 0);
		g_sCamBufA.operator()(idx, 0).b = Vec4f(state.r.dir(), FLT_MAX);
	}
	g_RNGData(rng);
}

CUDA_GLOBAL void performPPMEstimate(unsigned int N, float a_Radius, float nPhotons)
{
	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	CudaRNG rng = g_RNGData();
	unsigned int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		auto ent = g_sCamBufA(idx);
		auto res = g_sCamBufA.res(idx, 0);
		if (res.dist)
		{
			TraceResult r2;
			res.toResult(&r2, g_SceneData);

			r2.getBsdfSample(ent.state.r, bRec, ETransportMode::ERadiance, &rng);

			ent.state.dVCM *= r2.m_fDist * r2.m_fDist;
			ent.state.dVCM /= math::abs(Frame::cosTheta(bRec.wi));
			ent.state.dVC /= math::abs(Frame::cosTheta(bRec.wi));
			ent.state.dVM /= math::abs(Frame::cosTheta(bRec.wi));

			Spectrum phL;
			if (!r2.getMat().bsdf.hasComponent(EGlossy))
				phL = L_Surface2<false>(g_NextMap2, ent.state, bRec, a_Radius, &r2.getMat(), mMisVcWeightFactor, nPhotons, true);
			else phL = L_Surface2<true>(g_NextMap2, ent.state, bRec, a_Radius, &r2.getMat(), mMisVcWeightFactor, nPhotons, true);
			g_sCamBufA(idx).acc = ent.acc + ent.state.throughput * phL;
		}
	}
	g_RNGData(rng);
}

CUDA_GLOBAL void extendCameraRays(unsigned int N, Image I, int iteration, bool lastIteration, float a_Radius, unsigned int lightOff, unsigned int numLightPaths, BPTVertex* g_pLightVertices)
{
	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	CudaRNG rng = g_RNGData();
	unsigned int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		bool extended = false;
		auto ent = g_sCamBufA(idx);
		auto res = g_sCamBufA.res(idx, 0);
		if (res.dist)
		{
			TraceResult r2;
			res.toResult(&r2, g_SceneData);

			r2.getBsdfSample(ent.state.r, bRec, ETransportMode::ERadiance, &rng);

			ent.state.dVCM *= r2.m_fDist * r2.m_fDist;
			ent.state.dVCM /= math::abs(Frame::cosTheta(bRec.wi));
			ent.state.dVC /= math::abs(Frame::cosTheta(bRec.wi));
			ent.state.dVM /= math::abs(Frame::cosTheta(bRec.wi));

			if (r2.LightIndex() != UINT_MAX)
				ent.acc += ent.state.throughput * gatherLight(ent.state, bRec, r2, rng, iteration + 1, true);

			if (r2.getMat().bsdf.hasComponent(ESmooth))
			{
				ent.acc += ent.state.throughput * connectToLight(ent.state, bRec, r2.getMat(), rng, mMisVmWeightFactor, true);

				unsigned int vOff = ((lightOff + idx) % numLightPaths) * MAX_LIGHT_SUB_PATH_LENGTH, i = 0;
				while (i < MAX_LIGHT_SUB_PATH_LENGTH && g_pLightVertices[vOff + i].mat)
				{
					BPTVertex& v = g_pLightVertices[vOff + i];
					BPTVertex lv;
					lv.dg = v.dg;
					lv.dVC = v.dVC;
					lv.dVM = v.dVM;
					lv.dVCM = v.dVCM;
					lv.mat = v.mat;
					lv.subPathLength = v.subPathLength;
					lv.throughput = v.throughput;
					lv.bRec = BSDFSamplingRecord(lv.dg);
					lv.bRec.eta = v.bRec.eta;
					lv.bRec.mode = v.bRec.mode;
					lv.bRec.rng = v.bRec.rng;
					lv.bRec.sampledType = v.bRec.sampledType;
					lv.bRec.typeMask = v.bRec.typeMask;
					lv.bRec.wi = v.bRec.wi;
					lv.bRec.wo = v.bRec.wo;
					ent.acc += ent.state.throughput * lv.throughput * connectVertices(lv, ent.state, bRec, r2.getMat(), mMisVcWeightFactor, mMisVmWeightFactor, true);
					i++;
				}
			}

			if (!lastIteration && sampleScattering(ent.state, bRec, r2.getMat(), rng, mMisVcWeightFactor, mMisVmWeightFactor))
			{
				extended = true;
				unsigned int newRayIdx = g_sCamBufB.insertRay(0);
				g_sCamBufB(newRayIdx) = ent;
				g_sCamBufB(newRayIdx, 0).a = Vec4f(ent.state.r.ori(), 0);
				g_sCamBufB(newRayIdx, 0).b = Vec4f(ent.state.r.dir(), FLT_MAX);
			}
		}
		if (!extended)
			I.AddSample(ent.x, ent.y, ent.acc);
	}
	g_RNGData(rng);
}

void WavefrontVCM::RenderBlock(Image* I, int x, int y, int blockW, int blockH)
{
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_sCamBufA), &m_sCamBufA, sizeof(m_sCamBufA)));
	m_sCamBufA.Clear();
	createCameraRays << <BLOCK_SAMPLER_LAUNCH_CONFIG >> >(x, y, blockW, blockH, w, h);
	ThrowCudaErrors(hipDeviceSynchronize());
	m_sCamBufA.setNumRays(BLOCK_SAMPLER_BlockSize * BLOCK_SAMPLER_BlockSize, 0);

	k_WVCM_CamBuffer* srcBuf = &m_sCamBufA, *destBuf = &m_sCamBufB;
	int i = 0;
	do
	{
		destBuf->Clear();
		srcBuf->IntersectBuffers<false>(false);
		ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_sCamBufA), srcBuf, sizeof(*srcBuf)));
		ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_sCamBufB), destBuf, sizeof(*destBuf)));
		performPPMEstimate << <srcBuf->getNumRays(0) / (32 * 6) + 1, dim3(32, 6) >> >(srcBuf->getNumRays(0), getCurrentRadius(2), (float)m_uNumLightRays);
		extendCameraRays << <srcBuf->getNumRays(0) / (32 * 6) + 1, dim3(32, 6) >> >(srcBuf->getNumRays(0), *I, i++, i == 4, getCurrentRadius(2), m_uLightOff, m_uNumLightRays, m_pDeviceLightVertices);
		ThrowCudaErrors(hipDeviceSynchronize());
		ThrowCudaErrors(hipMemcpyFromSymbol(srcBuf, HIP_SYMBOL(g_sCamBufA), sizeof(*srcBuf)));
		ThrowCudaErrors(hipMemcpyFromSymbol(destBuf, HIP_SYMBOL(g_sCamBufB), sizeof(*destBuf)));
		swapk(srcBuf, destBuf);

	} while (srcBuf->getNumRays(0) && i < 6);
	m_uLightOff += blockW * blockH;
}

float WavefrontVCM::getSplatScale()
{
	return Tracer<true, true>::getSplatScale() * (w * h) / m_uNumLightRays;
}

}