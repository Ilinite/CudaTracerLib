#include "hip/hip_runtime.h"
#include "BDPT.h"
#include "VCMHelper.h"

namespace CudaTracerLib {

CUDA_FUNC_IN float pathWeight(int force_s, int force_t, int s, int t)
{
	if (force_s != -1 && force_t != -1 && (s != force_s || t != force_t))
		return 0;
	else return 1;
}

CUDA_FUNC_IN void BPT(const Vec2f& pixelPosition, Image& img, Sampler& rng, unsigned int w, unsigned int h,
	bool use_mis, int force_s, int force_t, float LScale)
{
	float mLightSubPathCount = 1 * 1;
	const float etaVCM = (PI * 1) * mLightSubPathCount;
	float mMisVmWeightFactor = 0;
	float mMisVcWeightFactor = Mis(1.f / etaVCM);

	BPTVertex lightPath[NUM_V_PER_PATH];
	BPTSubPathState lightPathState;
	sampleEmitter(lightPathState, rng, mMisVcWeightFactor);
	int emitterPathLength = 1, emitterVerticesStored = 0;
	for (; emitterVerticesStored < NUM_V_PER_PATH && emitterPathLength < MAX_SUB_PATH_LENGTH; emitterPathLength++)
	{
		TraceResult r2 = traceRay(lightPathState.r);
		if (!r2.hasHit())
			break;

		BPTVertex& v = lightPath[emitterVerticesStored];
		r2.getBsdfSample(lightPathState.r, v.bRec, ETransportMode::EImportance, &lightPathState.throughput);

		if (emitterPathLength > 1 || true)
			lightPathState.dVCM *= r2.m_fDist * r2.m_fDist;
		lightPathState.dVCM /= math::abs(Frame::cosTheta(v.bRec.wi));
		lightPathState.dVC /= math::abs(Frame::cosTheta(v.bRec.wi));

		//store in list
		if (r2.getMat().bsdf.hasComponent(ESmooth))
		{
			v.dVCM = lightPathState.dVCM;
			v.dVC = lightPathState.dVC;
			v.throughput = lightPathState.throughput;
			v.mat = &r2.getMat();
			v.subPathLength = emitterPathLength + 1;
			emitterVerticesStored++;
		}

		//connect to camera
		if (r2.getMat().bsdf.hasComponent(ESmooth))
			connectToCamera(lightPathState, v.bRec, r2.getMat(), img, rng, mLightSubPathCount, mMisVmWeightFactor, LScale * pathWeight(force_s, force_t, emitterPathLength, 1), use_mis);

		if (!sampleScattering(lightPathState, v.bRec, r2.getMat(), rng, mMisVcWeightFactor, mMisVmWeightFactor))
			break;
	}

	BPTSubPathState cameraState;
	sampleCamera(cameraState, rng, pixelPosition, mLightSubPathCount);
	Spectrum acc(0.0f);
	for (int camPathLength = 1; camPathLength <= NUM_V_PER_PATH; camPathLength++)
	{
		TraceResult r2 = traceRay(cameraState.r);
		if (!r2.hasHit())
		{
			//sample environment map
			acc += pathWeight(force_s, force_t, 0, camPathLength) * cameraState.throughput * gatherEnvironmentMap(cameraState, camPathLength, use_mis);
			break;
		}

		DifferentialGeometry dg;
		BSDFSamplingRecord bRec(dg);
		r2.getBsdfSample(cameraState.r, bRec, ETransportMode::ERadiance);

		cameraState.dVCM *= r2.m_fDist * r2.m_fDist;
		cameraState.dVCM /= math::abs(Frame::cosTheta(bRec.wi));
		cameraState.dVC /= math::abs(Frame::cosTheta(bRec.wi));

		if (r2.LightIndex() != UINT_MAX)
		{
			acc += pathWeight(force_s, force_t, 0, camPathLength) * cameraState.throughput * gatherLight(cameraState, bRec, r2, rng, camPathLength, use_mis);
			break;
		}

		if (r2.getMat().bsdf.hasComponent(ESmooth))
			acc += pathWeight(force_s, force_t, 1, camPathLength) * cameraState.throughput * connectToLight(cameraState, bRec, r2.getMat(), rng, mMisVmWeightFactor, use_mis);

		if (r2.getMat().bsdf.hasComponent(ESmooth))
			for (int emitterVertexIdx = 0; emitterVertexIdx < emitterVerticesStored; emitterVertexIdx++)
			{
				BPTVertex lv = lightPath[emitterVertexIdx];
				acc += pathWeight(force_s, force_t, lv.subPathLength, camPathLength) * cameraState.throughput * lv.throughput * connectVertices(lv, cameraState, bRec, r2.getMat(), mMisVcWeightFactor, mMisVmWeightFactor, use_mis);
			}

		if (!sampleScattering(cameraState, bRec, r2.getMat(), rng, mMisVcWeightFactor, mMisVmWeightFactor))
			break;
	}

	img.AddSample(pixelPosition.x, pixelPosition.y, acc * LScale);
}

__global__ void pathKernel(unsigned int w, unsigned int h, int xoff, int yoff, Image img,
	bool use_mis, int force_s, int force_t, float LScale)
{
	Vec2i pixel = TracerBase::getPixelPos(xoff, yoff);
	auto rng = g_SamplerData();
	if (pixel.x < w && pixel.y < h)
		BPT(Vec2f(pixel.x + rng.randomFloat(), pixel.y + rng.randomFloat()), img, rng, w, h, use_mis, force_s, force_t, LScale);
	g_SamplerData(rng);
}

void BDPT::RenderBlock(Image* I, int x, int y, int blockW, int blockH)
{
	pathKernel << < BLOCK_SAMPLER_LAUNCH_CONFIG >> >(w, h, x, y, *I,
													 m_sParameters.getValue(KEY_UseMis()), m_sParameters.getValue(KEY_Force_s()), m_sParameters.getValue(KEY_Force_t()), m_sParameters.getValue(KEY_ResultMultiplier()));
}

void BDPT::DebugInternal(Image* I, const Vec2i& pixel)
{
	//Li(*gI, g_RNGData(), pixel.x, pixel.y);
	auto rng = g_SamplerData();
	BPT(Vec2f(pixel), *I, rng, w, h, 
		m_sParameters.getValue(KEY_UseMis()), m_sParameters.getValue(KEY_Force_s()), m_sParameters.getValue(KEY_Force_t()), m_sParameters.getValue(KEY_ResultMultiplier()));
	g_SamplerData(rng);
}

}