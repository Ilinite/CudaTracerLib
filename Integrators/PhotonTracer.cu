#include "hip/hip_runtime.h"
#include "PhotonTracer.h"
#include <Kernel/TraceHelper.h>
#include <Kernel/TraceAlgorithms.h>
#include <Engine/Light.h>
#include <Engine/Sensor.h>

namespace CudaTracerLib {

enum
{
	MaxBlockHeight = 6,
};

CUDA_DEVICE unsigned int g_NextRayCounter3;

CUDA_FUNC_IN void handleEmission(const Spectrum& weight, const PositionSamplingRecord& pRec, Image& g_Image, CudaRNG& rng)
{
	DirectSamplingRecord dRec(pRec.p, pRec.n);
	Spectrum value = weight * g_SceneData.sampleSensorDirect(dRec, rng.randomFloat2());
	if (!value.isZero() && V(dRec.p, dRec.ref))
	{
		const KernelLight* emitter = (const KernelLight*)pRec.object;
		value *= emitter->evalDirection(DirectionSamplingRecord(dRec.d), pRec);
		g_Image.Splat(dRec.uv.x, dRec.uv.y, value);
	}
}

CUDA_FUNC_IN void handleSurfaceInteraction(const Spectrum& weight, BSDFSamplingRecord& bRec, const TraceResult& r2, Image& g_Image, CudaRNG& rng)
{
	DirectSamplingRecord dRec(bRec.dg.P, bRec.dg.sys.n);
	Spectrum value = weight * g_SceneData.sampleSensorDirect(dRec, rng.randomFloat2());
	if (!value.isZero() && V(dRec.p, dRec.ref))
	{
		bRec.wo = bRec.dg.toLocal(dRec.d);
		value *= r2.getMat().bsdf.f(bRec);
		g_Image.Splat(dRec.uv.x, dRec.uv.y, value);
	}
}

CUDA_FUNC_IN Vec3f refract(const Vec3f &wi, float cosThetaT, float eta)
{
	float scale = -(cosThetaT < 0 ? (1.0f / eta) : eta);
	return Vec3f(scale*wi.x, scale*wi.y, cosThetaT);
}
CUDA_FUNC_IN Vec3f reflect(const Vec3f &wi)
{
	return Vec3f(-wi.x, -wi.y, wi.z);
}
CUDA_FUNC_IN Spectrum sample(const Spectrum& s_, BSDFSamplingRecord& bRec, CudaRNG& rng)
{
	float w;
	Spectrum s = s_.SampleSpectrum(w, rng.randomFloat());
	for (int i = 0; i < 3; i++)
		if (s_[i] != 0)
			s[i] /= s_[i];

	Vec3f B(1.03961212f, 0.231792344f, 1.01046945f), C(6.00069867e-3f, 2.00179144e-2f, 1.03560653e2f);
	float w_mu = w / 1e3;
	float eta = math::safe_sqrt(1 + ((B * w_mu * w_mu) / (Vec3f(w_mu * w_mu) - C)).sum());
	//float eta = math::lerp(1.4f, 1.8f, (w - 300) / (600));
	//float eta = 1.5f;

	float cosThetaT;
	float F = MonteCarlo::fresnelDielectricExt(Frame::cosTheta(bRec.wi), cosThetaT, eta);
	Vec2f sample = rng.randomFloat2();
	if (sample.x <= F) {
		bRec.sampledType = EDeltaReflection;
		bRec.wo = reflect(bRec.wi);
		bRec.eta = 1.0f;

		return Spectrum(1.0f);
	}
	else {
		bRec.sampledType = EDeltaTransmission;
		bRec.wo = refract(bRec.wi, cosThetaT, eta);
		bRec.eta = cosThetaT < 0 ? eta : (1.0f / eta);

		float factor = (bRec.mode == ERadiance) ? (cosThetaT < 0 ? (1.0f / eta) : eta) : 1.0f;

		return s * (factor * factor);
	}
}

CUDA_FUNC_IN void doWork(Image& g_Image, CudaRNG& rng)
{
	PositionSamplingRecord pRec;
	Spectrum power = g_SceneData.sampleEmitterPosition(pRec, rng.randomFloat2()), throughput = Spectrum(1.0f);

	handleEmission(power, pRec, g_Image, rng);

	DirectionSamplingRecord dRec;
	power *= ((const KernelLight*)pRec.object)->sampleDirection(dRec, pRec, rng.randomFloat2());

	Ray r(pRec.p, dRec.d);
	TraceResult r2;
	r2.Init();
	int depth = -1;
	DifferentialGeometry dg;
	BSDFSamplingRecord bRec(dg);
	while (++depth < 12 && Traceray(r.direction, r.origin, &r2))
	{
		r2.getBsdfSample(r, bRec, ETransportMode::EImportance, &rng);

		if (r2.getMat().bsdf.getTypeToken() != UINT_MAX)
			handleSurfaceInteraction(power * throughput, bRec, r2, g_Image, rng);

		Spectrum bsdfWeight = r2.getMat().bsdf.getTypeToken() == UINT_MAX ? sample(power * throughput, bRec, rng) : r2.getMat().bsdf.sample(bRec, rng.randomFloat2());

		r = Ray(bRec.dg.P, bRec.getOutgoing());
		r2.Init();
		if (bsdfWeight.isZero())
			break;
		throughput *= bsdfWeight;
		if (depth > 5)
		{
			float q = min(throughput.max(), 0.95f);
			if (rng.randomFloat() >= q)
				break;
			throughput /= q;
		}
	}
}

__global__ void pathKernel(unsigned int N, Image g_Image)
{
	CudaRNG rng = g_RNGData();
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	volatile int& rayBase = nextRayArray[threadIdx.y];
	do
	{
		if (threadIdx.x == 0)
			rayBase = atomicAdd(&g_NextRayCounter3, blockDim.x);

		int rayidx = rayBase + threadIdx.x;
		if (rayidx >= N)
			break;

		doWork(g_Image, rng);
	} while (true);
	g_RNGData(rng);
}

void k_PhotonTracer::DoRender(Image* I)
{
	unsigned int zero = 0;
	ThrowCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_NextRayCounter3), &zero, sizeof(unsigned int)));
	k_INITIALIZE(m_pScene, g_sRngs);
	pathKernel << < 180, dim3(32, MaxBlockHeight, 1) >> >(w * h, *I);
	ThrowCudaErrors(hipDeviceSynchronize());
}

void k_PhotonTracer::Debug(Image* I, const Vec2i& pixel)
{
	k_INITIALIZE(m_pScene, g_sRngs);
	CudaRNG rng = g_RNGData();
	doWork(*I, rng);
	g_RNGData(rng);
}

}