#include "hip/hip_runtime.h"
#include "k_PathTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include <time.h>
#include "..\Kernel\k_TraceAlgorithms.h"

CUDA_ALIGN(16) CUDA_DEVICE unsigned int g_NextRayCounter;

template<bool DIRECT> __global__ void pathKernel(unsigned int width, unsigned int height, unsigned int a_PassIndex, e_Image g_Image)
{
	CudaRNG rng = g_RNGData();
	int rayidx;
	int N = width * height;
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        const bool          terminated     = 1;//nodeAddr == EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));	

        if(terminated)
        {			
            if (idxTerminated == 0)
				rayBase = atomicAdd(&g_NextRayCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
			if (rayidx >= N)
                break;
		}

		unsigned int x = rayidx % width, y = rayidx / width;
		Ray r;
		Spectrum imp = g_SceneData.sampleSensorRay(r, make_float2(x, y), rng.randomFloat2());

		Spectrum col = imp * PathTrace<DIRECT>(r.direction, r.origin, rng);
		
		g_Image.AddSample(x, y, col);
	}
	while(true);
	g_RNGData(rng);
}

__global__ void debugPixel(unsigned int width, unsigned int height, int2 p)
{
	CudaRNG rng = g_RNGData();
	Ray r = g_SceneData.GenerateSensorRay(p.x, p.y);	
	PathTrace<true>(r.direction, r.origin, rng);
}

void k_PathTracer::DoRender(e_Image* I)
{
	k_ProgressiveTracer::DoRender(I);
	ZeroSymbol(g_NextRayCounter);
	k_INITIALIZE(m_pScene, g_sRngs);
	if(m_Direct)
		pathKernel<true><<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, m_uPassesDone, *I);
	else pathKernel<false><<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, m_uPassesDone, *I);
	m_uPassesDone++;
	k_TracerBase_update_TracedRays
	I->DoUpdateDisplay(m_uPassesDone);
}

void k_PathTracer::Debug(int2 p)
{
	k_INITIALIZE(m_pScene, g_sRngs);
	//debugPixel<<<1,1>>>(w,h,p);
	CudaRNG rng = g_RNGData();
	Ray r = g_SceneData.GenerateSensorRay(p.x, p.y);	
	PathTrace<true>(r.direction, r.origin, rng);
}

template<bool DIRECT> __global__ void pathKernel2(unsigned int width, unsigned int height, e_Image g_Image, k_BlockSampler sampler)
{
	uint2 pixel = sampler.pixelCoord();
	CudaRNG rng = g_RNGData();
	if(pixel.x < width && pixel.y < height)
	{
		Ray r;
		Spectrum imp = g_SceneData.sampleSensorRay(r, make_float2(pixel.x, pixel.y), rng.randomFloat2());
		Spectrum col = imp * PathTrace<DIRECT>(r.direction, r.origin, rng);
		g_Image.AddSample(pixel.x, pixel.y, col);
	}
	g_RNGData(rng);
}

void k_BlockPathTracer::DoRender(e_Image* I)
{
	k_ProgressiveTracer::DoRender(I);
	k_INITIALIZE(m_pScene, g_sRngs);
	if(m_Direct)
		pathKernel2<true><<< sampler.blockDim(), sampler.threadDim()>>>(w, h, *I, sampler);
	else pathKernel2<false><<< sampler.blockDim(), sampler.threadDim()>>>(w, h, *I, sampler);
	k_TracerBase_update_TracedRays
	I->DoUpdateDisplay(1);
	sampler.AddPass(*I);
}