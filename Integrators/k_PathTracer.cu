#include "hip/hip_runtime.h"
#include "k_PathTracer.h"
#include "..\Kernel\k_TraceHelper.h"
#include <time.h>
#include "..\Kernel\k_TraceAlgorithms.h"

CUDA_ALIGN(16) CUDA_DEVICE unsigned int g_NextRayCounter;

CUDA_FUNC_IN float G(const float3& N_x, const float3& N_y, const float3& x, const float3& y)
{
	float3 theta = normalize(y - x);
	return AbsDot(N_x, theta) * AbsDot(N_y, -theta) / DistanceSquared(x, y);
}

template<bool DIRECT> CUDA_FUNC_IN Spectrum PathTraceTTT(float3& a_Dir, float3& a_Ori, CudaRNG& rnd, float* distTravalled = 0)
{
	Ray r0 = Ray(a_Ori, a_Dir);
	TraceResult r;
	r.Init();
	Spectrum cl = Spectrum(0.0f);   // accumulated color
	Spectrum cf = Spectrum(1.0f);  // accumulated reflectance
	int depth = 0;
	BSDFSamplingRecord bRec;
	while (k_TraceRay(r0.direction, r0.origin, &r) && depth++ < 7)
	{
		r.getBsdfSample(r0, rnd, &bRec); //return (Spectrum(bRec.map.sys.n) + Spectrum(1)) / 2.0f; //return bRec.map.sys.n;
		cl += cf * r.Le(r0(r.m_fDist), bRec.map.sys, -r0.direction);
		Spectrum f;
		if (DIRECT)
		{
			DirectSamplingRecord dRec(bRec.map.P, bRec.map.sys.n);
			Spectrum value = g_SceneData.m_sLightData[0].sampleDirect(dRec, rnd.randomFloat2());
			bRec.wo = normalize(bRec.map.sys.toLocal(dRec.d));
			f = r.getMat().bsdf.f(bRec) / g_SceneData.m_sLightData[0].As<e_DiffuseLight>()->shapeSet.Pdf(dRec) * G(bRec.map.sys.n, dRec.n, bRec.map.P, dRec.p);
		}
		else f = r.getMat().bsdf.sample(bRec, rnd.randomFloat2());

		cf = cf * f;
		r0 = Ray(r0(r.m_fDist), bRec.getOutgoing());
		r.Init();
	}
	return cl;
}

template<bool DIRECT> __global__ void pathKernel(unsigned int width, unsigned int height, unsigned int a_PassIndex, e_Image g_Image)
{
	CudaRNG rng = g_RNGData();
	int rayidx;
	int N = width * height;
	__shared__ volatile int nextRayArray[MaxBlockHeight];
	do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        const bool          terminated     = 1;//nodeAddr == EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));	

        if(terminated)
        {			
            if (idxTerminated == 0)
				rayBase = atomicAdd(&g_NextRayCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
			if (rayidx >= N)
                break;
		}

		unsigned int x = rayidx % width, y = rayidx / width;
		Ray r;
		Spectrum imp = g_SceneData.sampleSensorRay(r, make_float2(x, y), rng.randomFloat2());

		Spectrum col = imp * PathTrace<DIRECT>(r.direction, r.origin, rng);
		
		g_Image.AddSample(x, y, col);
	}
	while(true);
	g_RNGData(rng);
}

__global__ void debugPixel(unsigned int width, unsigned int height, int2 p)
{
	CudaRNG rng = g_RNGData();
	Ray r = g_SceneData.GenerateSensorRay(p.x, p.y);	
	PathTrace<true>(r.direction, r.origin, rng);
}

void k_PathTracer::DoRender(e_Image* I)
{
	k_ProgressiveTracer::DoRender(I);
	ZeroSymbol(g_NextRayCounter);
	k_INITIALIZE(m_pScene, g_sRngs);
	if(m_Direct)
		pathKernel<true><<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, m_uPassesDone, *I);
	else pathKernel<false><<< 180, dim3(32, MaxBlockHeight, 1)>>>(w, h, m_uPassesDone, *I);
	m_uPassesDone++;
	k_TracerBase_update_TracedRays
	I->DoUpdateDisplay(m_uPassesDone);
}

void k_PathTracer::Debug(int2 p)
{
	k_INITIALIZE(m_pScene, g_sRngs);
	//debugPixel<<<1,1>>>(w,h,p);
	CudaRNG rng = g_RNGData();
	Ray r = g_SceneData.GenerateSensorRay(p.x, p.y);	
	PathTrace<true>(r.direction, r.origin, rng);
}

template<bool DIRECT> __global__ void pathKernel2(unsigned int width, unsigned int height, e_Image g_Image, k_BlockSampler sampler)
{
	uint2 pixel = sampler.pixelCoord();
	CudaRNG rng = g_RNGData();
	if(pixel.x < width && pixel.y < height)
	{
		Ray r;
		Spectrum imp = g_SceneData.sampleSensorRay(r, make_float2(pixel.x, pixel.y), rng.randomFloat2());
		Spectrum col = imp * PathTrace<DIRECT>(r.direction, r.origin, rng);
		g_Image.AddSample(pixel.x, pixel.y, col);
	}
	g_RNGData(rng);
}

void k_BlockPathTracer::DoRender(e_Image* I)
{
	k_ProgressiveTracer::DoRender(I);
	k_INITIALIZE(m_pScene, g_sRngs);
	if(m_Direct)
		pathKernel2<true><<< sampler.blockDim(), sampler.threadDim()>>>(w, h, *I, sampler);
	else pathKernel2<false><<< sampler.blockDim(), sampler.threadDim()>>>(w, h, *I, sampler);
	k_TracerBase_update_TracedRays
	I->DoUpdateDisplay(1);
	sampler.AddPass(*I);
}