#include "hip/hip_runtime.h"
#include <StdAfx.h>
#include "Platform.h"
#include <string.h>

#ifdef ISWINDOWS
#include <Windows.h>
#endif

namespace CudaTracerLib {

unsigned int Platform::Increment(unsigned int* add)
{
#if defined(ISCUDA)
	return atomicInc(add, UINT_MAX);
#elif defined(ISWINDOWS)
	return InterlockedExchangeAdd(add, 1);
#else
	return __sync_fetch_and_add(add, 1);
#endif
}

unsigned int Platform::Add(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
	return atomicAdd(add, val);
#elif defined(ISWINDOWS)
	return InterlockedExchangeAdd(add, val);
#else
	return __sync_fetch_and_add(add, val);
#endif
}

unsigned int Platform::Exchange(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
	return atomicExch(add, val);
#elif defined(ISWINDOWS)
	return InterlockedExchange(add, val);
#else
	return __atomic_exchange_n(add, val, __ATOMIC_SEQ_CST);
#endif
}

float Platform::Add(float* add, float val)
{
#if defined(ISCUDA)
	return atomicAdd(add, val);
#else
	float f = *add;
	*add = f + val;
	return f;
#endif
}

void Platform::SetMemory(void* dest, unsigned long long length, unsigned int val)
{
	memset(dest, val, length);
}

void Platform::OutputDebug(const std::string& msg)
{
#if defined(ISWINDOWS)
	OutputDebugString(msg.c_str());
#else

#endif
}

std::string vformat(const char *fmt, va_list ap)
{
	int l = vsnprintf(0, 0, fmt, ap);
	std::string str;
	str.resize(l);
	int n = vsnprintf((char*)str.c_str(), l, fmt, ap);
	if (n != l)
		throw std::runtime_error("Error formating string!");
	return str;
}

}
