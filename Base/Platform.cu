#include "hip/hip_runtime.h"
#include <StdAfx.h>
#include "Platform.h"

#ifdef ISWINDOWS
#include <Windows.h>
#elif ISUNIX

#endif

unsigned int Platform::Increment(unsigned int* add)
{
#if defined(ISCUDA)
	return atomicInc(add, 0xffffffff);
#elif defined(ISWINDOWS)
	return InterlockedIncrement(add);
#elif defined(ISUNIX)
	unsigned int v = *add;
	*add++;
	return v;
#endif
}

unsigned int Platform::Add(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
return atomicAdd(add, val);
#elif defined(ISWINDOWS)
return InterlockedAdd((long*)add, val);	
#elif defined(ISUNIX)
	unsigned int v = *add;
	*add += val;
	return v;
#endif
}

void Platform::SetMemory(void* dest, unsigned long long length, unsigned int val)
{
#if defined(ISWINDOWS)
	ZeroMemory(dest, length);
#elif defined(ISUNIX)

#endif
}

void Platform::OutputDebug(const char* msg)
{
#if defined(ISWINDOWS)
	OutputDebugString(msg);
#elif defined(ISUNIX)

#endif
}