#include "hip/hip_runtime.h"
#include <StdAfx.h>
#include "Platform.h"
#include <string.h>
#include <cstdint>

#ifdef ISWINDOWS
#include <Windows.h>
#endif

namespace CudaTracerLib {

unsigned int Platform::Increment(unsigned int* add)
{
#if defined(ISCUDA)
	return atomicInc(add, UINT_MAX);
#elif defined(ISWINDOWS)
	return InterlockedExchangeAdd(add, 1);
#else
	return __sync_fetch_and_add(add, 1);
#endif
}

unsigned int Platform::Add(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
	return atomicAdd(add, val);
#elif defined(ISWINDOWS)
	return InterlockedExchangeAdd(add, val);
#else
	return __sync_fetch_and_add(add, val);
#endif
}

unsigned int Platform::Exchange(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
	return atomicExch(add, val);
#elif defined(ISWINDOWS)
	return InterlockedExchange(add, val);
#else
	return __atomic_exchange_n(add, val, __ATOMIC_SEQ_CST);
#endif
}

float Platform::Add(float* add, float val)
{
#if defined(ISCUDA)
	return atomicAdd(add, val);
#else
	float f = *add;
	*add = f + val;
	return f;
#endif
}

void Platform::SetMemory(void* dest, size_t length, unsigned char val)
{
	if (dest && length)
		memset(dest, val, length);
}

void Platform::SetMemoryExt(void* p, size_t n, unsigned int c)
{
	if (!p || !n)
		return;

	//code from here : http://www.xs-labs.com/en/blog/2013/08/06/optimising-memset/

	uint8_t  * sp;
	uint64_t * lp;
	uint64_t   u64;
	uint8_t    u8;

	u8 = (uint8_t)c;
	u64 = (uint64_t)c;
	u64 = (u64 << 32) | u64;
	sp = (uint8_t *)p;

	while (n-- && (((uint64_t)sp & (uint64_t)-8) < (uint64_t)sp))
	{
		*(sp++) = u8;
	}

	lp = (uint64_t *)((void *)sp);

	while ((n / 8) > 0)
	{
		*(lp++) = u64;
		n -= 8;
	}

	sp = (uint8_t *)((void *)lp);

	while (n--)
	{
		*(sp++) = u8;
	}
}

void Platform::OutputDebug(const std::string& msg)
{
#if defined(ISWINDOWS)
	OutputDebugString(msg.c_str());
#else

#endif
}

std::string vformat(const char *fmt, va_list ap)
{
	va_list ap2;
	va_copy(ap2, ap);
	int l = vsnprintf(0, 0, fmt, ap);
	std::string str;
	str.resize(l);
	int n = vsnprintf((char*)str.c_str(), l, fmt, ap2);
	if (n != l)
		throw std::runtime_error("Error formating string!");
	return str;
}

}
