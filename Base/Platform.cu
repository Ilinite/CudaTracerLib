#include "hip/hip_runtime.h"
#include <StdAfx.h>
#include "Platform.h"
#include <string.h>

#ifdef ISWINDOWS
#include <Windows.h>
#elif ISUNIX

#endif

namespace CudaTracerLib {

unsigned int Platform::Increment(unsigned int* add)
{
#if defined(ISCUDA)
	return atomicInc(add, UINT_MAX);
#elif defined(ISWINDOWS)
	return InterlockedIncrement(add);
#elif defined(ISUNIX)
	unsigned int v = *add;
	*add++;
	return v;
#endif
}

unsigned int Platform::Add(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
	return atomicAdd(add, val);
#elif defined(ISWINDOWS)
	return InterlockedAdd((long*)add, val);
#elif defined(ISUNIX)
	unsigned int v = *add;
	*add += val;
	return v;
#endif
}

unsigned int Platform::Exchange(unsigned int* add, unsigned int val)
{
#if defined(ISCUDA)
	return atomicExch(add, val);
#elif defined(ISWINDOWS)
	return InterlockedExchange(add, val);
#elif defined(ISUNIX)
	unsigned int old = *add;
	*add = val;
	return old;
#endif
}

float Platform::Add(float* add, float val)
{
#if defined(ISCUDA)
	return atomicAdd(add, val);
#else
	float f = *add;
	*add = f + val;
	return f;
#endif
}

void Platform::SetMemory(void* dest, unsigned long long length, unsigned int val)
{
	memset(dest, val, length);
}

void Platform::OutputDebug(const std::string& msg)
{
#if defined(ISWINDOWS)
	OutputDebugString(msg.c_str());
#elif defined(ISUNIX)

#endif
}

std::string vformat(const char *fmt, va_list ap)
{
	int l = vsnprintf(0, 0, fmt, ap);
	std::string str;
	str.resize(l);
	int n = vsnprintf((char*)str.c_str(), l, fmt, ap);
	if (n != l)
		throw std::runtime_error("Error formating string!");
	return str;
}

}