#include "CudaRandom.h"
#include "..\MathTypes.h"
#include "../CudaMemoryManager.h"

float CudaRNG::randomFloat()
{
#ifdef ISCUDA
	return hiprand_uniform(&state);
#else
	return curand_uniform2(curand2(&state));
#endif
}

unsigned long CudaRNG::randomUint()
{
#ifdef ISCUDA
	return hiprand(&state);
#else
	return curand2(&state);
#endif
}

void CudaRNG::Initialize(unsigned int a_Index, unsigned int a_Spacing, unsigned int a_Offset)
{
#ifdef ISCUDA
	hiprand_init(a_Index * a_Spacing, a_Index * a_Offset, 0, &state);
#else
	curand_init2(a_Index * a_Spacing, a_Index * a_Offset, 0, &state);
#endif
}

CudaRNGBuffer::CudaRNGBuffer(unsigned int a_Length, unsigned int a_Spacing, unsigned int a_Offset)
{
	m_uNumGenerators = a_Length;
	CUDA_MALLOC(&m_pDeviceGenerators, a_Length * sizeof(CudaRNG));
	m_pHostGenerators = new CudaRNG[a_Length];
	createGenerators(a_Spacing, a_Offset);
}

void CudaRNGBuffer::Free()
{
	CUDA_FREE(m_pDeviceGenerators);
	delete[] m_pHostGenerators;
}

void CudaRNGBuffer::createGenerators(unsigned int a_Spacing, unsigned int a_Offset)
{
	for(unsigned int i = 0; i < m_uNumGenerators; i++)
	{
		(m_pHostGenerators + i)->Initialize(i, a_Spacing, a_Offset);
	}
	CUDA_MEMCPY_TO_DEVICE(m_pDeviceGenerators, m_pHostGenerators, sizeof(CudaRNG) * m_uNumGenerators);
}

CUDA_FUNC_IN unsigned int getGlobalIdx_2D_2D()
{
#ifdef ISCUDA
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
#else
	return 0u;
#endif
}


CudaRNG CudaRNGBuffer::operator()()
{
	unsigned int idx = getGlobalIdx_2D_2D();
	unsigned int i = idx % m_uNumGenerators;
	CudaRNG rng;
#ifdef ISCUDA
	rng = m_pDeviceGenerators[i];
	if (idx >= m_uNumGenerators)
	{
		//skipahead_sequence(idx / m_uNumGenerators, &rng.state);
	}
#else
	rng = m_pHostGenerators[i];
#endif
	return rng;
}

void CudaRNGBuffer::operator()(CudaRNG& val)
{
	unsigned int i = getGlobalIdx_2D_2D();
#ifdef ISCUDA
	if(i < m_uNumGenerators)
		m_pDeviceGenerators[i] = val;
#else
	m_pHostGenerators[i % m_uNumGenerators] = val;
#endif
}

void CudaRNGBuffer::NextPass()
{

}