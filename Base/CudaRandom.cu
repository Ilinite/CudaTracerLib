#include "CudaRandom.h"
#include <Math/Vector.h>
#include <CudaMemoryManager.h>

namespace CudaTracerLib {

float Curand_GENERATOR::randomFloat()
{
	float f;
#ifdef ISCUDA
	f = hiprand_uniform(&state);
#else
	f = curand_uniform2(curand2(&state));
#endif
	return f * (1 - 1e-5f);//hiprand_uniform := (0, 1] -> [0, 1)
}

unsigned long Curand_GENERATOR::randomUint()
{
#ifdef ISCUDA
	return hiprand(&state);
#else
	return curand2(&state);
#endif
}

void Curand_GENERATOR::Initialize(unsigned int a_Index)
{
#ifdef ISCUDA
	hiprand_init(1234, a_Index, 0, &state);
#else
	curand_init2(1234, a_Index, 0, &state);
#endif
}

CudaRNGBuffer::CudaRNGBuffer(unsigned int a_Length)
	: m_uNumGenerators(a_Length)
{
	CUDA_MALLOC(&m_pDeviceGenerators, a_Length * sizeof(CudaRNG));
	m_pHostGenerators = new CudaRNG[a_Length];
	createGenerators();
}

void CudaRNGBuffer::Free()
{
	CUDA_FREE(m_pDeviceGenerators);
	m_pDeviceGenerators = 0;
	delete[] m_pHostGenerators;
	m_pHostGenerators = 0;
}

void CudaRNGBuffer::createGenerators()
{
	for (unsigned int i = 0; i < m_uNumGenerators; i++)
	{
		m_pHostGenerators[i] = CudaRNG(i);
	}
	CUDA_MEMCPY_TO_DEVICE(m_pDeviceGenerators, m_pHostGenerators, sizeof(CudaRNG) * m_uNumGenerators);
}

CUDA_FUNC_IN unsigned int getGlobalIdx_2D_2D()
{
#ifdef ISCUDA
	unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
#else
	return 0u;
#endif
}


CudaRNG CudaRNGBuffer::operator()()
{
	unsigned int idx = getGlobalIdx_2D_2D();
	unsigned int i = idx % m_uNumGenerators;
#ifdef ISCUDA
	CudaRNG rng = m_pDeviceGenerators[i];
	if (idx >= m_uNumGenerators)
	{
		//skipahead_sequence(idx - m_uNumGenerators, &rng.state);
	}
#else
	CudaRNG rng = m_pHostGenerators[i];
#endif
	return rng;
}

void CudaRNGBuffer::operator()(CudaRNG& val)
{
	unsigned int i = getGlobalIdx_2D_2D();
#ifdef ISCUDA
	if(i < m_uNumGenerators)
		m_pDeviceGenerators[i] = val;
#else
	m_pHostGenerators[i % m_uNumGenerators] = val;
#endif
}

void CudaRNGBuffer::NextPass()
{

}

}